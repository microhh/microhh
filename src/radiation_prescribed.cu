#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2020 Chiel van Heerwaarden
 * Copyright (c) 2011-2020 Thijs Heus
 * Copyright (c) 2014-2020 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <iostream>

#include "radiation_prescribed.h"
#include "grid.h"
#include "tools.h"

namespace
{
    template<typename TF> __global__
    void set_surface_fluxes_g(
            TF* const __restrict__ sw_flux_dn,
            TF* const __restrict__ sw_flux_up,
            TF* const __restrict__ lw_flux_dn,
            TF* const __restrict__ lw_flux_up,
            const TF sw_flux_dn_in,
            const TF sw_flux_up_in,
            const TF lw_flux_dn_in,
            const TF lw_flux_up_in,
            const int icells, const int jcells)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        const int j = blockIdx.y*blockDim.y + threadIdx.y;

        if (i < icells && j < jcells)
        {
            const int ij  = i + j*icells;

            sw_flux_dn[ij] = sw_flux_dn_in;
            sw_flux_up[ij] = sw_flux_up_in;
            lw_flux_dn[ij] = lw_flux_dn_in;
            lw_flux_up[ij] = lw_flux_up_in;
        }
    }
}

#ifdef USECUDA
template<typename TF>
void Radiation_prescribed<TF>::exec(
        Thermo<TF>& thermo, const double time, Timeloop<TF>& timeloop, Stats<TF>& stats)
{
    if (swtimedep_prescribed)
    {
        auto& gd = grid.get_grid_data();

        const int blocki = gd.ithread_block;
        const int blockj = gd.jthread_block;
        const int gridi = gd.icells/blocki + (gd.icells%blocki > 0);
        const int gridj = gd.jcells/blockj + (gd.jcells%blockj > 0);
        dim3 gridGPU (gridi,  gridj,  1);
        dim3 blockGPU(blocki, blockj, 1);

        set_surface_fluxes_g<<<gridGPU, blockGPU>>>(
            sw_flux_dn_g,
            sw_flux_up_g,
            lw_flux_dn_g,
            lw_flux_up_g,
            sw_flux_dn_value,
            sw_flux_up_value,
            lw_flux_dn_value,
            lw_flux_up_value,
            gd.icells, gd.jcells);
        cuda_check_error();
    }
}

template<typename TF>
TF* Radiation_prescribed<TF>::get_surface_radiation_g(const std::string& name)
{
    if (name == "sw_down")
        return sw_flux_dn_g;
    else if (name == "sw_up")
        return sw_flux_up_g;
    else if (name == "lw_down")
        return lw_flux_dn_g;
    else if (name == "lw_up")
        return lw_flux_up_g;
    else
    {
        std::string error = "Variable \"" + name + "\" is not a valid surface radiation field";
        throw std::runtime_error(error);
    }
}

template<typename TF>
void Radiation_prescribed<TF>::prepare_device()
{
    auto& gd = grid.get_grid_data();
    const int memsize = gd.ijcells*sizeof(TF);

    // Allocate surface radiation fields.
    cuda_safe_call(hipMalloc(&lw_flux_dn_g, memsize));
    cuda_safe_call(hipMalloc(&lw_flux_up_g, memsize));
    cuda_safe_call(hipMalloc(&sw_flux_dn_g, memsize));
    cuda_safe_call(hipMalloc(&sw_flux_up_g, memsize));

    // Send data to GPU, in case timedep is disabled.
    forward_device();
}

template<typename TF>
void Radiation_prescribed<TF>::clear_device()
{
    cuda_safe_call(hipFree(lw_flux_dn_g));
    cuda_safe_call(hipFree(lw_flux_up_g));
    cuda_safe_call(hipFree(sw_flux_dn_g));
    cuda_safe_call(hipFree(sw_flux_up_g));
}

template<typename TF>
void Radiation_prescribed<TF>::forward_device()
{
    auto& gd = grid.get_grid_data();
    const int memsize = gd.ijcells*sizeof(TF);

    cuda_safe_call(hipMemcpy(lw_flux_dn_g, lw_flux_dn.data(), memsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(lw_flux_up_g, lw_flux_up.data(), memsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(sw_flux_dn_g, sw_flux_dn.data(), memsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(sw_flux_up_g, sw_flux_up.data(), memsize, hipMemcpyHostToDevice));
}

template<typename TF>
void Radiation_prescribed<TF>::backward_device()
{
    auto& gd = grid.get_grid_data();
    const int memsize = gd.ijcells*sizeof(TF);

    cuda_safe_call(hipMemcpy(lw_flux_dn.data(), lw_flux_dn_g, memsize, hipMemcpyDeviceToHost));
    cuda_safe_call(hipMemcpy(lw_flux_up.data(), lw_flux_up_g, memsize, hipMemcpyDeviceToHost));
    cuda_safe_call(hipMemcpy(sw_flux_dn.data(), sw_flux_dn_g, memsize, hipMemcpyDeviceToHost));
    cuda_safe_call(hipMemcpy(sw_flux_up.data(), sw_flux_up_g, memsize, hipMemcpyDeviceToHost));
}
#endif

#ifdef FLOAT_SINGLE
template class Radiation_prescribed<float>;
#else
template class Radiation_prescribed<double>;
#endif
