#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2020 Chiel van Heerwaarden
 * Copyright (c) 2011-2020 Thijs Heus
 * Copyright (c) 2014-2020 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "grid.h"
#include "tools.h"
#include "math.h"

template<typename TF>
void Grid<TF>::prepare_device()
{
    // Calculate optimal size thread blocks based on grid
    gd.ithread_block = min(256, 16 * ((gd.itot / 16) + (gd.itot % 16 > 0)));
    gd.jthread_block = 256 / gd.ithread_block;

    const int imemsize = gd.icells*sizeof(TF);
    const int jmemsize = gd.jcells*sizeof(TF);
    const int kmemsize = gd.kcells*sizeof(TF);

    cuda_safe_call(hipMalloc((void**)&gd.x_g,     imemsize));
    cuda_safe_call(hipMalloc((void**)&gd.y_g,     jmemsize));
    cuda_safe_call(hipMalloc((void**)&gd.z_g,     kmemsize));
    cuda_safe_call(hipMalloc((void**)&gd.zh_g,    kmemsize));
    cuda_safe_call(hipMalloc((void**)&gd.dz_g,    kmemsize));
    cuda_safe_call(hipMalloc((void**)&gd.dzh_g,   kmemsize));
    cuda_safe_call(hipMalloc((void**)&gd.dzi_g,   kmemsize));
    cuda_safe_call(hipMalloc((void**)&gd.dzhi_g,  kmemsize));
    cuda_safe_call(hipMalloc((void**)&gd.dzi4_g,  kmemsize));
    cuda_safe_call(hipMalloc((void**)&gd.dzhi4_g, kmemsize));

    cuda_safe_call(hipMemcpy(gd.x_g,     gd.x.data(),     imemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(gd.y_g,     gd.y.data(),     jmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(gd.z_g,     gd.z.data(),     kmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(gd.zh_g,    gd.zh.data(),    kmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(gd.dz_g,    gd.dz.data(),    kmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(gd.dzh_g,   gd.dzh.data(),   kmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(gd.dzi_g,   gd.dzi.data(),   kmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(gd.dzhi_g,  gd.dzhi.data(),  kmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(gd.dzi4_g,  gd.dzi4.data(),  kmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(gd.dzhi4_g, gd.dzhi4.data(), kmemsize, hipMemcpyHostToDevice));
}

template<typename TF>
void Grid<TF>::clear_device()
{
    cuda_safe_call(hipFree(gd.y_g    ));
    cuda_safe_call(hipFree(gd.z_g    ));
    cuda_safe_call(hipFree(gd.zh_g   ));
    cuda_safe_call(hipFree(gd.dz_g   ));
    cuda_safe_call(hipFree(gd.dzh_g  ));
    cuda_safe_call(hipFree(gd.dzi_g  ));
    cuda_safe_call(hipFree(gd.dzhi_g ));
    cuda_safe_call(hipFree(gd.dzi4_g ));
    cuda_safe_call(hipFree(gd.dzhi4_g));
}

template class Grid<double>;
template class Grid<float>;
