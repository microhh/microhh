#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2020 Chiel van Heerwaarden
 * Copyright (c) 2011-2020 Thijs Heus
 * Copyright (c) 2014-2020 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <algorithm>
#include <iostream>
#include "master.h"
#include "grid.h"
#include "fields.h"
#include "stats.h"
#include "limiter.h"
#include "tools.h"

namespace
{
    template<typename TF>__global__
    void tendency_limiter(
            TF* const __restrict__ at,
            const TF* const __restrict__ a,
            const TF dt, const TF dti, const TF eps,
            const int istart, const int iend,
            const int jstart, const int jend,
            const int kstart, const int kend,
            const int jj, const int kk)
    {
        const int i  = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j  = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k  = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;

            const TF a_new = a[ijk] + dt*at[ijk];
            at[ijk] += (a_new < TF(0.)) ? (-a_new + eps) * dti : TF(0.);
        }
    }
}

#ifdef USECUDA
template <typename TF>
void Limiter<TF>::exec(double dt, Stats<TF>& stats)
{
    if (limit_list.empty())
        return;

    const Grid_data<TF>& gd = grid.get_grid_data();
    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, gd.kmax);
    dim3 blockGPU(blocki, blockj, 1);

    const TF dti = 1./dt;

    constexpr TF eps = std::numeric_limits<TF>::epsilon();

    for (auto& name : limit_list)
    {
        tendency_limiter<TF><<<gridGPU, blockGPU>>>(
            fields.at.at(name)->fld_g, fields.ap.at(name)->fld_g,
            dt, dti, eps,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            gd.kstart, gd.kend,
            gd.icells, gd.ijcells);
        cuda_check_error();

        stats.calc_tend(*fields.at.at(name), tend_name);
    }

}
#endif

template class Limiter<double>;
template class Limiter<float>;
