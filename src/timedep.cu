#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2020 Chiel van Heerwaarden
 * Copyright (c) 2011-2020 Thijs Heus
 * Copyright (c) 2014-2020 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "tools.h"
#include "grid.h"
#include "timedep.h"
#include "timeloop.h"

namespace
{
    template<typename TF> __global__
    void calc_time_dependent_prof_g(
            TF* const __restrict__ prof,
            const TF* const __restrict__ data,
            const TF fac0, const TF fac1,
            const int index0, const int index1,
            const int kmax, const int kgc)
    {
        const int k = blockIdx.x*blockDim.x + threadIdx.x;
        const int kk = kmax;

        if (k < kmax)
            prof[k+kgc] = fac0*data[index0*kk+k] + fac1*data[index1*kk+k];
    }
}


#ifdef USECUDA
template<typename TF>
void Timedep<TF>::clear_device()
{
    if(sw == Timedep_switch::Enabled)
        cuda_safe_call(hipFree(data_g));
}
#endif

#ifdef USECUDA
template<typename TF>
void Timedep<TF>::prepare_device()
{
    const int nmemsize = data.size()*sizeof(TF);
    cuda_safe_call(hipMalloc(&data_g, nmemsize));
    cuda_safe_call(hipMemcpy(data_g, data.data(), nmemsize, hipMemcpyHostToDevice));
}
#endif

#ifdef USECUDA
template <typename TF>
void Timedep<TF>::update_time_dependent_prof_g(TF* prof, Timeloop<TF>& timeloop)
{
    if (sw == Timedep_switch::Disabled)
        return;

    auto& gd = grid.get_grid_data();
    const int blockk = 128;
    const int gridk  = gd.kmax/blockk + (gd.kmax%blockk > 0);

    // Get/calculate the interpolation indexes/factors
    Interpolation_factors<TF> ifac = timeloop.get_interpolation_factors(time);

    // Calculate the new vertical profile
    calc_time_dependent_prof_g<<<gridk, blockk>>>(
            prof, data_g, ifac.fac0, ifac.fac1, ifac.index0, ifac.index1, gd.kmax, gd.kgc);
    cuda_check_error();
}
#endif

template class Timedep<double>;
template class Timedep<float>;
