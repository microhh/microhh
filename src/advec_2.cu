#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2020 Chiel van Heerwaarden
 * Copyright (c) 2011-2020 Thijs Heus
 * Copyright (c) 2014-2020 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "advec_2.h"
#include "grid.h"
#include "fields.h"
#include "tools.h"
#include "constants.h"
#include "tools.h"
#include "stats.h"
#include "finite_difference.h"
#include "field3d_operators.h"

using namespace Finite_difference::O2;

namespace
{
    template<typename TF>__global__
    void advec_uvw_g(TF* __restrict__ ut, TF* __restrict__ vt, TF * __restrict__ wt,
                     TF* __restrict__ u,  TF* __restrict__ v,  TF * __restrict__ w,
                     const TF* __restrict__ rhoref, const TF* __restrict__ rhorefh,
                     const TF* __restrict__ dzi,    const TF* __restrict__ dzhi, TF dxi, TF dyi,
                     int jj, int kk,
                     int istart, int jstart, int kstart,
                     int iend,   int jend,   int kend)
    {
        const int i  = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j  = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k  = blockIdx.z + kstart;
        const int ii = 1;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            ut[ijk] +=
                - (  interp2(u[ijk   ], u[ijk+ii]) * interp2(u[ijk   ], u[ijk+ii])
                   - interp2(u[ijk-ii], u[ijk   ]) * interp2(u[ijk-ii], u[ijk   ]) ) * dxi

                - (  interp2(v[ijk-ii+jj], v[ijk+jj]) * interp2(u[ijk   ], u[ijk+jj])
                   - interp2(v[ijk-ii   ], v[ijk   ]) * interp2(u[ijk-jj], u[ijk   ]) ) * dyi

                - (  rhorefh[k+1] * interp2(w[ijk-ii+kk], w[ijk+kk]) * interp2(u[ijk   ], u[ijk+kk])
                   - rhorefh[k  ] * interp2(w[ijk-ii   ], w[ijk   ]) * interp2(u[ijk-kk], u[ijk   ]) ) / rhoref[k] * dzi[k];

            vt[ijk] +=
                - (  interp2(u[ijk+ii-jj], u[ijk+ii]) * interp2(v[ijk   ], v[ijk+ii])
                   - interp2(u[ijk   -jj], u[ijk   ]) * interp2(v[ijk-ii], v[ijk   ]) ) * dxi

                - (  interp2(v[ijk   ], v[ijk+jj]) * interp2(v[ijk   ], v[ijk+jj])
                   - interp2(v[ijk-jj], v[ijk   ]) * interp2(v[ijk-jj], v[ijk   ]) ) * dyi

                - (  rhorefh[k+1] * interp2(w[ijk-jj+kk], w[ijk+kk]) * interp2(v[ijk   ], v[ijk+kk])
                   - rhorefh[k  ] * interp2(w[ijk-jj   ], w[ijk   ]) * interp2(v[ijk-kk], v[ijk   ]) ) / rhoref[k] * dzi[k];

            if (k > kstart)
            {
                wt[ijk] +=
                    - (  interp2(u[ijk+ii-kk], u[ijk+ii]) * interp2(w[ijk   ], w[ijk+ii])
                       - interp2(u[ijk   -kk], u[ijk   ]) * interp2(w[ijk-ii], w[ijk   ]) ) * dxi

                    - (  interp2(v[ijk+jj-kk], v[ijk+jj]) * interp2(w[ijk   ], w[ijk+jj])
                       - interp2(v[ijk   -kk], v[ijk   ]) * interp2(w[ijk-jj], w[ijk   ]) ) * dyi

                    - (  rhoref[k  ] * interp2(w[ijk   ], w[ijk+kk]) * interp2(w[ijk   ], w[ijk+kk])
                       - rhoref[k-1] * interp2(w[ijk-kk], w[ijk   ]) * interp2(w[ijk-kk], w[ijk   ]) ) / rhorefh[k] * dzhi[k];
            }
        }
    }

    template<typename TF>__global__
    void advec_s_g(TF* __restrict__ st, TF* __restrict__ s,
                   TF* __restrict__ u,  TF* __restrict__ v, TF* __restrict__ w,
                   const TF* __restrict__ rhoref, const TF* __restrict__ rhorefh,
                   const TF* __restrict__ dzi, TF dxi, TF dyi,
                   int jj, int kk,
                   int istart, int jstart, int kstart,
                   int iend,   int jend,   int kend)
    {
        const int i  = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j  = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k  = blockIdx.z + kstart;
        const int ii = 1;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            st[ijk] +=
                - (  u[ijk+ii] * interp2(s[ijk   ], s[ijk+ii])
                   - u[ijk   ] * interp2(s[ijk-ii], s[ijk   ]) ) * dxi

                - (  v[ijk+jj] * interp2(s[ijk   ], s[ijk+jj])
                   - v[ijk   ] * interp2(s[ijk-jj], s[ijk   ]) ) * dyi

                - (  rhorefh[k+1] * w[ijk+kk] * interp2(s[ijk   ], s[ijk+kk])
                   - rhorefh[k  ] * w[ijk   ] * interp2(s[ijk-kk], s[ijk   ]) ) / rhoref[k] * dzi[k];
        }
    }

    template<typename TF>__global__
    void calc_cfl_g(TF* __restrict__ u, TF* __restrict__ v, TF* __restrict__ w,
                    TF* __restrict__ cfl, const TF* __restrict__ dzi, TF dxi, TF dyi,
                    int jj, int kk,
                    int istart, int jstart, int kstart,
                    int iend, int jend, int kend)
    {
        const int i  = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j  = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k  = blockIdx.z + kstart;
        const int ii = 1;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            cfl[ijk] = std::abs(interp2(u[ijk], u[ijk+ii]))*dxi +
                       std::abs(interp2(v[ijk], v[ijk+jj]))*dyi +
                       std::abs(interp2(w[ijk], w[ijk+kk]))*dzi[k];
        }
    }
}

#ifdef USECUDA
template<typename TF>
unsigned long Advec_2<TF>::get_time_limit(unsigned long idt, double dt)
{
    // Calculate cfl and prevent zero divisons.
    double cfl = get_cfl(dt);
    cfl = std::max(cflmin, cfl);
    const unsigned long idtlim = idt * cflmax / cfl;

    return idtlim;
}

template<typename TF>
double Advec_2<TF>::get_cfl(const double dt)
{
    const Grid_data<TF>& gd = grid.get_grid_data();
    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, gd.kcells);
    dim3 blockGPU(blocki, blockj, 1);

    const TF dxi = 1./gd.dx;
    const TF dyi = 1./gd.dy;

    auto tmp1 = fields.get_tmp_g();

    calc_cfl_g<TF><<<gridGPU, blockGPU>>>(
        fields.mp.at("u")->fld_g, fields.mp.at("v")->fld_g, fields.mp.at("w")->fld_g,
        tmp1->fld_g, gd.dzi_g, dxi, dyi,
        gd.icells, gd.ijcells,
        gd.istart,  gd.jstart, gd.kstart,
        gd.iend,    gd.jend,   gd.kend);
    cuda_check_error();

    TF cfl = field3d_operators.calc_max_g(tmp1->fld_g);
    fields.release_tmp_g(tmp1);

    cfl = cfl*dt;

    return static_cast<double>(cfl);
}

template<typename TF>
void Advec_2<TF>::exec(Stats<TF>& stats)
{
    const Grid_data<TF>& gd = grid.get_grid_data();
    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, gd.kmax);
    dim3 blockGPU(blocki, blockj, 1);

    const TF dxi = 1./gd.dx;
    const TF dyi = 1./gd.dy;

    advec_uvw_g<TF><<<gridGPU, blockGPU>>>(
        fields.mt.at("u")->fld_g, fields.mt.at("v")->fld_g, fields.mt.at("w")->fld_g,
        fields.mp.at("u")->fld_g, fields.mp.at("v")->fld_g, fields.mp.at("w")->fld_g,
        fields.rhoref_g, fields.rhorefh_g, gd.dzi_g, gd.dzhi_g, dxi, dyi,
        gd.icells, gd.ijcells,
        gd.istart,  gd.jstart, gd.kstart,
        gd.iend,    gd.jend,   gd.kend);
    cuda_check_error();

    for (auto& it : fields.st)
        advec_s_g<TF><<<gridGPU, blockGPU>>>(
            it.second->fld_g, fields.sp.at(it.first)->fld_g,
            fields.mp.at("u")->fld_g, fields.mp.at("v")->fld_g, fields.mp.at("w")->fld_g,
            fields.rhoref_g, fields.rhorefh_g, gd.dzi_g, dxi, dyi,
            gd.icells, gd.ijcells,
            gd.istart,  gd.jstart, gd.kstart,
            gd.iend,    gd.jend,   gd.kend);
    cuda_check_error();

    hipDeviceSynchronize();
    stats.calc_tend(*fields.mt.at("u"), tend_name);
    stats.calc_tend(*fields.mt.at("v"), tend_name);
    stats.calc_tend(*fields.mt.at("w"), tend_name);
    for (auto it : fields.st)
        stats.calc_tend(*it.second, tend_name);
}
#endif

template class Advec_2<double>;
template class Advec_2<float>;
