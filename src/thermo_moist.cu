#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2024 Chiel van Heerwaarden
 * Copyright (c) 2011-2024 Thijs Heus
 * Copyright (c) 2014-2024 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <cstdio>
#include "grid.h"
#include "fields.h"
#include "thermo_moist.h"
#include "defines.h"
#include "constants.h"
#include "finite_difference.h"
#include "master.h"
#include "tools.h"
#include "column.h"
#include "stats.h"
#include "thermo_moist_functions.h"
#include <iostream>

namespace
{
    using namespace Constants;
    using namespace Finite_difference::O2;
    using namespace Thermo_moist_functions;

    template<typename TF>
    inline __device__ Struct_sat_adjust<TF> sat_adjust_g(
            const TF thl, const TF qt, const TF p, const TF exn)
    {
        using Fast_math::pow2;

        int niter = 0;
        const int nitermax = 10;

        TF tnr_old = TF(1.e9);

        const TF tl = thl * exn;
        TF qs = qsat_liq(p, tl);

        Struct_sat_adjust<TF> ans =
        {
            TF(0.), // ql
            TF(0.), // qi
            tl, // t
            qs, // qs
        };

        // Calculate if q-qs(Tl) <= 0. If so, return 0. Else continue with saturation adjustment.
        if (qt-ans.qs <= TF(0.))
            return ans;

        /* Saturation adjustment solver.
         * Root finding function is f(T) = T - tnr - Lv/cp*qt + alpha_w * Lv/cp*qs(T) + alpha_i*Ls/cp*qs(T)
         * dq_sat/dT derivatives can be rewritten using Claussius-Clapeyron (desat/dT = L{v,s}*esat / (Rv*T^2)).
         */

        TF tnr = tl;

        // Warm adjustment.
        if (tl >= T0<TF>)
        {
            while (fabs(tnr-tnr_old)/tnr_old > TF(1.e-5) && niter < nitermax)
            {
                tnr_old = tnr;
                qs = qsat_liq(p, tnr);
                const TF f =
                    tnr - tl - Lv<TF>/cp<TF>*(qt - qs);

                const TF f_prime = TF(1.) + Lv<TF>/cp<TF>*dqsatdT_liq(p, tnr);

                tnr -= f / f_prime;

                niter += 1;
            }

            qs = qsat_liq(p, tnr);
            ans.ql = fmax(TF(0.), qt - qs);
            ans.t  = tnr;
            ans.qs = qs;
        }
        // Cold adjustment.
        else
        {
            while (fabs(tnr-tnr_old)/tnr_old > TF(1.e-5) && niter < nitermax)
            {
                tnr_old = tnr;
                qs = qsat(p, tnr);
                const TF alpha_w = water_fraction(tnr);
                const TF alpha_i = TF(1.) - alpha_w;
                const TF dalphadT = (alpha_w > TF(0.) && alpha_w < TF(1.)) ? TF(0.025) : TF(0.);
                const TF dqsatdT_w = dqsatdT_liq(p, tnr);
                const TF dqsatdT_i = dqsatdT_ice(p, tnr);

                const TF f =
                    tnr - tl - alpha_w*Lv<TF>/cp<TF>*qt - alpha_i*Ls<TF>/cp<TF>*qt
                             + alpha_w*Lv<TF>/cp<TF>*qs + alpha_i*Ls<TF>/cp<TF>*qs;

                const TF f_prime = TF(1.)
                    - dalphadT*Lv<TF>/cp<TF>*qt + dalphadT*Ls<TF>/cp<TF>*qt
                    + dalphadT*Lv<TF>/cp<TF>*qs - dalphadT*Ls<TF>/cp<TF>*qs
                    + alpha_w*Lv<TF>/cp<TF>*dqsatdT_w
                    + alpha_i*Ls<TF>/cp<TF>*dqsatdT_i;

                tnr -= f / f_prime;

                niter += 1;
            }

            const TF alpha_w = water_fraction(tnr);
            const TF alpha_i = TF(1.) - alpha_w;

            qs = qsat(p, tnr);
            const TF qlqi = fmax(TF(0.), qt - qs);

            ans.ql = alpha_w*qlqi;
            ans.qi = alpha_i*qlqi;
            ans.t  = tnr;
            ans.qs = qs;
        }

        // Raise exception if nitermax is reached.
        if (niter == nitermax)
        {
            printf("ERROR: saturation adjustment did not converge: thl=%f, qt=%f, p=%f\n", thl, qt, p);
            asm("trap;");
        }

        return ans;
    }

    template<typename TF> __global__
    void calc_buoyancy_tend_2nd_g(TF* __restrict__ wt, TF* __restrict__ th, TF* __restrict__ qt,
                                  TF* __restrict__ thvrefh, TF* __restrict__ exnh, TF* __restrict__ ph,
                                  int istart, int jstart, int kstart,
                                  int iend,   int jend,   int kend,
                                  int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;

            // Half level temperature and moisture content
            const TF thh = static_cast<TF>(0.5) * (th[ijk-kk] + th[ijk]); // Half level liq. water pot. temp.
            const TF qth = static_cast<TF>(0.5) * (qt[ijk-kk] + qt[ijk]); // Half level specific hum.

            Struct_sat_adjust<TF> ssa = sat_adjust_g(thh, qth, ph[k], exnh[k]);

            // Calculate tendency.
            if (ssa.ql + ssa.qi > 0)
                wt[ijk] += buoyancy(exnh[k], thh, qth, ssa.ql, ssa.qi, thvrefh[k]);
            else
                wt[ijk] += buoyancy_no_ql(thh, qth, thvrefh[k]);
        }
    }

    template<typename TF> __global__
    void calc_buoyancy_g(TF* __restrict__ b,  TF* __restrict__ th,
                         TF* __restrict__ qt, TF* __restrict__ thvref,
                         TF* __restrict__ p,  TF* __restrict__ exn,
                         int istart, int jstart, int kstart,
                         int iend,   int jend,   int kcells,
                         int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z;

        if (i < iend && j < jend && k < kstart)
        {
            const int ijk   = i + j*jj + k*kk;
            b[ijk] = buoyancy_no_ql(th[ijk], qt[ijk], thvref[k]);
        }
        else if (i < iend && j < jend && k < kcells)
        {
            const int ijk = i + j*jj + k*kk;

            Struct_sat_adjust<TF> ssa = sat_adjust_g(th[ijk], qt[ijk], p[k], exn[k]);

            if (ssa.ql + ssa.qi > 0)
                b[ijk] = buoyancy(exn[k], th[ijk], qt[ijk], ssa.ql, ssa.qi, thvref[k]);
            else
                b[ijk] = buoyancy_no_ql(th[ijk], qt[ijk], thvref[k]);
        }
    }

    template<typename TF> __global__
    void calc_buoyancy_h_g(TF* __restrict__ bh,  TF* __restrict__ th,
                         TF* __restrict__ qt, TF* __restrict__ thvrefh,
                         TF* __restrict__ ph,  TF* __restrict__ exnh,
                         int istart, int jstart, int kstart,
                         int iend,   int jend,   int kend,
                         int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            const int kk  = i + j*jj;

            // Half level temperature and moisture content
            const TF thh = static_cast<TF>(0.5) * (th[ijk-kk] + th[ijk]); // Half level liq. water pot. temp.
            const TF qth = static_cast<TF>(0.5) * (qt[ijk-kk] + qt[ijk]); // Half level specific hum.

            Struct_sat_adjust<TF> ssa = sat_adjust_g(thh, qth, ph[k], exnh[k]);

            // Calculate tendency
            if (ssa.ql + ssa.qi > 0)
                bh[ijk] += buoyancy(exnh[k], thh, qth, ssa.ql, ssa.qi, thvrefh[k]);
            else
                bh[ijk] += buoyancy_no_ql(thh, qth, thvrefh[k]);
        }
    }

    template<typename TF> __global__
    void calc_buoyancy_bot_g(TF* __restrict__ b,      TF* __restrict__ bbot,
                             TF* __restrict__ th,     TF* __restrict__ thbot,
                             TF* __restrict__ qt,     TF* __restrict__ qtbot,
                             TF* __restrict__ thvref, TF* __restrict__ thvrefh,
                             int kstart, int icells, int jcells,
                             int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        const int j = blockIdx.y*blockDim.y + threadIdx.y;

        if (i < icells && j < jcells)
        {
            const int ij  = i + j*jj;
            const int ijk = i + j*jj + kstart*kk;

            bbot[ij ] = buoyancy_no_ql(thbot[ij], qtbot[ij], thvrefh[kstart]);
            b   [ijk] = buoyancy_no_ql(th[ijk],   qt[ijk],   thvref[kstart]);
        }
    }

    template<typename TF> __global__
    void calc_buoyancy_bot_g(TF* __restrict__ bbot,
                             TF* __restrict__ thbot,
                             TF* __restrict__ qtbot,
                             TF* __restrict__ thvrefh,
                             int icells, int jcells, int kstart)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        const int j = blockIdx.y*blockDim.y + threadIdx.y;

        if (i < icells && j < jcells)
        {
            const int ij  = i + j*icells;
            bbot[ij] = buoyancy_no_ql(thbot[ij], qtbot[ij], thvrefh[kstart]);
        }
    }

    template<typename TF> __global__
    void calc_buoyancy_flux_bot_g(TF* __restrict__ bfluxbot,
                                  TF* __restrict__ th, TF* __restrict__ thfluxbot,
                                  TF* __restrict__ qt, TF* __restrict__ qtfluxbot,
                                  TF* __restrict__ thvrefh,
                                  int kstart, int icells, int jcells,
                                  int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        const int j = blockIdx.y*blockDim.y + threadIdx.y;

        // Calculate the surface buoyancy flux using the first model level temperature and humidity
        // to ensure bitwise identical restarts.
        if (i < icells && j < jcells)
        {
            const int ij  = i + j*jj;
            const int ijk = i + j*jj + kstart*kk;

            bfluxbot[ij] = buoyancy_flux_no_ql(th[ijk], thfluxbot[ij], qt[ijk], qtfluxbot[ij], thvrefh[kstart]);
        }
    }

    template<typename TF> __global__
    void calc_N2_g(TF* __restrict__ N2, TF* __restrict__ th,
                   TF* __restrict__ thvref, const TF* __restrict__ dzi,
                   int istart, int jstart, int kstart,
                   int iend,   int jend,   int kend,
                   int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            N2[ijk] = grav<TF>/thvref[k]*static_cast<TF>(0.5)*(th[ijk+kk] - th[ijk-kk])*dzi[k];
        }
    }

    template<typename TF> __global__
    void calc_liquid_water_g(TF* __restrict__ ql, TF* __restrict__ th, TF* __restrict__ qt,
                             TF* __restrict__ exn, TF* __restrict__ p,
                             int istart, int jstart, int kstart,
                             int iend,   int jend,   int kend,
                             int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            ql[ijk] = sat_adjust_g(th[ijk], qt[ijk], p[k], exn[k]).ql;
        }
    }

    template<typename TF> __global__
    void calc_liquid_and_ice_g(
            TF* __restrict__ qlqi,
            TF* __restrict__ thl,
            TF* __restrict__ qt,
            TF* __restrict__ exn,
            TF* __restrict__ p,
            int istart, int jstart, int kstart,
            int iend,   int jend,   int kend,
            int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;

            Struct_sat_adjust<TF> ssa = sat_adjust_g(thl[ijk], qt[ijk], p[k], exn[k]);
            qlqi[ijk] = ssa.ql + ssa.qi;
        }
    }

    template<typename TF> __global__
    void calc_liquid_water_h_g(TF* __restrict__ qlh, TF* __restrict__ th, TF* __restrict__ qt,
                             TF* __restrict__ exnh, TF* __restrict__ ph,
                             int istart, int jstart, int kstart,
                             int iend,   int jend,   int kend,
                             int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            const int kk  = i + j*jj;

            const TF thh = static_cast<TF>(0.5) * (th[ijk-kk] + th[ijk]); // Half level liq. water pot. temp.
            const TF qth = static_cast<TF>(0.5) * (qt[ijk-kk] + qt[ijk]); // Half level specific hum.

            qlh[ijk] = sat_adjust_g(thh, qth, ph[k], exnh[k]).ql; // Half level liquid water content
        }
    }

    template<typename TF> __global__
    void calc_ice_g(TF* __restrict__ qi, TF* __restrict__ th, TF* __restrict__ qt,
                             TF* __restrict__ exn, TF* __restrict__ p,
                             int istart, int jstart, int kstart,
                             int iend,   int jend,   int kend,
                             int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            qi[ijk] = sat_adjust_g(th[ijk], qt[ijk], p[k], exn[k]).qi;
        }
    }

    template<typename TF> __global__
    void calc_condensate_g(TF* __restrict__ qc, TF* __restrict__ th, TF* __restrict__ qt,
                           TF* __restrict__ exn, TF* __restrict__ p,
                           int istart, int jstart, int kstart,
                           int iend,   int jend,   int kend,
                           int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            qc[ijk] = fmax(qt[ijk] - sat_adjust_g(th[ijk], qt[ijk], p[k], exn[k]).qs, TF(0.));
        }
    }

    template<typename TF> __global__
    void calc_thv_g(
            TF* const __restrict__ thv,
            const TF* const __restrict__ thl,
            const TF* const __restrict__ qt,
            const TF* const __restrict__ p,
            const TF* const __restrict__ exn,
            int istart, int jstart, int kstart,
            int iend,   int jend,   int kend,
            int icells, int ijcells)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*icells + k*ijcells;

            Struct_sat_adjust<TF> ssa = sat_adjust_g(thl[ijk], qt[ijk], p[k], exn[k]);
            thv[ijk] = virtual_temperature(exn[k], thl[ijk], qt[ijk], ssa.ql, ssa.qi);
        }
    }


    template<typename TF> __global__
    void calc_land_surface_fields(
        TF* const __restrict__ T_bot,
        TF* const __restrict__ T_a,
        TF* const __restrict__ vpd,
        TF* const __restrict__ qsat_bot,
        TF* const __restrict__ dqsatdT_bot,
        const TF* const __restrict__ thl_bot,
        const TF* const __restrict__ thl,
        const TF* const __restrict__ qt,
        const TF* const __restrict__ exner,
        const TF* const __restrict__ exnerh,
        const TF* const __restrict__ p,
        const TF* const __restrict__ ph,
        const int istart, const int iend,
        const int jstart, const int jend,
        const int kstart,
        const int icells, const int ijcells)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = kstart;

        if (i < iend && j < jend)
        {
            const int ij = i + j*icells;
            const int ijk = ij + k*ijcells;

            // Saturation adjustment for first model level
            Struct_sat_adjust<TF> ssa = sat_adjust_g(thl[ijk], qt[ijk], p[k], exner[k]);
            T_bot[ij] = exnerh[k] * thl_bot[ij];
            T_a[ij] = ssa.t;

            // Vapor pressure deficit first model level
            const TF es = esat(ssa.t);
            const TF e = qt[ijk]/ssa.qs * es;
            vpd[ij] = es-e;

            // qsat(T_bot) + dqsatdT(T_bot)
            qsat_bot[ij] = qsat(ph[k], T_bot[ij]);
            dqsatdT_bot[ij] = dqsatdT(ph[k], T_bot[ij]);
        }
    }


    template<typename TF> __global__
    void calc_radiation_fields_g(
            TF* restrict T, TF* restrict T_h, TF* restrict vmr_h2o,
            TF* restrict clwp, TF* restrict ciwp, TF* restrict T_sfc,
            const TF* restrict thl, const TF* restrict qt, const TF* restrict thl_bot,
            const TF* restrict p, const TF* restrict ph,
            const int istart, const int iend,
            const int jstart, const int jend,
            const int kstart, const int kend,
            const int igc, const int jgc, const int kgc,
            const int jj, const int kk,
            const int jj_nogc, const int kk_nogc)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        // This routine strips off the ghost cells, because of the data handling in radiation.
        using Finite_difference::O2::interp2;

        if (i < iend && j < jend && k < kend)
        {
            const TF ex = exner(p[k]);
            const TF dpg = (ph[k] - ph[k+1]) / Constants::grav<TF>;

            const int ijk = i + j*jj + k*kk;
            const int ijk_nogc = (i-igc) + (j-jgc)*jj_nogc + (k-kgc)*kk_nogc;
            const Struct_sat_adjust<TF> ssa = sat_adjust_g(thl[ijk], qt[ijk], p[k], ex);

            clwp[ijk_nogc] = ssa.ql * dpg;
            ciwp[ijk_nogc] = ssa.qi * dpg;

            const TF qv = qt[ijk] - ssa.ql - ssa.qi;
            vmr_h2o[ijk_nogc] = qv / (ep<TF> - ep<TF>*qv);

            T[ijk_nogc] = ssa.t;
        }

        // Exclude surface, is calculated below without saturation adjustment.
        if (i < iend && j < jend && k > kstart && k < kend+1)
        {
            const TF exnh = exner(ph[k]);
            const int ijk = i + j*jj + k*kk;

            const TF thlh = interp2(thl[ijk-kk], thl[ijk]);
            const TF qth  = interp2(qt [ijk-kk], qt [ijk]);

            const int ijk_nogc = (i-igc) + (j-jgc)*jj_nogc + (k-kgc)*kk_nogc;
            T_h[ijk_nogc] = sat_adjust_g(thlh, qth, ph[k], exnh).t;
        }

        if (i < iend && j < jend && k == kstart)
        {
            // Calculate surface temperature (assuming no liquid water)
            const TF exn_bot = exner(ph[kstart]);
            const int ij = i + j*jj;
            const int ij_nogc = (i-igc) + (j-jgc)*jj_nogc;
            const int ijk_nogc = (i-igc) + (j-jgc)*jj_nogc + (kstart-kgc)*kk_nogc;

            T_sfc[ij_nogc] = thl_bot[ij] * exn_bot;
            T_h[ijk_nogc] = T_sfc[ij_nogc];
        }
    }

    template<typename TF> __global__
    void calc_radiation_fields_g(
            TF* restrict T, TF* restrict T_h, TF* restrict vmr_h2o, TF* restrict rh,
            TF* restrict clwp, TF* restrict ciwp, TF* restrict T_sfc,
            const TF* restrict thl, const TF* restrict qt, const TF* restrict thl_bot,
            const TF* restrict p, const TF* restrict ph,
            const int istart, const int iend,
            const int jstart, const int jend,
            const int kstart, const int kend,
            const int igc, const int jgc, const int kgc,
            const int jj, const int kk,
            const int jj_nogc, const int kk_nogc)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        // This routine strips off the ghost cells, because of the data handling in radiation.
        using Finite_difference::O2::interp2;

        if (i < iend && j < jend && k < kend)
        {
            const TF ex = exner(p[k]);
            const TF dpg = (ph[k] - ph[k+1]) / Constants::grav<TF>;

            const int ijk = i + j*jj + k*kk;
            const int ijk_nogc = (i-igc) + (j-jgc)*jj_nogc + (k-kgc)*kk_nogc;
            const Struct_sat_adjust<TF> ssa = sat_adjust_g(thl[ijk], qt[ijk], p[k], ex);

            clwp[ijk_nogc] = ssa.ql * dpg;
            ciwp[ijk_nogc] = ssa.qi * dpg;

            const TF qv = qt[ijk] - ssa.ql - ssa.qi;
            vmr_h2o[ijk_nogc] = qv / (ep<TF> - ep<TF>*qv);
            rh[ijk_nogc] = min(qt[ijk] / ssa.qs, TF(1.));

            T[ijk_nogc] = ssa.t;
        }

        // Exclude surface, is calculated below without saturation adjustment.
        if (i < iend && j < jend && k > kstart && k < kend+1)
        {
            const TF exnh = exner(ph[k]);
            const int ijk = i + j*jj + k*kk;

            const TF thlh = interp2(thl[ijk-kk], thl[ijk]);
            const TF qth  = interp2(qt [ijk-kk], qt [ijk]);

            const int ijk_nogc = (i-igc) + (j-jgc)*jj_nogc + (k-kgc)*kk_nogc;
            T_h[ijk_nogc] = sat_adjust_g(thlh, qth, ph[k], exnh).t;
        }

        if (i < iend && j < jend && k == kstart)
        {
            // Calculate surface temperature (assuming no liquid water)
            const TF exn_bot = exner(ph[kstart]);
            const int ij = i + j*jj;
            const int ij_nogc = (i-igc) + (j-jgc)*jj_nogc;
            const int ijk_nogc = (i-igc) + (j-jgc)*jj_nogc + (kstart-kgc)*kk_nogc;

            T_sfc[ij_nogc] = thl_bot[ij] * exn_bot;
            T_h[ijk_nogc] = T_sfc[ij_nogc];
        }
    }

    template<typename TF> __global__
    void calc_radiation_columns_g(
            TF* const restrict T, TF* const restrict T_h, TF* const restrict vmr_h2o, TF* const restrict rh,
            TF* const restrict clwp, TF* const restrict ciwp, TF* const restrict T_sfc,
            const TF* const restrict thl, const TF* const restrict qt, const TF* const restrict thl_bot,
            const TF* const restrict p, const TF* const restrict ph,
            const int* const col_i, const int* const col_j,
            const int n_cols,
            const int kgc, const int kstart, const int kend,
            const int icells, const int ijcells)
    {
        // This routine strips off the ghost cells, because of the data handling in radiation.
        using Finite_difference::O2::interp2;

        const int n = blockIdx.x*blockDim.x + threadIdx.x;
        const int k = blockIdx.y*blockDim.y + threadIdx.y + kstart;

        if (n < n_cols)
        {
            const int i = col_i[n];
            const int j = col_j[n];


            const int ij = i + j*icells;
            const int ijk = i + j*icells + k*ijcells;

            const int ij_out = n;
            const int ijk_out = n + (k-kgc) * n_cols;

            if (k < kend)
            {
                const Struct_sat_adjust<TF> ssa = sat_adjust_g(thl[ijk], qt[ijk], p[k], exner(p[k]));

                const TF dpg = (ph[k] - ph[k+1]) / Constants::grav<TF>;
                clwp[ijk_out] = ssa.ql * dpg;
                ciwp[ijk_out] = ssa.qi * dpg;

                const TF qv = qt[ijk] - ssa.ql - ssa.qi;
                vmr_h2o[ijk_out] = qv / (ep<TF> - ep<TF>*qv);
                rh[ijk_out] = min(qt[ijk] / ssa.qs, TF(1.));
                T[ijk_out] = ssa.t;
            }

            if (k > kstart && k < kend+1)
            {
                const TF thlh = interp2(thl[ijk-ijcells], thl[ijk]);
                const TF qth  = interp2(qt [ijk-ijcells], qt [ijk]);

                T_h[ijk_out] = sat_adjust_g(thlh, qth, ph[k], exner(ph[k])).t;
            }

            if (k == kstart)
            {
                T_sfc[ij_out] = thl_bot[ij] * exner(ph[kstart]);
                T_h[ijk_out] = T_sfc[ij_out];
            }
        }
    }


    template<typename TF> __global__
    void calc_path_g(
        TF* const restrict path,
        const TF* const restrict fld,
        const TF* const restrict rhoref,
        const TF* const restrict dz,
        const int istart, const int iend,
        const int jstart, const int jend,
        const int kstart, const int kend,
        const int icells, const int ijcells)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;

        if (i < iend && j < jend)
        {
            const int ij = i + j*icells;
            path[ij] = TF(0);

            // Bit of a cheap solution, but this function is only called for statistics..
            for (int k=kstart; k<kend; ++k)
            {
                const int ijk = ij + k*ijcells;
                path[ij] += rhoref[k] * fld[ijk] * dz[k];
            }
        }
    }

    /*
    // BvS: no longer used, base state is calculated at the host
    // CvH: This unused code does not take into account ice
    template <typename TF> __global__
    void calc_base_state_g(TF* __restrict__ pref,     TF* __restrict__ prefh,
                           TF* __restrict__ rho,      TF* __restrict__ rhoh,
                           TF* __restrict__ thv,      TF* __restrict__ thvh,
                           TF* __restrict__ ex,       TF* __restrict__ exh,
                           TF* __restrict__ thlmean,  TF* __restrict__ qtmean,
                           TF* __restrict__ z,        TF* __restrict__ dz,
                           TF* __restrict__ dzh,
                           TF pbot, int kstart, int kend)
    {
        TF ql, si, qti, qli;
        TF rdcp = Rd<TF>/cp<TF>;

        const TF ssurf  = interp2(thlmean[kstart-1], thlmean[kstart]);
        const TF qtsurf = interp2(qtmean[kstart-1],  qtmean[kstart]);

        // Calculate surface (half=kstart) values
        exh[kstart]   = exner(pbot);
        ql            = sat_adjust_g(ssurf,qtsurf,pbot,exh[kstart]).ql;
        thvh[kstart]  = (ssurf + Lv<TF>*ql/(cp<TF>*exh[kstart])) * (1. - (1. - Rv<TF>/Rd<TF>)*qtsurf - Rv<TF>/Rd<TF>*ql);
        prefh[kstart] = pbot;
        rhoh[kstart]  = pbot / (Rd<TF> * exh[kstart] * thvh[kstart]);

        // First full grid level pressure
        pref[kstart] = pow((pow(pbot,rdcp) - grav<TF> * pow(p0<TF>,rdcp) * z[kstart] / (cp<TF> * thvh[kstart])),(1./rdcp));

        for (int k=kstart+1; k<kend+1; k++)
        {
            // 1. Calculate values at full level below zh[k]
            ex[k-1]  = exner(pref[k-1]);
            ql       = sat_adjust_g(thlmean[k-1],qtmean[k-1],pref[k-1],ex[k-1]).ql;
            thv[k-1] = (thlmean[k-1] + Lv<TF>*ql/(cp<TF>*ex[k-1])) * (1. - (1. - Rv<TF>/Rd<TF>)*qtmean[k-1] - Rv<TF>/Rd<TF>*ql);
            rho[k-1] = pref[k-1] / (Rd<TF> * ex[k-1] * thv[k-1]);

            // 2. Calculate half level pressure at zh[k] using values at z[k-1]
            prefh[k] = pow((pow(prefh[k-1],rdcp) - grav<TF> * pow(p0<TF>,rdcp) * dz[k-1] / (cp<TF> * thv[k-1])),(1./rdcp));

            // 3. Interpolate conserved variables to zh[k] and calculate virtual temp and ql
            si     = interp2(thlmean[k-1],thlmean[k]);
            qti    = interp2(qtmean[k-1],qtmean[k]);

            exh[k]   = exner(prefh[k]);
            qli      = sat_adjust_g(si,qti,prefh[k],exh[k]).ql;
            thvh[k]  = (si + Lv<TF>*qli/(cp<TF>*exh[k])) * (1. - (1. - Rv<TF>/Rd<TF>)*qti - Rv<TF>/Rd<TF>*qli);
            rhoh[k]  = prefh[k] / (Rd<TF> * exh[k] * thvh[k]);

            // 4. Calculate full level pressure at z[k]
            pref[k]  = pow((pow(pref[k-1],rdcp) - grav<TF> * pow(p0<TF>,rdcp) * dzh[k] / (cp<TF> * thvh[k])),(1./rdcp));
        }

        // Fill bottom and top full level ghost cells
        pref[kstart-1] = static_cast<TF>(2.)*prefh[kstart] - pref[kstart];
        pref[kend]     = static_cast<TF>(2.)*prefh[kend]   - pref[kend-1];
    }


    // BvS: no longer used, base state is calculated at the host
    template <typename TF> __global__
    void calc_hydrostatic_pressure_g(TF* __restrict__ pref,     TF* __restrict__ prefh,
                                     TF* __restrict__ ex,       TF* __restrict__ exh,
                                     TF* __restrict__ thlmean,  TF* __restrict__ qtmean,
                                     const TF* const __restrict__ z,        const TF* const __restrict__ dz,
                                     const TF* const __restrict__ dzh,
                                     const TF pbot, int kstart, int kend)
    {
        TF ql, si, qti, qli, thvh, thv;
        TF rdcp = Rd<TF>/cp<TF>;

        const TF ssurf  = interp2(thlmean[kstart-1], thlmean[kstart]);
        const TF qtsurf = interp2(qtmean[kstart-1],  qtmean[kstart]);

        // Calculate surface (half=kstart) values
        ql            = sat_adjust_g(ssurf,qtsurf,pbot,exh[kstart]).ql;
        thvh          = (ssurf + Lv<TF>*ql/(cp<TF>*exh[kstart])) * (1. - (1. - Rv<TF>/Rd<TF>)*qtsurf - Rv<TF>/Rd<TF>*ql);
        prefh[kstart] = pbot;

        // First full grid level pressure
        pref[kstart] = pow((pow(pbot,rdcp) - grav<TF> * pow(p0<TF>,rdcp) * z[kstart] / (cp<TF> * thvh)),(1./rdcp));

        for (int k=kstart+1; k<kend+1; k++)
        {
            // 1. Calculate values at full level below zh[k]
            ex[k-1]  = exner(pref[k-1]);
            ql       = sat_adjust_g(thlmean[k-1],qtmean[k-1],pref[k-1],ex[k-1]).ql;
            thv      = (thlmean[k-1] + Lv<TF>*ql/(cp<TF>*ex[k-1])) * (1. - (1. - Rv<TF>/Rd<TF>)*qtmean[k-1] - Rv<TF>/Rd<TF>*ql);

            // 2. Calculate half level pressure at zh[k] using values at z[k-1]
            prefh[k] = pow((pow(prefh[k-1],rdcp) - grav<TF> * pow(p0<TF>,rdcp) * dz[k-1] / (cp<TF> * thv)),(1./rdcp));

            // 3. Interpolate conserved variables to zh[k] and calculate virtual temp and ql
            si     = interp2(thlmean[k-1],thlmean[k]);
            qti    = interp2(qtmean[k-1],qtmean[k]);

            exh[k]   = exner(prefh[k]);
            qli      = sat_adjust_g(si,qti,prefh[k],exh[k]).ql;
            thvh     = (si + Lv<TF>*qli/(cp<TF>*exh[k])) * (1. - (1. - Rv<TF>/Rd<TF>)*qti - Rv<TF>/Rd<TF>*qli);

            // 4. Calculate full level pressure at z[k]
            pref[k]  = pow((pow(pref[k-1],rdcp) - grav<TF> * pow(p0<TF>,rdcp) * dzh[k] / (cp<TF> * thvh)),(1./rdcp));
        }

        // Fill bottom and top full level ghost cells
        pref[kstart-1] = static_cast<TF>(2.)*prefh[kstart] - pref[kstart];
        pref[kend]     = static_cast<TF>(2.)*prefh[kend]   - pref[kend-1];
    }
    */
} // end name    space

template<typename TF>
void Thermo_moist<TF>::prepare_device()
{
    auto& gd = grid.get_grid_data();
    const int nmemsize = gd.kcells*sizeof(TF);

    // Allocate fields for Boussinesq and anelastic solver
    cuda_safe_call(hipMalloc(&bs.thvref_g,  nmemsize));
    cuda_safe_call(hipMalloc(&bs.thvrefh_g, nmemsize));
    cuda_safe_call(hipMalloc(&bs.pref_g,    nmemsize));
    cuda_safe_call(hipMalloc(&bs.prefh_g,   nmemsize));
    cuda_safe_call(hipMalloc(&bs.exnref_g,  nmemsize));
    cuda_safe_call(hipMalloc(&bs.exnrefh_g, nmemsize));
    cuda_safe_call(hipMalloc(&bs.rhoref_g,  nmemsize));
    cuda_safe_call(hipMalloc(&bs.rhorefh_g, nmemsize));

    // Copy fields to device
    cuda_safe_call(hipMemcpy(bs.thvref_g,  bs.thvref.data(),  nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(bs.thvrefh_g, bs.thvrefh.data(), nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(bs.pref_g,    bs.pref.data(),    nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(bs.prefh_g,   bs.prefh.data(),   nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(bs.exnref_g,  bs.exnref.data(),  nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(bs.exnrefh_g, bs.exnrefh.data(), nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(bs.rhoref_g,  bs.rhoref.data(),  nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(bs.rhorefh_g, bs.rhorefh.data(), nmemsize, hipMemcpyHostToDevice));
}

template<typename TF>
void Thermo_moist<TF>::clear_device()
{
    cuda_safe_call(hipFree(bs.thvref_g ));
    cuda_safe_call(hipFree(bs.thvrefh_g));
    cuda_safe_call(hipFree(bs.pref_g   ));
    cuda_safe_call(hipFree(bs.prefh_g  ));
    cuda_safe_call(hipFree(bs.exnref_g ));
    cuda_safe_call(hipFree(bs.exnrefh_g));
    cuda_safe_call(hipFree(bs.rhoref_g ));
    cuda_safe_call(hipFree(bs.rhorefh_g));
    tdep_pbot->clear_device();
}

template<typename TF>
void Thermo_moist<TF>::forward_device()
{
    // Copy fields to device
    auto& gd = grid.get_grid_data();
    const int nmemsize = gd.kcells*sizeof(TF);
    cuda_safe_call(hipMemcpy(bs.pref_g,    bs.pref.data(),    nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(bs.prefh_g,   bs.prefh.data(),   nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(bs.exnref_g,  bs.exnref.data(),  nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(bs.exnrefh_g, bs.exnrefh.data(), nmemsize, hipMemcpyHostToDevice));
}

template<typename TF>
void Thermo_moist<TF>::backward_device()
{
    auto& gd = grid.get_grid_data();
    const int nmemsize = gd.kcells*sizeof(TF);
    hipMemcpy(bs.pref_g,    bs.pref.data(),    nmemsize, hipMemcpyHostToDevice);
    hipMemcpy(bs.prefh_g,   bs.prefh.data(),   nmemsize, hipMemcpyHostToDevice);
    hipMemcpy(bs.exnref_g,  bs.exnref.data(),  nmemsize, hipMemcpyHostToDevice);
    hipMemcpy(bs.exnrefh_g, bs.exnrefh.data(), nmemsize, hipMemcpyHostToDevice);

    bs_stats = bs;
}

#ifdef USECUDA
template<typename TF>
void Thermo_moist<TF>::exec(const double dt, Stats<TF>& stats)
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, gd.kmax);
    dim3 blockGPU(blocki, blockj, 1);

    // Re-calculate hydrostatic pressure and exner
    if (bs.swupdatebasestate)
    {
        //calc_hydrostatic_pressure<TF><<<1, 1>>>(bs.pref_g, bs.prefh_g, bs.exnref_g, bs.exnrefh_g,
        //                                        fields.sp.at("thl")->fld_mean_g, fields.sp.at("qt")->fld_mean_g,
        //                                        gd.z_g, gd.dz_g, gd.dzh_g, bs.pbot, gd.kstart, gd.kend);
        //cuda_check_error();

        // BvS: Calculating hydrostatic pressure on GPU is extremely slow. As temporary solution, copy back mean profiles to host,
        //      calculate pressure there and copy back the required profiles.
        hipMemcpy(fields.sp.at("thl")->fld_mean.data(), fields.sp.at("thl")->fld_mean_g, gd.kcells*sizeof(TF), hipMemcpyDeviceToHost);
        hipMemcpy(fields.sp.at("qt")->fld_mean.data(),  fields.sp.at("qt")->fld_mean_g,  gd.kcells*sizeof(TF), hipMemcpyDeviceToHost);

        auto tmp = fields.get_tmp();

        calc_base_state(
                bs.pref.data(), bs.prefh.data(),
                bs.rhoref.data(), bs.rhorefh.data(),
                bs.thvref.data(), bs.thvrefh.data(),
                bs.exnref.data(), bs.exnrefh.data(),
                fields.sp.at("thl")->fld_mean.data(),
                fields.sp.at("qt")->fld_mean.data(),
                bs.pbot, gd.kstart, gd.kend,
                gd.z.data(), gd.dz.data(), gd.dzh.data());

        fields.release_tmp(tmp);

        hipMemcpy(bs.pref_g,    bs.pref.data(),    gd.kcells*sizeof(TF), hipMemcpyHostToDevice);
        hipMemcpy(bs.prefh_g,   bs.prefh.data(),   gd.kcells*sizeof(TF), hipMemcpyHostToDevice);

        hipMemcpy(bs.exnref_g,  bs.exnref.data(),  gd.kcells*sizeof(TF), hipMemcpyHostToDevice);
        hipMemcpy(bs.exnrefh_g, bs.exnrefh.data(), gd.kcells*sizeof(TF), hipMemcpyHostToDevice);

        hipMemcpy(bs.thvref_g,  bs.thvref.data(),  gd.kcells*sizeof(TF), hipMemcpyHostToDevice);
        hipMemcpy(bs.thvrefh_g, bs.thvrefh.data(), gd.kcells*sizeof(TF), hipMemcpyHostToDevice);

        hipMemcpy(bs.rhoref_g,  bs.rhoref.data(),  gd.kcells*sizeof(TF), hipMemcpyHostToDevice);
        hipMemcpy(bs.rhorefh_g, bs.rhorefh.data(), gd.kcells*sizeof(TF), hipMemcpyHostToDevice);
    }

    calc_buoyancy_tend_2nd_g<TF><<<gridGPU, blockGPU>>>(
            fields.mt.at("w")->fld_g, fields.sp.at("thl")->fld_g,
            fields.sp.at("qt")->fld_g, bs.thvrefh_g, bs.exnrefh_g, bs.prefh_g,
            gd.istart, gd.jstart, gd.kstart+1,
            gd.iend,   gd.jend,   gd.kend,
            gd.icells, gd.ijcells);
    cuda_check_error();

    hipDeviceSynchronize();
    stats.calc_tend(*fields.mt.at("w"), tend_name);

}

template<typename TF>
void Thermo_moist<TF>::get_thermo_field_g(
        Field3d<TF>& fld, const std::string& name, const bool cyclic )
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, gd.kcells);
    dim3 blockGPU(blocki, blockj, 1);

    dim3 gridGPU2 (gridi, gridj, gd.kmax);
    dim3 blockGPU2(blocki, blockj, 1);

    // BvS: getthermofield() is called from subgrid-model, before thermo(), so re-calculate the hydrostatic pressure
    if (bs.swupdatebasestate && (name != "N2"))
    {
        //calc_hydrostatic_pressure_g<TF><<<1, 1>>>(bs.pref_g, bs.prefh_g, bs.exnref_g, bs.exnrefh_g,
        //                                          fields.sp.at("thl")->fld_mean_g, fields.sp.at("qt")->fld_mean_g,
        //                                          gd.z_g, gd.dz_g, gd.dzh_g, bs.pbot, gd.kstart, gd.kend);
        //cuda_check_error();

        // BvS: Calculating hydrostatic pressure on GPU is extremely slow. As temporary solution, copy back mean profiles to host,
        //      calculate pressure there and copy back the required profiles.
        hipMemcpy(fields.sp.at("thl")->fld_mean.data(), fields.sp.at("thl")->fld_mean_g, gd.kcells*sizeof(TF), hipMemcpyDeviceToHost);
        hipMemcpy(fields.sp.at("qt")->fld_mean.data(),  fields.sp.at("qt")->fld_mean_g,  gd.kcells*sizeof(TF), hipMemcpyDeviceToHost);

        auto tmp = fields.get_tmp();

        calc_base_state(
                bs.pref.data(), bs.prefh.data(),
                &tmp->fld[0*gd.kcells], &tmp->fld[1*gd.kcells],
                &tmp->fld[2*gd.kcells], &tmp->fld[3*gd.kcells],
                bs.exnref.data(), bs.exnrefh.data(),
                fields.sp.at("thl")->fld_mean.data(),
                fields.sp.at("qt")->fld_mean.data(),
                bs.pbot, gd.kstart, gd.kend,
                gd.z.data(), gd.dz.data(), gd.dzh.data());

        fields.release_tmp(tmp);

        // Only full level pressure and bs.exner needed for calculating buoyancy of ql
        hipMemcpy(bs.pref_g,   bs.pref.data(),     gd.kcells*sizeof(TF), hipMemcpyHostToDevice);
        hipMemcpy(bs.prefh_g,  bs.prefh.data(),    gd.kcells*sizeof(TF), hipMemcpyHostToDevice);
        hipMemcpy(bs.exnref_g, bs.exnref.data(),   gd.kcells*sizeof(TF), hipMemcpyHostToDevice);
        hipMemcpy(bs.exnrefh_g, bs.exnrefh.data(), gd.kcells*sizeof(TF), hipMemcpyHostToDevice);
    }

    if (name == "b")
    {
        calc_buoyancy_g<TF><<<gridGPU, blockGPU>>>(
            fld.fld_g, fields.sp.at("thl")->fld_g, fields.sp.at("qt")->fld_g,
            bs.thvref_g, bs.pref_g, bs.exnref_g,
            gd.istart,  gd.jstart, gd.kstart,
            gd.iend, gd.jend, gd.kcells,
            gd.icells, gd.ijcells);
        cuda_check_error();
    }
    else if (name == "b_h")
    {
        calc_buoyancy_g<TF><<<gridGPU, blockGPU>>>(
            fld.fld_g, fields.sp.at("thl")->fld_g, fields.sp.at("qt")->fld_g,
            bs.thvrefh_g, bs.prefh_g, bs.exnrefh_g,
            gd.istart,  gd.jstart, gd.kstart,
            gd.iend, gd.jend, gd.kcells,
            gd.icells, gd.ijcells);
        cuda_check_error();
    }
    else if (name == "ql")
    {
        calc_liquid_water_g<TF><<<gridGPU2, blockGPU2>>>(
            fld.fld_g, fields.sp.at("thl")->fld_g, fields.sp.at("qt")->fld_g,
            bs.exnref_g, bs.pref_g,
            gd.istart,  gd.jstart,  gd.kstart,
            gd.iend,    gd.jend,    gd.kend,
            gd.icells, gd.ijcells);
        cuda_check_error();
    }
    else if (name == "ql_h")
    {
        calc_liquid_water_h_g<TF><<<gridGPU2, blockGPU2>>>(
            fld.fld_g, fields.sp.at("thl")->fld_g, fields.sp.at("qt")->fld_g,
            bs.exnrefh_g, bs.prefh_g,
            gd.istart,  gd.jstart,  gd.kstart,
            gd.iend,    gd.jend,    gd.kend,
            gd.icells, gd.ijcells);
        cuda_check_error();
    }
    else if (name == "qi")
    {
        calc_ice_g<TF><<<gridGPU2, blockGPU2>>>(
            fld.fld_g, fields.sp.at("thl")->fld_g, fields.sp.at("qt")->fld_g,
            bs.exnref_g, bs.pref_g,
            gd.istart,  gd.jstart,  gd.kstart,
            gd.iend,    gd.jend,    gd.kend,
            gd.icells, gd.ijcells);
        cuda_check_error();
    }
    else if (name == "qlqi")
    {
        calc_liquid_and_ice_g<TF><<<gridGPU2, blockGPU2>>>(
            fld.fld_g,
            fields.sp.at("thl")->fld_g,
            fields.sp.at("qt")->fld_g,
            bs.exnref_g, bs.pref_g,
            gd.istart,  gd.jstart,  gd.kstart,
            gd.iend,    gd.jend,    gd.kend,
            gd.icells, gd.ijcells);
        cuda_check_error();
    }
    else if (name == "qlqi")
    {
        calc_condensate_g<TF><<<gridGPU2, blockGPU2>>>(
            fld.fld_g, fields.sp.at("thl")->fld_g, fields.sp.at("qt")->fld_g,
            bs.exnrefh_g, bs.prefh_g,
            gd.istart,  gd.jstart,  gd.kstart,
            gd.iend,    gd.jend,    gd.kend,
            gd.icells, gd.ijcells);
        cuda_check_error();
    }
    else if (name == "N2")
    {
        calc_N2_g<TF><<<gridGPU2, blockGPU2>>>(
            fld.fld_g, fields.sp.at("thl")->fld_g,
            bs.thvref_g, gd.dzi_g,
            gd.istart,  gd.jstart, gd.kstart,
            gd.iend,    gd.jend,   gd.kend,
            gd.icells, gd.ijcells);
        cuda_check_error();
    }
    else if (name == "thv")
    {
        calc_thv_g<TF><<<gridGPU2, blockGPU2>>>(
            fld.fld_g,
            fields.sp.at("thl")->fld_g,
            fields.sp.at("qt")->fld_g,
            bs.pref_g,
            bs.exnref_g,
            gd.istart, gd.jstart, gd.kstart,
            gd.iend,   gd.jend,   gd.kend,
            gd.icells, gd.ijcells);
        cuda_check_error();
    }
    else
    {
        std::string msg = "get_thermo_field_g \"" + name + "\" not supported";
        throw std::runtime_error(msg);
    }

    if (cyclic)
        boundary_cyclic.exec_g(fld.fld_g);
}

template<typename TF>
TF* Thermo_moist<TF>::get_basestate_fld_g(std::string name)
{
    // BvS TO-DO: change std::string to enum
    if (name == "pref")
        return bs.pref_g;
    else if (name == "prefh")
        return bs.prefh_g;
    else if (name == "exner")
        return bs.exnref_g;
    else if (name == "exnerh")
        return bs.exnrefh_g;
    else if (name == "rhoh")
        return bs.rhorefh_g;
    else if (name == "thvh")
        return bs.thvrefh_g;
    else
    {
        std::string error_message = "Can not get basestate field \"" + name + "\" from thermo_moist";
        throw std::runtime_error(error_message);
    }
}

template<typename TF>
void Thermo_moist<TF>::get_buoyancy_fluxbot_g(Field3d<TF>& bfield)
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.icells/blocki + (gd.icells%blocki > 0);
    const int gridj  = gd.jcells/blockj + (gd.jcells%blockj > 0);

    dim3 gridGPU (gridi, gridj, 1);
    dim3 blockGPU(blocki, blockj, 1);

    calc_buoyancy_flux_bot_g<TF><<<gridGPU, blockGPU>>>(
        bfield.flux_bot_g,
        fields.sp.at("thl")->fld_g, fields.sp.at("thl")->flux_bot_g,
        fields.sp.at("qt")->fld_g, fields.sp.at("qt")->flux_bot_g,
        bs.thvrefh_g, gd.kstart, gd.icells, gd.jcells,
        gd.icells, gd.ijcells);
    cuda_check_error();
}

template<typename TF>
void Thermo_moist<TF>::get_buoyancy_surf_g(Field3d<TF>& bfield)
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.icells/blocki + (gd.icells%blocki > 0);
    const int gridj  = gd.jcells/blockj + (gd.jcells%blockj > 0);

    dim3 gridGPU (gridi, gridj, 1);
    dim3 blockGPU(blocki, blockj, 1);

    calc_buoyancy_bot_g<TF><<<gridGPU, blockGPU>>>(
        bfield.fld_g, bfield.fld_bot_g,
        fields.sp.at("thl")->fld_g, fields.sp.at("thl")->fld_bot_g,
        fields.sp.at("qt")->fld_g, fields.sp.at("qt")->fld_bot_g,
        bs.thvref_g, bs.thvrefh_g, gd.kstart, gd.icells, gd.jcells,
        gd.icells, gd.ijcells);
    cuda_check_error();

    calc_buoyancy_flux_bot_g<TF><<<gridGPU, blockGPU>>>(
        bfield.flux_bot_g,
        fields.sp.at("thl")->fld_g, fields.sp.at("thl")->flux_bot_g,
        fields.sp.at("qt")->fld_g, fields.sp.at("qt")->flux_bot_g,
        bs.thvrefh_g, gd.kstart, gd.icells, gd.jcells,
        gd.icells, gd.ijcells);
    cuda_check_error();
}

template<typename TF>
void Thermo_moist<TF>::get_buoyancy_surf_g(
    TF* const restrict b_bot,
    TF* const restrict thl_bot,
    TF* const restrict qt_bot)
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.icells/blocki + (gd.icells%blocki > 0);
    const int gridj  = gd.jcells/blockj + (gd.jcells%blockj > 0);

    dim3 gridGPU (gridi, gridj);
    dim3 blockGPU(blocki, blockj);

    calc_buoyancy_bot_g<<<gridGPU, blockGPU>>>(
        b_bot, thl_bot, qt_bot,
        bs.thvrefh_g,
        gd.icells, gd.jcells, gd.kstart);
    cuda_check_error();
}

template<typename TF>
void Thermo_moist<TF>::exec_column(Column<TF>& column)
{
    auto& gd = grid.get_grid_data();
    auto output = fields.get_tmp_g();
    const TF no_offset = 0.;

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);
    dim3 gridGPU (gridi, gridj);
    dim3 blockGPU(blocki, blockj);

    get_thermo_field_g(*output, "thv", false);
    column.calc_column("thv", output->fld_g, no_offset);

    // Liquid water
    get_thermo_field_g(*output, "ql", false);

    calc_path_g<TF><<<gridGPU, blockGPU>>>(
        output->fld_bot_g,
        output->fld_g,
        bs.rhoref_g,
        gd.dz_g,
        gd.istart, gd.iend,
        gd.jstart, gd.jend,
        gd.kstart, gd.kend,
        gd.icells, gd.ijcells);

    column.calc_column("ql", output->fld_g, no_offset);
    column.calc_time_series("ql_path", output->fld_bot_g, no_offset);

    // Ice ice baby
    get_thermo_field_g(*output, "qi", false);

    calc_path_g<TF><<<gridGPU, blockGPU>>>(
        output->fld_bot_g,
        output->fld_g,
        bs.rhoref_g,
        gd.dz_g,
        gd.istart, gd.iend,
        gd.jstart, gd.jend,
        gd.kstart, gd.kend,
        gd.icells, gd.ijcells);

    column.calc_column("qi", output->fld_g, no_offset);
    column.calc_time_series("qi_path", output->fld_bot_g, no_offset);

    // Time series
    column.calc_time_series("thl_bot", fields.ap.at("thl")->fld_bot_g, no_offset);
    column.calc_time_series("qt_bot",  fields.ap.at("qt")->fld_bot_g,  no_offset);

    fields.release_tmp_g(output);
}
#endif

#ifdef USECUDA
template<typename TF>
void Thermo_moist<TF>::get_radiation_fields_g(
        Field3d<TF>& T, Field3d<TF>& T_h, Field3d<TF>& qv, Field3d<TF>& clwp, Field3d<TF>& ciwp) const
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU(gridi, gridj, gd.ktot+1);
    dim3 blockGPU(blocki, blockj, 1);

    calc_radiation_fields_g<TF><<<gridGPU, blockGPU>>>(
            T.fld_g, T_h.fld_g, qv.fld_g,
            clwp.fld_g, ciwp.fld_g, T_h.fld_bot_g,
            fields.sp.at("thl")->fld_g, fields.sp.at("qt")->fld_g,
            fields.sp.at("thl")->fld_bot_g,
            bs.pref_g, bs.prefh_g,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            gd.kstart, gd.kend,
            gd.igc, gd.jgc, gd.kgc,
            gd.icells, gd.ijcells,
            gd.imax, gd.imax*gd.jmax);
    cuda_check_error();
}
#endif

#ifdef USECUDA
template<typename TF>
void Thermo_moist<TF>::get_radiation_fields_g(
        Field3d<TF>& T, Field3d<TF>& T_h, Field3d<TF>& qv, Field3d<TF>& rh, Field3d<TF>& clwp, Field3d<TF>& ciwp) const
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU(gridi, gridj, gd.ktot+1);
    dim3 blockGPU(blocki, blockj, 1);

    calc_radiation_fields_g<TF><<<gridGPU, blockGPU>>>(
            T.fld_g, T_h.fld_g, qv.fld_g, rh.fld_g,
            clwp.fld_g, ciwp.fld_g, T_h.fld_bot_g,
            fields.sp.at("thl")->fld_g, fields.sp.at("qt")->fld_g,
            fields.sp.at("thl")->fld_bot_g,
            bs.pref_g, bs.prefh_g,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            gd.kstart, gd.kend,
            gd.igc, gd.jgc, gd.kgc,
            gd.icells, gd.ijcells,
            gd.imax, gd.imax*gd.jmax);
    cuda_check_error();
}

template<typename TF>
void Thermo_moist<TF>::get_radiation_columns_g(
    Field3d<TF>& tmp,
    const int* const __restrict__ col_i_g,
    const int* const __restrict__ col_j_g,
    const int n_cols) const
{
    auto& gd = grid.get_grid_data();

    // Get slices from tmp field.
    const int n_full = gd.ktot;
    const int n_half = gd.ktot+1;

    int offset = 0;
    TF* t_lay_a = &tmp.fld_g[offset]; offset += n_cols * n_full;
    TF* t_lev_a = &tmp.fld_g[offset]; offset += n_cols * n_half;
    TF* t_sfc_a = &tmp.fld_g[offset]; offset += n_cols;
    TF* h2o_a   = &tmp.fld_g[offset]; offset += n_cols * n_full;
    TF* rh_a    = &tmp.fld_g[offset]; offset += n_cols * n_full;
    TF* clwp_a  = &tmp.fld_g[offset]; offset += n_cols * n_full;
    TF* ciwp_a  = &tmp.fld_g[offset];

    const int blocki = 4;
    const int blockj = 32;
    const int gridi = n_cols/blocki + (n_cols%blocki > 0);
    const int gridj = n_half/blockj + (n_half%blockj > 0);

    dim3 gridGPU(gridi, gridj);
    dim3 blockGPU(blocki, blockj);

    calc_radiation_columns_g<TF><<<gridGPU, blockGPU>>>(
            t_lay_a, t_lev_a, h2o_a, rh_a, clwp_a, ciwp_a, t_sfc_a,
            fields.sp.at("thl")->fld_g,
            fields.sp.at("qt")->fld_g,
            fields.sp.at("thl")->fld_bot_g,
            bs.pref_g,
            bs.prefh_g,
            col_i_g,
            col_j_g,
            n_cols,
            gd.kgc, gd.kstart, gd.kend,
            gd.icells, gd.ijcells);
    cuda_check_error();
}

template<typename TF>
void Thermo_moist<TF>::get_land_surface_fields_g(
    TF* const __restrict__ T_bot,
    TF* const __restrict__ T_a,
    TF* const __restrict__ vpd,
    TF* const __restrict__ qsat_bot,
    TF* const __restrict__ dqsatdT_bot)
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, 1);
    dim3 blockGPU(blocki, blockj, 1);

    calc_land_surface_fields<TF><<<gridGPU, blockGPU>>>(
        T_bot, T_a, vpd, qsat_bot, dqsatdT_bot,
        fields.sp.at("thl")->fld_bot_g,
        fields.sp.at("thl")->fld_g,
        fields.sp.at("qt")->fld_g,
        bs.exnref_g, bs.exnrefh_g,
        bs.pref_g, bs.prefh_g,
        gd.istart, gd.iend,
        gd.jstart, gd.jend,
        gd.kstart,
        gd.icells, gd.ijcells);
    cuda_check_error();
}
#endif


#ifdef FLOAT_SINGLE
template class Thermo_moist<float>;
#else
template class Thermo_moist<double>;
#endif
