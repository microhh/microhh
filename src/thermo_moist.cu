#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2020 Chiel van Heerwaarden
 * Copyright (c) 2011-2020 Thijs Heus
 * Copyright (c) 2014-2020 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <cstdio>
#include "grid.h"
#include "fields.h"
#include "thermo_moist.h"
#include "defines.h"
#include "constants.h"
#include "finite_difference.h"
#include "master.h"
#include "tools.h"
#include "column.h"
#include "stats.h"
#include "thermo_moist_functions.h"
#include <iostream>

namespace
{
    using namespace Constants;
    using namespace Finite_difference::O2;
    using namespace Thermo_moist_functions;

    template<typename TF>
    inline __device__ Struct_sat_adjust<TF> sat_adjust_g(
            const TF thl, const TF qt, const TF p, const TF exn)
    {
        using Fast_math::pow2;

        int niter = 0;
        const int nitermax = 10;

        TF tnr_old = TF(1.e9);

        const TF tl = thl * exn;
        TF qs = qsat_liq(p, tl);

        Struct_sat_adjust<TF> ans =
        {
            TF(0.), // ql
            TF(0.), // qi
            tl, // t
            qs, // qs
        };

        // Calculate if q-qs(Tl) <= 0. If so, return 0. Else continue with saturation adjustment.
        if (qt-ans.qs <= TF(0.))
            return ans;

        /* Saturation adjustment solver.
         * Root finding function is f(T) = T - tnr - Lv/cp*qt + alpha_w * Lv/cp*qs(T) + alpha_i*Ls/cp*qs(T)
         * dq_sat/dT derivatives can be rewritten using Claussius-Clapeyron (desat/dT = L{v,s}*esat / (Rv*T^2)).
         */

        TF tnr = tl;

        // Warm adjustment.
        if (tl >= T0<TF>)
        {
            while (fabs(tnr-tnr_old)/tnr_old > TF(1.e-5) && niter < nitermax)
            {
                tnr_old = tnr;
                qs = qsat_liq(p, tnr);
                const TF f =
                    tnr - tl - Lv<TF>/cp<TF>*(qt - qs);

                const TF f_prime = TF(1.) + Lv<TF>/cp<TF>*dqsatdT_liq(p, tnr);

                tnr -= f / f_prime;

                niter += 1;
            }

            qs = qsat_liq(p, tnr);
            ans.ql = fmax(TF(0.), qt - qs);
            ans.t  = tnr;
            ans.qs = qs;
        }
        // Cold adjustment.
        else
        {
            while (fabs(tnr-tnr_old)/tnr_old > TF(1.e-5) && niter < nitermax)
            {
                tnr_old = tnr;
                qs = qsat(p, tnr);
                const TF alpha_w = water_fraction(tnr);
                const TF alpha_i = TF(1.) - alpha_w;
                const TF dalphadT = (alpha_w > TF(0.) && alpha_w < TF(1.)) ? TF(0.025) : TF(0.);
                const TF dqsatdT_w = dqsatdT_liq(p, tnr);
                const TF dqsatdT_i = dqsatdT_ice(p, tnr);

                const TF f =
                    tnr - tl - alpha_w*Lv<TF>/cp<TF>*qt - alpha_i*Ls<TF>/cp<TF>*qt
                             + alpha_w*Lv<TF>/cp<TF>*qs + alpha_i*Ls<TF>/cp<TF>*qs;

                const TF f_prime = TF(1.)
                    - dalphadT*Lv<TF>/cp<TF>*qt + dalphadT*Ls<TF>/cp<TF>*qt
                    + dalphadT*Lv<TF>/cp<TF>*qs - dalphadT*Ls<TF>/cp<TF>*qs
                    + alpha_w*Lv<TF>/cp<TF>*dqsatdT_w
                    + alpha_i*Ls<TF>/cp<TF>*dqsatdT_i;

                tnr -= f / f_prime;

                niter += 1;
            }

            const TF alpha_w = water_fraction(tnr);
            const TF alpha_i = TF(1.) - alpha_w;

            qs = qsat(p, tnr);
            const TF ql_qi = fmax(TF(0.), qt - qs);

            ans.ql = alpha_w*ql_qi;
            ans.qi = alpha_i*ql_qi;
            ans.t  = tnr;
            ans.qs = qs;
        }

        // Raise exception if nitermax is reached.
        if (niter == nitermax)
        {
            printf("ERROR: saturation adjustment did not converge!\n");
            asm("trap;");
        }

        return ans;
    }

    template<typename TF> __global__
    void calc_buoyancy_tend_2nd_g(TF* __restrict__ wt, TF* __restrict__ th, TF* __restrict__ qt,
                                  TF* __restrict__ thvrefh, TF* __restrict__ exnh, TF* __restrict__ ph,
                                  int istart, int jstart, int kstart,
                                  int iend,   int jend,   int kend,
                                  int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;

            // Half level temperature and moisture content
            const TF thh = static_cast<TF>(0.5) * (th[ijk-kk] + th[ijk]); // Half level liq. water pot. temp.
            const TF qth = static_cast<TF>(0.5) * (qt[ijk-kk] + qt[ijk]); // Half level specific hum.

            Struct_sat_adjust<TF> ssa = sat_adjust_g(thh, qth, ph[k], exnh[k]);

            // Calculate tendency.
            if (ssa.ql > 0)
                wt[ijk] += buoyancy(exnh[k], thh, qth, ssa.ql, ssa.qi, thvrefh[k]);
            else
                wt[ijk] += buoyancy_no_ql(thh, qth, thvrefh[k]);
        }
    }

    template<typename TF> __global__
    void calc_buoyancy_g(TF* __restrict__ b,  TF* __restrict__ th,
                         TF* __restrict__ qt, TF* __restrict__ thvref,
                         TF* __restrict__ p,  TF* __restrict__ exn,
                         int istart, int jstart, int kstart,
                         int iend,   int jend,   int kcells,
                         int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z;

        if (i < iend && j < jend && k < kstart)
        {
            const int ijk   = i + j*jj + k*kk;
            b[ijk] = buoyancy_no_ql(th[ijk], qt[ijk], thvref[k]);
        }
        else if (i < iend && j < jend && k < kcells)
        {
            const int ijk = i + j*jj + k*kk;

            Struct_sat_adjust<TF> ssa = sat_adjust_g(th[ijk], qt[ijk], p[k], exn[k]);

            if (ssa.ql > 0)
                b[ijk] = buoyancy(exn[k], th[ijk], qt[ijk], ssa.ql, ssa.qi, thvref[k]);
            else
                b[ijk] = buoyancy_no_ql(th[ijk], qt[ijk], thvref[k]);
        }
    }

    template<typename TF> __global__
    void calc_buoyancy_h_g(TF* __restrict__ bh,  TF* __restrict__ th,
                         TF* __restrict__ qt, TF* __restrict__ thvrefh,
                         TF* __restrict__ ph,  TF* __restrict__ exnh,
                         int istart, int jstart, int kstart,
                         int iend,   int jend,   int kend,
                         int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            const int kk  = i + j*jj;

            // Half level temperature and moisture content
            const TF thh = static_cast<TF>(0.5) * (th[ijk-kk] + th[ijk]); // Half level liq. water pot. temp.
            const TF qth = static_cast<TF>(0.5) * (qt[ijk-kk] + qt[ijk]); // Half level specific hum.

            Struct_sat_adjust<TF> ssa = sat_adjust_g(thh, qth, ph[k], exnh[k]);

            // Calculate tendency
            if (ssa.ql > 0)
                bh[ijk] += buoyancy(exnh[k], thh, qth, ssa.ql, ssa.qi, thvrefh[k]);
            else
                bh[ijk] += buoyancy_no_ql(thh, qth, thvrefh[k]);
        }
    }

    template<typename TF> __global__
    void calc_buoyancy_bot_g(TF* __restrict__ b,      TF* __restrict__ bbot,
                             TF* __restrict__ th,     TF* __restrict__ thbot,
                             TF* __restrict__ qt,     TF* __restrict__ qtbot,
                             TF* __restrict__ thvref, TF* __restrict__ thvrefh,
                             int kstart, int icells, int jcells,
                             int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        const int j = blockIdx.y*blockDim.y + threadIdx.y;

        if (i < icells && j < jcells)
        {
            const int ij  = i + j*jj;
            const int ijk = i + j*jj + kstart*kk;

            bbot[ij ] = buoyancy_no_ql(thbot[ij], qtbot[ij], thvrefh[kstart]);
            b   [ijk] = buoyancy_no_ql(th[ijk],   qt[ijk],   thvref[kstart]);
        }
    }

    template<typename TF> __global__
    void calc_buoyancy_flux_bot_g(TF* __restrict__ bfluxbot,
                                  TF* __restrict__ th, TF* __restrict__ thfluxbot,
                                  TF* __restrict__ qt, TF* __restrict__ qtfluxbot,
                                  TF* __restrict__ thvrefh,
                                  int kstart, int icells, int jcells,
                                  int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        const int j = blockIdx.y*blockDim.y + threadIdx.y;

        // Calculate the surface buoyancy flux using the first model level temperature and humidity
        // to ensure bitwise identical restarts.
        if (i < icells && j < jcells)
        {
            const int ij  = i + j*jj;
            const int ijk = i + j*jj + kstart*kk;
            bfluxbot[ij] = buoyancy_flux_no_ql(th[ijk], thfluxbot[ij], qt[ijk], qtfluxbot[ij], thvrefh[kstart]);
        }
    }

    template<typename TF> __global__
    void calc_N2_g(TF* __restrict__ N2, TF* __restrict__ th,
                   TF* __restrict__ thvref, TF* __restrict__ dzi,
                   int istart, int jstart, int kstart,
                   int iend,   int jend,   int kend,
                   int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            N2[ijk] = grav<TF>/thvref[k]*static_cast<TF>(0.5)*(th[ijk+kk] - th[ijk-kk])*dzi[k];
        }
    }

    template<typename TF> __global__
    void calc_liquid_water_g(TF* __restrict__ ql, TF* __restrict__ th, TF* __restrict__ qt,
                             TF* __restrict__ exn, TF* __restrict__ p,
                             int istart, int jstart, int kstart,
                             int iend,   int jend,   int kend,
                             int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            ql[ijk] = sat_adjust_g(th[ijk], qt[ijk], p[k], exn[k]).ql;
        }
    }

    template<typename TF> __global__
    void calc_liquid_water_h_g(TF* __restrict__ qlh, TF* __restrict__ th, TF* __restrict__ qt,
                             TF* __restrict__ exnh, TF* __restrict__ ph,
                             int istart, int jstart, int kstart,
                             int iend,   int jend,   int kend,
                             int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            const int kk  = i + j*jj;

            const TF thh = static_cast<TF>(0.5) * (th[ijk-kk] + th[ijk]); // Half level liq. water pot. temp.
            const TF qth = static_cast<TF>(0.5) * (qt[ijk-kk] + qt[ijk]); // Half level specific hum.
            qlh[ijk] = sat_adjust_g(thh, qth, ph[k], exnh[k]).ql; // Half level liquid water content
        }
    }

    /*
    // BvS: no longer used, base state is calculated at the host
    // CvH: This unused code does not take into account ice
    template <typename TF> __global__
    void calc_base_state_g(TF* __restrict__ pref,     TF* __restrict__ prefh,
                           TF* __restrict__ rho,      TF* __restrict__ rhoh,
                           TF* __restrict__ thv,      TF* __restrict__ thvh,
                           TF* __restrict__ ex,       TF* __restrict__ exh,
                           TF* __restrict__ thlmean,  TF* __restrict__ qtmean,
                           TF* __restrict__ z,        TF* __restrict__ dz,
                           TF* __restrict__ dzh,
                           TF pbot, int kstart, int kend)
    {
        TF ql, si, qti, qli;
        TF rdcp = Rd<TF>/cp<TF>;

        const TF ssurf  = interp2(thlmean[kstart-1], thlmean[kstart]);
        const TF qtsurf = interp2(qtmean[kstart-1],  qtmean[kstart]);

        // Calculate surface (half=kstart) values
        exh[kstart]   = exner(pbot);
        ql            = sat_adjust_g(ssurf,qtsurf,pbot,exh[kstart]).ql;
        thvh[kstart]  = (ssurf + Lv<TF>*ql/(cp<TF>*exh[kstart])) * (1. - (1. - Rv<TF>/Rd<TF>)*qtsurf - Rv<TF>/Rd<TF>*ql);
        prefh[kstart] = pbot;
        rhoh[kstart]  = pbot / (Rd<TF> * exh[kstart] * thvh[kstart]);

        // First full grid level pressure
        pref[kstart] = pow((pow(pbot,rdcp) - grav<TF> * pow(p0<TF>,rdcp) * z[kstart] / (cp<TF> * thvh[kstart])),(1./rdcp));

        for (int k=kstart+1; k<kend+1; k++)
        {
            // 1. Calculate values at full level below zh[k]
            ex[k-1]  = exner(pref[k-1]);
            ql       = sat_adjust_g(thlmean[k-1],qtmean[k-1],pref[k-1],ex[k-1]).ql;
            thv[k-1] = (thlmean[k-1] + Lv<TF>*ql/(cp<TF>*ex[k-1])) * (1. - (1. - Rv<TF>/Rd<TF>)*qtmean[k-1] - Rv<TF>/Rd<TF>*ql);
            rho[k-1] = pref[k-1] / (Rd<TF> * ex[k-1] * thv[k-1]);

            // 2. Calculate half level pressure at zh[k] using values at z[k-1]
            prefh[k] = pow((pow(prefh[k-1],rdcp) - grav<TF> * pow(p0<TF>,rdcp) * dz[k-1] / (cp<TF> * thv[k-1])),(1./rdcp));

            // 3. Interpolate conserved variables to zh[k] and calculate virtual temp and ql
            si     = interp2(thlmean[k-1],thlmean[k]);
            qti    = interp2(qtmean[k-1],qtmean[k]);

            exh[k]   = exner(prefh[k]);
            qli      = sat_adjust_g(si,qti,prefh[k],exh[k]).ql;
            thvh[k]  = (si + Lv<TF>*qli/(cp<TF>*exh[k])) * (1. - (1. - Rv<TF>/Rd<TF>)*qti - Rv<TF>/Rd<TF>*qli);
            rhoh[k]  = prefh[k] / (Rd<TF> * exh[k] * thvh[k]);

            // 4. Calculate full level pressure at z[k]
            pref[k]  = pow((pow(pref[k-1],rdcp) - grav<TF> * pow(p0<TF>,rdcp) * dzh[k] / (cp<TF> * thvh[k])),(1./rdcp));
        }

        // Fill bottom and top full level ghost cells
        pref[kstart-1] = static_cast<TF>(2.)*prefh[kstart] - pref[kstart];
        pref[kend]     = static_cast<TF>(2.)*prefh[kend]   - pref[kend-1];
    }


    // BvS: no longer used, base state is calculated at the host
    template <typename TF> __global__
    void calc_hydrostatic_pressure_g(TF* __restrict__ pref,     TF* __restrict__ prefh,
                                     TF* __restrict__ ex,       TF* __restrict__ exh,
                                     TF* __restrict__ thlmean,  TF* __restrict__ qtmean,
                                     const TF* const __restrict__ z,        const TF* const __restrict__ dz,
                                     const TF* const __restrict__ dzh,
                                     const TF pbot, int kstart, int kend)
    {
        TF ql, si, qti, qli, thvh, thv;
        TF rdcp = Rd<TF>/cp<TF>;

        const TF ssurf  = interp2(thlmean[kstart-1], thlmean[kstart]);
        const TF qtsurf = interp2(qtmean[kstart-1],  qtmean[kstart]);

        // Calculate surface (half=kstart) values
        ql            = sat_adjust_g(ssurf,qtsurf,pbot,exh[kstart]).ql;
        thvh          = (ssurf + Lv<TF>*ql/(cp<TF>*exh[kstart])) * (1. - (1. - Rv<TF>/Rd<TF>)*qtsurf - Rv<TF>/Rd<TF>*ql);
        prefh[kstart] = pbot;

        // First full grid level pressure
        pref[kstart] = pow((pow(pbot,rdcp) - grav<TF> * pow(p0<TF>,rdcp) * z[kstart] / (cp<TF> * thvh)),(1./rdcp));

        for (int k=kstart+1; k<kend+1; k++)
        {
            // 1. Calculate values at full level below zh[k]
            ex[k-1]  = exner(pref[k-1]);
            ql       = sat_adjust_g(thlmean[k-1],qtmean[k-1],pref[k-1],ex[k-1]).ql;
            thv      = (thlmean[k-1] + Lv<TF>*ql/(cp<TF>*ex[k-1])) * (1. - (1. - Rv<TF>/Rd<TF>)*qtmean[k-1] - Rv<TF>/Rd<TF>*ql);

            // 2. Calculate half level pressure at zh[k] using values at z[k-1]
            prefh[k] = pow((pow(prefh[k-1],rdcp) - grav<TF> * pow(p0<TF>,rdcp) * dz[k-1] / (cp<TF> * thv)),(1./rdcp));

            // 3. Interpolate conserved variables to zh[k] and calculate virtual temp and ql
            si     = interp2(thlmean[k-1],thlmean[k]);
            qti    = interp2(qtmean[k-1],qtmean[k]);

            exh[k]   = exner(prefh[k]);
            qli      = sat_adjust_g(si,qti,prefh[k],exh[k]).ql;
            thvh     = (si + Lv<TF>*qli/(cp<TF>*exh[k])) * (1. - (1. - Rv<TF>/Rd<TF>)*qti - Rv<TF>/Rd<TF>*qli);

            // 4. Calculate full level pressure at z[k]
            pref[k]  = pow((pow(pref[k-1],rdcp) - grav<TF> * pow(p0<TF>,rdcp) * dzh[k] / (cp<TF> * thvh)),(1./rdcp));
        }

        // Fill bottom and top full level ghost cells
        pref[kstart-1] = static_cast<TF>(2.)*prefh[kstart] - pref[kstart];
        pref[kend]     = static_cast<TF>(2.)*prefh[kend]   - pref[kend-1];
    }
    */
} // end name    space

template<typename TF>
void Thermo_moist<TF>::prepare_device()
{
    auto& gd = grid.get_grid_data();
    const int nmemsize = gd.kcells*sizeof(TF);

    // Allocate fields for Boussinesq and anelastic solver
    cuda_safe_call(hipMalloc(&bs.thvref_g,  nmemsize));
    cuda_safe_call(hipMalloc(&bs.thvrefh_g, nmemsize));
    cuda_safe_call(hipMalloc(&bs.pref_g,    nmemsize));
    cuda_safe_call(hipMalloc(&bs.prefh_g,   nmemsize));
    cuda_safe_call(hipMalloc(&bs.exnref_g,  nmemsize));
    cuda_safe_call(hipMalloc(&bs.exnrefh_g, nmemsize));

    // Copy fields to device
    cuda_safe_call(hipMemcpy(bs.thvref_g,  bs.thvref.data(),  nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(bs.thvrefh_g, bs.thvrefh.data(), nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(bs.pref_g,    bs.pref.data(),    nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(bs.prefh_g,   bs.prefh.data(),   nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(bs.exnref_g,  bs.exnref.data(),  nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(bs.exnrefh_g, bs.exnrefh.data(), nmemsize, hipMemcpyHostToDevice));
}

template<typename TF>
void Thermo_moist<TF>::clear_device()
{
    cuda_safe_call(hipFree(bs.thvref_g ));
    cuda_safe_call(hipFree(bs.thvrefh_g));
    cuda_safe_call(hipFree(bs.pref_g   ));
    cuda_safe_call(hipFree(bs.prefh_g  ));
    cuda_safe_call(hipFree(bs.exnref_g ));
    cuda_safe_call(hipFree(bs.exnrefh_g));
    tdep_pbot->clear_device();
}

template<typename TF>
void Thermo_moist<TF>::forward_device()
{
    // Copy fields to device
    auto& gd = grid.get_grid_data();
    const int nmemsize = gd.kcells*sizeof(TF);
    cuda_safe_call(hipMemcpy(bs.pref_g,    bs.pref.data(),    nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(bs.prefh_g,   bs.prefh.data(),   nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(bs.exnref_g,  bs.exnref.data(),  nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(bs.exnrefh_g, bs.exnrefh.data(), nmemsize, hipMemcpyHostToDevice));
}

template<typename TF>
void Thermo_moist<TF>::backward_device()
{
    auto& gd = grid.get_grid_data();
    const int nmemsize = gd.kcells*sizeof(TF);
    hipMemcpy(bs.pref_g,    bs.pref.data(),    nmemsize, hipMemcpyHostToDevice);
    hipMemcpy(bs.prefh_g,   bs.prefh.data(),   nmemsize, hipMemcpyHostToDevice);
    hipMemcpy(bs.exnref_g,  bs.exnref.data(),  nmemsize, hipMemcpyHostToDevice);
    hipMemcpy(bs.exnrefh_g, bs.exnrefh.data(), nmemsize, hipMemcpyHostToDevice);

    bs_stats = bs;
}

#ifdef USECUDA
template<typename TF>
void Thermo_moist<TF>::exec(const double dt, Stats<TF>& stats)
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, gd.kmax);
    dim3 blockGPU(blocki, blockj, 1);

    // Re-calculate hydrostatic pressure and exner
    if (bs.swupdatebasestate)
    {
        //calc_hydrostatic_pressure<TF><<<1, 1>>>(bs.pref_g, bs.prefh_g, bs.exnref_g, bs.exnrefh_g,
        //                                        fields.sp.at("thl")->fld_mean_g, fields.sp.at("qt")->fld_mean_g,
        //                                        gd.z_g, gd.dz_g, gd.dzh_g, bs.pbot, gd.kstart, gd.kend);
        //cuda_check_error();

        // BvS: Calculating hydrostatic pressure on GPU is extremely slow. As temporary solution, copy back mean profiles to host,
        //      calculate pressure there and copy back the required profiles.
        hipMemcpy(fields.sp.at("thl")->fld_mean.data(), fields.sp.at("thl")->fld_mean_g, gd.kcells*sizeof(TF), hipMemcpyDeviceToHost);
        hipMemcpy(fields.sp.at("qt")->fld_mean.data(),  fields.sp.at("qt")->fld_mean_g,  gd.kcells*sizeof(TF), hipMemcpyDeviceToHost);

        auto tmp = fields.get_tmp();

        calc_base_state(
                bs.pref.data(), bs.prefh.data(),
                bs.rhoref.data(), bs.rhorefh.data(), bs.thvref.data(), bs.thvrefh.data(),
                bs.exnref.data(), bs.exnrefh.data(),
                fields.sp.at("thl")->fld_mean.data(), fields.sp.at("qt")->fld_mean.data(),
                bs.pbot, gd.kstart, gd.kend, gd.z.data(), gd.dz.data(), gd.dzh.data());

        fields.release_tmp(tmp);

        // Only half level pressure, exner and virtual potential temperature are needed for calc_buoyancy_tend
        hipMemcpy(bs.prefh_g,   bs.prefh.data(),   gd.kcells*sizeof(TF), hipMemcpyHostToDevice);
        hipMemcpy(bs.exnrefh_g, bs.exnrefh.data(), gd.kcells*sizeof(TF), hipMemcpyHostToDevice);
        hipMemcpy(bs.thvrefh_g, bs.thvrefh.data(), gd.kcells*sizeof(TF), hipMemcpyHostToDevice);
    }

    calc_buoyancy_tend_2nd_g<<<gridGPU, blockGPU>>>(
            fields.mt.at("w")->fld_g, fields.sp.at("thl")->fld_g,
            fields.sp.at("qt")->fld_g, bs.thvrefh_g, bs.exnrefh_g, bs.prefh_g,
            gd.istart, gd.jstart, gd.kstart+1,
            gd.iend,   gd.jend,   gd.kend,
            gd.icells, gd.ijcells);
    cuda_check_error();

    hipDeviceSynchronize();
    stats.calc_tend(*fields.mt.at("w"), tend_name);

}
#endif

#ifdef USECUDA
template<typename TF>
void Thermo_moist<TF>::get_thermo_field_g(
        Field3d<TF>& fld, const std::string& name, const bool cyclic )
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, gd.kcells);
    dim3 blockGPU(blocki, blockj, 1);

    dim3 gridGPU2 (gridi, gridj, gd.kmax);
    dim3 blockGPU2(blocki, blockj, 1);

    // BvS: getthermofield() is called from subgrid-model, before thermo(), so re-calculate the hydrostatic pressure
    if (bs.swupdatebasestate && (name == "b" || name == "ql" || name == "bh" || name == "qlh"))
    {
        //calc_hydrostatic_pressure_g<TF><<<1, 1>>>(bs.pref_g, bs.prefh_g, bs.exnref_g, bs.exnrefh_g,
        //                                          fields.sp.at("thl")->fld_mean_g, fields.sp.at("qt")->fld_mean_g,
        //                                          gd.z_g, gd.dz_g, gd.dzh_g, bs.pbot, gd.kstart, gd.kend);
        //cuda_check_error();

        // BvS: Calculating hydrostatic pressure on GPU is extremely slow. As temporary solution, copy back mean profiles to host,
        //      calculate pressure there and copy back the required profiles.
        hipMemcpy(fields.sp.at("thl")->fld_mean.data(), fields.sp.at("thl")->fld_mean_g, gd.kcells*sizeof(TF), hipMemcpyDeviceToHost);
        hipMemcpy(fields.sp.at("qt")->fld_mean.data(),  fields.sp.at("qt")->fld_mean_g,  gd.kcells*sizeof(TF), hipMemcpyDeviceToHost);

        auto tmp = fields.get_tmp();

        calc_base_state(bs.pref.data(), bs.prefh.data(),
                        &tmp->fld[0*gd.kcells], &tmp->fld[1*gd.kcells], &tmp->fld[2*gd.kcells], &tmp->fld[3*gd.kcells],
                        bs.exnref.data(), bs.exnrefh.data(), fields.sp.at("thl")->fld_mean.data(), fields.sp.at("qt")->fld_mean.data(),
                        bs.pbot, gd.kstart, gd.kend, gd.z.data(), gd.dz.data(), gd.dzh.data());

        fields.release_tmp(tmp);

        // Only full level pressure and bs.exner needed for calculating buoyancy of ql
        hipMemcpy(bs.pref_g,   bs.pref.data(),   gd.kcells*sizeof(TF), hipMemcpyHostToDevice);
        hipMemcpy(bs.prefh_g,  bs.prefh.data(),  gd.kcells*sizeof(TF), hipMemcpyHostToDevice);
        hipMemcpy(bs.exnref_g, bs.exnref.data(), gd.kcells*sizeof(TF), hipMemcpyHostToDevice);
        hipMemcpy(bs.exnref_g, bs.exnref.data(), gd.kcells*sizeof(TF), hipMemcpyHostToDevice);
    }

    if (name == "b")
    {
        calc_buoyancy_g<<<gridGPU, blockGPU>>>(
            fld.fld_g, fields.sp.at("thl")->fld_g, fields.sp.at("qt")->fld_g,
            bs.thvref_g, bs.pref_g, bs.exnref_g,
            gd.istart,  gd.jstart, gd.kstart,
            gd.iend, gd.jend, gd.kcells,
            gd.icells, gd.ijcells);
        cuda_check_error();
    }
    else if (name == "b_h")
    {
        calc_buoyancy_g<<<gridGPU, blockGPU>>>(
            fld.fld_g, fields.sp.at("thl")->fld_g, fields.sp.at("qt")->fld_g,
            bs.thvrefh_g, bs.prefh_g, bs.exnrefh_g,
            gd.istart,  gd.jstart, gd.kstart,
            gd.iend, gd.jend, gd.kcells,
            gd.icells, gd.ijcells);
        cuda_check_error();
    }
    else if (name == "ql")
    {
        calc_liquid_water_g<<<gridGPU2, blockGPU2>>>(
            fld.fld_g, fields.sp.at("thl")->fld_g, fields.sp.at("qt")->fld_g,
            bs.exnref_g, bs.pref_g,
            gd.istart,  gd.jstart,  gd.kstart,
            gd.iend,    gd.jend,    gd.kend,
            gd.icells, gd.ijcells);
        cuda_check_error();
    }
    else if (name == "ql_h")
    {
        calc_liquid_water_h_g<<<gridGPU2, blockGPU2>>>(
            fld.fld_g, fields.sp.at("thl")->fld_g, fields.sp.at("qt")->fld_g,
            bs.exnrefh_g, bs.prefh_g,
            gd.istart,  gd.jstart,  gd.kstart,
            gd.iend,    gd.jend,    gd.kend,
            gd.icells, gd.ijcells);
        cuda_check_error();
    }
    else if (name == "N2")
    {
        calc_N2_g<<<gridGPU2, blockGPU2>>>(
            fld.fld_g, fields.sp.at("thl")->fld_g, bs.thvref_g, gd.dzi_g,
            gd.istart,  gd.jstart, gd.kstart,
            gd.iend,    gd.jend,   gd.kend,
            gd.icells, gd.ijcells);
        cuda_check_error();
    }
    else
    {
        std::string msg = "get_thermo_field_g \"" + name + "\" not supported";
        throw std::runtime_error(msg);
    }

    if (cyclic)
        boundary_cyclic.exec_g(fld.fld_g);
}
#endif

#ifdef USECUDA
template<typename TF>
TF* Thermo_moist<TF>::get_basestate_fld_g(std::string name)
{
    // BvS TO-DO: change std::string to enum
    if (name == "pref")
        return bs.pref_g;
    else if (name == "prefh")
        return bs.prefh_g;
    else if (name == "exner")
        return bs.exnref_g;
    else if (name == "exnerh")
        return bs.exnrefh_g;
    else
    {
        std::string error_message = "Can not get basestate field \"" + name + "\" from thermo_moist";
        throw std::runtime_error(error_message);
    }
}
#endif

#ifdef USECUDA
template<typename TF>
void Thermo_moist<TF>::get_buoyancy_fluxbot_g(Field3d<TF>& bfield)
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.icells/blocki + (gd.icells%blocki > 0);
    const int gridj  = gd.jcells/blockj + (gd.jcells%blockj > 0);

    dim3 gridGPU (gridi, gridj, 1);
    dim3 blockGPU(blocki, blockj, 1);

    calc_buoyancy_flux_bot_g<<<gridGPU, blockGPU>>>(
        bfield.flux_bot_g,
        fields.sp.at("thl")->fld_g, fields.sp.at("thl")->flux_bot_g,
        fields.sp.at("qt")->fld_g, fields.sp.at("qt")->flux_bot_g,
        bs.thvrefh_g, gd.kstart, gd.icells, gd.jcells,
        gd.icells, gd.ijcells);
    cuda_check_error();
}
#endif

#ifdef USECUDA
template<typename TF>
void Thermo_moist<TF>::get_buoyancy_surf_g(Field3d<TF>& bfield)
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.icells/blocki + (gd.icells%blocki > 0);
    const int gridj  = gd.jcells/blockj + (gd.jcells%blockj > 0);

    dim3 gridGPU (gridi, gridj, 1);
    dim3 blockGPU(blocki, blockj, 1);

    calc_buoyancy_bot_g<<<gridGPU, blockGPU>>>(
        bfield.fld_g, bfield.fld_bot_g,
        fields.sp.at("thl")->fld_g, fields.sp.at("thl")->fld_bot_g,
        fields.sp.at("qt")->fld_g, fields.sp.at("qt")->fld_bot_g,
        bs.thvref_g, bs.thvrefh_g, gd.kstart, gd.icells, gd.jcells,
        gd.icells, gd.ijcells);
    cuda_check_error();

    calc_buoyancy_flux_bot_g<<<gridGPU, blockGPU>>>(
        bfield.flux_bot_g,
        fields.sp.at("thl")->fld_g, fields.sp.at("thl")->flux_bot_g,
        fields.sp.at("qt")->fld_g, fields.sp.at("qt")->flux_bot_g,
        bs.thvrefh_g, gd.kstart, gd.icells, gd.jcells,
        gd.icells, gd.ijcells);
    cuda_check_error();
}
#endif

#ifdef USECUDA
template<typename TF>
void Thermo_moist<TF>::exec_column(Column<TF>& column)
{
    const TF no_offset = 0.;
    auto output = fields.get_tmp_g();

    get_thermo_field_g(*output, "b", false);
    column.calc_column("b", output->fld_g, no_offset);

    get_thermo_field_g(*output, "ql", false);
    column.calc_column("ql", output->fld_g, no_offset);

    fields.release_tmp_g(output);
}
#endif

#ifdef USECUDA
template<typename TF>
void Thermo_moist<TF>::get_mask(Stats<TF>& stats, std::string mask_name)
{
    if (mask_name == "ql")
    {
        auto ql  = fields.get_tmp();
        auto qlh = fields.get_tmp();
        auto ql_g = fields.get_tmp_g();

        get_thermo_field_g(*ql_g, "ql", true);

        fields.backward_field_device_3d(ql->fld.data(), ql_g->fld_g);
        get_thermo_field_g(*ql_g, "ql_h", true);
        fields.backward_field_device_3d(qlh->fld.data(), ql_g->fld_g);

        stats.set_mask_thres(mask_name, *ql, *qlh, 0., Stats_mask_type::Plus);

        fields.release_tmp_g(ql_g);
        fields.release_tmp(ql);
        fields.release_tmp(qlh);
    }
    else if (mask_name == "qlcore")
    {
        auto ql  = fields.get_tmp();
        auto qlh = fields.get_tmp();
        auto tmp_g = fields.get_tmp_g();

        get_thermo_field_g(*tmp_g, "ql", true);
        fields.backward_field_device_3d(ql->fld.data(), tmp_g->fld_g);
        get_thermo_field_g(*tmp_g, "ql_h", true);
        fields.backward_field_device_3d(qlh->fld.data(), tmp_g->fld_g);

        stats.set_mask_thres(mask_name, *ql, *qlh, 0., Stats_mask_type::Plus);

        fields.release_tmp(ql);
        fields.release_tmp(qlh);

        auto b = fields.get_tmp();
        auto bh = fields.get_tmp();

        get_thermo_field_g(*tmp_g, "b", true);
        fields.backward_field_device_3d(b->fld.data(), tmp_g->fld_g);
        get_thermo_field_g(*tmp_g, "b_h", true);
        fields.backward_field_device_3d(bh->fld.data(), tmp_g->fld_g);

        field3d_operators.calc_mean_profile(b->fld_mean.data(), b->fld.data());
        field3d_operators.subtract_mean_profile(b->fld.data(), b->fld_mean.data());

        field3d_operators.calc_mean_profile(bh->fld_mean.data(), bh->fld.data());
        field3d_operators.subtract_mean_profile(bh->fld.data(), bh->fld_mean.data());

        stats.set_mask_thres(mask_name, *b, *bh, 0., Stats_mask_type::Plus);

        fields.release_tmp(b);
        fields.release_tmp(bh);
        fields.release_tmp_g(tmp_g);
    }
    else
    {
        std::string message = "Moist thermodynamics can not provide mask: \"" + mask_name +"\"";
        throw std::runtime_error(message);
    }
}
#endif

template class Thermo_moist<double>;
template class Thermo_moist<float>;
