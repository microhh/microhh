#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2024 Chiel van Heerwaarden
 * Copyright (c) 2011-2024 Thijs Heus
 * Copyright (c) 2014-2024 Bart van Stratum
 * Copyright (c) 2020-2024 Menno Veerman
 * Copyright (c) 2022-2024 Mirjam Tijhuis
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <numeric>
#include <boost/algorithm/string.hpp>

#include "radiation_rrtmgp_rt.h"
#include "radiation_rrtmgp_functions.h"

#include "grid.h"
#include "fields.h"
#include "timeloop.h"
#include "timedep.h"
#include "thermo.h"
#include "microphys.h"
#include "stats.h"
#include "netcdf_interface.h"
#include "constants.h"
#include "stats.h"
#include "cross.h"
#include "column.h"
#include "tools.h"

#include "Array.h"
#include "Fluxes.h"
#include "Fluxes_rt.h"

#include "raytracer_definitions.h"
#include "subset_kernels_cuda.h"
#include "gas_optics_rrtmgp_kernels_cuda_rt.h"
#include "gpt_combine_kernels_cuda_rt.h"


namespace
{
    using namespace Radiation_rrtmgp_functions;
    using namespace Raytracer_definitions;

    __global__
    void calc_tendency_rt(
            Float* __restrict__ thlt_rad,
            Float* __restrict__ rt_flux_abs_dir, Float* __restrict__ rt_flux_abs_dif,
            const Float* __restrict__ rho, const Float* __restrict__ exner, const Float* __restrict__ dz,
            const int istart, const int jstart, const int kstart,
            const int iend, const int jend, const int kend,
            const int igc, const int jgc, const int kgc,
            const int jj, const int kk,
            const int jj_nogc, const int kk_nogc)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if ( (i < iend) && (j < jend) && (k < kend) )
        {
            const Float fac = Float(1.) / (rho[k] * Constants::cp<Float> * exner[k]);

            const int ijk = i + j*jj + k*kk;
            const int ijk_nogc = (i-igc) + (j-jgc)*jj_nogc + (k-kgc)*kk_nogc;

            rt_flux_abs_dir[ijk_nogc] *= fac;
            rt_flux_abs_dif[ijk_nogc] *= fac;

            thlt_rad[ijk] += rt_flux_abs_dir[ijk_nogc] + rt_flux_abs_dif[ijk_nogc];
        }
    }

    __global__
    void calc_tendency(
            Float* __restrict__ thlt_rad,  const Float* __restrict__ flux_up,
            const Float* __restrict flux_dn, const Float* __restrict__ rho,
            const Float* __restrict__ exner, const Float* __restrict__ dz,
            const int istart, const int jstart, const int kstart,
            const int iend, const int jend, const int kend,
            const int igc, const int jgc, const int kgc,
            const int jj, const int kk,
            const int jj_nogc, const int kk_nogc)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if ( (i < iend) && (j < jend) && (k < kend) )
        {
            const Float fac = Float(1.) / (rho[k] * Constants::cp<Float> * exner[k] * dz[k]);

            const int ijk = i + j*jj + k*kk;
            const int ijk_nogc = (i-igc) + (j-jgc)*jj_nogc + (k-kgc)*kk_nogc;

            thlt_rad[ijk] -= fac * ( flux_up[ijk_nogc + kk_nogc] - flux_up[ijk_nogc]
                                   - flux_dn[ijk_nogc + kk_nogc] + flux_dn[ijk_nogc] );
        }
    }

    __global__
    void add_tendency(
            Float* __restrict__ thlt,  const Float* __restrict__ thlt_rad,
            const int istart, const int jstart, const int kstart,
            const int iend, const int jend, const int kend,
            const int jj, const int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if ( (i < iend) && (j < jend) && (k < kend) )
        {
            const int ijk = i + j*jj + k*kk;
            thlt[ijk] += thlt_rad[ijk];
        }
    }

    __global__
    void store_surface_fluxes_rt(
            Float* __restrict__ flux_up_sfc, Float* __restrict__ flux_dn_sfc,
            const Float* __restrict__ rt_flux_sfc_dir, const Float* __restrict__ rt_flux_sfc_dif,
            const Float* __restrict__ rt_flux_sfc_up,
            const int istart, const int iend,
            const int jstart, const int jend,
            const int igc, const int jgc,
            const int jj, const int kk,
            const int jj_nogc)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;

        if ( (i < iend) && (j < jend) )
        {
            const int ij = i + j*jj;
            const int ij_nogc = (i-igc) + (j-jgc)*jj_nogc;
            flux_up_sfc[ij] = rt_flux_sfc_up[ij_nogc];
            flux_dn_sfc[ij] = rt_flux_sfc_dir[ij_nogc] + rt_flux_sfc_dif[ij_nogc];
        }
    }

    __global__
    void set_to_value(Float* __restrict__ fld, const int nsize, const Float value)
    {
        const int n = blockIdx.x*blockDim.x + threadIdx.x;
        if (n < nsize)
            fld[n] = value;
    }

    __global__
    void add_profile(
            Float* __restrict__ fld,
            const Float* __restrict__ profile,
            const int istart, const int iend,
            const int jstart, const int jend,
            const int kstart, const int kend,
            const int icells, const int ijcells)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if ((i < iend) && (j < jend) && (k < kend))
        {
            const int ijk = i + j*icells + k*ijcells;
            fld[ijk] += profile[k];
        }
    }

    __global__
    void store_surface_fluxes(
            Float* __restrict__ flux_up_sfc, Float* __restrict__ flux_dn_sfc,
            const Float* __restrict__ flux_up, const Float* __restrict__ flux_dn,
            const int istart, const int iend,
            const int jstart, const int jend,
            const int igc, const int jgc,
            const int jj, const int kk,
            const int jj_nogc)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;

        if ( (i < iend) && (j < jend) )
        {
            const int ij = i + j*jj;
            const int ij_nogc = (i-igc) + (j-jgc)*jj_nogc;
            flux_up_sfc[ij] = flux_up[ij_nogc];
            flux_dn_sfc[ij] = flux_dn[ij_nogc];
        }
    }

    __global__
    void effective_radius_and_ciwp_to_gm2(
            Float* __restrict__ rel, Float* __restrict__ rei,
            Float* __restrict__ clwp, Float* __restrict__ ciwp,
            const Float* __restrict__ dz,
            const int ncol, const int nlay, const int kstart,
            const Float four_third_pi_N0_rho_w,
            const Float four_third_pi_N0_rho_i,
            const Float sig_g_fac)
    {
        const int icol = blockIdx.x*blockDim.x + threadIdx.x;
        const int ilay = blockIdx.y*blockDim.y + threadIdx.y;

        if ( (icol < ncol) && (ilay < nlay) )
        {
            const int idx = icol + ilay*ncol;
            const int idx_z = ilay + kstart;
            const Float rel_local = clwp[idx] > Float(0.) ? Float(1.e6) * sig_g_fac * pow(clwp[idx] / dz[idx_z] / four_third_pi_N0_rho_w, Float(1.)/Float(3.)) : Float(0.);
            const Float rei_local = ciwp[idx] > Float(0.) ? Float(1.e6) * sig_g_fac * pow(ciwp[idx] / dz[idx_z] / four_third_pi_N0_rho_i, Float(1.)/Float(3.)) : Float(0.);

            rel[idx] = max(Float(2.5), min(rel_local, Float(21.5)));
            rei[idx] = max(Float(10.), min(rei_local, Float(180.)));

            clwp[idx] *= Float(1.e3);
            ciwp[idx] *= Float(1.e3);
        }
    }

    __global__
    void add_ghost_cells_g(
            Float* __restrict__ out, const Float* __restrict__ in,
            const int istart, const int jstart, const int kstart,
            const int iend, const int jend, const int kendh,
            const int jj, const int kk,
            const int jj_nogc, const int kk_nogc)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kendh)
        {
            const int ijk_nogc  = (i-istart) + (j-jstart)*jj_nogc + (k-kstart)*kk_nogc;
            const int ijk = i + j*jj + k*kk;

            out[ijk] = in[ijk_nogc];
        }
    }

    __global__
    void add_ghost_cells_2d_g(
            Float* __restrict__ out, const Float* __restrict__ in,
            const int istart, const int jstart,
            const int iend, const int jend,
            const int jj, const int jj_nogc)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;

        if (i < iend && j < jend)
        {
            const int ij_nogc  = (i-istart) + (j-jstart)*jj_nogc;
            const int ij = i + j*jj;

            out[ij] = in[ij_nogc];
        }
    }

    __global__
    void sum_tau_kernel(
            const int ncol, const int nlev, const int col_s_in,
            const Float* __restrict__ tau, const int ibnd, Float* __restrict__ aod)
    {
        const int icol = blockIdx.x*blockDim.x + threadIdx.x;

        if ( icol < ncol)
        {
            Float aod_s = 0;
            for (int ilev=0; ilev < nlev; ++ilev)
            {
                const int idx_in = icol + ilev*ncol + ibnd*nlev*ncol;
                aod_s += tau[idx_in];
            }
            aod[col_s_in + icol - 1] = aod_s;
        }
    }

    void sum_tau(
            int ncol, int nlev, int col_s_in,
            const Float* tau, const int ibnd, Float* aod)
    {
        const int block_lev = 16;
        const int block_col = 16;

        const int grid_col = ncol/block_col + (ncol%block_col > 0);
        const int grid_lev = nlev/block_lev + (nlev%block_lev > 0);

        dim3 grid_gpu(grid_col);
        dim3 block_gpu(block_col);

        sum_tau_kernel<<<grid_gpu, block_gpu>>>(ncol, nlev, col_s_in, tau, ibnd-1, aod);
    }

    std::vector<std::string> get_variable_string(
            const std::string& var_name,
            std::vector<int> i_count,
            Netcdf_handle& input_nc,
            const int string_len,
            bool trim=true)
    {
        // Multiply all elements in i_count.
        int total_count = std::accumulate(i_count.begin(), i_count.end(), 1, std::multiplies<>());

        // Add the string length as the rightmost dimension.
        i_count.push_back(string_len);

        // Multiply all elements in i_count.
        // int total_count_char = std::accumulate(i_count.begin(), i_count.end(), 1, std::multiplies<>());

        // Read the entire char array;
        std::vector<char> var_char;
        var_char = input_nc.get_variable<char>(var_name, i_count);

        std::vector<std::string> var;

        for (int n=0; n<total_count; ++n)
        {
            std::string s(var_char.begin()+n*string_len, var_char.begin()+(n+1)*string_len);
            if (trim)
                boost::trim(s);
            var.push_back(s);
        }

        return var;
    }

    Gas_optics_rrtmgp_gpu load_and_init_gas_optics(
            Master& master,
            const Gas_concs_gpu& gas_concs,
            const std::string& coef_file)
    {
        // READ THE COEFFICIENTS FOR THE OPTICAL SOLVER.
        Netcdf_file coef_nc(master, coef_file, Netcdf_mode::Read);

        // Read k-distribution information.
        const int n_temps = coef_nc.get_dimension_size("temperature");
        const int n_press = coef_nc.get_dimension_size("pressure");
        const int n_absorbers = coef_nc.get_dimension_size("absorber");
        const int n_char = coef_nc.get_dimension_size("string_len");
        const int n_minorabsorbers = coef_nc.get_dimension_size("minor_absorber");
        const int n_extabsorbers = coef_nc.get_dimension_size("absorber_ext");
        const int n_mixingfracs = coef_nc.get_dimension_size("mixing_fraction");
        const int n_layers = coef_nc.get_dimension_size("atmos_layer");
        const int n_bnds = coef_nc.get_dimension_size("bnd");
        const int n_gpts = coef_nc.get_dimension_size("gpt");
        const int n_pairs = coef_nc.get_dimension_size("pair");
        const int n_minor_absorber_intervals_lower = coef_nc.get_dimension_size("minor_absorber_intervals_lower");
        const int n_minor_absorber_intervals_upper = coef_nc.get_dimension_size("minor_absorber_intervals_upper");
        const int n_contributors_lower = coef_nc.get_dimension_size("contributors_lower");
        const int n_contributors_upper = coef_nc.get_dimension_size("contributors_upper");

        // Read gas names.
        Array<std::string,1> gas_names(
                get_variable_string("gas_names", {n_absorbers}, coef_nc, n_char, true), {n_absorbers});

        Array<int,3> key_species(
                coef_nc.get_variable<int>("key_species", {n_bnds, n_layers, 2}),
                {2, n_layers, n_bnds});
        Array<Float,2> band_lims(coef_nc.get_variable<Float>("bnd_limits_wavenumber", {n_bnds, 2}), {2, n_bnds});
        Array<int,2> band2gpt(coef_nc.get_variable<int>("bnd_limits_gpt", {n_bnds, 2}), {2, n_bnds});
        Array<Float,1> press_ref(coef_nc.get_variable<Float>("press_ref", {n_press}), {n_press});
        Array<Float,1> temp_ref(coef_nc.get_variable<Float>("temp_ref", {n_temps}), {n_temps});

        Float temp_ref_p = coef_nc.get_variable<Float>("absorption_coefficient_ref_P");
        Float temp_ref_t = coef_nc.get_variable<Float>("absorption_coefficient_ref_T");
        Float press_ref_trop = coef_nc.get_variable<Float>("press_ref_trop");

        Array<Float,3> kminor_lower(
                coef_nc.get_variable<Float>("kminor_lower", {n_temps, n_mixingfracs, n_contributors_lower}),
                {n_contributors_lower, n_mixingfracs, n_temps});
        Array<Float,3> kminor_upper(
                coef_nc.get_variable<Float>("kminor_upper", {n_temps, n_mixingfracs, n_contributors_upper}),
                {n_contributors_upper, n_mixingfracs, n_temps});

        Array<std::string,1> gas_minor(get_variable_string("gas_minor", {n_minorabsorbers}, coef_nc, n_char),
                                       {n_minorabsorbers});

        Array<std::string,1> identifier_minor(
                get_variable_string("identifier_minor", {n_minorabsorbers}, coef_nc, n_char), {n_minorabsorbers});

        Array<std::string,1> minor_gases_lower(
                get_variable_string("minor_gases_lower", {n_minor_absorber_intervals_lower}, coef_nc, n_char),
                {n_minor_absorber_intervals_lower});
        Array<std::string,1> minor_gases_upper(
                get_variable_string("minor_gases_upper", {n_minor_absorber_intervals_upper}, coef_nc, n_char),
                {n_minor_absorber_intervals_upper});

        Array<int,2> minor_limits_gpt_lower(
                coef_nc.get_variable<int>("minor_limits_gpt_lower", {n_minor_absorber_intervals_lower, n_pairs}),
                {n_pairs, n_minor_absorber_intervals_lower});
        Array<int,2> minor_limits_gpt_upper(
                coef_nc.get_variable<int>("minor_limits_gpt_upper", {n_minor_absorber_intervals_upper, n_pairs}),
                {n_pairs, n_minor_absorber_intervals_upper});

        Array<Bool,1> minor_scales_with_density_lower(
                coef_nc.get_variable<Bool>("minor_scales_with_density_lower", {n_minor_absorber_intervals_lower}),
                {n_minor_absorber_intervals_lower});
        Array<Bool,1> minor_scales_with_density_upper(
                coef_nc.get_variable<Bool>("minor_scales_with_density_upper", {n_minor_absorber_intervals_upper}),
                {n_minor_absorber_intervals_upper});

        Array<Bool,1> scale_by_complement_lower(
                coef_nc.get_variable<Bool>("scale_by_complement_lower", {n_minor_absorber_intervals_lower}),
                {n_minor_absorber_intervals_lower});
        Array<Bool,1> scale_by_complement_upper(
                coef_nc.get_variable<Bool>("scale_by_complement_upper", {n_minor_absorber_intervals_upper}),
                {n_minor_absorber_intervals_upper});

        Array<std::string,1> scaling_gas_lower(
                get_variable_string("scaling_gas_lower", {n_minor_absorber_intervals_lower}, coef_nc, n_char),
                {n_minor_absorber_intervals_lower});
        Array<std::string,1> scaling_gas_upper(
                get_variable_string("scaling_gas_upper", {n_minor_absorber_intervals_upper}, coef_nc, n_char),
                {n_minor_absorber_intervals_upper});

        Array<int,1> kminor_start_lower(
                coef_nc.get_variable<int>("kminor_start_lower", {n_minor_absorber_intervals_lower}),
                {n_minor_absorber_intervals_lower});
        Array<int,1> kminor_start_upper(
                coef_nc.get_variable<int>("kminor_start_upper", {n_minor_absorber_intervals_upper}),
                {n_minor_absorber_intervals_upper});

        Array<Float,3> vmr_ref(
                coef_nc.get_variable<Float>("vmr_ref", {n_temps, n_extabsorbers, n_layers}),
                {n_layers, n_extabsorbers, n_temps});

        Array<Float,4> kmajor(
                coef_nc.get_variable<Float>("kmajor", {n_temps, n_press+1, n_mixingfracs, n_gpts}),
                {n_gpts, n_mixingfracs, n_press+1, n_temps});

        // Keep the size at zero, if it does not exist.
        Array<Float,3> rayl_lower;
        Array<Float,3> rayl_upper;

        if (coef_nc.variable_exists("rayl_lower"))
        {
            rayl_lower.set_dims({n_gpts, n_mixingfracs, n_temps});
            rayl_upper.set_dims({n_gpts, n_mixingfracs, n_temps});
            rayl_lower = coef_nc.get_variable<Float>("rayl_lower", {n_temps, n_mixingfracs, n_gpts});
            rayl_upper = coef_nc.get_variable<Float>("rayl_upper", {n_temps, n_mixingfracs, n_gpts});
        }

        // Is it really LW if so read these variables as well.
        if (coef_nc.variable_exists("totplnk"))
        {
            int n_internal_sourcetemps = coef_nc.get_dimension_size("temperature_Planck");

            Array<Float,2> totplnk(
                    coef_nc.get_variable<Float>( "totplnk", {n_bnds, n_internal_sourcetemps}),
                    {n_internal_sourcetemps, n_bnds});
            Array<Float,4> planck_frac(
                    coef_nc.get_variable<Float>("plank_fraction", {n_temps, n_press+1, n_mixingfracs, n_gpts}),
                    {n_gpts, n_mixingfracs, n_press+1, n_temps});

            // Construct the k-distribution.
            return Gas_optics_rrtmgp_gpu(
                    gas_concs,
                    gas_names,
                    key_species,
                    band2gpt,
                    band_lims,
                    press_ref,
                    press_ref_trop,
                    temp_ref,
                    temp_ref_p,
                    temp_ref_t,
                    vmr_ref,
                    kmajor,
                    kminor_lower,
                    kminor_upper,
                    gas_minor,
                    identifier_minor,
                    minor_gases_lower,
                    minor_gases_upper,
                    minor_limits_gpt_lower,
                    minor_limits_gpt_upper,
                    minor_scales_with_density_lower,
                    minor_scales_with_density_upper,
                    scaling_gas_lower,
                    scaling_gas_upper,
                    scale_by_complement_lower,
                    scale_by_complement_upper,
                    kminor_start_lower,
                    kminor_start_upper,
                    totplnk,
                    planck_frac,
                    rayl_lower,
                    rayl_upper);
        }
        else
        {
            Array<Float,1> solar_src_quiet(
                    coef_nc.get_variable<Float>("solar_source_quiet", {n_gpts}), {n_gpts});
            Array<Float,1> solar_src_facular(
                    coef_nc.get_variable<Float>("solar_source_facular", {n_gpts}), {n_gpts});
            Array<Float,1> solar_src_sunspot(
                    coef_nc.get_variable<Float>("solar_source_sunspot", {n_gpts}), {n_gpts});

            Float tsi = coef_nc.get_variable<Float>("tsi_default");
            Float mg_index = coef_nc.get_variable<Float>("mg_default");
            Float sb_index = coef_nc.get_variable<Float>("sb_default");

            return Gas_optics_rrtmgp_gpu(
                    gas_concs,
                    gas_names,
                    key_species,
                    band2gpt,
                    band_lims,
                    press_ref,
                    press_ref_trop,
                    temp_ref,
                    temp_ref_p,
                    temp_ref_t,
                    vmr_ref,
                    kmajor,
                    kminor_lower,
                    kminor_upper,
                    gas_minor,
                    identifier_minor,
                    minor_gases_lower,
                    minor_gases_upper,
                    minor_limits_gpt_lower,
                    minor_limits_gpt_upper,
                    minor_scales_with_density_lower,
                    minor_scales_with_density_upper,
                    scaling_gas_lower,
                    scaling_gas_upper,
                    scale_by_complement_lower,
                    scale_by_complement_upper,
                    kminor_start_lower,
                    kminor_start_upper,
                    solar_src_quiet,
                    solar_src_facular,
                    solar_src_sunspot,
                    tsi,
                    mg_index,
                    sb_index,
                    rayl_lower,
                    rayl_upper);
        }
        // End reading of k-distribution.
    }

    Cloud_optics_gpu load_and_init_cloud_optics(
            Master& master,
            const std::string& coef_file)
    {
        // READ THE COEFFICIENTS FOR THE OPTICAL SOLVER.
        Netcdf_file coef_nc(master, coef_file, Netcdf_mode::Read);

        // Read look-up table coefficient dimensions
        int n_band     = coef_nc.get_dimension_size("nband");
        int n_rghice   = coef_nc.get_dimension_size("nrghice");
        int n_size_liq = coef_nc.get_dimension_size("nsize_liq");
        int n_size_ice = coef_nc.get_dimension_size("nsize_ice");

        Array<Float,2> band_lims_wvn(coef_nc.get_variable<Float>("bnd_limits_wavenumber", {n_band, 2}), {2, n_band});

        // Read look-up table constants.
        Float radliq_lwr = coef_nc.get_variable<Float>("radliq_lwr");
        Float radliq_upr = coef_nc.get_variable<Float>("radliq_upr");
        Float radliq_fac = coef_nc.get_variable<Float>("radliq_fac");

        Float radice_lwr = coef_nc.get_variable<Float>("radice_lwr");
        Float radice_upr = coef_nc.get_variable<Float>("radice_upr");
        Float radice_fac = coef_nc.get_variable<Float>("radice_fac");

        Array<Float,2> lut_extliq(
                coef_nc.get_variable<Float>("lut_extliq", {n_band, n_size_liq}), {n_size_liq, n_band});
        Array<Float,2> lut_ssaliq(
                coef_nc.get_variable<Float>("lut_ssaliq", {n_band, n_size_liq}), {n_size_liq, n_band});
        Array<Float,2> lut_asyliq(
                coef_nc.get_variable<Float>("lut_asyliq", {n_band, n_size_liq}), {n_size_liq, n_band});

        Array<Float,3> lut_extice(
                coef_nc.get_variable<Float>("lut_extice", {n_rghice, n_band, n_size_ice}), {n_size_ice, n_band, n_rghice});
        Array<Float,3> lut_ssaice(
                coef_nc.get_variable<Float>("lut_ssaice", {n_rghice, n_band, n_size_ice}), {n_size_ice, n_band, n_rghice});
        Array<Float,3> lut_asyice(
                coef_nc.get_variable<Float>("lut_asyice", {n_rghice, n_band, n_size_ice}), {n_size_ice, n_band, n_rghice});

        return Cloud_optics_gpu(
                band_lims_wvn,
                radliq_lwr, radliq_upr, radliq_fac,
                radice_lwr, radice_upr, radice_fac,
                lut_extliq, lut_ssaliq, lut_asyliq,
                lut_extice, lut_ssaice, lut_asyice);
    }

    Aerosol_optics_gpu load_and_init_aerosol_optics(
            Master& master,
            const std::string& coef_file)
    {
        // READ THE COEFFICIENTS FOR THE OPTICAL SOLVER.
        Netcdf_file coef_nc(master, coef_file, Netcdf_mode::Read);

        // Read look-up table coefficient dimensions
        int n_band     = coef_nc.get_dimension_size("band_sw");
        int n_hum      = coef_nc.get_dimension_size("relative_humidity");
        int n_philic = coef_nc.get_dimension_size("hydrophilic");
        int n_phobic = coef_nc.get_dimension_size("hydrophobic");

        Array<Float,2> band_lims_wvn({2, n_band});

        Array<Float,2> mext_phobic(
                coef_nc.get_variable<Float>("mass_ext_sw_hydrophobic", {n_phobic, n_band}), {n_band, n_phobic});
        Array<Float,2> ssa_phobic(
                coef_nc.get_variable<Float>("ssa_sw_hydrophobic", {n_phobic, n_band}), {n_band, n_phobic});
        Array<Float,2> g_phobic(
                coef_nc.get_variable<Float>("asymmetry_sw_hydrophobic", {n_phobic, n_band}), {n_band, n_phobic});

        Array<Float,3> mext_philic(
                coef_nc.get_variable<Float>("mass_ext_sw_hydrophilic", {n_philic, n_hum, n_band}), {n_band, n_hum, n_philic});
        Array<Float,3> ssa_philic(
                coef_nc.get_variable<Float>("ssa_sw_hydrophilic", {n_philic, n_hum, n_band}), {n_band, n_hum, n_philic});
        Array<Float,3> g_philic(
                coef_nc.get_variable<Float>("asymmetry_sw_hydrophilic", {n_philic, n_hum, n_band}), {n_band, n_hum, n_philic});

        Array<Float,1> rh_upper(
                coef_nc.get_variable<Float>("relative_humidity2", {n_hum}), {n_hum});

        return Aerosol_optics_gpu(
                band_lims_wvn, rh_upper,
                mext_phobic, ssa_phobic, g_phobic,
                mext_philic, ssa_philic, g_philic);
    }

    // extremely unnecessary copy of load_init_gas_optics - we definitely need a proper construction
    Gas_optics_rrtmgp_rt load_and_init_gas_optics_rt(
            Master& master,
            const Gas_concs_gpu& gas_concs,
            const std::string& coef_file)
    {
        // READ THE COEFFICIENTS FOR THE OPTICAL SOLVER.
        Netcdf_file coef_nc(master, coef_file, Netcdf_mode::Read);

        // Read k-distribution information.
        const int n_temps = coef_nc.get_dimension_size("temperature");
        const int n_press = coef_nc.get_dimension_size("pressure");
        const int n_absorbers = coef_nc.get_dimension_size("absorber");
        const int n_char = coef_nc.get_dimension_size("string_len");
        const int n_minorabsorbers = coef_nc.get_dimension_size("minor_absorber");
        const int n_extabsorbers = coef_nc.get_dimension_size("absorber_ext");
        const int n_mixingfracs = coef_nc.get_dimension_size("mixing_fraction");
        const int n_layers = coef_nc.get_dimension_size("atmos_layer");
        const int n_bnds = coef_nc.get_dimension_size("bnd");
        const int n_gpts = coef_nc.get_dimension_size("gpt");
        const int n_pairs = coef_nc.get_dimension_size("pair");
        const int n_minor_absorber_intervals_lower = coef_nc.get_dimension_size("minor_absorber_intervals_lower");
        const int n_minor_absorber_intervals_upper = coef_nc.get_dimension_size("minor_absorber_intervals_upper");
        const int n_contributors_lower = coef_nc.get_dimension_size("contributors_lower");
        const int n_contributors_upper = coef_nc.get_dimension_size("contributors_upper");

        // Read gas names.
        Array<std::string,1> gas_names(
                get_variable_string("gas_names", {n_absorbers}, coef_nc, n_char, true), {n_absorbers});

        Array<int,3> key_species(
                coef_nc.get_variable<int>("key_species", {n_bnds, n_layers, 2}),
                {2, n_layers, n_bnds});
        Array<Float,2> band_lims(coef_nc.get_variable<Float>("bnd_limits_wavenumber", {n_bnds, 2}), {2, n_bnds});
        Array<int,2> band2gpt(coef_nc.get_variable<int>("bnd_limits_gpt", {n_bnds, 2}), {2, n_bnds});
        Array<Float,1> press_ref(coef_nc.get_variable<Float>("press_ref", {n_press}), {n_press});
        Array<Float,1> temp_ref(coef_nc.get_variable<Float>("temp_ref", {n_temps}), {n_temps});

        Float temp_ref_p = coef_nc.get_variable<Float>("absorption_coefficient_ref_P");
        Float temp_ref_t = coef_nc.get_variable<Float>("absorption_coefficient_ref_T");
        Float press_ref_trop = coef_nc.get_variable<Float>("press_ref_trop");

        Array<Float,3> kminor_lower(
                coef_nc.get_variable<Float>("kminor_lower", {n_temps, n_mixingfracs, n_contributors_lower}),
                {n_contributors_lower, n_mixingfracs, n_temps});
        Array<Float,3> kminor_upper(
                coef_nc.get_variable<Float>("kminor_upper", {n_temps, n_mixingfracs, n_contributors_upper}),
                {n_contributors_upper, n_mixingfracs, n_temps});

        Array<std::string,1> gas_minor(get_variable_string("gas_minor", {n_minorabsorbers}, coef_nc, n_char),
                                       {n_minorabsorbers});

        Array<std::string,1> identifier_minor(
                get_variable_string("identifier_minor", {n_minorabsorbers}, coef_nc, n_char), {n_minorabsorbers});

        Array<std::string,1> minor_gases_lower(
                get_variable_string("minor_gases_lower", {n_minor_absorber_intervals_lower}, coef_nc, n_char),
                {n_minor_absorber_intervals_lower});
        Array<std::string,1> minor_gases_upper(
                get_variable_string("minor_gases_upper", {n_minor_absorber_intervals_upper}, coef_nc, n_char),
                {n_minor_absorber_intervals_upper});

        Array<int,2> minor_limits_gpt_lower(
                coef_nc.get_variable<int>("minor_limits_gpt_lower", {n_minor_absorber_intervals_lower, n_pairs}),
                {n_pairs, n_minor_absorber_intervals_lower});
        Array<int,2> minor_limits_gpt_upper(
                coef_nc.get_variable<int>("minor_limits_gpt_upper", {n_minor_absorber_intervals_upper, n_pairs}),
                {n_pairs, n_minor_absorber_intervals_upper});

        Array<Bool,1> minor_scales_with_density_lower(
                coef_nc.get_variable<Bool>("minor_scales_with_density_lower", {n_minor_absorber_intervals_lower}),
                {n_minor_absorber_intervals_lower});
        Array<Bool,1> minor_scales_with_density_upper(
                coef_nc.get_variable<Bool>("minor_scales_with_density_upper", {n_minor_absorber_intervals_upper}),
                {n_minor_absorber_intervals_upper});

        Array<Bool,1> scale_by_complement_lower(
                coef_nc.get_variable<Bool>("scale_by_complement_lower", {n_minor_absorber_intervals_lower}),
                {n_minor_absorber_intervals_lower});
        Array<Bool,1> scale_by_complement_upper(
                coef_nc.get_variable<Bool>("scale_by_complement_upper", {n_minor_absorber_intervals_upper}),
                {n_minor_absorber_intervals_upper});

        Array<std::string,1> scaling_gas_lower(
                get_variable_string("scaling_gas_lower", {n_minor_absorber_intervals_lower}, coef_nc, n_char),
                {n_minor_absorber_intervals_lower});
        Array<std::string,1> scaling_gas_upper(
                get_variable_string("scaling_gas_upper", {n_minor_absorber_intervals_upper}, coef_nc, n_char),
                {n_minor_absorber_intervals_upper});

        Array<int,1> kminor_start_lower(
                coef_nc.get_variable<int>("kminor_start_lower", {n_minor_absorber_intervals_lower}),
                {n_minor_absorber_intervals_lower});
        Array<int,1> kminor_start_upper(
                coef_nc.get_variable<int>("kminor_start_upper", {n_minor_absorber_intervals_upper}),
                {n_minor_absorber_intervals_upper});

        Array<Float,3> vmr_ref(
                coef_nc.get_variable<Float>("vmr_ref", {n_temps, n_extabsorbers, n_layers}),
                {n_layers, n_extabsorbers, n_temps});

        Array<Float,4> kmajor(
                coef_nc.get_variable<Float>("kmajor", {n_temps, n_press+1, n_mixingfracs, n_gpts}),
                {n_gpts, n_mixingfracs, n_press+1, n_temps});

        // Keep the size at zero, if it does not exist.
        Array<Float,3> rayl_lower;
        Array<Float,3> rayl_upper;

        if (coef_nc.variable_exists("rayl_lower"))
        {
            rayl_lower.set_dims({n_gpts, n_mixingfracs, n_temps});
            rayl_upper.set_dims({n_gpts, n_mixingfracs, n_temps});
            rayl_lower = coef_nc.get_variable<Float>("rayl_lower", {n_temps, n_mixingfracs, n_gpts});
            rayl_upper = coef_nc.get_variable<Float>("rayl_upper", {n_temps, n_mixingfracs, n_gpts});
        }

        // Is it really LW if so read these variables as well.
        if (coef_nc.variable_exists("totplnk"))
        {
            int n_internal_sourcetemps = coef_nc.get_dimension_size("temperature_Planck");

            Array<Float,2> totplnk(
                    coef_nc.get_variable<Float>( "totplnk", {n_bnds, n_internal_sourcetemps}),
                    {n_internal_sourcetemps, n_bnds});
            Array<Float,4> planck_frac(
                    coef_nc.get_variable<Float>("plank_fraction", {n_temps, n_press+1, n_mixingfracs, n_gpts}),
                    {n_gpts, n_mixingfracs, n_press+1, n_temps});

            // Construct the k-distribution.
            return Gas_optics_rrtmgp_rt(
                    gas_concs,
                    gas_names,
                    key_species,
                    band2gpt,
                    band_lims,
                    press_ref,
                    press_ref_trop,
                    temp_ref,
                    temp_ref_p,
                    temp_ref_t,
                    vmr_ref,
                    kmajor,
                    kminor_lower,
                    kminor_upper,
                    gas_minor,
                    identifier_minor,
                    minor_gases_lower,
                    minor_gases_upper,
                    minor_limits_gpt_lower,
                    minor_limits_gpt_upper,
                    minor_scales_with_density_lower,
                    minor_scales_with_density_upper,
                    scaling_gas_lower,
                    scaling_gas_upper,
                    scale_by_complement_lower,
                    scale_by_complement_upper,
                    kminor_start_lower,
                    kminor_start_upper,
                    totplnk,
                    planck_frac,
                    rayl_lower,
                    rayl_upper);
        }
        else
        {
            Array<Float,1> solar_src_quiet(
                    coef_nc.get_variable<Float>("solar_source_quiet", {n_gpts}), {n_gpts});
            Array<Float,1> solar_src_facular(
                    coef_nc.get_variable<Float>("solar_source_facular", {n_gpts}), {n_gpts});
            Array<Float,1> solar_src_sunspot(
                    coef_nc.get_variable<Float>("solar_source_sunspot", {n_gpts}), {n_gpts});

            Float tsi = coef_nc.get_variable<Float>("tsi_default");
            Float mg_index = coef_nc.get_variable<Float>("mg_default");
            Float sb_index = coef_nc.get_variable<Float>("sb_default");

            return Gas_optics_rrtmgp_rt(
                    gas_concs,
                    gas_names,
                    key_species,
                    band2gpt,
                    band_lims,
                    press_ref,
                    press_ref_trop,
                    temp_ref,
                    temp_ref_p,
                    temp_ref_t,
                    vmr_ref,
                    kmajor,
                    kminor_lower,
                    kminor_upper,
                    gas_minor,
                    identifier_minor,
                    minor_gases_lower,
                    minor_gases_upper,
                    minor_limits_gpt_lower,
                    minor_limits_gpt_upper,
                    minor_scales_with_density_lower,
                    minor_scales_with_density_upper,
                    scaling_gas_lower,
                    scaling_gas_upper,
                    scale_by_complement_lower,
                    scale_by_complement_upper,
                    kminor_start_lower,
                    kminor_start_upper,
                    solar_src_quiet,
                    solar_src_facular,
                    solar_src_sunspot,
                    tsi,
                    mg_index,
                    sb_index,
                    rayl_lower,
                    rayl_upper);
        }
        // End reading of k-distribution.
    }

    Cloud_optics_rt load_and_init_cloud_optics_rt(
            Master& master,
            const std::string& coef_file)
    {
        // READ THE COEFFICIENTS FOR THE OPTICAL SOLVER.
        Netcdf_file coef_nc(master, coef_file, Netcdf_mode::Read);

        // Read look-up table coefficient dimensions
        int n_band     = coef_nc.get_dimension_size("nband");
        int n_rghice   = coef_nc.get_dimension_size("nrghice");
        int n_size_liq = coef_nc.get_dimension_size("nsize_liq");
        int n_size_ice = coef_nc.get_dimension_size("nsize_ice");

        Array<Float,2> band_lims_wvn(coef_nc.get_variable<Float>("bnd_limits_wavenumber", {n_band, 2}), {2, n_band});

        // Read look-up table constants.
        Float radliq_lwr = coef_nc.get_variable<Float>("radliq_lwr");
        Float radliq_upr = coef_nc.get_variable<Float>("radliq_upr");
        Float radliq_fac = coef_nc.get_variable<Float>("radliq_fac");

        Float radice_lwr = coef_nc.get_variable<Float>("radice_lwr");
        Float radice_upr = coef_nc.get_variable<Float>("radice_upr");
        Float radice_fac = coef_nc.get_variable<Float>("radice_fac");

        Array<Float,2> lut_extliq(
                coef_nc.get_variable<Float>("lut_extliq", {n_band, n_size_liq}), {n_size_liq, n_band});
        Array<Float,2> lut_ssaliq(
                coef_nc.get_variable<Float>("lut_ssaliq", {n_band, n_size_liq}), {n_size_liq, n_band});
        Array<Float,2> lut_asyliq(
                coef_nc.get_variable<Float>("lut_asyliq", {n_band, n_size_liq}), {n_size_liq, n_band});

        Array<Float,3> lut_extice(
                coef_nc.get_variable<Float>("lut_extice", {n_rghice, n_band, n_size_ice}), {n_size_ice, n_band, n_rghice});
        Array<Float,3> lut_ssaice(
                coef_nc.get_variable<Float>("lut_ssaice", {n_rghice, n_band, n_size_ice}), {n_size_ice, n_band, n_rghice});
        Array<Float,3> lut_asyice(
                coef_nc.get_variable<Float>("lut_asyice", {n_rghice, n_band, n_size_ice}), {n_size_ice, n_band, n_rghice});

        return Cloud_optics_rt(
                band_lims_wvn,
                radliq_lwr, radliq_upr, radliq_fac,
                radice_lwr, radice_upr, radice_fac,
                lut_extliq, lut_ssaliq, lut_asyliq,
                lut_extice, lut_ssaice, lut_asyice);
    }

    Aerosol_optics_rt load_and_init_aerosol_optics_rt(
            Master& master,
            const std::string& coef_file)
    {
        // READ THE COEFFICIENTS FOR THE OPTICAL SOLVER.
        Netcdf_file coef_nc(master, coef_file, Netcdf_mode::Read);

        // Read look-up table coefficient dimensions
        int n_band     = coef_nc.get_dimension_size("band_sw");
        int n_hum      = coef_nc.get_dimension_size("relative_humidity");
        int n_philic = coef_nc.get_dimension_size("hydrophilic");
        int n_phobic = coef_nc.get_dimension_size("hydrophobic");

        Array<Float,2> band_lims_wvn({2, n_band});

        Array<Float,2> mext_phobic(
                coef_nc.get_variable<Float>("mass_ext_sw_hydrophobic", {n_phobic, n_band}), {n_band, n_phobic});
        Array<Float,2> ssa_phobic(
                coef_nc.get_variable<Float>("ssa_sw_hydrophobic", {n_phobic, n_band}), {n_band, n_phobic});
        Array<Float,2> g_phobic(
                coef_nc.get_variable<Float>("asymmetry_sw_hydrophobic", {n_phobic, n_band}), {n_band, n_phobic});

        Array<Float,3> mext_philic(
                coef_nc.get_variable<Float>("mass_ext_sw_hydrophilic", {n_philic, n_hum, n_band}), {n_band, n_hum, n_philic});
        Array<Float,3> ssa_philic(
                coef_nc.get_variable<Float>("ssa_sw_hydrophilic", {n_philic, n_hum, n_band}), {n_band, n_hum, n_philic});
        Array<Float,3> g_philic(
                coef_nc.get_variable<Float>("asymmetry_sw_hydrophilic", {n_philic, n_hum, n_band}), {n_band, n_hum, n_philic});

        Array<Float,1> rh_upper(
                coef_nc.get_variable<Float>("relative_humidity2", {n_hum}), {n_hum});

        return Aerosol_optics_rt(
                band_lims_wvn, rh_upper,
                mext_phobic, ssa_phobic, g_phobic,
                mext_philic, ssa_philic, g_philic);
    }

    void configure_memory_pool(int nlays, int ncols, int nchunks, int ngpts, int nbnds)
    {
        #ifdef RTE_RRTMGP_GPU_MEMPOOL_OWN
        /* Heuristic way to set up memory pool queues */
        std::map<std::size_t, std::size_t> pool_queues = {
            {64, 20},
            {128, 20},
            {256, 10},
            {512, 10},
            {1024, 5},
            {2048, 5},
            {nchunks * ngpts * sizeof(Float), 16},
            {nchunks * nbnds * sizeof(Float), 16},
            {(nlays + 1) * ncols * sizeof(Float), 14},
            {(nlays + 1) * nchunks * sizeof(Float), 10},
            {(nlays + 1) * nchunks * nbnds * sizeof(Float), 4},
            {(nlays + 1) * nchunks * ngpts * sizeof(int)/2, 6},
            {(nlays + 1) * nchunks * ngpts * sizeof(Float), 18}
        };

        Memory_pool_gpu::init_instance(pool_queues);
        #endif
    }
}

#ifdef USECUDA
template<typename TF>
Float Radiation_rrtmgp_rt<TF>::calc_max_nogc(
            const Float* const restrict fld,
            const int itot, const int jtot, const int ktot)
{
    using namespace Tools_g;

    const Float scalefac = 1.;
    Float max_value;

    auto tmp = fields.get_tmp_g();
    //reduce_interior<TF>(
    //    fld, tmp->fld_g, gd.itot, gd.istart, gd.iend, gd.jtot,
    //    gd.jstart, gd.jend, gd.kcells, 0, gd.icells, gd.ijcells, Sum_type);

    // Reduce 3D field excluding ghost cells and padding to jtot*ktot values
    reduce_interior<Float>(fld, tmp->fld_g, itot, 0, itot, jtot, 0, jtot, ktot, 0, itot, itot*jtot, Max_type);
    // Reduce jtot*ktot to ktot values
    reduce_all<Float>     (tmp->fld_g, &tmp->fld_g[jtot*ktot], jtot*ktot, ktot, jtot, Max_type, scalefac);
    // Reduce ktot values to a single value
    reduce_all<Float>     (&tmp->fld_g[jtot*ktot], tmp->fld_g, ktot, 1, ktot, Max_type, scalefac);
    // Copy back result from GPU
    cuda_safe_call(hipMemcpy(&max_value, tmp->fld_g, sizeof(Float), hipMemcpyDeviceToHost));

    fields.release_tmp_g(tmp);

    return max_value;
}
#endif

#ifdef USECUDA
template<typename TF>
void Radiation_rrtmgp_rt<TF>::prepare_device()
{
    auto& gd = grid.get_grid_data();

    // Set the memory pool.
    int ngpt_pool = 0;
    int nbnd_pool = 0;

    if (sw_longwave)
    {
        Netcdf_file coef_nc_lw(master, "coefficients_lw.nc", Netcdf_mode::Read);
        nbnd_pool = std::max(coef_nc_lw.get_dimension_size("bnd"), nbnd_pool);
        ngpt_pool = std::max(coef_nc_lw.get_dimension_size("gpt"), ngpt_pool);
    }

    if (sw_shortwave)
    {
        Netcdf_file coef_nc_sw(master, "coefficients_sw.nc", Netcdf_mode::Read);
        nbnd_pool = std::max(coef_nc_sw.get_dimension_size("bnd"), nbnd_pool);
        ngpt_pool = std::max(coef_nc_sw.get_dimension_size("gpt"), ngpt_pool);
    }

    configure_memory_pool(gd.ktot, gd.imax*gd.jmax, 512, ngpt_pool, nbnd_pool);

    // Transfer the surface properties to the GPU
    emis_sfc_g = emis_sfc;
    sfc_alb_dir_g = sfc_alb_dir;
    sfc_alb_dif_g = sfc_alb_dif;

    // Initialize the pointers.
    this->gas_concs_gpu = std::make_unique<Gas_concs_gpu>(gas_concs);
    this->aerosol_concs_gpu = std::make_unique<Aerosol_concs_gpu>(aerosol_concs);

    const int nlaysize  = gd.ktot*sizeof(TF);
    for (auto& it : gaslist)
    {
        gasprofs_g.emplace(it, nullptr);
        cuda_safe_call(hipMalloc(&gasprofs_g.at(it), nlaysize));
        cuda_safe_call(hipMemcpy(gasprofs_g.at(it), gasprofs.at(it).data(), nlaysize, hipMemcpyHostToDevice));
    }

    if (sw_longwave)
    {
        this->kdist_lw_gpu = std::make_unique<Gas_optics_rrtmgp_gpu>(
                load_and_init_gas_optics(master, *gas_concs_gpu, "coefficients_lw.nc"));

        this->cloud_lw_gpu = std::make_unique<Cloud_optics_gpu>(
                load_and_init_cloud_optics(master, "cloud_coefficients_lw.nc"));

        const int nsfcsize = gd.ijcells*sizeof(Float);
        cuda_safe_call(hipMalloc(&lw_flux_dn_sfc_g, nsfcsize));
        cuda_safe_call(hipMalloc(&lw_flux_up_sfc_g, nsfcsize));

        const int ncolgptsize = n_col*kdist_lw_gpu->get_ngpt()*sizeof(Float);
        cuda_safe_call(hipMalloc(&lw_flux_dn_inc_g, ncolgptsize));

        cuda_safe_call(hipMemcpy(lw_flux_dn_inc_g, lw_flux_dn_inc.ptr(), ncolgptsize, hipMemcpyHostToDevice));
    }

    if (sw_shortwave)
    {
        this->kdist_sw_gpu = std::make_unique<Gas_optics_rrtmgp_gpu>(
                load_and_init_gas_optics(master, *gas_concs_gpu, "coefficients_sw.nc"));

        this->cloud_sw_gpu = std::make_unique<Cloud_optics_gpu>(
                load_and_init_cloud_optics(master, "cloud_coefficients_sw.nc"));

        if (sw_aerosol)
        {
            this->aerosol_sw_gpu = std::make_unique<Aerosol_optics_gpu>(
                    load_and_init_aerosol_optics(master, "aerosol_optics.nc"));
            cuda_safe_call(hipMalloc(&aod550_g, gd.imax*gd.jmax*sizeof(Float)));
        }
        this->kdist_sw_rt = std::make_unique<Gas_optics_rrtmgp_rt>(
                load_and_init_gas_optics_rt(master, *gas_concs_gpu, "coefficients_sw.nc"));

        this->cloud_sw_rt = std::make_unique<Cloud_optics_rt>(
                load_and_init_cloud_optics_rt(master, "cloud_coefficients_sw.nc"));

        if (sw_aerosol)
            this->aerosol_sw_rt = std::make_unique<Aerosol_optics_rt>(
                    load_and_init_aerosol_optics_rt(master, "aerosol_optics.nc"));

        const int nsfcsize = gd.ijcells*sizeof(Float);
        cuda_safe_call(hipMalloc(&sw_flux_dn_sfc_g, nsfcsize));
        cuda_safe_call(hipMalloc(&sw_flux_up_sfc_g, nsfcsize));

        cuda_safe_call(hipMalloc(&sw_flux_sfc_dir_rt_g, nsfcsize));
        cuda_safe_call(hipMalloc(&sw_flux_sfc_dif_rt_g, nsfcsize));
        cuda_safe_call(hipMalloc(&sw_flux_sfc_up_rt_g, nsfcsize));
        cuda_safe_call(hipMalloc(&sw_flux_tod_dn_rt_g, nsfcsize));
        cuda_safe_call(hipMalloc(&sw_flux_tod_up_rt_g, nsfcsize));

        const int ncolgptsize = n_col*kdist_sw_rt->get_ngpt()*sizeof(Float);
        cuda_safe_call(hipMalloc(&sw_flux_dn_dir_inc_g, ncolgptsize));
        cuda_safe_call(hipMalloc(&sw_flux_dn_dif_inc_g, ncolgptsize));

        cuda_safe_call(hipMemcpy(sw_flux_dn_dir_inc_g, sw_flux_dn_dir_inc.ptr(), ncolgptsize, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(sw_flux_dn_dif_inc_g, sw_flux_dn_dif_inc.ptr(), ncolgptsize, hipMemcpyHostToDevice));
    }
}
#endif


#ifdef USECUDA
template<typename TF>
void Radiation_rrtmgp_rt<TF>::exec_longwave(
        Thermo<TF>& thermo, Microphys<TF>& microphys, Timeloop<TF>& timeloop, Stats<TF>& stats,
        Array_gpu<Float,2>& flux_up, Array_gpu<Float,2>& flux_dn, Array_gpu<Float,2>& flux_net,
        const Array_gpu<Float,2>& t_lay, const Array_gpu<Float,2>& t_lev, const Array_gpu<Float,1>& t_sfc,
        const Array_gpu<Float,2>& h2o, const Array_gpu<Float,2>& clwp, const Array_gpu<Float,2>& ciwp,
        const bool compute_clouds)
{
    constexpr int n_col_block = 1024;

    auto& gd = grid.get_grid_data();

    const int n_col = gd.imax*gd.jmax;
    const int n_lay = gd.ktot;
    const int n_lev = gd.ktot+1;

    const int n_blocks = n_col / n_col_block;
    const int n_col_block_residual = n_col % n_col_block;

    const int n_gpt = this->kdist_lw_gpu->get_ngpt();
    const int n_bnd = this->kdist_lw_gpu->get_nband();

    const Bool top_at_1 = 0;

    // Define the pointers for the subsetting.
    std::unique_ptr<Optical_props_arry_gpu> optical_props_subset =
            std::make_unique<Optical_props_1scl_gpu>(n_col_block, n_lay, *kdist_lw_gpu);
    std::unique_ptr<Source_func_lw_gpu> sources_subset =
            std::make_unique<Source_func_lw_gpu>(n_col_block, n_lay, *kdist_lw_gpu);
    std::unique_ptr<Optical_props_1scl_gpu> cloud_optical_props_subset =
            std::make_unique<Optical_props_1scl_gpu>(n_col_block, n_lay, *cloud_lw_gpu);

    std::unique_ptr<Optical_props_arry_gpu> optical_props_residual =
            std::make_unique<Optical_props_1scl_gpu>(n_col_block_residual, n_lay, *kdist_lw_gpu);
    std::unique_ptr<Source_func_lw_gpu> sources_residual =
            std::make_unique<Source_func_lw_gpu>(n_col_block_residual, n_lay, *kdist_lw_gpu);
    std::unique_ptr<Optical_props_1scl_gpu> cloud_optical_props_residual =
            std::make_unique<Optical_props_1scl_gpu>(n_col_block_residual, n_lay, *cloud_lw_gpu);

    // Make view to the TOD flux pointers
    auto lw_flux_dn_inc_local = Array_gpu<Float,2>(lw_flux_dn_inc_g, {1, n_gpt});

    // Make views to the base state pointer.
    auto p_lay = Array_gpu<Float,2>(thermo.get_basestate_fld_g("pref") + gd.kstart, {1, n_lay});
    auto p_lev = Array_gpu<Float,2>(thermo.get_basestate_fld_g("prefh") + gd.kstart, {1, n_lev});

    gas_concs_gpu->set_vmr("h2o", h2o);

    // CvH: This can be done better: we now allocate a complete array.
    Array_gpu<Float,2> col_dry({n_col, n_lay});
    Gas_optics_rrtmgp_gpu::get_col_dry(col_dry, gas_concs_gpu->get_vmr("h2o"), p_lev.subset({{ {1, n_col}, {1, n_lev} }}));

    // Constants for computation of liquid and ice droplet effective radius
    const Float sig_g = 1.34;
    const Float fac = std::exp(std::log(sig_g)*std::log(sig_g)); // no conversion to micron yet.

    const TF Nc0 = microphys.get_Nc0();
    const TF Ni0 = microphys.get_Ni0();

    const Float four_third_pi_N0_rho_w = (4./3.)*M_PI*Nc0*Constants::rho_w<Float>;
    const Float four_third_pi_N0_rho_i = (4./3.)*M_PI*Ni0*Constants::rho_i<Float>;

    const int block_col = 16;
    const int block_lay = 16;
    const int grid_col  = n_col_block/block_col + (n_col_block%block_col > 0);
    const int grid_lay  = n_lay/block_lay + (n_lay%block_lay > 0);

    dim3 gridGPU_re (grid_col, grid_lay, 1);
    dim3 blockGPU_re (block_col, block_lay, 1);

    // Lambda function for solving optical properties subset.
    auto call_kernels = [&](
            const int col_s_in, const int col_e_in,
            std::unique_ptr<Optical_props_arry_gpu>& optical_props_subset_in,
            std::unique_ptr<Optical_props_1scl_gpu>& cloud_optical_props_subset_in,
            Source_func_lw_gpu& sources_subset_in,
            const Array_gpu<Float,2>& emis_sfc_subset_in,
            const Array_gpu<Float,2>& lw_flux_dn_inc_subset_in,
            Fluxes_broadband_gpu& fluxes,
            Fluxes_broadband_gpu& bnd_fluxes)
    {
        const int n_col_in = col_e_in - col_s_in + 1;
        Gas_concs_gpu gas_concs_subset(*gas_concs_gpu, col_s_in, n_col_in);

        auto p_lev_subset = p_lev.subset({{ {col_s_in, col_e_in}, {1, n_lev} }});

        kdist_lw_gpu->gas_optics(
                p_lay.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                p_lev_subset,
                t_lay.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                t_sfc.subset({{ {col_s_in, col_e_in} }}),
                gas_concs_subset,
                optical_props_subset_in,
                sources_subset_in,
                col_dry.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                t_lev.subset({{ {col_s_in, col_e_in}, {1, n_lev} }}) );


        if (compute_clouds)
        {
            auto clwp_subset = clwp.subset({{ {col_s_in, col_e_in}, {1, n_lay} }});
            auto ciwp_subset = ciwp.subset({{ {col_s_in, col_e_in}, {1, n_lay} }});
            Array_gpu<Float,2> rel({n_col_in, n_lay});
            Array_gpu<Float,2> rei({n_col_in, n_lay});

            effective_radius_and_ciwp_to_gm2<<<gridGPU_re, blockGPU_re>>>(
                    rel.ptr(), rei.ptr(),
                    clwp_subset.ptr(), ciwp_subset.ptr(),
                    gd.dz_g,
                    n_col_in, n_lay, gd.kstart,
                    four_third_pi_N0_rho_w, four_third_pi_N0_rho_i, fac);

            cloud_lw_gpu->cloud_optics(
                    clwp_subset,
                    ciwp_subset,
                    rel,
                    rei,
                    *cloud_optical_props_subset_in);

            // Add the cloud optical props to the gas optical properties.
            add_to(
                    dynamic_cast<Optical_props_1scl_gpu&>(*optical_props_subset_in),
                    dynamic_cast<Optical_props_1scl_gpu&>(*cloud_optical_props_subset_in));
        }

        Array_gpu<Float,3> gpt_flux_up({n_col_in, n_lev, n_gpt});
        Array_gpu<Float,3> gpt_flux_dn({n_col_in, n_lev, n_gpt});

        constexpr int n_ang = 1;

        rte_lw_gpu.rte_lw(
                optical_props_subset_in,
                top_at_1,
                sources_subset_in,
                emis_sfc_subset_in,
                lw_flux_dn_inc_subset_in,
                gpt_flux_up,
                gpt_flux_dn,
                n_ang);

        fluxes.reduce(gpt_flux_up, gpt_flux_dn, optical_props_subset_in, top_at_1);

        // Copy the data to the output.
        Subset_kernels_cuda::get_from_subset(
                n_col, n_lev, n_col_in, col_s_in, flux_up.ptr(), flux_dn.ptr(), flux_net.ptr(),
                fluxes.get_flux_up().ptr(), fluxes.get_flux_dn().ptr(), fluxes.get_flux_net().ptr());
    };

    for (int b=1; b<=n_blocks; ++b)
    {
        const int col_s = (b-1) * n_col_block + 1;
        const int col_e =  b    * n_col_block;

        Array_gpu<Float,2> emis_sfc_subset = emis_sfc_g.subset({{ {1, n_bnd}, {col_s, col_e} }});
        Array_gpu<Float,2> lw_flux_dn_inc_subset = lw_flux_dn_inc_local.subset({{ {col_s, col_e}, {1, n_gpt} }});

        std::unique_ptr<Fluxes_broadband_gpu> fluxes_subset =
                std::make_unique<Fluxes_broadband_gpu>(n_col_block, n_lev);
        std::unique_ptr<Fluxes_broadband_gpu> bnd_fluxes_subset =
                std::make_unique<Fluxes_byband_gpu>(n_col_block, n_lev, n_bnd);

        call_kernels(
                col_s, col_e,
                optical_props_subset,
                cloud_optical_props_subset,
                *sources_subset,
                emis_sfc_subset,
                lw_flux_dn_inc_subset,
                *fluxes_subset,
                *bnd_fluxes_subset);
    }

    if (n_col_block_residual > 0)
    {
        const int col_s = n_col - n_col_block_residual + 1;
        const int col_e = n_col;

        Array_gpu<Float,2> emis_sfc_residual = emis_sfc_g.subset({{ {1, n_bnd}, {col_s, col_e} }});
        Array_gpu<Float,2> lw_flux_dn_inc_residual = lw_flux_dn_inc_local.subset({{ {col_s, col_e}, {1, n_gpt} }});

        std::unique_ptr<Fluxes_broadband_gpu> fluxes_residual =
                std::make_unique<Fluxes_broadband_gpu>(n_col_block_residual, n_lev);
        std::unique_ptr<Fluxes_broadband_gpu> bnd_fluxes_residual =
                std::make_unique<Fluxes_byband_gpu>(n_col_block_residual, n_lev, n_bnd);

        call_kernels(
                col_s, col_e,
                optical_props_residual,
                cloud_optical_props_residual,
                *sources_residual,
                emis_sfc_residual,
                lw_flux_dn_inc_residual,
                *fluxes_residual,
                *bnd_fluxes_residual);
    }
}
#endif

#ifdef USECUDA
template<typename TF>
void Radiation_rrtmgp_rt<TF>::exec_shortwave(
        Thermo<TF>& thermo, Microphys<TF>& microphys, Timeloop<TF>& timeloop, Stats<TF>& stats,
        Array_gpu<Float,2>& flux_up, Array_gpu<Float,2>& flux_dn, Array_gpu<Float,2>& flux_dn_dir, Array_gpu<Float,2>& flux_net,
        const Array_gpu<Float,2>& t_lay, const Array_gpu<Float,2>& t_lev,
        const Array_gpu<Float,2>& h2o, const Array_gpu<Float,2>& rh,
        const Array_gpu<Float,2>& clwp, const Array_gpu<Float,2>& ciwp,
        const bool compute_clouds)
{
    constexpr int n_col_block = 1024;

    auto& gd = grid.get_grid_data();

    const int n_col = gd.imax*gd.jmax;
    const int n_lay = gd.ktot;
    const int n_lev = gd.ktot+1;

    const int n_blocks = n_col / n_col_block;
    const int n_col_block_residual = n_col % n_col_block;

    const int n_gpt = this->kdist_sw_gpu->get_ngpt();
    const int n_bnd = this->kdist_sw_gpu->get_nband();

    const Bool top_at_1 = 0;
    const bool do_radiation_stats = timeloop.is_stats_step();

    // Define the pointers for the subsetting.
    std::unique_ptr<Optical_props_arry_gpu> optical_props_subset =
            std::make_unique<Optical_props_2str_gpu>(n_col_block, n_lay, *kdist_sw_gpu);
    std::unique_ptr<Optical_props_2str_gpu> cloud_optical_props_subset =
            std::make_unique<Optical_props_2str_gpu>(n_col_block, n_lay, *cloud_sw_gpu);
    std::unique_ptr<Optical_props_2str_gpu> aerosol_optical_props_subset;
    if (sw_aerosol)
        aerosol_optical_props_subset = std::make_unique<Optical_props_2str_gpu>(n_col_block, n_lay, *aerosol_sw_gpu);

    std::unique_ptr<Optical_props_arry_gpu> optical_props_residual =
            std::make_unique<Optical_props_2str_gpu>(n_col_block_residual, n_lay, *kdist_sw_gpu);
    std::unique_ptr<Optical_props_2str_gpu> cloud_optical_props_residual =
            std::make_unique<Optical_props_2str_gpu>(n_col_block_residual, n_lay, *cloud_sw_gpu);
    std::unique_ptr<Optical_props_2str_gpu> aerosol_optical_props_residual;
    if (sw_aerosol)
        aerosol_optical_props_residual = std::make_unique<Optical_props_2str_gpu>(n_col_block_residual, n_lay, *aerosol_sw_gpu);

    // Make views to the base state pointer.
    auto p_lay = Array_gpu<Float,2>(thermo.get_basestate_fld_g("pref") + gd.kstart, {1, n_lay});
    auto p_lev = Array_gpu<Float,2>(thermo.get_basestate_fld_g("prefh") + gd.kstart, {1, n_lev});

    // Make views to the TOD flux pointers
    auto sw_flux_dn_dir_inc_local = Array_gpu<Float,2>(sw_flux_dn_dir_inc_g, {1, n_gpt});
    auto sw_flux_dn_dif_inc_local = Array_gpu<Float,2>(sw_flux_dn_dif_inc_g, {1, n_gpt});

    // Create the boundary conditions
    Array<Float,1> mu0_cpu(std::vector<Float>(1, this->mu0), {1});
    Array_gpu<Float,1> mu0(mu0_cpu);

    gas_concs_gpu->set_vmr("h2o", h2o);

    // CvH: This can be done better: we now allocate a complete array.
    Array_gpu<Float,2> col_dry({n_col, n_lay});
    Gas_optics_rrtmgp_gpu::get_col_dry(col_dry, gas_concs_gpu->get_vmr("h2o"), p_lev.subset({{ {1, n_col}, {1, n_lev} }}));

    // Constants for computation of liquid and ice droplet effective radius
    const Float sig_g = 1.34;
    const Float fac = std::exp(std::log(sig_g)*std::log(sig_g)); // no conversion to micron yet.

    const TF Nc0 = microphys.get_Nc0();
    const TF Ni0 = microphys.get_Ni0();

    const Float four_third_pi_N0_rho_w = (4./3.)*M_PI*Nc0*Constants::rho_w<Float>;
    const Float four_third_pi_N0_rho_i = (4./3.)*M_PI*Ni0*Constants::rho_i<Float>;

    const int block_col = 16;
    const int block_lay = 16;
    const int grid_col  = n_col_block/block_col + (n_col_block%block_col > 0);
    const int grid_lay  = n_lay/block_lay + (n_lay%block_lay > 0);

    dim3 gridGPU_re (grid_col, grid_lay, 1);
    dim3 blockGPU_re (block_col, block_lay, 1);

    // Lambda function for solving optical properties subset.
    auto call_kernels = [&](
            const int col_s_in, const int col_e_in,
            std::unique_ptr<Optical_props_arry_gpu>& optical_props_subset_in,
            std::unique_ptr<Optical_props_2str_gpu>& cloud_optical_props_subset_in,
            std::unique_ptr<Optical_props_2str_gpu>& aerosol_optical_props_subset_in,
            const Array_gpu<Float,1>& mu0_subset_in,
            const Array_gpu<Float,2>& sw_flux_dn_dir_inc_subset_in,
            const Array_gpu<Float,2>& sfc_alb_dir_subset_in,
            const Array_gpu<Float,2>& sfc_alb_dif_subset_in,
            const Array_gpu<Float,2>& sw_flux_dn_dif_inc_subset_in,
            Fluxes_broadband_gpu& fluxes,
            Fluxes_broadband_gpu& bnd_fluxes)
    {
        const int n_col_in = col_e_in - col_s_in + 1;
        Gas_concs_gpu gas_concs_subset(*gas_concs_gpu, col_s_in, n_col_in);
        Array_gpu<Float,2> toa_src_dummy({n_col_in, n_gpt});

        auto p_lev_subset = p_lev.subset({{ {col_s_in, col_e_in}, {1, n_lev} }});
        kdist_sw_gpu->gas_optics(
                p_lay.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                p_lev_subset,
                t_lay.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                gas_concs_subset,
                optical_props_subset_in,
                toa_src_dummy,
                col_dry.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}) );


        if (compute_clouds)
        {
            auto clwp_subset = clwp.subset({{ {col_s_in, col_e_in}, {1, n_lay} }});
            auto ciwp_subset = ciwp.subset({{ {col_s_in, col_e_in}, {1, n_lay} }});
            Array_gpu<Float,2> rel({n_col_in, n_lay});
            Array_gpu<Float,2> rei({n_col_in, n_lay});

            effective_radius_and_ciwp_to_gm2<<<gridGPU_re, blockGPU_re>>>(
                    rel.ptr(), rei.ptr(),
                    clwp_subset.ptr(), ciwp_subset.ptr(),
                    gd.dz_g,
                    n_col_in, n_lay, gd.kstart,
                    four_third_pi_N0_rho_w, four_third_pi_N0_rho_i, fac);

            cloud_sw_gpu->cloud_optics(
                    clwp_subset,
                    ciwp_subset,
                    rel,
                    rei,
                    *cloud_optical_props_subset_in);

            if (sw_delta_cloud)
                cloud_optical_props_subset_in->delta_scale();

            // Add the cloud optical props to the gas optical properties.
            add_to(
                    dynamic_cast<Optical_props_2str_gpu&>(*optical_props_subset_in),
                    dynamic_cast<Optical_props_2str_gpu&>(*cloud_optical_props_subset_in));
        }

        if (sw_aerosol)
        {
            Aerosol_concs_gpu aerosol_concs_subset(*aerosol_concs_gpu, col_s_in, n_col_in);
            aerosol_sw_gpu->aerosol_optics(
                    aerosol_concs_subset,
                    rh.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                    p_lev_subset,
                    *aerosol_optical_props_subset_in);

            if (sw_delta_aer)
                aerosol_optical_props_subset_in->delta_scale();

            // Add the cloud optical props to the gas optical properties.
            add_to(
                    dynamic_cast<Optical_props_2str_gpu&>(*optical_props_subset_in),
                    dynamic_cast<Optical_props_2str_gpu&>(*aerosol_optical_props_subset_in));

            if (do_radiation_stats)
                sum_tau(n_col_in, n_lay, col_s_in, aerosol_optical_props_subset_in->get_tau().ptr(), ibnd_550, aod550_g);
        }

        Array_gpu<Float,3> gpt_flux_up({n_col_in, n_lev, n_gpt});
        Array_gpu<Float,3> gpt_flux_dn({n_col_in, n_lev, n_gpt});
        Array_gpu<Float,3> gpt_flux_dn_dir({n_col_in, n_lev, n_gpt});

        rte_sw_gpu.rte_sw(
                optical_props_subset_in,
                top_at_1,
                mu0_subset_in,
                sw_flux_dn_dir_inc_subset_in,
                sfc_alb_dir_subset_in,
                sfc_alb_dif_subset_in,
                sw_flux_dn_dif_inc_subset_in,
                gpt_flux_up,
                gpt_flux_dn,
                gpt_flux_dn_dir);

        fluxes.reduce(gpt_flux_up, gpt_flux_dn, gpt_flux_dn_dir, optical_props_subset_in, top_at_1);

        // Copy the data to the output.
        Subset_kernels_cuda::get_from_subset(
                n_col, n_lev, n_col_in, col_s_in, flux_up.ptr(), flux_dn.ptr(), flux_dn_dir.ptr(), flux_net.ptr(),
                fluxes.get_flux_up().ptr(), fluxes.get_flux_dn().ptr(), fluxes.get_flux_dn_dir().ptr(), fluxes.get_flux_net().ptr());
    };

    if (sw_aerosol && do_radiation_stats)
    {
        const int nmemsize = gd.imax*gd.jmax * sizeof(TF);
        cuda_safe_call(hipMemcpy(aod550.ptr(), aod550_g, nmemsize, hipMemcpyDeviceToHost));
    }

    for (int b=1; b<=n_blocks; ++b)
    {
        const int col_s = (b-1) * n_col_block + 1;
        const int col_e =  b    * n_col_block;

        Array_gpu<Float,1> mu0_subset = mu0.subset({{ {col_s, col_e} }});
        Array_gpu<Float,2> sw_flux_dn_dir_inc_subset = sw_flux_dn_dir_inc_local.subset({{ {col_s, col_e}, {1, n_gpt} }});
        Array_gpu<Float,2> sfc_alb_dir_subset = sfc_alb_dir_g.subset({{ {1, n_bnd}, {col_s, col_e} }});
        Array_gpu<Float,2> sfc_alb_dif_subset = sfc_alb_dif_g.subset({{ {1, n_bnd}, {col_s, col_e} }});
        Array_gpu<Float,2> sw_flux_dn_dif_inc_subset = sw_flux_dn_dif_inc_local.subset({{ {col_s, col_e}, {1, n_gpt} }});

        std::unique_ptr<Fluxes_broadband_gpu> fluxes_subset =
                std::make_unique<Fluxes_broadband_gpu>(n_col_block, n_lev);
        std::unique_ptr<Fluxes_broadband_gpu> bnd_fluxes_subset =
                std::make_unique<Fluxes_byband_gpu>(n_col_block, n_lev, n_bnd);
        call_kernels(
                col_s, col_e,
                optical_props_subset,
                cloud_optical_props_subset,
                aerosol_optical_props_subset,
                mu0_subset,
                sw_flux_dn_dir_inc_subset,
                sfc_alb_dir_subset,
                sfc_alb_dif_subset,
                sw_flux_dn_dif_inc_subset,
                *fluxes_subset,
                *bnd_fluxes_subset);
    }

    if (n_col_block_residual > 0)
    {
        const int col_s = n_col - n_col_block_residual + 1;
        const int col_e = n_col;

        Array_gpu<Float,1> mu0_residual = mu0.subset({{ {col_s, col_e} }});
        Array_gpu<Float,2> sw_flux_dn_dir_inc_residual = sw_flux_dn_dir_inc_local.subset({{ {col_s, col_e}, {1, n_gpt} }});
        Array_gpu<Float,2> sfc_alb_dir_residual = sfc_alb_dir_g.subset({{ {1, n_bnd}, {col_s, col_e} }});
        Array_gpu<Float,2> sfc_alb_dif_residual = sfc_alb_dif_g.subset({{ {1, n_bnd}, {col_s, col_e} }});
        Array_gpu<Float,2> sw_flux_dn_dif_inc_residual = sw_flux_dn_dif_inc_local.subset({{ {col_s, col_e}, {1, n_gpt} }});

        std::unique_ptr<Fluxes_broadband_gpu> fluxes_residual =
                std::make_unique<Fluxes_broadband_gpu>(n_col_block_residual, n_lev);
        std::unique_ptr<Fluxes_broadband_gpu> bnd_fluxes_residual =
                std::make_unique<Fluxes_byband_gpu>(n_col_block_residual, n_lev, n_bnd);

        call_kernels(
                col_s, col_e,
                optical_props_residual,
                cloud_optical_props_residual,
                aerosol_optical_props_residual,
                mu0_residual,
                sw_flux_dn_dir_inc_residual,
                sfc_alb_dir_residual,
                sfc_alb_dif_residual,
                sw_flux_dn_dif_inc_residual,
                *fluxes_residual,
                *bnd_fluxes_residual);
    }
}
#endif

#ifdef USECUDA
template<typename TF>
void Radiation_rrtmgp_rt<TF>::exec_shortwave_rt(
        Thermo<TF>& thermo, Microphys<TF>& microphys, Timeloop<TF>& timeloop, Stats<TF>& stats,
        Array_gpu<Float,2>& flux_up, Array_gpu<Float,2>& flux_dn, Array_gpu<Float,2>& flux_dn_dir, Array_gpu<Float,2>& flux_net,
        Array_gpu<Float,2>& rt_flux_tod_dn, Array_gpu<Float,2>& rt_flux_tod_up, Array_gpu<Float,2>& rt_flux_sfc_dir, Array_gpu<Float,2>& rt_flux_sfc_dif,
        Array_gpu<Float,2>& rt_flux_sfc_up, Array_gpu<Float,3>& rt_flux_abs_dir, Array_gpu<Float,3>& rt_flux_abs_dif,
        const Array_gpu<Float,2>& t_lay, const Array_gpu<Float,2>& t_lev,
        const Array_gpu<Float,2>& h2o, const Array_gpu<Float,2>& rh,
        Array_gpu<Float,2>& clwp, Array_gpu<Float,2>& ciwp,
        const bool compute_clouds, const bool run_raytracer)
{
    auto& gd = grid.get_grid_data();

    const int n_col = gd.imax*gd.jmax;
    const int n_lay = gd.ktot;
    const int n_lev = gd.ktot+1;

    const int n_gpt = this->kdist_sw_rt->get_ngpt();
    const int n_bnd = this->kdist_sw_rt->get_nband();

    const Bool top_at_1 = 0;

    const Vector<int> grid_cells = {gd.imax, gd.jmax, gd.ktot};
    const Vector<Float> grid_d = {gd.dx, gd.dy, gd.dz[gd.kstart]};
    const Vector<int> kn_grid = {kngrid_i, kngrid_j, kngrid_k};

    // initiate flux & heating rate arrays to 0
    Gas_optics_rrtmgp_kernels_cuda_rt::zero_array(gd.jmax, gd.imax, rt_flux_tod_dn.ptr());
    Gas_optics_rrtmgp_kernels_cuda_rt::zero_array(gd.jmax, gd.imax, rt_flux_tod_up.ptr());
    Gas_optics_rrtmgp_kernels_cuda_rt::zero_array(gd.jmax, gd.imax, rt_flux_sfc_dir.ptr());
    Gas_optics_rrtmgp_kernels_cuda_rt::zero_array(gd.jmax, gd.imax, rt_flux_sfc_dif.ptr());
    Gas_optics_rrtmgp_kernels_cuda_rt::zero_array(gd.jmax, gd.imax, rt_flux_sfc_up.ptr());
    Gas_optics_rrtmgp_kernels_cuda_rt::zero_array(n_lay, gd.jmax, gd.imax, rt_flux_abs_dir.ptr());
    Gas_optics_rrtmgp_kernels_cuda_rt::zero_array(n_lay, gd.jmax, gd.imax, rt_flux_abs_dif.ptr());

    Gas_optics_rrtmgp_kernels_cuda_rt::zero_array(n_lev, gd.jmax, gd.imax, flux_up.ptr());
    Gas_optics_rrtmgp_kernels_cuda_rt::zero_array(n_lev, gd.jmax, gd.imax, flux_dn.ptr());
    Gas_optics_rrtmgp_kernels_cuda_rt::zero_array(n_lev, gd.jmax, gd.imax, flux_dn_dir.ptr());
    Gas_optics_rrtmgp_kernels_cuda_rt::zero_array(n_lev, gd.jmax, gd.imax, flux_net.ptr());


    // Define the pointers for the subsetting.
    std::unique_ptr<Optical_props_arry_rt> optical_props =
            std::make_unique<Optical_props_2str_rt>(n_col, n_lay, *kdist_sw_rt);
    std::unique_ptr<Optical_props_2str_rt> cloud_optical_props =
            std::make_unique<Optical_props_2str_rt>(n_col, n_lay, *cloud_sw_rt);
    std::unique_ptr<Optical_props_2str_rt> aerosol_optical_props;
    if (sw_aerosol)
        aerosol_optical_props = std::make_unique<Optical_props_2str_rt>(n_col, n_lay, *aerosol_sw_rt);
    else //initialise with cloud optics, pointer must exist here
        aerosol_optical_props = std::make_unique<Optical_props_2str_rt>(n_col, n_lay, *cloud_sw_rt);

    // Make views to the base state pointer.
    auto p_lay_tmp = Array_gpu<Float,2>(thermo.get_basestate_fld_g("pref") + gd.kstart, {1, n_lay});
    auto p_lev_tmp = Array_gpu<Float,2>(thermo.get_basestate_fld_g("prefh") + gd.kstart, {1, n_lev});

    // // Make TOD flux arrays
    Array_gpu<Float,1> sw_flux_dn_dir_inc_local({n_col});// = Array_gpu<Float,2>(sw_flux_dn_dir_inc_g, {1, n_gpt});
    Array_gpu<Float,1> sw_flux_dn_dif_inc_local({n_col});// = Array_gpu<Float,2>(sw_flux_dn_dif_inc_g, {1, n_gpt});

    // Create the boundary conditions
    Array<Float,1> mu0_cpu(std::vector<Float>(1, this->mu0), {1});
    Array_gpu<Float,1> mu0(mu0_cpu);

    gas_concs_gpu->set_vmr("h2o", h2o);

    // plev and play need column dimension
    auto p_lay = p_lay_tmp.subset({{ {1, n_col}, {1, n_lay} }});
    auto p_lev = p_lev_tmp.subset({{ {1, n_col}, {1, n_lev} }});

    // CvH: This can be done better: we now allocate a complete array.
    Array_gpu<Float,2> col_dry({n_col, n_lay});
    Gas_optics_rrtmgp_rt::get_col_dry(col_dry, gas_concs_gpu->get_vmr("h2o"), p_lev);

    // Array_gpu<Float,1> toa_src_dummy({n_col});

    // compute cloud eff radius and convert ice/liquid water path to g/m2
    Array_gpu<Float,2> rel;
    Array_gpu<Float,2> rei;

    if (compute_clouds)
    {
        // Constants for computation of liquid and ice droplet effective radius
        const Float sig_g = 1.34;
        const Float fac = std::exp(std::log(sig_g)*std::log(sig_g)); // no conversion to micron yet.

        const TF Nc0 = microphys.get_Nc0();
        const TF Ni0 = microphys.get_Ni0();

        const Float four_third_pi_N0_rho_w = (4./3.)*M_PI*Nc0*Constants::rho_w<Float>;
        const Float four_third_pi_N0_rho_i = (4./3.)*M_PI*Ni0*Constants::rho_i<Float>;

        const int block_col = 16;
        const int block_lay = 16;
        const int grid_col  = n_col/block_col + (n_col%block_col > 0);
        const int grid_lay  = n_lay/block_lay + (n_lay%block_lay > 0);

        dim3 gridGPU_re (grid_col, grid_lay, 1);
        dim3 blockGPU_re (block_col, block_lay, 1);

        rel.set_dims({n_col, n_lay});
        rei.set_dims({n_col, n_lay});

        effective_radius_and_ciwp_to_gm2<<<gridGPU_re, blockGPU_re>>>(
                rel.ptr(), rei.ptr(),
                clwp.ptr(), ciwp.ptr(),
                gd.dz_g,
                n_col, n_lay, gd.kstart,
                four_third_pi_N0_rho_w, four_third_pi_N0_rho_i, fac);
    }

    // main g-point loop
    const Array<int, 2>& band_limits_gpt(this->kdist_sw_rt->get_band_lims_gpoint());
    for (int igpt=1; igpt<=n_gpt; ++igpt)
    {
        int band = 0;
        for (int ibnd=1; ibnd<=n_bnd; ++ibnd)
        {
            if (igpt <= band_limits_gpt({2, ibnd}))
            {
                band = ibnd;
                break;
            }
        }

        /*
        kdist_sw_rt->gas_optics(
                igpt-1,
                p_lay,
                p_lev,
                t_lay,
                *gas_concs_gpu,
                optical_props,
                toa_src_dummy,
                col_dry);
                */

        // We loop over the gas optics, due to memory constraints
        constexpr int n_col_block = 1<<14; // 2^14

        Array_gpu<Float,1> toa_src_temp({n_col_block});
        auto gas_optics_subset = [&](
                const int col_s, const int n_col_subset)
        {
            // Run the gas_optics on a subset.
            kdist_sw_rt->gas_optics(
                    igpt,
                    col_s,
                    n_col_subset,
                    n_col,
                    p_lay,
                    p_lev,
                    t_lay,
                    gas_concs,
                    optical_props,
                    toa_src_temp,
                    col_dry);
        };

        const int n_blocks = n_col / n_col_block;
        const int n_col_residual = n_col % n_col_block;

        if (n_blocks > 0)
        {
            for (int n=0; n<n_blocks; ++n)
            {
                const int col_s = n*n_col_block;
                gas_optics_subset(col_s, n_col_block);
            }
        }

        if (n_col_residual > 0)
        {
            const int col_s = n_blocks*n_col_block;
            gas_optics_subset(col_s, n_col_residual);
        }

        if (compute_clouds)
        {
            cloud_sw_rt->cloud_optics(
                    band,
                    clwp,
                    ciwp,
                    rel,
                    rei,
                    *cloud_optical_props);

            if (sw_delta_cloud)
                cloud_optical_props->delta_scale();

            // Add the cloud optical props to the gas optical properties.
            add_to(
                    dynamic_cast<Optical_props_2str_rt&>(*optical_props),
                    dynamic_cast<Optical_props_2str_rt&>(*cloud_optical_props));
        }
        else
        {
            Gas_optics_rrtmgp_kernels_cuda_rt::zero_array(n_col, n_lay, cloud_optical_props->get_tau().ptr());
            Gas_optics_rrtmgp_kernels_cuda_rt::zero_array(n_col, n_lay, cloud_optical_props->get_ssa().ptr());
            Gas_optics_rrtmgp_kernels_cuda_rt::zero_array(n_col, n_lay, cloud_optical_props->get_g().ptr());
        }

        if (sw_aerosol)
        {
            aerosol_sw_rt->aerosol_optics(
                    band,
                    *aerosol_concs_gpu,
                    rh, p_lev,
                    *aerosol_optical_props);

            if (sw_delta_aer)
                aerosol_optical_props->delta_scale();

            // Add the cloud optical props to the gas optical properties.
            add_to(
                    dynamic_cast<Optical_props_2str_rt&>(*optical_props),
                    dynamic_cast<Optical_props_2str_rt&>(*aerosol_optical_props));
        }
        else
        {
            Gas_optics_rrtmgp_kernels_cuda_rt::zero_array(n_col, n_lay, aerosol_optical_props->get_tau().ptr());
            Gas_optics_rrtmgp_kernels_cuda_rt::zero_array(n_col, n_lay, aerosol_optical_props->get_ssa().ptr());
            Gas_optics_rrtmgp_kernels_cuda_rt::zero_array(n_col, n_lay, aerosol_optical_props->get_g().ptr());
        }

        std::unique_ptr<Fluxes_broadband_rt> fluxes =
                std::make_unique<Fluxes_broadband_rt>(gd.imax, gd.jmax, n_lev);

        sw_flux_dn_dir_inc_local.fill(sw_flux_dn_dir_inc({1, igpt}));
        sw_flux_dn_dif_inc_local.fill(sw_flux_dn_dif_inc({1, igpt}));

        rte_sw_rt.rte_sw(
                optical_props,
                top_at_1,
                mu0.subset({{ {1, n_col} }}),
                sw_flux_dn_dir_inc_local,
                sfc_alb_dir_g.subset({{ {band, band}, {1, n_col}} }),
                sfc_alb_dif_g.subset({{ {band, band}, {1, n_col}} }),
                sw_flux_dn_dif_inc_local,
                fluxes->get_flux_up(),
                fluxes->get_flux_dn(),
                fluxes->get_flux_dn_dir());

        fluxes->net_flux();

        Gpt_combine_kernels_cuda_rt::add_from_gpoint(
                  n_col, n_lev, flux_up.ptr(), flux_dn.ptr(), flux_dn_dir.ptr(), flux_net.ptr(),
                  fluxes->get_flux_up().ptr(), fluxes->get_flux_dn().ptr(), fluxes->get_flux_dn_dir().ptr(), fluxes->get_flux_net().ptr());


        // if we are here during night, it is just for tuning, no need to run ray tracer then
        if (!is_day(this->mu0))
            return;

        if (run_raytracer)
        {
            // CvH: this computation assumes that mu0 and azimuth are constant over the entire subset. Works for small LES only.
            Float zenith_angle = std::acos(mu0({1}));
            Float azimuth_angle = this->azimuth;

            Array_gpu<Float,2> mie_cdfs_sub;
            Array_gpu<Float,3> mie_angs_sub;

            const Int qrng_offset = Int(igpt - 1) + this->time_idx * Int(n_gpt);
            raytracer.trace_rays(
                    igpt,
                    this->rays_per_pixel,
                    grid_cells, grid_d, kn_grid,
                    mie_cdfs_sub,
                    mie_angs_sub,
                    dynamic_cast<Optical_props_2str_rt&>(*optical_props).get_tau(),
                    dynamic_cast<Optical_props_2str_rt&>(*optical_props).get_ssa(),
                    dynamic_cast<Optical_props_2str_rt&>(*cloud_optical_props).get_tau(),
                    dynamic_cast<Optical_props_2str_rt&>(*cloud_optical_props).get_ssa(),
                    dynamic_cast<Optical_props_2str_rt&>(*cloud_optical_props).get_g(),
                    dynamic_cast<Optical_props_2str_rt&>(*aerosol_optical_props).get_tau(),
                    dynamic_cast<Optical_props_2str_rt&>(*aerosol_optical_props).get_ssa(),
                    dynamic_cast<Optical_props_2str_rt&>(*aerosol_optical_props).get_g(),
                    rel, sfc_alb_dir.subset({{ {band, band}, {1, n_col} }}), zenith_angle,
                    azimuth_angle,
                    sw_flux_dn_dir_inc({1,igpt}) * mu0({1}), sw_flux_dn_dif_inc({1,igpt}),
                    fluxes->get_flux_tod_dn(),
                    fluxes->get_flux_tod_up(),
                    fluxes->get_flux_sfc_dir(),
                    fluxes->get_flux_sfc_dif(),
                    fluxes->get_flux_sfc_up(),
                    fluxes->get_flux_abs_dir(),
                    fluxes->get_flux_abs_dif());

            Gpt_combine_kernels_cuda_rt::add_from_gpoint(
                      gd.imax, gd.jmax, rt_flux_tod_dn.ptr(), rt_flux_tod_up.ptr(), rt_flux_sfc_dir.ptr(), rt_flux_sfc_dif.ptr(), rt_flux_sfc_up.ptr(),
                      fluxes->get_flux_tod_dn().ptr(), fluxes->get_flux_tod_up().ptr(), fluxes->get_flux_sfc_dir().ptr(), fluxes->get_flux_sfc_dif().ptr(), fluxes->get_flux_sfc_up().ptr());

            Gpt_combine_kernels_cuda_rt::add_from_gpoint(
                      n_col, n_lay, rt_flux_abs_dir.ptr(), rt_flux_abs_dif.ptr(),
                      fluxes->get_flux_abs_dir().ptr(), fluxes->get_flux_abs_dif().ptr());

        }
    }
    this->time_idx += Int(1);
}
#endif

#ifdef USECUDA
template <typename TF>
void Radiation_rrtmgp_rt<TF>::exec(
        Thermo<TF>& thermo, double time, Timeloop<TF>& timeloop, Stats<TF>& stats,
        Aerosol<TF>& aerosol, Background<TF>& background, Microphys<TF>& microphys)
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU_3d (gridi, gridj, gd.kmax+1);
    dim3 blockGPU_3d(blocki, blockj, 1);
    dim3 gridGPU_2d (gridi, gridj, 1);
    dim3 blockGPU_2d(blocki, blockj, 1);

    const bool do_radiation = ((timeloop.get_itime() % idt_rad == 0) && !timeloop.in_substep()) ;
    const bool do_radiation_stats = timeloop.is_stats_step();

    if (do_radiation)
    {
        auto homogenize = [&](TF* const __restrict__ field)
        {
            // Lambda function to homogenize 2D field
            const int blockGPU = 256;
            const int gridGPU = gd.ijcells/blockGPU + (gd.ijcells%blockGPU > 0);

            const TF mean_value = field3d_operators.calc_mean_2d_g(field);
            set_to_value<<<gridGPU, blockGPU>>>(field, gd.ijcells, mean_value);
        };

        // Set the tendency to zero.
        hipMemset(fields.sd.at("thlt_rad")->fld_g, 0, gd.ncells*sizeof(Float));

        auto t_lay = fields.get_tmp_g();
        auto t_lev = fields.get_tmp_g();
        auto h2o   = fields.get_tmp_g(); // This is the volume mixing ratio, not the specific humidity of vapor.
        auto rh    = fields.get_tmp_g();
        auto clwp  = fields.get_tmp_g();
        auto ciwp  = fields.get_tmp_g();

        // Set the input to the radiation on a 3D grid without ghost cells.
        //thermo.get_radiation_fields_g(*t_lay, *t_lev, *h2o, *clwp, *ciwp);
        thermo.get_radiation_fields_g(*t_lay, *t_lev, *h2o, *rh, *clwp, *ciwp);

        const Float lwp_max = calc_max_nogc(clwp->fld_g, gd.imax, gd.jmax, gd.ktot);
        const Float iwp_max = calc_max_nogc(ciwp->fld_g, gd.imax, gd.jmax, gd.ktot);
        Float liwp_max = lwp_max + iwp_max;

        const bool run_raytracer = (liwp_max > 0 || !sw_2str_when_no_clouds) ? true : false;

        const int nmaxh = gd.imax*gd.jmax*(gd.ktot+1);
        const int ijmax = gd.imax*gd.jmax;

        // Create views on existing variables.
        Array_gpu<Float,2> t_lay_a(t_lay->fld_g, {gd.imax*gd.jmax, gd.ktot});
        Array_gpu<Float,2> t_lev_a(t_lev->fld_g, {gd.imax*gd.jmax, gd.ktot+1});
        Array_gpu<Float,1> t_sfc_a(t_lev->fld_bot_g, {gd.imax*gd.jmax});
        Array_gpu<Float,2> h2o_a(h2o->fld_g, {gd.imax*gd.jmax, gd.ktot});
        Array_gpu<Float,2> rh_a(rh->fld_g, {gd.imax*gd.jmax, gd.ktot});
        Array_gpu<Float,2> clwp_a(clwp->fld_g, {gd.imax*gd.jmax, gd.ktot});
        Array_gpu<Float,2> ciwp_a(ciwp->fld_g, {gd.imax*gd.jmax, gd.ktot});


        const bool compute_clouds = true;

        // get aerosol mixing ratios
        if (sw_aerosol && swtimedep_aerosol)
            aerosol.get_radiation_fields(aerosol_concs_gpu);

        try
        {
            if (swtimedep_background)
            {
                // Temperature, pressure and moisture
                background.get_tpm(t_lay_col, t_lev_col, p_lay_col, p_lev_col, gas_concs_col);
                Gas_optics_rrtmgp::get_col_dry(col_dry, gas_concs_col.get_vmr("h2o"), p_lev_col);
                // gasses
                background.get_gasses(gas_concs_col);
                // aerosols
                if (sw_aerosol && swtimedep_aerosol)
                    background.get_aerosols(aerosol_concs_col);
            }

            if (sw_longwave)
            {
                // Flux fields.
                Array_gpu<Float,2> flux_up ({gd.imax*gd.jmax, gd.ktot+1});
                Array_gpu<Float,2> flux_dn ({gd.imax*gd.jmax, gd.ktot+1});
                Array_gpu<Float,2> flux_net({gd.imax*gd.jmax, gd.ktot+1});

                set_background_column_longwave(thermo);

                // Copy TOD flux to GPU
                const int n_gpt = kdist_lw->get_ngpt();
                const int ncolgptsize = n_col * n_gpt * sizeof(Float);
                cuda_safe_call(hipMemcpy(lw_flux_dn_inc_g, lw_flux_dn_inc.ptr(), ncolgptsize, hipMemcpyHostToDevice));

                exec_longwave(
                        thermo, microphys, timeloop, stats,
                        flux_up, flux_dn, flux_net,
                        t_lay_a, t_lev_a, t_sfc_a, h2o_a, clwp_a, ciwp_a,
                        compute_clouds);
                cuda_check_error();

                if (sw_homogenize_hr_lw)
                {
                    auto thlt = fields.get_tmp_g();
                    hipMemset(thlt->fld_g, 0, gd.ncells*sizeof(Float));

                    calc_tendency<<<gridGPU_3d, blockGPU_3d>>>(
                            thlt->fld_g,
                            flux_up.ptr(), flux_dn.ptr(),
                            fields.rhoref_g, thermo.get_basestate_fld_g("exner"),
                            gd.dz_g,
                            gd.istart, gd.jstart, gd.kstart,
                            gd.iend, gd.jend, gd.kend,
                            gd.igc, gd.jgc, gd.kgc,
                            gd.icells, gd.ijcells,
                            gd.imax, gd.imax*gd.jmax);
                    cuda_check_error();

                    field3d_operators.calc_mean_profile_g(thlt->fld_mean_g, thlt->fld_g);

                    add_profile<<<gridGPU_3d, blockGPU_3d>>>(
                            fields.sd.at("thlt_rad")->fld_g,
                            thlt->fld_mean_g,
                            gd.istart, gd.iend,
                            gd.jstart, gd.jend,
                            gd.kstart, gd.kend,
                            gd.icells, gd.ijcells);
                    cuda_check_error();

                    fields.release_tmp_g(thlt);
                }
                else
                {
                    calc_tendency<<<gridGPU_3d, blockGPU_3d>>>(
                            fields.sd.at("thlt_rad")->fld_g,
                            flux_up.ptr(), flux_dn.ptr(),
                            fields.rhoref_g, thermo.get_basestate_fld_g("exner"),
                            gd.dz_g,
                            gd.istart, gd.jstart, gd.kstart,
                            gd.iend, gd.jend, gd.kend,
                            gd.igc, gd.jgc, gd.kgc,
                            gd.icells, gd.ijcells,
                            gd.imax, gd.imax*gd.jmax);
                    cuda_check_error();
                }

                store_surface_fluxes<<<gridGPU_2d, blockGPU_2d>>>(
                        lw_flux_up_sfc_g, lw_flux_dn_sfc_g,
                        flux_up.ptr(), flux_dn.ptr(),
                        gd.istart, gd.iend,
                        gd.jstart, gd.jend,
                        gd.igc, gd.jgc,
                        gd.icells, gd.ijcells,
                        gd.imax);
                cuda_check_error();

                if (sw_homogenize_sfc_lw)
                {
                    homogenize(lw_flux_up_sfc_g);
                    homogenize(lw_flux_dn_sfc_g);
                }

                if (do_radiation_stats)
                {
                    // Make sure that the top boundary is taken into account in case of fluxes.
                    auto do_gcs = [&](Field3d<Float>& out, const Array_gpu<Float,2>& in)
                    {
                        add_ghost_cells_g<<<gridGPU_3d, blockGPU_3d>>>(
                                out.fld_g, in.ptr(),
                                gd.istart, gd.jstart, gd.kstart,
                                gd.iend, gd.jend, gd.kend+1,
                                gd.icells, gd.ijcells,
                                gd.imax, gd.imax*gd.jmax);
                    };

                    do_gcs(*fields.sd.at("lw_flux_up"), flux_up);
                    do_gcs(*fields.sd.at("lw_flux_dn"), flux_dn);

                    // clear sky
                    if (sw_clear_sky_stats)
                    {
                        exec_longwave(
                                thermo, microphys, timeloop, stats,
                                flux_up, flux_dn, flux_net,
                                t_lay_a, t_lev_a, t_sfc_a, h2o_a, clwp_a, ciwp_a,
                                !compute_clouds);

                        do_gcs(*fields.sd.at("lw_flux_up_clear"), flux_up);
                        do_gcs(*fields.sd.at("lw_flux_dn_clear"), flux_dn);
                    }
                }
            }

            if (sw_shortwave)
            {
                // Flux fields.
                Array_gpu<Float,2> flux_up ({gd.imax*gd.jmax, gd.ktot+1});
                Array_gpu<Float,2> flux_dn ({gd.imax*gd.jmax, gd.ktot+1});
                Array_gpu<Float,2> flux_net({gd.imax*gd.jmax, gd.ktot+1});
                Array_gpu<Float,2> flux_dn_dir({gd.imax*gd.jmax, gd.ktot+1});

                // RT fields.
                Array_gpu<Float,2> rt_flux_tod_dn({gd.imax,gd.jmax});
                Array_gpu<Float,2> rt_flux_tod_up({gd.imax,gd.jmax});
                Array_gpu<Float,2> rt_flux_sfc_dir({gd.imax,gd.jmax});
                Array_gpu<Float,2> rt_flux_sfc_dif({gd.imax,gd.jmax});
                Array_gpu<Float,2> rt_flux_sfc_up({gd.imax,gd.jmax});
                Array_gpu<Float,3> rt_flux_abs_dir({gd.imax,gd.jmax, gd.ktot});
                Array_gpu<Float,3> rt_flux_abs_dif({gd.imax,gd.jmax, gd.ktot});

                // Single column solve of background profile for TOA conditions
                if (!sw_fixed_sza)
                {
                    // Update the solar zenith angle, and calculate new shortwave reference column
                    const int day_of_year = int(timeloop.calc_day_of_year());
                    const int year = timeloop.get_year();
                    const Float seconds_after_midnight = Float(timeloop.calc_hour_of_day()*3600);
                    std::tie(this->mu0, this->azimuth) = calc_cos_zenith_angle(
                            gd.lat, gd.lon, day_of_year, seconds_after_midnight, year);

                    // Calculate correction factor for impact Sun's distance on the solar "constant"
                    const Float frac_day_of_year = Float(day_of_year) + seconds_after_midnight / Float(86400);
                    this->tsi_scaling = calc_sun_distance_factor(frac_day_of_year);
                }

                if (!sw_fixed_sza || swtimedep_background)
                {
                    if (is_day(this->mu0) || !sw_is_tuned)
                    {
                        const int n_bnd = kdist_sw->get_nband();
                        const int n_gpt = kdist_sw->get_ngpt();

                        // Set the solar zenith angle and albedo.
                        Array<Float,2> sfc_alb_dir({n_bnd, n_col});
                        Array<Float,2> sfc_alb_dif({n_bnd, n_col});

                        for (int ibnd=1; ibnd<=n_bnd; ++ibnd)
                        {
                            sfc_alb_dir({ibnd, 1}) = this->sfc_alb_dir_hom;
                            sfc_alb_dif({ibnd, 1}) = this->sfc_alb_dif_hom;
                        }

                        Array<Float,1> mu0({n_col});
                        mu0({1}) = this->mu0;

                        // sw column solve on cpu for TOD fluxes
                        solve_shortwave_column(
                                optical_props_sw,
                                aerosol_props_sw,
                                sw_flux_up_col, sw_flux_dn_col, sw_flux_dn_dir_col, sw_flux_net_col,
                                sw_flux_dn_dir_inc, sw_flux_dn_dif_inc, thermo.get_basestate_vector("ph")[gd.kend],
                                gas_concs_col,
                                *kdist_sw,
                                col_dry,
                                p_lay_col, p_lev_col,
                                t_lay_col, t_lev_col,
                                aerosol_concs_col,
                                mu0,
                                sfc_alb_dir, sfc_alb_dif,
                                tsi_scaling,
                                n_lay_col);

                        //TOD fluxes to CPU
                        const int ncolgptsize = n_col*n_gpt*sizeof(Float);
                        cuda_safe_call(hipMemcpy(sw_flux_dn_dir_inc_g,  sw_flux_dn_dir_inc.ptr(),  ncolgptsize, hipMemcpyHostToDevice));
                        cuda_safe_call(hipMemcpy(sw_flux_dn_dif_inc_g,  sw_flux_dn_dif_inc.ptr(),  ncolgptsize, hipMemcpyHostToDevice));
                    }
                }

                if (is_day(this->mu0) || !sw_is_tuned)
                {
                    exec_shortwave_rt(
                            thermo, microphys, timeloop, stats,
                            flux_up, flux_dn, flux_dn_dir, flux_net,
                            rt_flux_tod_dn, rt_flux_tod_up, rt_flux_sfc_dir, rt_flux_sfc_dif,
                            rt_flux_sfc_up, rt_flux_abs_dir, rt_flux_abs_dif,
                            t_lay_a, t_lev_a, h2o_a, rh_a, clwp_a, ciwp_a,
                            compute_clouds, run_raytracer);
                    cuda_check_error();

                    if (sw_homogenize_hr_sw)
                    {
                        auto thlt = fields.get_tmp_g();
                        hipMemset(thlt->fld_g, 0, gd.ncells*sizeof(Float));

                        if (run_raytracer)
                        {
                            calc_tendency_rt<<<gridGPU_3d, blockGPU_3d>>>(
                                    thlt->fld_g,
                                    rt_flux_abs_dir.ptr(), rt_flux_abs_dif.ptr(),
                                    fields.rhoref_g, thermo.get_basestate_fld_g("exner"),
                                    gd.dz_g,
                                    gd.istart, gd.jstart, gd.kstart,
                                    gd.iend, gd.jend, gd.kend,
                                    gd.igc, gd.jgc, gd.kgc,
                                    gd.icells, gd.ijcells,
                                    gd.imax, gd.imax*gd.jmax);
                            cuda_check_error();
                        }
                        else
                        {
                            calc_tendency<<<gridGPU_3d, blockGPU_3d>>>(
                                    thlt->fld_g,
                                    flux_up.ptr(), flux_dn.ptr(),
                                    fields.rhoref_g, thermo.get_basestate_fld_g("exner"),
                                    gd.dz_g,
                                    gd.istart, gd.jstart, gd.kstart,
                                    gd.iend, gd.jend, gd.kend,
                                    gd.igc, gd.jgc, gd.kgc,
                                    gd.icells, gd.ijcells,
                                    gd.imax, gd.imax*gd.jmax);
                            cuda_check_error();
                        }
                        field3d_operators.calc_mean_profile_g(thlt->fld_mean_g, thlt->fld_g);

                        add_profile<<<gridGPU_3d, blockGPU_3d>>>(
                                fields.sd.at("thlt_rad")->fld_g,
                                thlt->fld_mean_g,
                                gd.istart, gd.iend,
                                gd.jstart, gd.jend,
                                gd.kstart, gd.kend,
                                gd.icells, gd.ijcells);
                        cuda_check_error();

                        fields.release_tmp_g(thlt);
                    }
                    else
                    {
                        if (run_raytracer)
                        {
                            calc_tendency_rt<<<gridGPU_3d, blockGPU_3d>>>(
                                    fields.sd.at("thlt_rad")->fld_g,
                                    rt_flux_abs_dir.ptr(), rt_flux_abs_dif.ptr(),
                                    fields.rhoref_g, thermo.get_basestate_fld_g("exner"),
                                    gd.dz_g,
                                    gd.istart, gd.jstart, gd.kstart,
                                    gd.iend, gd.jend, gd.kend,
                                    gd.igc, gd.jgc, gd.kgc,
                                    gd.icells, gd.ijcells,
                                    gd.imax, gd.imax*gd.jmax);
                            cuda_check_error();
                        }
                        else
                        {
                            calc_tendency<<<gridGPU_3d, blockGPU_3d>>>(
                                    fields.sd.at("thlt_rad")->fld_g,
                                    flux_up.ptr(), flux_dn.ptr(),
                                    fields.rhoref_g, thermo.get_basestate_fld_g("exner"),
                                    gd.dz_g,
                                    gd.istart, gd.jstart, gd.kstart,
                                    gd.iend, gd.jend, gd.kend,
                                    gd.igc, gd.jgc, gd.kgc,
                                    gd.icells, gd.ijcells,
                                    gd.imax, gd.imax*gd.jmax);
                            cuda_check_error();


                        }
                    }

                    if (run_raytracer)
                    {
                        store_surface_fluxes_rt<<<gridGPU_2d, blockGPU_2d>>>(
                                sw_flux_up_sfc_g, sw_flux_dn_sfc_g,
                                rt_flux_sfc_dir.ptr(), rt_flux_sfc_dif.ptr(),
                                rt_flux_sfc_up.ptr(),
                                gd.istart, gd.iend,
                                gd.jstart, gd.jend,
                                gd.igc, gd.jgc,
                                gd.icells, gd.ijcells,
                                gd.imax);
                        cuda_check_error();
                    }
                    else
                    {
                        store_surface_fluxes<<<gridGPU_2d, blockGPU_2d>>>(
                                sw_flux_up_sfc_g, sw_flux_dn_sfc_g,
                                flux_up.ptr(), flux_dn.ptr(),
                                gd.istart, gd.iend,
                                gd.jstart, gd.jend,
                                gd.igc, gd.jgc,
                                gd.icells, gd.ijcells,
                                gd.imax);
                        cuda_check_error();
                    }

                    if (sw_homogenize_sfc_sw)
                    {
                        homogenize(sw_flux_up_sfc_g);
                        homogenize(sw_flux_dn_sfc_g);
                    }
                }
                // Note: keep this as a separate `if()` instead of an `else`,
                // we still want to zero everything if radiation was just calculated to tune the model.
                if(!is_day(this->mu0))
                {
                    // Set the surface fluxes to zero, for (e.g.) the land-surface model.
                    hipMemset(sw_flux_dn_sfc_g, 0, gd.ijcells*sizeof(Float));
                    hipMemset(sw_flux_up_sfc_g, 0, gd.ijcells*sizeof(Float));

                    // Set tendency to zero if sw was calculated just for tuning..
                    if(!sw_is_tuned)
                        hipMemset(fields.sd.at("thlt_rad")->fld_g, 0, gd.ncells*sizeof(Float));
                }

                if (!sw_is_tuned)
                    sw_is_tuned = true;

                if (do_radiation_stats)
                {
                    // Make sure that the top boundary is taken into account in case of fluxes.
                    auto do_gcs = [&](Field3d<Float>& out, const Array_gpu<Float,2>& in)
                    {
                        add_ghost_cells_g<<<gridGPU_3d, blockGPU_3d>>>(
                                out.fld_g, in.ptr(),
                                gd.istart, gd.jstart, gd.kstart,
                                gd.iend, gd.jend, gd.kend+1,
                                gd.icells, gd.ijcells,
                                gd.imax, gd.imax*gd.jmax);
                    };
                    auto do_gcs_rt = [&](Field3d<Float>& out, const Array_gpu<Float,3>& in)
                    {
                        add_ghost_cells_g<<<gridGPU_3d, blockGPU_3d>>>(
                                out.fld_g, in.ptr(),
                                gd.istart, gd.jstart, gd.kstart,
                                gd.iend, gd.jend, gd.kend,
                                gd.icells, gd.ijcells,
                                gd.imax, gd.imax*gd.jmax);
                    };

                    auto do_gcs_2d = [&](Float* out, const Array_gpu<Float,2>& in)
                    {
                        add_ghost_cells_2d_g<<<gridGPU_3d, blockGPU_3d>>>(
                                out, in.ptr(),
                                gd.istart, gd.jstart,
                                gd.iend, gd.jend,
                                gd.icells, gd.imax);
                    };

                    if (!is_day(this->mu0))
                    {
                        flux_up.fill(Float(0.));
                        flux_dn.fill(Float(0.));
                        flux_dn_dir.fill(Float(0.));
                    }
                    if (!is_day(this->mu0) || !run_raytracer)
                    {
                        rt_flux_abs_dir.fill(Float(0.));
                        rt_flux_abs_dif.fill(Float(0.));
                        rt_flux_sfc_dir.fill(Float(0.));
                        rt_flux_sfc_dif.fill(Float(0.));
                        rt_flux_sfc_up.fill(Float(0.));
                        rt_flux_tod_dn.fill(Float(0.));
                        rt_flux_tod_up.fill(Float(0.));
                    }

                    do_gcs(*fields.sd.at("sw_flux_up"), flux_up);
                    do_gcs(*fields.sd.at("sw_flux_dn"), flux_dn);
                    do_gcs(*fields.sd.at("sw_flux_dn_dir"), flux_dn_dir);

                    do_gcs_rt(*fields.sd.at("sw_heat_dir_rt"), rt_flux_abs_dir);
                    do_gcs_rt(*fields.sd.at("sw_heat_dif_rt"), rt_flux_abs_dif);

                    do_gcs_2d(sw_flux_sfc_dir_rt_g,rt_flux_sfc_dir);
                    do_gcs_2d(sw_flux_sfc_dif_rt_g,rt_flux_sfc_dif);
                    do_gcs_2d(sw_flux_sfc_up_rt_g,rt_flux_sfc_up);
                    do_gcs_2d(sw_flux_tod_dn_rt_g,rt_flux_tod_dn);
                    do_gcs_2d(sw_flux_tod_up_rt_g,rt_flux_tod_up);

                    // clear sky
                    if (sw_clear_sky_stats)
                    {
                        if (is_day(this->mu0))
                        {
                            exec_shortwave(
                                    thermo, microphys, timeloop, stats,
                                    flux_up, flux_dn, flux_dn_dir, flux_net,
                                    t_lay_a, t_lev_a, h2o_a, rh_a, clwp_a, ciwp_a,
                                    !compute_clouds);
                        }
                        do_gcs(*fields.sd.at("sw_flux_up_clear"), flux_up);
                        do_gcs(*fields.sd.at("sw_flux_dn_clear"), flux_dn);
                        do_gcs(*fields.sd.at("sw_flux_dn_dir_clear"), flux_dn_dir);
                    }
                }
            }
        } // End try block.
        catch (std::exception& e)
        {
             #ifdef USEMPI
            std::cout << "SINGLE PROCESS EXCEPTION: " << e.what() << std::endl;
            MPI_Abort(MPI_COMM_WORLD, 1);
            #else
            throw;
            #endif
        }

        fields.release_tmp_g(t_lay);
        fields.release_tmp_g(t_lev);
        fields.release_tmp_g(h2o);
        fields.release_tmp_g(rh);
        fields.release_tmp_g(clwp);
        fields.release_tmp_g(ciwp);
    }

    // Always add the tendency.
    add_tendency<<<gridGPU_3d, blockGPU_3d>>>(
            fields.st.at("thl")->fld_g,
            fields.sd.at("thlt_rad")->fld_g,
            gd.istart, gd.jstart, gd.kstart,
            gd.iend, gd.jend, gd.kend,
            gd.icells, gd.ijcells);
    cuda_check_error();

    hipDeviceSynchronize();
    stats.calc_tend(*fields.st.at("thl"), tend_name);
}

#ifdef USECUDA
template <typename TF>
void Radiation_rrtmgp_rt<TF>::update_time_dependent(Timeloop<TF>& timeloop)
{
    auto& gd = grid.get_grid_data();

    for (auto& it : tdep_gases)
    {
        it.second->update_time_dependent_prof_g(gasprofs_g.at(it.first), timeloop, gd.ktot);
        Array_gpu<Float,2> tmp_array(gasprofs_g.at(it.first), {1, int(gd.ktot)});
        gas_concs_gpu->set_vmr(it.first, tmp_array);
    }
}
#endif

template <typename TF>
std::vector<TF>& Radiation_rrtmgp_rt<TF>::get_surface_radiation(const std::string& name)
{
    throw std::runtime_error("Radiation_rrtmgp_rt is not implemented yet on the GPU");
}


template <typename TF>
TF* Radiation_rrtmgp_rt<TF>::get_surface_radiation_g(const std::string& name)
{
    // Check if short/longwave is active, otherwise the fields below are not allocated.
    if ((name == "sw_down" || name == "sw_up") && !sw_shortwave)
        throw std::runtime_error("get_surface_radiation_g() requires swshortwave=true & swlongwave=true.");
    else if ((name == "lw_down" || name == "lw_up") && !sw_longwave)
        throw std::runtime_error("get_surface_radiation_g() requires swshortwave=true & swlongwave=true.");

    if (name == "sw_down")
        return sw_flux_dn_sfc_g;
    else if (name == "sw_up")
        return sw_flux_up_sfc_g;
    else if (name == "lw_down")
        return lw_flux_dn_sfc_g;
    else if (name == "lw_up")
        return lw_flux_up_sfc_g;
    else
    {
        std::string error = "Variable \"" + name + "\" is not a valid surface radiation field";
        throw std::runtime_error(error);
    }
}


template <typename TF>
void Radiation_rrtmgp_rt<TF>::clear_device()
{
    cuda_safe_call(hipFree(lw_flux_dn_sfc_g));
    cuda_safe_call(hipFree(lw_flux_up_sfc_g));
    cuda_safe_call(hipFree(sw_flux_dn_sfc_g));
    cuda_safe_call(hipFree(sw_flux_up_sfc_g));

    for (auto& it : gasprofs_g)
        cuda_safe_call(hipFree(it.second));

    cuda_safe_call(hipFree(sw_flux_sfc_dir_rt_g));
    cuda_safe_call(hipFree(sw_flux_sfc_dif_rt_g));
    cuda_safe_call(hipFree(sw_flux_sfc_up_rt_g));
    cuda_safe_call(hipFree(sw_flux_tod_dn_rt_g));
    cuda_safe_call(hipFree(sw_flux_tod_up_rt_g));

    cuda_safe_call(hipFree(lw_flux_dn_inc_g));
    cuda_safe_call(hipFree(sw_flux_dn_dir_inc_g));
    cuda_safe_call(hipFree(sw_flux_dn_dif_inc_g));
}


template<typename TF>
void Radiation_rrtmgp_rt<TF>::exec_all_stats(
        Stats<TF>& stats, Cross<TF>& cross,
        Dump<TF>& dump, Column<TF>& column,
        Thermo<TF>& thermo, Timeloop<TF>& timeloop,
        const unsigned long itime, const int iotime)
{
    const bool do_stats  = stats.do_statistics(itime);
    const bool do_cross  = cross.do_cross(itime) && crosslist.size() > 0;
    const bool do_column = column.do_column(itime);

    // Return in case of no stats or cross section.
    if ( !(do_stats || do_cross || do_column) )
        return;

    const Float no_offset = 0.;
    const Float no_threshold = 0.;

    // CvH: lots of code repetition with exec()
    auto& gd = grid.get_grid_data();

    const bool compute_clouds = true;

    // Use a lambda function to avoid code repetition.
    auto save_stats_and_cross = [&](
            Field3d<TF>& array, const std::string& name, const std::array<int,3>& loc)
    {
        if (do_stats)
            stats.calc_stats(name, array, no_offset, no_threshold);

        if (do_cross)
        {
            if (std::find(crosslist.begin(), crosslist.end(), name) != crosslist.end())
                cross.cross_simple(array.fld.data(), no_offset, name, iotime, loc);
        }

        if (do_column)
            column.calc_column(name, array.fld_g.data(), no_offset);
    };

    if (sw_longwave)
    {
        save_stats_and_cross(*fields.sd.at("lw_flux_up"), "lw_flux_up", gd.wloc);
        save_stats_and_cross(*fields.sd.at("lw_flux_dn"), "lw_flux_dn", gd.wloc);

        if (sw_clear_sky_stats)
        {
            save_stats_and_cross(*fields.sd.at("lw_flux_up_clear"), "lw_flux_up_clear", gd.wloc);
            save_stats_and_cross(*fields.sd.at("lw_flux_dn_clear"), "lw_flux_dn_clear", gd.wloc);
        }

        if (do_stats && swtimedep_background)
        {
            stats.set_prof_background("lw_flux_up_ref", lw_flux_up_col.v());
            stats.set_prof_background("lw_flux_dn_ref", lw_flux_dn_col.v());
        }
    }

    if (sw_shortwave)
    {
        save_stats_and_cross(*fields.sd.at("sw_flux_up"), "sw_flux_up", gd.wloc);
        save_stats_and_cross(*fields.sd.at("sw_flux_dn"), "sw_flux_dn", gd.wloc);
        save_stats_and_cross(*fields.sd.at("sw_flux_dn_dir"), "sw_flux_dn_dir", gd.wloc);

        save_stats_and_cross(*fields.sd.at("sw_heat_dir_rt"), "sw_heat_dir_rt", gd.sloc);
        save_stats_and_cross(*fields.sd.at("sw_heat_dif_rt"), "sw_heat_dif_rt", gd.sloc);

        if (sw_clear_sky_stats)
        {
            save_stats_and_cross(*fields.sd.at("sw_flux_up_clear"), "sw_flux_up_clear", gd.wloc);
            save_stats_and_cross(*fields.sd.at("sw_flux_dn_clear"), "sw_flux_dn_clear", gd.wloc);
            save_stats_and_cross(*fields.sd.at("sw_flux_dn_dir_clear"), "sw_flux_dn_dir_clear", gd.wloc);
        }

        if (do_stats)
        {
            if (sw_aerosol)
            {
                // calc mean aod
                int ncol = gd.imax*gd.jmax;
                Float total_aod = 0;
                for (int icol = 1; icol <= ncol; ++icol)
                {
                    total_aod += aod550({icol});
                }
                Float mean_aod = total_aod/ncol;
                stats.set_time_series("AOD550", mean_aod);
            }

            if ((swtimedep_background || !sw_fixed_sza))
            {
                stats.set_prof_background("sw_flux_up_ref", sw_flux_up_col.v());
                stats.set_prof_background("sw_flux_dn_ref", sw_flux_dn_col.v());
                stats.set_prof_background("sw_flux_dn_dir_ref", sw_flux_dn_dir_col.v());
            }
        }

        const int nsfcsize = gd.ijcells*sizeof(Float);
        cuda_safe_call(hipMemcpy(sw_flux_sfc_dir_rt.data(), sw_flux_sfc_dir_rt_g, nsfcsize, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(sw_flux_sfc_dif_rt.data(), sw_flux_sfc_dif_rt_g, nsfcsize, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(sw_flux_sfc_up_rt.data(), sw_flux_sfc_up_rt_g, nsfcsize, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(sw_flux_tod_dn_rt.data(), sw_flux_tod_dn_rt_g, nsfcsize, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(sw_flux_tod_up_rt.data(), sw_flux_tod_up_rt_g, nsfcsize, hipMemcpyDeviceToHost));

        if (do_stats)
        {
            stats.calc_stats_2d("sw_flux_sfc_dir_rt", sw_flux_sfc_dir_rt, no_offset);
            stats.calc_stats_2d("sw_flux_sfc_dif_rt", sw_flux_sfc_dif_rt, no_offset);
            stats.calc_stats_2d("sw_flux_sfc_up_rt", sw_flux_sfc_up_rt, no_offset);
            stats.calc_stats_2d("sw_flux_tod_dn_rt", sw_flux_tod_dn_rt, no_offset);
            stats.calc_stats_2d("sw_flux_tod_up_rt", sw_flux_tod_up_rt, no_offset);
        }

        if (do_cross)
        {
            if (std::find(crosslist.begin(), crosslist.end(), "sw_flux_sfc_dir_rt") != crosslist.end())
                cross.cross_plane(sw_flux_sfc_dir_rt.data(), no_offset, "sw_flux_sfc_dir_rt", iotime);
            if (std::find(crosslist.begin(), crosslist.end(), "sw_flux_sfc_dif_rt") != crosslist.end())
                cross.cross_plane(sw_flux_sfc_dif_rt.data(), no_offset, "sw_flux_sfc_dif_rt", iotime);
            if (std::find(crosslist.begin(), crosslist.end(), "sw_flux_sfc_up_rt") != crosslist.end())
                cross.cross_plane(sw_flux_sfc_up_rt.data(), no_offset, "sw_flux_sfc_up_rt", iotime);
            if (std::find(crosslist.begin(), crosslist.end(), "sw_flux_tod_dn_rt") != crosslist.end())
                cross.cross_plane(sw_flux_tod_dn_rt.data(), no_offset, "sw_flux_tod_dn_rt", iotime);
            if (std::find(crosslist.begin(), crosslist.end(), "sw_flux_tod_up_rt") != crosslist.end())
                cross.cross_plane(sw_flux_tod_up_rt.data(), no_offset, "sw_flux_tod_up_rt", iotime);
        }
    }

    if (do_stats)
    {
        stats.set_time_series("sza", std::acos(mu0));
        stats.set_time_series("saa", azimuth);
        stats.set_time_series("tsi_scaling", this->tsi_scaling);
        stats.set_time_series("sw_flux_dn_toa", sw_flux_dn_col({1,n_lev_col}));
    }
}
#endif


#ifdef FLOAT_SINGLE
template class Radiation_rrtmgp_rt<float>;
#else
template class Radiation_rrtmgp_rt<double>;
#endif
