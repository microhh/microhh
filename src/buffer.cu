#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2020 Chiel van Heerwaarden
 * Copyright (c) 2011-2020 Thijs Heus
 * Copyright (c) 2014-2020 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <cstdio>
#include <iostream>
#include <cmath>
#include <stdlib.h>
#include "grid.h"
#include "fields.h"
#include "buffer.h"
#include "constants.h"
#include "stats.h"
#include "tools.h"

namespace
{
    template<typename TF>__global__
    void buffer_g(TF* __restrict__ at,   TF* __restrict__ a,
                  TF* __restrict__ abuf, TF* __restrict__ z,
                  TF zstart, TF zsizebufi, TF sigma,  TF beta,
                  int istart, int jstart, int bufferkstart,
                  int iend,   int jend,   int kend,
                  int jj, int kk)
    {
        __shared__ TF sigmaz;

        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + bufferkstart;

        /* sigmaz only depends on height. Let one thread calculate it to shared memory,
           other threads re-use value */
        if (threadIdx.x == 0 && threadIdx.y == 0)
            sigmaz = sigma * pow((z[k]-zstart)*zsizebufi, beta);
        __syncthreads();

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            at[ijk] -= sigmaz*(a[ijk]-abuf[k]);
        }
    }
}

template<typename TF>
void Buffer<TF>::prepare_device()
{
    const Grid_data<TF>& gd = grid.get_grid_data();

    if (swbuffer)
    {
        const int nmemsize = gd.kcells*sizeof(TF);

        // Allocate the buffer arrays at GPU.
        for (auto& it : fields.ap)
        {
            bufferprofs_g.emplace(it.first, nullptr);
            cuda_safe_call(hipMalloc(&bufferprofs_g.at(it.first), nmemsize));
        }

        // Copy buffers to GPU.
        for (auto& it : fields.ap)
            cuda_safe_call(hipMemcpy(bufferprofs_g.at(it.first), bufferprofs.at(it.first).data(), nmemsize, hipMemcpyHostToDevice));
    }
}

template<typename TF>
void Buffer<TF>::clear_device()
{
    if (swbuffer)
    {
        for (auto& it : bufferprofs_g)
            cuda_safe_call(hipFree(it.second));
    }
}

#ifdef USECUDA
template<typename TF>
void Buffer<TF>::exec(Stats<TF>& stats)
{
    if (swbuffer)
    {
        const Grid_data<TF>& gd = grid.get_grid_data();

        const int blocki = gd.ithread_block;
        const int blockj = gd.jthread_block;
        const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
        const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);
        const int gridk  = gd.kmax - (bufferkstart-gd.kgc);

        dim3 gridGPU (gridi, gridj, gridk);
        dim3 blockGPU(blocki, blockj, 1);

        const TF zsizebufi = 1./(gd.zsize-zstart);

        if (swupdate)
        {
            buffer_g<TF><<<gridGPU, blockGPU>>>(
                fields.mt.at("u")->fld_g, fields.mp.at("u")->fld_g,
                fields.mp.at("u")->fld_mean_g, gd.z_g,
                zstart, zsizebufi, sigma, beta,
                gd.istart, gd.jstart, bufferkstart,
                gd.iend,   gd.jend,   gd.kend,
                gd.icells, gd.ijcells);
            cuda_check_error();

            buffer_g<TF><<<gridGPU, blockGPU>>>(
                fields.mt.at("v")->fld_g, fields.mp.at("v")->fld_g,
                fields.mp.at("v")->fld_mean_g, gd.z_g,
                zstart, zsizebufi, sigma, beta,
                gd.istart, gd.jstart, bufferkstart,
                gd.iend,   gd.jend,   gd.kend,
                gd.icells, gd.ijcells);
            cuda_check_error();

            buffer_g<TF><<<gridGPU, blockGPU>>>(
                fields.mt.at("w")->fld_g, fields.mp.at("w")->fld_g,
                fields.mp.at("w")->fld_mean_g, gd.zh_g,
                zstart, zsizebufi, sigma, beta,
                gd.istart, gd.jstart, bufferkstarth,
                gd.iend,   gd.jend,   gd.kend,
                gd.icells, gd.ijcells);
            cuda_check_error();

            for (auto& it : fields.sp)
                buffer_g<TF><<<gridGPU, blockGPU>>>(
                    fields.st.at(it.first)->fld_g, fields.sp.at(it.first)->fld_g,
                    fields.sp.at(it.first)->fld_mean_g, gd.z_g,
                    zstart, zsizebufi, sigma, beta,
                    gd.istart, gd.jstart, bufferkstart,
                    gd.iend,   gd.jend,   gd.kend,
                    gd.icells, gd.ijcells);
            cuda_check_error();
        }
        else
        {
            buffer_g<TF><<<gridGPU, blockGPU>>>(
                fields.mt.at("u")->fld_g, fields.mp.at("u")->fld_g,
                bufferprofs_g.at("u"), gd.z_g,
                zstart, zsizebufi, sigma, beta,
                gd.istart, gd.jstart, bufferkstart,
                gd.iend,   gd.jend,   gd.kend,
                gd.icells, gd.ijcells);
            cuda_check_error();

            buffer_g<TF><<<gridGPU, blockGPU>>>(
                fields.mt.at("v")->fld_g, fields.mp.at("v")->fld_g,
                bufferprofs_g.at("v"), gd.z_g,
                zstart, zsizebufi, sigma, beta,
                gd.istart, gd.jstart, bufferkstart,
                gd.iend,   gd.jend,   gd.kend,
                gd.icells, gd.ijcells);
            cuda_check_error();

            buffer_g<TF><<<gridGPU, blockGPU>>>(
                fields.mt.at("w")->fld_g, fields.mp.at("w")->fld_g,
                bufferprofs_g.at("w"), gd.zh_g,
                zstart, zsizebufi, sigma, beta,
                gd.istart, gd.jstart, bufferkstarth,
                gd.iend,   gd.jend,   gd.kend,
                gd.icells, gd.ijcells);
            cuda_check_error();

            for (auto& it : fields.sp)
                buffer_g<TF><<<gridGPU, blockGPU>>>(
                    fields.st.at(it.first)->fld_g, fields.sp.at(it.first)->fld_g,
                    bufferprofs_g.at(it.first), gd.z_g,
                    zstart, zsizebufi, sigma, beta,
                    gd.istart, gd.jstart, bufferkstart,
                    gd.iend,   gd.jend,   gd.kend,
                    gd.icells, gd.ijcells);
            cuda_check_error();
        }

        hipDeviceSynchronize();

        stats.calc_tend(*fields.mt.at("u"), tend_name);
        stats.calc_tend(*fields.mt.at("v"), tend_name);
        stats.calc_tend(*fields.mt.at("w"), tend_name);
        for (auto it : fields.st)
            stats.calc_tend(*it.second, tend_name);
    }
}
#endif

template class Buffer<double>;
template class Buffer<float>;
