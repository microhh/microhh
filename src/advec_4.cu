#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2020 Chiel van Heerwaarden
 * Copyright (c) 2011-2020 Thijs Heus
 * Copyright (c) 2014-2020 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "advec_4.h"
#include "grid.h"
#include "fields.h"
#include "stats.h"
#include "tools.h"
#include "constants.h"
#include "finite_difference.h"
#include "field3d_operators.h"

using namespace Finite_difference::O4;

namespace
{
    template<typename TF> __global__
    void advec_u_g(TF* __restrict__ ut, const TF* __restrict__ u,
                   const TF* __restrict__ v,  const TF* __restrict__ w,
                   const TF* __restrict__ dzi4, const TF dxi, const TF dyi,
                   const int jj, const int kk,
                   const int istart, const int jstart, const int kstart,
                   const int iend,   const int jend,   const int kend)
        {
            const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
            const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
            const int k = blockIdx.z + kstart;

            const int ii1 = 1;
            const int ii2 = 2;
            const int ii3 = 3;
            const int jj1 = 1*jj;
            const int jj2 = 2*jj;
            const int jj3 = 3*jj;
            const int kk1 = 1*kk;
            const int kk2 = 2*kk;
            const int kk3 = 3*kk;

            if (i < iend && j < jend && k > kstart && k < kend-1)
            {
                const int ijk = i + j*jj + k*kk;
                ut[ijk] -= ( cg0<TF>*((ci0<TF>*u[ijk-ii3] + ci1<TF>*u[ijk-ii2] + ci2<TF>*u[ijk-ii1] + ci3<TF>*u[ijk    ]) * (ci0<TF>*u[ijk-ii3] + ci1<TF>*u[ijk-ii2] + ci2<TF>*u[ijk-ii1] + ci3<TF>*u[ijk    ]))
                           + cg1<TF>*((ci0<TF>*u[ijk-ii2] + ci1<TF>*u[ijk-ii1] + ci2<TF>*u[ijk    ] + ci3<TF>*u[ijk+ii1]) * (ci0<TF>*u[ijk-ii2] + ci1<TF>*u[ijk-ii1] + ci2<TF>*u[ijk    ] + ci3<TF>*u[ijk+ii1]))
                           + cg2<TF>*((ci0<TF>*u[ijk-ii1] + ci1<TF>*u[ijk    ] + ci2<TF>*u[ijk+ii1] + ci3<TF>*u[ijk+ii2]) * (ci0<TF>*u[ijk-ii1] + ci1<TF>*u[ijk    ] + ci2<TF>*u[ijk+ii1] + ci3<TF>*u[ijk+ii2]))
                           + cg3<TF>*((ci0<TF>*u[ijk    ] + ci1<TF>*u[ijk+ii1] + ci2<TF>*u[ijk+ii2] + ci3<TF>*u[ijk+ii3]) * (ci0<TF>*u[ijk    ] + ci1<TF>*u[ijk+ii1] + ci2<TF>*u[ijk+ii2] + ci3<TF>*u[ijk+ii3])) ) * dxi;

                ut[ijk] -= ( cg0<TF>*((ci0<TF>*v[ijk-ii2-jj1] + ci1<TF>*v[ijk-ii1-jj1] + ci2<TF>*v[ijk-jj1] + ci3<TF>*v[ijk+ii1-jj1]) * (ci0<TF>*u[ijk-jj3] + ci1<TF>*u[ijk-jj2] + ci2<TF>*u[ijk-jj1] + ci3<TF>*u[ijk    ]))
                           + cg1<TF>*((ci0<TF>*v[ijk-ii2    ] + ci1<TF>*v[ijk-ii1    ] + ci2<TF>*v[ijk    ] + ci3<TF>*v[ijk+ii1    ]) * (ci0<TF>*u[ijk-jj2] + ci1<TF>*u[ijk-jj1] + ci2<TF>*u[ijk    ] + ci3<TF>*u[ijk+jj1]))
                           + cg2<TF>*((ci0<TF>*v[ijk-ii2+jj1] + ci1<TF>*v[ijk-ii1+jj1] + ci2<TF>*v[ijk+jj1] + ci3<TF>*v[ijk+ii1+jj1]) * (ci0<TF>*u[ijk-jj1] + ci1<TF>*u[ijk    ] + ci2<TF>*u[ijk+jj1] + ci3<TF>*u[ijk+jj2]))
                           + cg3<TF>*((ci0<TF>*v[ijk-ii2+jj2] + ci1<TF>*v[ijk-ii1+jj2] + ci2<TF>*v[ijk+jj2] + ci3<TF>*v[ijk+ii1+jj2]) * (ci0<TF>*u[ijk    ] + ci1<TF>*u[ijk+jj1] + ci2<TF>*u[ijk+jj2] + ci3<TF>*u[ijk+jj3])) ) * dyi;

                ut[ijk] -= ( cg0<TF>*((ci0<TF>*w[ijk-ii2-kk1] + ci1<TF>*w[ijk-ii1-kk1] + ci2<TF>*w[ijk-kk1] + ci3<TF>*w[ijk+ii1-kk1]) * (ci0<TF>*u[ijk-kk3] + ci1<TF>*u[ijk-kk2] + ci2<TF>*u[ijk-kk1] + ci3<TF>*u[ijk    ]))
                           + cg1<TF>*((ci0<TF>*w[ijk-ii2    ] + ci1<TF>*w[ijk-ii1    ] + ci2<TF>*w[ijk    ] + ci3<TF>*w[ijk+ii1    ]) * (ci0<TF>*u[ijk-kk2] + ci1<TF>*u[ijk-kk1] + ci2<TF>*u[ijk    ] + ci3<TF>*u[ijk+kk1]))
                           + cg2<TF>*((ci0<TF>*w[ijk-ii2+kk1] + ci1<TF>*w[ijk-ii1+kk1] + ci2<TF>*w[ijk+kk1] + ci3<TF>*w[ijk+ii1+kk1]) * (ci0<TF>*u[ijk-kk1] + ci1<TF>*u[ijk    ] + ci2<TF>*u[ijk+kk1] + ci3<TF>*u[ijk+kk2]))
                           + cg3<TF>*((ci0<TF>*w[ijk-ii2+kk2] + ci1<TF>*w[ijk-ii1+kk2] + ci2<TF>*w[ijk+kk2] + ci3<TF>*w[ijk+ii1+kk2]) * (ci0<TF>*u[ijk    ] + ci1<TF>*u[ijk+kk1] + ci2<TF>*u[ijk+kk2] + ci3<TF>*u[ijk+kk3])) ) * dzi4[k];
            }
        }

    template<typename TF, int loc> __global__
    void advec_u_boundary_g(TF* __restrict__ ut, const TF* __restrict__ u,
                            const TF* __restrict__ v,  const TF* __restrict__ w,
                            const TF* __restrict__ dzi4, const TF dxi, const TF dyi,
                            const int jj, const int kk,
                            const int istart, const int jstart, const int kstart,
                            const int iend,   const int jend,   const int kend)
        {
            const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
            const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;

            const int ii1 = 1;
            const int ii2 = 2;
            const int ii3 = 3;
            const int jj1 = 1*jj;
            const int jj2 = 2*jj;
            const int jj3 = 3*jj;
            const int kk1 = 1*kk;
            const int kk2 = 2*kk;
            const int kk3 = 3*kk;

            if (i < iend && j < jend)
            {
                if (loc == 0)
                {
                    const int k = kstart;
                    const int ijk = i + j*jj + k*kk;

                    ut[ijk] -= ( cg0<TF>*((ci0<TF>*u[ijk-ii3] + ci1<TF>*u[ijk-ii2] + ci2<TF>*u[ijk-ii1] + ci3<TF>*u[ijk    ]) * (ci0<TF>*u[ijk-ii3] + ci1<TF>*u[ijk-ii2] + ci2<TF>*u[ijk-ii1] + ci3<TF>*u[ijk    ]))
                               + cg1<TF>*((ci0<TF>*u[ijk-ii2] + ci1<TF>*u[ijk-ii1] + ci2<TF>*u[ijk    ] + ci3<TF>*u[ijk+ii1]) * (ci0<TF>*u[ijk-ii2] + ci1<TF>*u[ijk-ii1] + ci2<TF>*u[ijk    ] + ci3<TF>*u[ijk+ii1]))
                               + cg2<TF>*((ci0<TF>*u[ijk-ii1] + ci1<TF>*u[ijk    ] + ci2<TF>*u[ijk+ii1] + ci3<TF>*u[ijk+ii2]) * (ci0<TF>*u[ijk-ii1] + ci1<TF>*u[ijk    ] + ci2<TF>*u[ijk+ii1] + ci3<TF>*u[ijk+ii2]))
                               + cg3<TF>*((ci0<TF>*u[ijk    ] + ci1<TF>*u[ijk+ii1] + ci2<TF>*u[ijk+ii2] + ci3<TF>*u[ijk+ii3]) * (ci0<TF>*u[ijk    ] + ci1<TF>*u[ijk+ii1] + ci2<TF>*u[ijk+ii2] + ci3<TF>*u[ijk+ii3])) ) * dxi;

                    ut[ijk] -= ( cg0<TF>*((ci0<TF>*v[ijk-ii2-jj1] + ci1<TF>*v[ijk-ii1-jj1] + ci2<TF>*v[ijk-jj1] + ci3<TF>*v[ijk+ii1-jj1]) * (ci0<TF>*u[ijk-jj3] + ci1<TF>*u[ijk-jj2] + ci2<TF>*u[ijk-jj1] + ci3<TF>*u[ijk    ]))
                               + cg1<TF>*((ci0<TF>*v[ijk-ii2    ] + ci1<TF>*v[ijk-ii1    ] + ci2<TF>*v[ijk    ] + ci3<TF>*v[ijk+ii1    ]) * (ci0<TF>*u[ijk-jj2] + ci1<TF>*u[ijk-jj1] + ci2<TF>*u[ijk    ] + ci3<TF>*u[ijk+jj1]))
                               + cg2<TF>*((ci0<TF>*v[ijk-ii2+jj1] + ci1<TF>*v[ijk-ii1+jj1] + ci2<TF>*v[ijk+jj1] + ci3<TF>*v[ijk+ii1+jj1]) * (ci0<TF>*u[ijk-jj1] + ci1<TF>*u[ijk    ] + ci2<TF>*u[ijk+jj1] + ci3<TF>*u[ijk+jj2]))
                               + cg3<TF>*((ci0<TF>*v[ijk-ii2+jj2] + ci1<TF>*v[ijk-ii1+jj2] + ci2<TF>*v[ijk+jj2] + ci3<TF>*v[ijk+ii1+jj2]) * (ci0<TF>*u[ijk    ] + ci1<TF>*u[ijk+jj1] + ci2<TF>*u[ijk+jj2] + ci3<TF>*u[ijk+jj3])) ) * dyi;

                    ut[ijk] -= ( cg0<TF>*((ci0<TF>*w[ijk-ii2-kk1] + ci1<TF>*w[ijk-ii1-kk1] + ci2<TF>*w[ijk-kk1] + ci3<TF>*w[ijk+ii1-kk1]) * (bi0<TF>*u[ijk-kk2] + bi1<TF>*u[ijk-kk1] + bi2<TF>*u[ijk    ] + bi3<TF>*u[ijk+kk1]))
                               + cg1<TF>*((ci0<TF>*w[ijk-ii2    ] + ci1<TF>*w[ijk-ii1    ] + ci2<TF>*w[ijk    ] + ci3<TF>*w[ijk+ii1    ]) * (ci0<TF>*u[ijk-kk2] + ci1<TF>*u[ijk-kk1] + ci2<TF>*u[ijk    ] + ci3<TF>*u[ijk+kk1]))
                               + cg2<TF>*((ci0<TF>*w[ijk-ii2+kk1] + ci1<TF>*w[ijk-ii1+kk1] + ci2<TF>*w[ijk+kk1] + ci3<TF>*w[ijk+ii1+kk1]) * (ci0<TF>*u[ijk-kk1] + ci1<TF>*u[ijk    ] + ci2<TF>*u[ijk+kk1] + ci3<TF>*u[ijk+kk2]))
                               + cg3<TF>*((ci0<TF>*w[ijk-ii2+kk2] + ci1<TF>*w[ijk-ii1+kk2] + ci2<TF>*w[ijk+kk2] + ci3<TF>*w[ijk+ii1+kk2]) * (ci0<TF>*u[ijk    ] + ci1<TF>*u[ijk+kk1] + ci2<TF>*u[ijk+kk2] + ci3<TF>*u[ijk+kk3])) ) * dzi4[k];
                }
                else if (loc == 1)
                {
                    const int k = kend-1;
                    const int ijk = i + j*jj + k*kk;

                    ut[ijk] -= ( cg0<TF>*((ci0<TF>*u[ijk-ii3] + ci1<TF>*u[ijk-ii2] + ci2<TF>*u[ijk-ii1] + ci3<TF>*u[ijk    ]) * (ci0<TF>*u[ijk-ii3] + ci1<TF>*u[ijk-ii2] + ci2<TF>*u[ijk-ii1] + ci3<TF>*u[ijk    ]))
                               + cg1<TF>*((ci0<TF>*u[ijk-ii2] + ci1<TF>*u[ijk-ii1] + ci2<TF>*u[ijk    ] + ci3<TF>*u[ijk+ii1]) * (ci0<TF>*u[ijk-ii2] + ci1<TF>*u[ijk-ii1] + ci2<TF>*u[ijk    ] + ci3<TF>*u[ijk+ii1]))
                               + cg2<TF>*((ci0<TF>*u[ijk-ii1] + ci1<TF>*u[ijk    ] + ci2<TF>*u[ijk+ii1] + ci3<TF>*u[ijk+ii2]) * (ci0<TF>*u[ijk-ii1] + ci1<TF>*u[ijk    ] + ci2<TF>*u[ijk+ii1] + ci3<TF>*u[ijk+ii2]))
                               + cg3<TF>*((ci0<TF>*u[ijk    ] + ci1<TF>*u[ijk+ii1] + ci2<TF>*u[ijk+ii2] + ci3<TF>*u[ijk+ii3]) * (ci0<TF>*u[ijk    ] + ci1<TF>*u[ijk+ii1] + ci2<TF>*u[ijk+ii2] + ci3<TF>*u[ijk+ii3])) ) * dxi;

                    ut[ijk] -= ( cg0<TF>*((ci0<TF>*v[ijk-ii2-jj1] + ci1<TF>*v[ijk-ii1-jj1] + ci2<TF>*v[ijk-jj1] + ci3<TF>*v[ijk+ii1-jj1]) * (ci0<TF>*u[ijk-jj3] + ci1<TF>*u[ijk-jj2] + ci2<TF>*u[ijk-jj1] + ci3<TF>*u[ijk    ]))
                               + cg1<TF>*((ci0<TF>*v[ijk-ii2    ] + ci1<TF>*v[ijk-ii1    ] + ci2<TF>*v[ijk    ] + ci3<TF>*v[ijk+ii1    ]) * (ci0<TF>*u[ijk-jj2] + ci1<TF>*u[ijk-jj1] + ci2<TF>*u[ijk    ] + ci3<TF>*u[ijk+jj1]))
                               + cg2<TF>*((ci0<TF>*v[ijk-ii2+jj1] + ci1<TF>*v[ijk-ii1+jj1] + ci2<TF>*v[ijk+jj1] + ci3<TF>*v[ijk+ii1+jj1]) * (ci0<TF>*u[ijk-jj1] + ci1<TF>*u[ijk    ] + ci2<TF>*u[ijk+jj1] + ci3<TF>*u[ijk+jj2]))
                               + cg3<TF>*((ci0<TF>*v[ijk-ii2+jj2] + ci1<TF>*v[ijk-ii1+jj2] + ci2<TF>*v[ijk+jj2] + ci3<TF>*v[ijk+ii1+jj2]) * (ci0<TF>*u[ijk    ] + ci1<TF>*u[ijk+jj1] + ci2<TF>*u[ijk+jj2] + ci3<TF>*u[ijk+jj3])) ) * dyi;

                    ut[ijk] -= ( cg0<TF>*((ci0<TF>*w[ijk-ii2-kk1] + ci1<TF>*w[ijk-ii1-kk1] + ci2<TF>*w[ijk-kk1] + ci3<TF>*w[ijk+ii1-kk1]) * (ci0<TF>*u[ijk-kk3] + ci1<TF>*u[ijk-kk2] + ci2<TF>*u[ijk-kk1] + ci3<TF>*u[ijk    ]))
                               + cg1<TF>*((ci0<TF>*w[ijk-ii2    ] + ci1<TF>*w[ijk-ii1    ] + ci2<TF>*w[ijk    ] + ci3<TF>*w[ijk+ii1    ]) * (ci0<TF>*u[ijk-kk2] + ci1<TF>*u[ijk-kk1] + ci2<TF>*u[ijk    ] + ci3<TF>*u[ijk+kk1]))
                               + cg2<TF>*((ci0<TF>*w[ijk-ii2+kk1] + ci1<TF>*w[ijk-ii1+kk1] + ci2<TF>*w[ijk+kk1] + ci3<TF>*w[ijk+ii1+kk1]) * (ci0<TF>*u[ijk-kk1] + ci1<TF>*u[ijk    ] + ci2<TF>*u[ijk+kk1] + ci3<TF>*u[ijk+kk2]))
                               + cg3<TF>*((ci0<TF>*w[ijk-ii2+kk2] + ci1<TF>*w[ijk-ii1+kk2] + ci2<TF>*w[ijk+kk2] + ci3<TF>*w[ijk+ii1+kk2]) * (ti0<TF>*u[ijk-kk1] + ti1<TF>*u[ijk    ] + ti2<TF>*u[ijk+kk1] + ti3<TF>*u[ijk+kk2])) ) * dzi4[k];
                }
            }
        }


    template<typename TF> __global__
    void advec_v_g(TF* __restrict__ vt, const TF* __restrict__ u,
                   const TF* __restrict__ v,  const TF* __restrict__ w,
                   const TF* __restrict__ dzi4, const TF dxi, const TF dyi,
                   const int jj, const int kk,
                   const int istart, const int jstart, const int kstart,
                   const int iend,   const int jend,   const int kend)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        const int ii1 = 1;
        const int ii2 = 2;
        const int ii3 = 3;
        const int jj1 = 1*jj;
        const int jj2 = 2*jj;
        const int jj3 = 3*jj;
        const int kk1 = 1*kk;
        const int kk2 = 2*kk;
        const int kk3 = 3*kk;

        if (i < iend && j < jend && k > kstart && k < kend-1)
        {
            const int ijk = i + j*jj + k*kk;

            vt[ijk] -= ( cg0<TF>*((ci0<TF>*u[ijk-ii1-jj2] + ci1<TF>*u[ijk-ii1-jj1] + ci2<TF>*u[ijk-ii1] + ci3<TF>*u[ijk-ii1+jj1]) * (ci0<TF>*v[ijk-ii3] + ci1<TF>*v[ijk-ii2] + ci2<TF>*v[ijk-ii1] + ci3<TF>*v[ijk    ]))
                       + cg1<TF>*((ci0<TF>*u[ijk    -jj2] + ci1<TF>*u[ijk    -jj1] + ci2<TF>*u[ijk    ] + ci3<TF>*u[ijk    +jj1]) * (ci0<TF>*v[ijk-ii2] + ci1<TF>*v[ijk-ii1] + ci2<TF>*v[ijk    ] + ci3<TF>*v[ijk+ii1]))
                       + cg2<TF>*((ci0<TF>*u[ijk+ii1-jj2] + ci1<TF>*u[ijk+ii1-jj1] + ci2<TF>*u[ijk+ii1] + ci3<TF>*u[ijk+ii1+jj1]) * (ci0<TF>*v[ijk-ii1] + ci1<TF>*v[ijk    ] + ci2<TF>*v[ijk+ii1] + ci3<TF>*v[ijk+ii2]))
                       + cg3<TF>*((ci0<TF>*u[ijk+ii2-jj2] + ci1<TF>*u[ijk+ii2-jj1] + ci2<TF>*u[ijk+ii2] + ci3<TF>*u[ijk+ii2+jj1]) * (ci0<TF>*v[ijk    ] + ci1<TF>*v[ijk+ii1] + ci2<TF>*v[ijk+ii2] + ci3<TF>*v[ijk+ii3])) ) * dxi;

            vt[ijk] -= ( cg0<TF>*((ci0<TF>*v[ijk-jj3] + ci1<TF>*v[ijk-jj2] + ci2<TF>*v[ijk-jj1] + ci3<TF>*v[ijk    ]) * (ci0<TF>*v[ijk-jj3] + ci1<TF>*v[ijk-jj2] + ci2<TF>*v[ijk-jj1] + ci3<TF>*v[ijk    ]))
                       + cg1<TF>*((ci0<TF>*v[ijk-jj2] + ci1<TF>*v[ijk-jj1] + ci2<TF>*v[ijk    ] + ci3<TF>*v[ijk+jj1]) * (ci0<TF>*v[ijk-jj2] + ci1<TF>*v[ijk-jj1] + ci2<TF>*v[ijk    ] + ci3<TF>*v[ijk+jj1]))
                       + cg2<TF>*((ci0<TF>*v[ijk-jj1] + ci1<TF>*v[ijk    ] + ci2<TF>*v[ijk+jj1] + ci3<TF>*v[ijk+jj2]) * (ci0<TF>*v[ijk-jj1] + ci1<TF>*v[ijk    ] + ci2<TF>*v[ijk+jj1] + ci3<TF>*v[ijk+jj2]))
                       + cg3<TF>*((ci0<TF>*v[ijk    ] + ci1<TF>*v[ijk+jj1] + ci2<TF>*v[ijk+jj2] + ci3<TF>*v[ijk+jj3]) * (ci0<TF>*v[ijk    ] + ci1<TF>*v[ijk+jj1] + ci2<TF>*v[ijk+jj2] + ci3<TF>*v[ijk+jj3])) ) * dyi;

            vt[ijk] -= ( cg0<TF>*((ci0<TF>*w[ijk-jj2-kk1] + ci1<TF>*w[ijk-jj1-kk1] + ci2<TF>*w[ijk-kk1] + ci3<TF>*w[ijk+jj1-kk1]) * (ci0<TF>*v[ijk-kk3] + ci1<TF>*v[ijk-kk2] + ci2<TF>*v[ijk-kk1] + ci3<TF>*v[ijk    ]))
                       + cg1<TF>*((ci0<TF>*w[ijk-jj2    ] + ci1<TF>*w[ijk-jj1    ] + ci2<TF>*w[ijk    ] + ci3<TF>*w[ijk+jj1    ]) * (ci0<TF>*v[ijk-kk2] + ci1<TF>*v[ijk-kk1] + ci2<TF>*v[ijk    ] + ci3<TF>*v[ijk+kk1]))
                       + cg2<TF>*((ci0<TF>*w[ijk-jj2+kk1] + ci1<TF>*w[ijk-jj1+kk1] + ci2<TF>*w[ijk+kk1] + ci3<TF>*w[ijk+jj1+kk1]) * (ci0<TF>*v[ijk-kk1] + ci1<TF>*v[ijk    ] + ci2<TF>*v[ijk+kk1] + ci3<TF>*v[ijk+kk2]))
                       + cg3<TF>*((ci0<TF>*w[ijk-jj2+kk2] + ci1<TF>*w[ijk-jj1+kk2] + ci2<TF>*w[ijk+kk2] + ci3<TF>*w[ijk+jj1+kk2]) * (ci0<TF>*v[ijk    ] + ci1<TF>*v[ijk+kk1] + ci2<TF>*v[ijk+kk2] + ci3<TF>*v[ijk+kk3])) ) * dzi4[k];
        }
    }

    template<typename TF, int loc> __global__
    void advec_v_boundary_g(TF* __restrict__ vt, const TF* __restrict__ u,
                            const TF* __restrict__ v,  const TF* __restrict__ w,
                            const TF* __restrict__ dzi4, const TF dxi, const TF dyi,
                            const int jj, const int kk,
                            const int istart, const int jstart, const int kstart,
                            const int iend,   const int jend,   const int kend)
        {
            const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
            const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;

            const int ii1 = 1;
            const int ii2 = 2;
            const int ii3 = 3;
            const int jj1 = 1*jj;
            const int jj2 = 2*jj;
            const int jj3 = 3*jj;
            const int kk1 = 1*kk;
            const int kk2 = 2*kk;
            const int kk3 = 3*kk;

            if (i < iend && j < jend)
            {
                if (loc == 0)
                {
                    const int k = kstart;
                    const int ijk = i + j*jj + k*kk;

                    vt[ijk] -= ( cg0<TF>*((ci0<TF>*u[ijk-ii1-jj2] + ci1<TF>*u[ijk-ii1-jj1] + ci2<TF>*u[ijk-ii1] + ci3<TF>*u[ijk-ii1+jj1]) * (ci0<TF>*v[ijk-ii3] + ci1<TF>*v[ijk-ii2] + ci2<TF>*v[ijk-ii1] + ci3<TF>*v[ijk    ]))
                               + cg1<TF>*((ci0<TF>*u[ijk    -jj2] + ci1<TF>*u[ijk    -jj1] + ci2<TF>*u[ijk    ] + ci3<TF>*u[ijk    +jj1]) * (ci0<TF>*v[ijk-ii2] + ci1<TF>*v[ijk-ii1] + ci2<TF>*v[ijk    ] + ci3<TF>*v[ijk+ii1]))
                               + cg2<TF>*((ci0<TF>*u[ijk+ii1-jj2] + ci1<TF>*u[ijk+ii1-jj1] + ci2<TF>*u[ijk+ii1] + ci3<TF>*u[ijk+ii1+jj1]) * (ci0<TF>*v[ijk-ii1] + ci1<TF>*v[ijk    ] + ci2<TF>*v[ijk+ii1] + ci3<TF>*v[ijk+ii2]))
                               + cg3<TF>*((ci0<TF>*u[ijk+ii2-jj2] + ci1<TF>*u[ijk+ii2-jj1] + ci2<TF>*u[ijk+ii2] + ci3<TF>*u[ijk+ii2+jj1]) * (ci0<TF>*v[ijk    ] + ci1<TF>*v[ijk+ii1] + ci2<TF>*v[ijk+ii2] + ci3<TF>*v[ijk+ii3])) ) * dxi;

                    vt[ijk] -= ( cg0<TF>*((ci0<TF>*v[ijk-jj3] + ci1<TF>*v[ijk-jj2] + ci2<TF>*v[ijk-jj1] + ci3<TF>*v[ijk    ]) * (ci0<TF>*v[ijk-jj3] + ci1<TF>*v[ijk-jj2] + ci2<TF>*v[ijk-jj1] + ci3<TF>*v[ijk    ]))
                               + cg1<TF>*((ci0<TF>*v[ijk-jj2] + ci1<TF>*v[ijk-jj1] + ci2<TF>*v[ijk    ] + ci3<TF>*v[ijk+jj1]) * (ci0<TF>*v[ijk-jj2] + ci1<TF>*v[ijk-jj1] + ci2<TF>*v[ijk    ] + ci3<TF>*v[ijk+jj1]))
                               + cg2<TF>*((ci0<TF>*v[ijk-jj1] + ci1<TF>*v[ijk    ] + ci2<TF>*v[ijk+jj1] + ci3<TF>*v[ijk+jj2]) * (ci0<TF>*v[ijk-jj1] + ci1<TF>*v[ijk    ] + ci2<TF>*v[ijk+jj1] + ci3<TF>*v[ijk+jj2]))
                               + cg3<TF>*((ci0<TF>*v[ijk    ] + ci1<TF>*v[ijk+jj1] + ci2<TF>*v[ijk+jj2] + ci3<TF>*v[ijk+jj3]) * (ci0<TF>*v[ijk    ] + ci1<TF>*v[ijk+jj1] + ci2<TF>*v[ijk+jj2] + ci3<TF>*v[ijk+jj3])) ) * dyi;

                    vt[ijk] -= ( cg0<TF>*((ci0<TF>*w[ijk-jj2-kk1] + ci1<TF>*w[ijk-jj1-kk1] + ci2<TF>*w[ijk-kk1] + ci3<TF>*w[ijk+jj1-kk1]) * (bi0<TF>*v[ijk-kk2] + bi1<TF>*v[ijk-kk1] + bi2<TF>*v[ijk    ] + bi3<TF>*v[ijk+kk1]))
                               + cg1<TF>*((ci0<TF>*w[ijk-jj2    ] + ci1<TF>*w[ijk-jj1    ] + ci2<TF>*w[ijk    ] + ci3<TF>*w[ijk+jj1    ]) * (ci0<TF>*v[ijk-kk2] + ci1<TF>*v[ijk-kk1] + ci2<TF>*v[ijk    ] + ci3<TF>*v[ijk+kk1]))
                               + cg2<TF>*((ci0<TF>*w[ijk-jj2+kk1] + ci1<TF>*w[ijk-jj1+kk1] + ci2<TF>*w[ijk+kk1] + ci3<TF>*w[ijk+jj1+kk1]) * (ci0<TF>*v[ijk-kk1] + ci1<TF>*v[ijk    ] + ci2<TF>*v[ijk+kk1] + ci3<TF>*v[ijk+kk2]))
                               + cg3<TF>*((ci0<TF>*w[ijk-jj2+kk2] + ci1<TF>*w[ijk-jj1+kk2] + ci2<TF>*w[ijk+kk2] + ci3<TF>*w[ijk+jj1+kk2]) * (ci0<TF>*v[ijk    ] + ci1<TF>*v[ijk+kk1] + ci2<TF>*v[ijk+kk2] + ci3<TF>*v[ijk+kk3])) ) * dzi4[k];
                }
                else if (loc == 1)
                {
                    const int k = kend-1;
                    const int ijk = i + j*jj + k*kk;

                    vt[ijk] -= ( cg0<TF>*((ci0<TF>*u[ijk-ii1-jj2] + ci1<TF>*u[ijk-ii1-jj1] + ci2<TF>*u[ijk-ii1] + ci3<TF>*u[ijk-ii1+jj1]) * (ci0<TF>*v[ijk-ii3] + ci1<TF>*v[ijk-ii2] + ci2<TF>*v[ijk-ii1] + ci3<TF>*v[ijk    ]))
                               + cg1<TF>*((ci0<TF>*u[ijk    -jj2] + ci1<TF>*u[ijk    -jj1] + ci2<TF>*u[ijk    ] + ci3<TF>*u[ijk    +jj1]) * (ci0<TF>*v[ijk-ii2] + ci1<TF>*v[ijk-ii1] + ci2<TF>*v[ijk    ] + ci3<TF>*v[ijk+ii1]))
                               + cg2<TF>*((ci0<TF>*u[ijk+ii1-jj2] + ci1<TF>*u[ijk+ii1-jj1] + ci2<TF>*u[ijk+ii1] + ci3<TF>*u[ijk+ii1+jj1]) * (ci0<TF>*v[ijk-ii1] + ci1<TF>*v[ijk    ] + ci2<TF>*v[ijk+ii1] + ci3<TF>*v[ijk+ii2]))
                               + cg3<TF>*((ci0<TF>*u[ijk+ii2-jj2] + ci1<TF>*u[ijk+ii2-jj1] + ci2<TF>*u[ijk+ii2] + ci3<TF>*u[ijk+ii2+jj1]) * (ci0<TF>*v[ijk    ] + ci1<TF>*v[ijk+ii1] + ci2<TF>*v[ijk+ii2] + ci3<TF>*v[ijk+ii3])) ) * dxi;

                    vt[ijk] -= ( cg0<TF>*((ci0<TF>*v[ijk-jj3] + ci1<TF>*v[ijk-jj2] + ci2<TF>*v[ijk-jj1] + ci3<TF>*v[ijk    ]) * (ci0<TF>*v[ijk-jj3] + ci1<TF>*v[ijk-jj2] + ci2<TF>*v[ijk-jj1] + ci3<TF>*v[ijk    ]))
                               + cg1<TF>*((ci0<TF>*v[ijk-jj2] + ci1<TF>*v[ijk-jj1] + ci2<TF>*v[ijk    ] + ci3<TF>*v[ijk+jj1]) * (ci0<TF>*v[ijk-jj2] + ci1<TF>*v[ijk-jj1] + ci2<TF>*v[ijk    ] + ci3<TF>*v[ijk+jj1]))
                               + cg2<TF>*((ci0<TF>*v[ijk-jj1] + ci1<TF>*v[ijk    ] + ci2<TF>*v[ijk+jj1] + ci3<TF>*v[ijk+jj2]) * (ci0<TF>*v[ijk-jj1] + ci1<TF>*v[ijk    ] + ci2<TF>*v[ijk+jj1] + ci3<TF>*v[ijk+jj2]))
                               + cg3<TF>*((ci0<TF>*v[ijk    ] + ci1<TF>*v[ijk+jj1] + ci2<TF>*v[ijk+jj2] + ci3<TF>*v[ijk+jj3]) * (ci0<TF>*v[ijk    ] + ci1<TF>*v[ijk+jj1] + ci2<TF>*v[ijk+jj2] + ci3<TF>*v[ijk+jj3])) ) * dyi;

                    vt[ijk] -= ( cg0<TF>*((ci0<TF>*w[ijk-jj2-kk1] + ci1<TF>*w[ijk-jj1-kk1] + ci2<TF>*w[ijk-kk1] + ci3<TF>*w[ijk+jj1-kk1]) * (ci0<TF>*v[ijk-kk3] + ci1<TF>*v[ijk-kk2] + ci2<TF>*v[ijk-kk1] + ci3<TF>*v[ijk    ]))
                               + cg1<TF>*((ci0<TF>*w[ijk-jj2    ] + ci1<TF>*w[ijk-jj1    ] + ci2<TF>*w[ijk    ] + ci3<TF>*w[ijk+jj1    ]) * (ci0<TF>*v[ijk-kk2] + ci1<TF>*v[ijk-kk1] + ci2<TF>*v[ijk    ] + ci3<TF>*v[ijk+kk1]))
                               + cg2<TF>*((ci0<TF>*w[ijk-jj2+kk1] + ci1<TF>*w[ijk-jj1+kk1] + ci2<TF>*w[ijk+kk1] + ci3<TF>*w[ijk+jj1+kk1]) * (ci0<TF>*v[ijk-kk1] + ci1<TF>*v[ijk    ] + ci2<TF>*v[ijk+kk1] + ci3<TF>*v[ijk+kk2]))
                               + cg3<TF>*((ci0<TF>*w[ijk-jj2+kk2] + ci1<TF>*w[ijk-jj1+kk2] + ci2<TF>*w[ijk+kk2] + ci3<TF>*w[ijk+jj1+kk2]) * (ti0<TF>*v[ijk-kk1] + ti1<TF>*v[ijk    ] + ti2<TF>*v[ijk+kk1] + ti3<TF>*v[ijk+kk2])) ) * dzi4[k];
                }
            }
        }

    template<typename TF> __global__
    void advec_w_g(TF* __restrict__ wt, const TF* __restrict__ u,
                   const TF* __restrict__ v,  const TF* __restrict__ w,
                   const TF* __restrict__ dzhi4, const TF dxi, const TF dyi,
                   const int jj, const int kk,
                   const int istart, const int jstart, const int kstart,
                   const int iend,   const int jend,   const int kend)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart + 1;

        const int ii1 = 1;
        const int ii2 = 2;
        const int ii3 = 3;
        const int jj1 = 1*jj;
        const int jj2 = 2*jj;
        const int jj3 = 3*jj;
        const int kk1 = 1*kk;
        const int kk2 = 2*kk;
        const int kk3 = 3*kk;

        if(i < iend && j < jend && k > kstart+1 && k < kend-1)
        {
            const int ijk = i + j*jj + k*kk;

            wt[ijk] -= ( cg0<TF>*((ci0<TF>*u[ijk-ii1-kk2] + ci1<TF>*u[ijk-ii1-kk1] + ci2<TF>*u[ijk-ii1] + ci3<TF>*u[ijk-ii1+kk1]) * (ci0<TF>*w[ijk-ii3] + ci1<TF>*w[ijk-ii2] + ci2<TF>*w[ijk-ii1] + ci3<TF>*w[ijk    ]))
                       + cg1<TF>*((ci0<TF>*u[ijk    -kk2] + ci1<TF>*u[ijk    -kk1] + ci2<TF>*u[ijk    ] + ci3<TF>*u[ijk    +kk1]) * (ci0<TF>*w[ijk-ii2] + ci1<TF>*w[ijk-ii1] + ci2<TF>*w[ijk    ] + ci3<TF>*w[ijk+ii1]))
                       + cg2<TF>*((ci0<TF>*u[ijk+ii1-kk2] + ci1<TF>*u[ijk+ii1-kk1] + ci2<TF>*u[ijk+ii1] + ci3<TF>*u[ijk+ii1+kk1]) * (ci0<TF>*w[ijk-ii1] + ci1<TF>*w[ijk    ] + ci2<TF>*w[ijk+ii1] + ci3<TF>*w[ijk+ii2]))
                       + cg3<TF>*((ci0<TF>*u[ijk+ii2-kk2] + ci1<TF>*u[ijk+ii2-kk1] + ci2<TF>*u[ijk+ii2] + ci3<TF>*u[ijk+ii2+kk1]) * (ci0<TF>*w[ijk    ] + ci1<TF>*w[ijk+ii1] + ci2<TF>*w[ijk+ii2] + ci3<TF>*w[ijk+ii3])) ) * dxi;

            wt[ijk] -= ( cg0<TF>*((ci0<TF>*v[ijk-jj1-kk2] + ci1<TF>*v[ijk-jj1-kk1] + ci2<TF>*v[ijk-jj1] + ci3<TF>*v[ijk-jj1+kk1]) * (ci0<TF>*w[ijk-jj3] + ci1<TF>*w[ijk-jj2] + ci2<TF>*w[ijk-jj1] + ci3<TF>*w[ijk    ]))
                       + cg1<TF>*((ci0<TF>*v[ijk    -kk2] + ci1<TF>*v[ijk    -kk1] + ci2<TF>*v[ijk    ] + ci3<TF>*v[ijk    +kk1]) * (ci0<TF>*w[ijk-jj2] + ci1<TF>*w[ijk-jj1] + ci2<TF>*w[ijk    ] + ci3<TF>*w[ijk+jj1]))
                       + cg2<TF>*((ci0<TF>*v[ijk+jj1-kk2] + ci1<TF>*v[ijk+jj1-kk1] + ci2<TF>*v[ijk+jj1] + ci3<TF>*v[ijk+jj1+kk1]) * (ci0<TF>*w[ijk-jj1] + ci1<TF>*w[ijk    ] + ci2<TF>*w[ijk+jj1] + ci3<TF>*w[ijk+jj2]))
                       + cg3<TF>*((ci0<TF>*v[ijk+jj2-kk2] + ci1<TF>*v[ijk+jj2-kk1] + ci2<TF>*v[ijk+jj2] + ci3<TF>*v[ijk+jj2+kk1]) * (ci0<TF>*w[ijk    ] + ci1<TF>*w[ijk+jj1] + ci2<TF>*w[ijk+jj2] + ci3<TF>*w[ijk+jj3])) ) * dyi;

            wt[ijk] -= ( cg0<TF>*((ci0<TF>*w[ijk-kk3] + ci1<TF>*w[ijk-kk2] + ci2<TF>*w[ijk-kk1] + ci3<TF>*w[ijk    ]) * (ci0<TF>*w[ijk-kk3] + ci1<TF>*w[ijk-kk2] + ci2<TF>*w[ijk-kk1] + ci3<TF>*w[ijk    ]))
                       + cg1<TF>*((ci0<TF>*w[ijk-kk2] + ci1<TF>*w[ijk-kk1] + ci2<TF>*w[ijk    ] + ci3<TF>*w[ijk+kk1]) * (ci0<TF>*w[ijk-kk2] + ci1<TF>*w[ijk-kk1] + ci2<TF>*w[ijk    ] + ci3<TF>*w[ijk+kk1]))
                       + cg2<TF>*((ci0<TF>*w[ijk-kk1] + ci1<TF>*w[ijk    ] + ci2<TF>*w[ijk+kk1] + ci3<TF>*w[ijk+kk2]) * (ci0<TF>*w[ijk-kk1] + ci1<TF>*w[ijk    ] + ci2<TF>*w[ijk+kk1] + ci3<TF>*w[ijk+kk2]))
                       + cg3<TF>*((ci0<TF>*w[ijk    ] + ci1<TF>*w[ijk+kk1] + ci2<TF>*w[ijk+kk2] + ci3<TF>*w[ijk+kk3]) * (ci0<TF>*w[ijk    ] + ci1<TF>*w[ijk+kk1] + ci2<TF>*w[ijk+kk2] + ci3<TF>*w[ijk+kk3])) ) * dzhi4[k];
        }
    }

    template<typename TF, int loc> __global__
    void advec_w_boundary_g(TF* __restrict__ wt, const TF* __restrict__ u,
                            const TF* __restrict__ v,  const TF* __restrict__ w,
                            const TF* __restrict__ dzhi4, const TF dxi, const TF dyi,
                            const int jj, const int kk,
                            const int istart, const int jstart, const int kstart,
                            const int iend,   const int jend,   const int kend)
        {
            const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
            const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;

            const int ii1 = 1;
            const int ii2 = 2;
            const int ii3 = 3;
            const int jj1 = 1*jj;
            const int jj2 = 2*jj;
            const int jj3 = 3*jj;
            const int kk1 = 1*kk;
            const int kk2 = 2*kk;
            const int kk3 = 3*kk;

            if (i < iend && j < jend)
            {
                if (loc == 0) //== kstart+1
                {
                    const int k = kstart+1;
                    const int ijk = i + j*jj + k*kk;

                    wt[ijk] -= ( cg0<TF>*((ci0<TF>*u[ijk-ii1-kk2] + ci1<TF>*u[ijk-ii1-kk1] + ci2<TF>*u[ijk-ii1] + ci3<TF>*u[ijk-ii1+kk1]) * (ci0<TF>*w[ijk-ii3] + ci1<TF>*w[ijk-ii2] + ci2<TF>*w[ijk-ii1] + ci3<TF>*w[ijk    ]))
                               + cg1<TF>*((ci0<TF>*u[ijk    -kk2] + ci1<TF>*u[ijk    -kk1] + ci2<TF>*u[ijk    ] + ci3<TF>*u[ijk    +kk1]) * (ci0<TF>*w[ijk-ii2] + ci1<TF>*w[ijk-ii1] + ci2<TF>*w[ijk    ] + ci3<TF>*w[ijk+ii1]))
                               + cg2<TF>*((ci0<TF>*u[ijk+ii1-kk2] + ci1<TF>*u[ijk+ii1-kk1] + ci2<TF>*u[ijk+ii1] + ci3<TF>*u[ijk+ii1+kk1]) * (ci0<TF>*w[ijk-ii1] + ci1<TF>*w[ijk    ] + ci2<TF>*w[ijk+ii1] + ci3<TF>*w[ijk+ii2]))
                               + cg3<TF>*((ci0<TF>*u[ijk+ii2-kk2] + ci1<TF>*u[ijk+ii2-kk1] + ci2<TF>*u[ijk+ii2] + ci3<TF>*u[ijk+ii2+kk1]) * (ci0<TF>*w[ijk    ] + ci1<TF>*w[ijk+ii1] + ci2<TF>*w[ijk+ii2] + ci3<TF>*w[ijk+ii3])) ) * dxi;

                    wt[ijk] -= ( cg0<TF>*((ci0<TF>*v[ijk-jj1-kk2] + ci1<TF>*v[ijk-jj1-kk1] + ci2<TF>*v[ijk-jj1] + ci3<TF>*v[ijk-jj1+kk1]) * (ci0<TF>*w[ijk-jj3] + ci1<TF>*w[ijk-jj2] + ci2<TF>*w[ijk-jj1] + ci3<TF>*w[ijk    ]))
                               + cg1<TF>*((ci0<TF>*v[ijk    -kk2] + ci1<TF>*v[ijk    -kk1] + ci2<TF>*v[ijk    ] + ci3<TF>*v[ijk    +kk1]) * (ci0<TF>*w[ijk-jj2] + ci1<TF>*w[ijk-jj1] + ci2<TF>*w[ijk    ] + ci3<TF>*w[ijk+jj1]))
                               + cg2<TF>*((ci0<TF>*v[ijk+jj1-kk2] + ci1<TF>*v[ijk+jj1-kk1] + ci2<TF>*v[ijk+jj1] + ci3<TF>*v[ijk+jj1+kk1]) * (ci0<TF>*w[ijk-jj1] + ci1<TF>*w[ijk    ] + ci2<TF>*w[ijk+jj1] + ci3<TF>*w[ijk+jj2]))
                               + cg3<TF>*((ci0<TF>*v[ijk+jj2-kk2] + ci1<TF>*v[ijk+jj2-kk1] + ci2<TF>*v[ijk+jj2] + ci3<TF>*v[ijk+jj2+kk1]) * (ci0<TF>*w[ijk    ] + ci1<TF>*w[ijk+jj1] + ci2<TF>*w[ijk+jj2] + ci3<TF>*w[ijk+jj3])) ) * dyi;

                    wt[ijk] -= ( cg0<TF>*((bi0<TF>*w[ijk-kk2] + bi1<TF>*w[ijk-kk1] + bi2<TF>*w[ijk    ] + bi3<TF>*w[ijk+kk1]) * (bi0<TF>*w[ijk-kk2] + bi1<TF>*w[ijk-kk1] + bi2<TF>*w[ijk    ] + bi3<TF>*w[ijk+kk1]))
                               + cg1<TF>*((ci0<TF>*w[ijk-kk2] + ci1<TF>*w[ijk-kk1] + ci2<TF>*w[ijk    ] + ci3<TF>*w[ijk+kk1]) * (ci0<TF>*w[ijk-kk2] + ci1<TF>*w[ijk-kk1] + ci2<TF>*w[ijk    ] + ci3<TF>*w[ijk+kk1]))
                               + cg2<TF>*((ci0<TF>*w[ijk-kk1] + ci1<TF>*w[ijk    ] + ci2<TF>*w[ijk+kk1] + ci3<TF>*w[ijk+kk2]) * (ci0<TF>*w[ijk-kk1] + ci1<TF>*w[ijk    ] + ci2<TF>*w[ijk+kk1] + ci3<TF>*w[ijk+kk2]))
                               + cg3<TF>*((ci0<TF>*w[ijk    ] + ci1<TF>*w[ijk+kk1] + ci2<TF>*w[ijk+kk2] + ci3<TF>*w[ijk+kk3]) * (ci0<TF>*w[ijk    ] + ci1<TF>*w[ijk+kk1] + ci2<TF>*w[ijk+kk2] + ci3<TF>*w[ijk+kk3])) ) * dzhi4[k];
                }
                else if (loc == 1) //==kend-1
                {
                    const int k = kend-1;
                    const int ijk = i + j*jj + k*kk;

                    wt[ijk] -= ( cg0<TF>*((ci0<TF>*u[ijk-ii1-kk2] + ci1<TF>*u[ijk-ii1-kk1] + ci2<TF>*u[ijk-ii1] + ci3<TF>*u[ijk-ii1+kk1]) * (ci0<TF>*w[ijk-ii3] + ci1<TF>*w[ijk-ii2] + ci2<TF>*w[ijk-ii1] + ci3<TF>*w[ijk    ]))
                               + cg1<TF>*((ci0<TF>*u[ijk    -kk2] + ci1<TF>*u[ijk    -kk1] + ci2<TF>*u[ijk    ] + ci3<TF>*u[ijk    +kk1]) * (ci0<TF>*w[ijk-ii2] + ci1<TF>*w[ijk-ii1] + ci2<TF>*w[ijk    ] + ci3<TF>*w[ijk+ii1]))
                               + cg2<TF>*((ci0<TF>*u[ijk+ii1-kk2] + ci1<TF>*u[ijk+ii1-kk1] + ci2<TF>*u[ijk+ii1] + ci3<TF>*u[ijk+ii1+kk1]) * (ci0<TF>*w[ijk-ii1] + ci1<TF>*w[ijk    ] + ci2<TF>*w[ijk+ii1] + ci3<TF>*w[ijk+ii2]))
                               + cg3<TF>*((ci0<TF>*u[ijk+ii2-kk2] + ci1<TF>*u[ijk+ii2-kk1] + ci2<TF>*u[ijk+ii2] + ci3<TF>*u[ijk+ii2+kk1]) * (ci0<TF>*w[ijk    ] + ci1<TF>*w[ijk+ii1] + ci2<TF>*w[ijk+ii2] + ci3<TF>*w[ijk+ii3])) ) * dxi;

                    wt[ijk] -= ( cg0<TF>*((ci0<TF>*v[ijk-jj1-kk2] + ci1<TF>*v[ijk-jj1-kk1] + ci2<TF>*v[ijk-jj1] + ci3<TF>*v[ijk-jj1+kk1]) * (ci0<TF>*w[ijk-jj3] + ci1<TF>*w[ijk-jj2] + ci2<TF>*w[ijk-jj1] + ci3<TF>*w[ijk    ]))
                               + cg1<TF>*((ci0<TF>*v[ijk    -kk2] + ci1<TF>*v[ijk    -kk1] + ci2<TF>*v[ijk    ] + ci3<TF>*v[ijk    +kk1]) * (ci0<TF>*w[ijk-jj2] + ci1<TF>*w[ijk-jj1] + ci2<TF>*w[ijk    ] + ci3<TF>*w[ijk+jj1]))
                               + cg2<TF>*((ci0<TF>*v[ijk+jj1-kk2] + ci1<TF>*v[ijk+jj1-kk1] + ci2<TF>*v[ijk+jj1] + ci3<TF>*v[ijk+jj1+kk1]) * (ci0<TF>*w[ijk-jj1] + ci1<TF>*w[ijk    ] + ci2<TF>*w[ijk+jj1] + ci3<TF>*w[ijk+jj2]))
                               + cg3<TF>*((ci0<TF>*v[ijk+jj2-kk2] + ci1<TF>*v[ijk+jj2-kk1] + ci2<TF>*v[ijk+jj2] + ci3<TF>*v[ijk+jj2+kk1]) * (ci0<TF>*w[ijk    ] + ci1<TF>*w[ijk+jj1] + ci2<TF>*w[ijk+jj2] + ci3<TF>*w[ijk+jj3])) ) * dyi;

                    wt[ijk] -= ( cg0<TF>*((ci0<TF>*w[ijk-kk3] + ci1<TF>*w[ijk-kk2] + ci2<TF>*w[ijk-kk1] + ci3<TF>*w[ijk    ]) * (ci0<TF>*w[ijk-kk3] + ci1<TF>*w[ijk-kk2] + ci2<TF>*w[ijk-kk1] + ci3<TF>*w[ijk    ]))
                               + cg1<TF>*((ci0<TF>*w[ijk-kk2] + ci1<TF>*w[ijk-kk1] + ci2<TF>*w[ijk    ] + ci3<TF>*w[ijk+kk1]) * (ci0<TF>*w[ijk-kk2] + ci1<TF>*w[ijk-kk1] + ci2<TF>*w[ijk    ] + ci3<TF>*w[ijk+kk1]))
                               + cg2<TF>*((ci0<TF>*w[ijk-kk1] + ci1<TF>*w[ijk    ] + ci2<TF>*w[ijk+kk1] + ci3<TF>*w[ijk+kk2]) * (ci0<TF>*w[ijk-kk1] + ci1<TF>*w[ijk    ] + ci2<TF>*w[ijk+kk1] + ci3<TF>*w[ijk+kk2]))
                               + cg3<TF>*((ti0<TF>*w[ijk-kk1] + ti1<TF>*w[ijk    ] + ti2<TF>*w[ijk+kk1] + ti3<TF>*w[ijk+kk2]) * (ti0<TF>*w[ijk-kk1] + ti1<TF>*w[ijk    ] + ti2<TF>*w[ijk+kk1] + ti3<TF>*w[ijk+kk2])) ) * dzhi4[k];
                }
            }
        }

    template<typename TF> __global__
    void advec_s_g(TF* __restrict__ st, const TF* __restrict__ s,
                   const TF* __restrict__ u,  const TF* __restrict__ v, const TF* __restrict__ w,
                   const TF* __restrict__ dzi4, const TF dxi, const TF dyi,
                   const int jj, const int kk,
                   const int istart, const int jstart, const int kstart,
                   const int iend,   const int jend,   const int kend)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        const int ii1 = 1;
        const int ii2 = 2;
        const int ii3 = 3;
        const int jj1 = 1*jj;
        const int jj2 = 2*jj;
        const int jj3 = 3*jj;
        const int kk1 = 1*kk;
        const int kk2 = 2*kk;
        const int kk3 = 3*kk;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;

            if (k == kstart)
            {
                st[ijk] -= ( cg0<TF>*(u[ijk-ii1] * (ci0<TF>*s[ijk-ii3] + ci1<TF>*s[ijk-ii2] + ci2<TF>*s[ijk-ii1] + ci3<TF>*s[ijk    ]))
                           + cg1<TF>*(u[ijk    ] * (ci0<TF>*s[ijk-ii2] + ci1<TF>*s[ijk-ii1] + ci2<TF>*s[ijk    ] + ci3<TF>*s[ijk+ii1]))
                           + cg2<TF>*(u[ijk+ii1] * (ci0<TF>*s[ijk-ii1] + ci1<TF>*s[ijk    ] + ci2<TF>*s[ijk+ii1] + ci3<TF>*s[ijk+ii2]))
                           + cg3<TF>*(u[ijk+ii2] * (ci0<TF>*s[ijk    ] + ci1<TF>*s[ijk+ii1] + ci2<TF>*s[ijk+ii2] + ci3<TF>*s[ijk+ii3])) ) * dxi;

                st[ijk] -= ( cg0<TF>*(v[ijk-jj1] * (ci0<TF>*s[ijk-jj3] + ci1<TF>*s[ijk-jj2] + ci2<TF>*s[ijk-jj1] + ci3<TF>*s[ijk    ]))
                           + cg1<TF>*(v[ijk    ] * (ci0<TF>*s[ijk-jj2] + ci1<TF>*s[ijk-jj1] + ci2<TF>*s[ijk    ] + ci3<TF>*s[ijk+jj1]))
                           + cg2<TF>*(v[ijk+jj1] * (ci0<TF>*s[ijk-jj1] + ci1<TF>*s[ijk    ] + ci2<TF>*s[ijk+jj1] + ci3<TF>*s[ijk+jj2]))
                           + cg3<TF>*(v[ijk+jj2] * (ci0<TF>*s[ijk    ] + ci1<TF>*s[ijk+jj1] + ci2<TF>*s[ijk+jj2] + ci3<TF>*s[ijk+jj3])) ) * dyi;

                st[ijk] -= ( cg0<TF>*(w[ijk-kk1] * (bi0<TF>*s[ijk-kk2] + bi1<TF>*s[ijk-kk1] + bi2<TF>*s[ijk    ] + bi3<TF>*s[ijk+kk1]))
                           + cg1<TF>*(w[ijk    ] * (ci0<TF>*s[ijk-kk2] + ci1<TF>*s[ijk-kk1] + ci2<TF>*s[ijk    ] + ci3<TF>*s[ijk+kk1]))
                           + cg2<TF>*(w[ijk+kk1] * (ci0<TF>*s[ijk-kk1] + ci1<TF>*s[ijk    ] + ci2<TF>*s[ijk+kk1] + ci3<TF>*s[ijk+kk2]))
                           + cg3<TF>*(w[ijk+kk2] * (ci0<TF>*s[ijk    ] + ci1<TF>*s[ijk+kk1] + ci2<TF>*s[ijk+kk2] + ci3<TF>*s[ijk+kk3])) ) * dzi4[k];
            }
            else if (k == kend-1)
            {
                st[ijk] -= ( cg0<TF>*(u[ijk-ii1] * (ci0<TF>*s[ijk-ii3] + ci1<TF>*s[ijk-ii2] + ci2<TF>*s[ijk-ii1] + ci3<TF>*s[ijk    ]))
                           + cg1<TF>*(u[ijk    ] * (ci0<TF>*s[ijk-ii2] + ci1<TF>*s[ijk-ii1] + ci2<TF>*s[ijk    ] + ci3<TF>*s[ijk+ii1]))
                           + cg2<TF>*(u[ijk+ii1] * (ci0<TF>*s[ijk-ii1] + ci1<TF>*s[ijk    ] + ci2<TF>*s[ijk+ii1] + ci3<TF>*s[ijk+ii2]))
                           + cg3<TF>*(u[ijk+ii2] * (ci0<TF>*s[ijk    ] + ci1<TF>*s[ijk+ii1] + ci2<TF>*s[ijk+ii2] + ci3<TF>*s[ijk+ii3])) ) * dxi;

                st[ijk] -= ( cg0<TF>*(v[ijk-jj1] * (ci0<TF>*s[ijk-jj3] + ci1<TF>*s[ijk-jj2] + ci2<TF>*s[ijk-jj1] + ci3<TF>*s[ijk    ]))
                           + cg1<TF>*(v[ijk    ] * (ci0<TF>*s[ijk-jj2] + ci1<TF>*s[ijk-jj1] + ci2<TF>*s[ijk    ] + ci3<TF>*s[ijk+jj1]))
                           + cg2<TF>*(v[ijk+jj1] * (ci0<TF>*s[ijk-jj1] + ci1<TF>*s[ijk    ] + ci2<TF>*s[ijk+jj1] + ci3<TF>*s[ijk+jj2]))
                           + cg3<TF>*(v[ijk+jj2] * (ci0<TF>*s[ijk    ] + ci1<TF>*s[ijk+jj1] + ci2<TF>*s[ijk+jj2] + ci3<TF>*s[ijk+jj3])) ) * dyi;

                st[ijk] -= ( cg0<TF>*(w[ijk-kk1] * (ci0<TF>*s[ijk-kk3] + ci1<TF>*s[ijk-kk2] + ci2<TF>*s[ijk-kk1] + ci3<TF>*s[ijk    ]))
                           + cg1<TF>*(w[ijk    ] * (ci0<TF>*s[ijk-kk2] + ci1<TF>*s[ijk-kk1] + ci2<TF>*s[ijk    ] + ci3<TF>*s[ijk+kk1]))
                           + cg2<TF>*(w[ijk+kk1] * (ci0<TF>*s[ijk-kk1] + ci1<TF>*s[ijk    ] + ci2<TF>*s[ijk+kk1] + ci3<TF>*s[ijk+kk2]))
                           + cg3<TF>*(w[ijk+kk2] * (ti0<TF>*s[ijk-kk1] + ti1<TF>*s[ijk    ] + ti2<TF>*s[ijk+kk1] + ti3<TF>*s[ijk+kk2])) ) * dzi4[k];
            }
            else
            {
                st[ijk] -= ( cg0<TF>*(u[ijk-ii1] * (ci0<TF>*s[ijk-ii3] + ci1<TF>*s[ijk-ii2] + ci2<TF>*s[ijk-ii1] + ci3<TF>*s[ijk    ]))
                           + cg1<TF>*(u[ijk    ] * (ci0<TF>*s[ijk-ii2] + ci1<TF>*s[ijk-ii1] + ci2<TF>*s[ijk    ] + ci3<TF>*s[ijk+ii1]))
                           + cg2<TF>*(u[ijk+ii1] * (ci0<TF>*s[ijk-ii1] + ci1<TF>*s[ijk    ] + ci2<TF>*s[ijk+ii1] + ci3<TF>*s[ijk+ii2]))
                           + cg3<TF>*(u[ijk+ii2] * (ci0<TF>*s[ijk    ] + ci1<TF>*s[ijk+ii1] + ci2<TF>*s[ijk+ii2] + ci3<TF>*s[ijk+ii3])) ) * dxi;

                st[ijk] -= ( cg0<TF>*(v[ijk-jj1] * (ci0<TF>*s[ijk-jj3] + ci1<TF>*s[ijk-jj2] + ci2<TF>*s[ijk-jj1] + ci3<TF>*s[ijk    ]))
                           + cg1<TF>*(v[ijk    ] * (ci0<TF>*s[ijk-jj2] + ci1<TF>*s[ijk-jj1] + ci2<TF>*s[ijk    ] + ci3<TF>*s[ijk+jj1]))
                           + cg2<TF>*(v[ijk+jj1] * (ci0<TF>*s[ijk-jj1] + ci1<TF>*s[ijk    ] + ci2<TF>*s[ijk+jj1] + ci3<TF>*s[ijk+jj2]))
                           + cg3<TF>*(v[ijk+jj2] * (ci0<TF>*s[ijk    ] + ci1<TF>*s[ijk+jj1] + ci2<TF>*s[ijk+jj2] + ci3<TF>*s[ijk+jj3])) ) * dyi;

                st[ijk] -= ( cg0<TF>*(w[ijk-kk1] * (ci0<TF>*s[ijk-kk3] + ci1<TF>*s[ijk-kk2] + ci2<TF>*s[ijk-kk1] + ci3<TF>*s[ijk    ]))
                           + cg1<TF>*(w[ijk    ] * (ci0<TF>*s[ijk-kk2] + ci1<TF>*s[ijk-kk1] + ci2<TF>*s[ijk    ] + ci3<TF>*s[ijk+kk1]))
                           + cg2<TF>*(w[ijk+kk1] * (ci0<TF>*s[ijk-kk1] + ci1<TF>*s[ijk    ] + ci2<TF>*s[ijk+kk1] + ci3<TF>*s[ijk+kk2]))
                           + cg3<TF>*(w[ijk+kk2] * (ci0<TF>*s[ijk    ] + ci1<TF>*s[ijk+kk1] + ci2<TF>*s[ijk+kk2] + ci3<TF>*s[ijk+kk3])) ) * dzi4[k];
            }
        }
    }

    template<typename TF> __global__
    void calc_cfl_g(TF* const __restrict__ tmp1,
                    const TF* __restrict__ u, const TF* __restrict__ v, const TF* __restrict__ w,
                    const TF* __restrict__ dzi, const TF dxi, const TF dyi,
                    const int jj, const int kk,
                    const int istart, const int jstart, const int kstart,
                    const int iend,   const int jend,   const int kend)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        const int ii1 = 1;
        const int ii2 = 2;
        const int jj1 = 1*jj;
        const int jj2 = 2*jj;
        const int kk1 = 1*kk;
        const int kk2 = 2*kk;

        const int ijk = i + j*jj + k*kk;

        if (i < iend && j < jend && k < kend)
            tmp1[ijk] = std::abs(ci0<TF>*u[ijk-ii1] + ci1<TF>*u[ijk] + ci2<TF>*u[ijk+ii1] + ci3<TF>*u[ijk+ii2])*dxi +
                        std::abs(ci0<TF>*v[ijk-jj1] + ci1<TF>*v[ijk] + ci2<TF>*v[ijk+jj1] + ci3<TF>*v[ijk+jj2])*dyi +
                        std::abs(ci0<TF>*w[ijk-kk1] + ci1<TF>*w[ijk] + ci2<TF>*w[ijk+kk1] + ci3<TF>*w[ijk+kk2])*dzi[k];
    }
}
#ifdef USECUDA
template<typename TF>
unsigned long Advec_4<TF>::get_time_limit(unsigned long idt, double dt)
{
    // Calculate cfl and prevent zero divisons.
    double cfl = get_cfl(dt);
    cfl = std::max(cflmin, cfl);
    const unsigned long idtlim = idt * cflmax / cfl;
    return idtlim;
}

template<typename TF>
double Advec_4<TF>::get_cfl(const double dt)
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.icells/blocki + (gd.icells%blocki > 0);
    const int gridj  = gd.jcells/blockj + (gd.jcells%blockj > 0);

    dim3 gridGPU (gridi, gridj, gd.kcells);
    dim3 blockGPU(blocki, blockj, 1);

    auto cfl_3d = fields.get_tmp_g();

    calc_cfl_g<TF><<<gridGPU, blockGPU>>>(
        cfl_3d->fld_g,
        fields.mp.at("u")->fld_g, fields.mp.at("v")->fld_g, fields.mp.at("w")->fld_g,
        gd.dzi_g, gd.dxi, gd.dyi,
        gd.icells, gd.ijcells,
        gd.istart, gd.jstart, gd.kstart,
        gd.iend,   gd.jend,   gd.kend);
    cuda_check_error();

    TF cfl = field3d_operators.calc_max_g(cfl_3d->fld_g);
    // TO DO communicate.

    cfl = cfl*dt;

    fields.release_tmp_g(cfl_3d);

    return cfl;
}

template<typename TF>
void Advec_4<TF>::exec(Stats<TF>& stats)
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, gd.kmax);
    dim3 blockGPU(blocki, blockj, 1);

    dim3 gridGPU2D (gridi, gridj, 1);
    dim3 blockGPU2D(blocki, blockj, 1);

    // Top and bottom boundary:
    advec_u_boundary_g<TF,0><<<gridGPU2D, blockGPU2D>>>(
        fields.mt.at("u")->fld_g, fields.mp.at("u")->fld_g, fields.mp.at("v")->fld_g,
        fields.mp.at("w")->fld_g, gd.dzi4_g, gd.dxi, gd.dyi,
        gd.icells, gd.ijcells,
        gd.istart, gd.jstart, gd.kstart,
        gd.iend,   gd.jend,   gd.kend);
    cuda_check_error();

    advec_u_boundary_g<TF,1><<<gridGPU2D, blockGPU2D>>>(
        fields.mt.at("u")->fld_g, fields.mp.at("u")->fld_g, fields.mp.at("v")->fld_g,
        fields.mp.at("w")->fld_g, gd.dzi4_g, gd.dxi, gd.dyi,
        gd.icells, gd.ijcells,
        gd.istart, gd.jstart, gd.kstart,
        gd.iend,   gd.jend,   gd.kend);
    cuda_check_error();

    // Interior:
    advec_u_g<TF><<<gridGPU, blockGPU>>>(
        fields.mt.at("u")->fld_g, fields.mp.at("u")->fld_g, fields.mp.at("v")->fld_g,
        fields.mp.at("w")->fld_g, gd.dzi4_g, gd.dxi, gd.dyi,
        gd.icells, gd.ijcells,
        gd.istart, gd.jstart, gd.kstart,
        gd.iend,   gd.jend,   gd.kend);
    cuda_check_error();

    // Top and bottom boundary:
    advec_v_boundary_g<TF,0><<<gridGPU2D, blockGPU2D>>>(
        fields.mt.at("v")->fld_g, fields.mp.at("u")->fld_g, fields.mp.at("v")->fld_g,
        fields.mp.at("w")->fld_g, gd.dzi4_g, gd.dxi, gd.dyi,
        gd.icells, gd.ijcells,
        gd.istart, gd.jstart, gd.kstart,
        gd.iend,   gd.jend,   gd.kend);
    cuda_check_error();

    advec_v_boundary_g<TF,1><<<gridGPU2D, blockGPU2D>>>(
        fields.mt.at("v")->fld_g, fields.mp.at("u")->fld_g, fields.mp.at("v")->fld_g,
        fields.mp.at("w")->fld_g, gd.dzi4_g, gd.dxi, gd.dyi,
        gd.icells, gd.ijcells,
        gd.istart, gd.jstart, gd.kstart,
        gd.iend,   gd.jend,   gd.kend);
    cuda_check_error();

    // Interior
    advec_v_g<TF><<<gridGPU, blockGPU>>>(
        fields.mt.at("v")->fld_g, fields.mp.at("u")->fld_g, fields.mp.at("v")->fld_g,
        fields.mp.at("w")->fld_g, gd.dzi4_g, gd.dxi, gd.dyi,
        gd.icells, gd.ijcells,
        gd.istart, gd.jstart, gd.kstart,
        gd.iend,   gd.jend,   gd.kend);
    cuda_check_error();

    // Top and bottom boundary:
    advec_w_boundary_g<TF,0><<<gridGPU2D, blockGPU2D>>>(
        fields.mt.at("w")->fld_g, fields.mp.at("u")->fld_g, fields.mp.at("v")->fld_g,
        fields.mp.at("w")->fld_g, gd.dzhi4_g, gd.dxi, gd.dyi,
        gd.icells, gd.ijcells,
        gd.istart, gd.jstart, gd.kstart,
        gd.iend,   gd.jend,   gd.kend);
    cuda_check_error();

    advec_w_boundary_g<TF,1><<<gridGPU2D, blockGPU2D>>>(
        fields.mt.at("w")->fld_g, fields.mp.at("u")->fld_g, fields.mp.at("v")->fld_g,
        fields.mp.at("w")->fld_g, gd.dzhi4_g, gd.dxi, gd.dyi,
        gd.icells, gd.ijcells,
        gd.istart, gd.jstart, gd.kstart,
        gd.iend,   gd.jend,   gd.kend);
    cuda_check_error();

    // Interior:
    advec_w_g<TF><<<gridGPU, blockGPU>>>(
        fields.mt.at("w")->fld_g, fields.mp.at("u")->fld_g, fields.mp.at("v")->fld_g,
        fields.mp.at("w")->fld_g, gd.dzhi4_g, gd.dxi, gd.dyi,
        gd.icells, gd.ijcells,
        gd.istart, gd.jstart, gd.kstart,
        gd.iend,   gd.jend,   gd.kend);
    cuda_check_error();

    for (auto& it : fields.st)
        advec_s_g<TF><<<gridGPU, blockGPU>>>(
            it.second->fld_g, fields.sp.at(it.first)->fld_g,
            fields.mp.at("u")->fld_g, fields.mp.at("v")->fld_g, fields.mp.at("w")->fld_g,
            gd.dzi4_g, gd.dxi, gd.dyi,
            gd.icells, gd.ijcells,
            gd.istart, gd.jstart, gd.kstart,
            gd.iend,   gd.jend,   gd.kend);
    cuda_check_error();

    hipDeviceSynchronize();

    stats.calc_tend(*fields.mt.at("u"), tend_name);
    stats.calc_tend(*fields.mt.at("v"), tend_name);
    stats.calc_tend(*fields.mt.at("w"), tend_name);

    for (auto it : fields.st)
        stats.calc_tend(*it.second, tend_name);
}
#endif

template class Advec_4<double>;
template class Advec_4<float>;
