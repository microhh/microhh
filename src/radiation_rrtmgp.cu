#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2020 Chiel van Heerwaarden
 * Copyright (c) 2011-2020 Thijs Heus
 * Copyright (c) 2014-2020 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <numeric>
#include <boost/algorithm/string.hpp>

#include "radiation_rrtmgp.h"
#include "radiation_rrtmgp_functions.h"
#include "grid.h"
#include "fields.h"
#include "timeloop.h"
#include "thermo.h"
#include "stats.h"
#include "netcdf_interface.h"
#include "constants.h"
#include "stats.h"
#include "cross.h"
#include "column.h"

#include "Array.h"
#include "Fluxes.h"
#include "subset_kernel_launcher_cuda.h"

using namespace Radiation_rrtmgp_functions;

namespace
{
    __global__
    void calc_tendency(
            Float* __restrict__ thlt_rad,  const Float* __restrict__ flux_up,
            const Float* __restrict flux_dn, const Float* __restrict__ rho,
            const Float* __restrict__ exner, const Float* __restrict__ dz,
            const int istart, const int jstart, const int kstart,
            const int iend, const int jend, const int kend,
            const int igc, const int jgc, const int kgc,
            const int jj, const int kk,
            const int jj_nogc, const int kk_nogc)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if ( (i < iend) && (j < jend) && (k < kend) )
        {
            const Float fac = Float(1.) / (rho[k] * Constants::cp<Float> * exner[k] * dz[k]);

            const int ijk = i + j*jj + k*kk;
            const int ijk_nogc = (i-igc) + (j-jgc)*jj_nogc + (k-kgc)*kk_nogc;

            thlt_rad[ijk] -= fac * ( flux_up[ijk_nogc + kk_nogc] - flux_up[ijk_nogc]
                                   - flux_dn[ijk_nogc + kk_nogc] + flux_dn[ijk_nogc] );
        }
    }

    __global__
    void add_tendency(
            Float* __restrict__ thlt,  const Float* __restrict__ thlt_rad,
            const int istart, const int jstart, const int kstart,
            const int iend, const int jend, const int kend,
            const int jj, const int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if ( (i < iend) && (j < jend) && (k < kend) )
        {
            const int ijk = i + j*jj + k*kk;
            thlt[ijk] += thlt_rad[ijk];
        }
    }

    __global__
    void store_surface_fluxes(
            Float* __restrict__ flux_up_sfc, Float* __restrict__ flux_dn_sfc,
            const Float* __restrict__ flux_up, const Float* __restrict__ flux_dn,
            const int istart, const int iend,
            const int jstart, const int jend,
            const int igc, const int jgc,
            const int jj, const int kk,
            const int jj_nogc)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;

        if ( (i < iend) && (j < jend) )
        {
            const int ij = i + j*jj;
            const int ij_nogc = (i-igc) + (j-jgc)*jj_nogc;
            flux_up_sfc[ij] = flux_up[ij_nogc];
            flux_dn_sfc[ij] = flux_dn[ij_nogc];
        }
    }

    __global__
    void effective_radius_and_ciwp_to_gm2(
            Float* __restrict__ rel, Float* __restrict__ rei,
            Float* __restrict__ clwp, Float* __restrict__ ciwp,
            const Float* __restrict__ dz,
            const int ncol, const int nlay, const int kstart,
            const Float four_third_pi_N0_rho_w,
            const Float four_third_pi_N0_rho_i,
            const Float sig_g_fac)
    {
        const int icol = blockIdx.x*blockDim.x + threadIdx.x;
        const int ilay = blockIdx.y*blockDim.y + threadIdx.y;

        if ( (icol < ncol) && (ilay < nlay) )
        {
            const int idx = icol + ilay*ncol;
            const int idx_z = ilay + kstart;
            const Float rel_local = clwp[idx] > Float(0.) ? Float(1.e6) * sig_g_fac * pow(clwp[idx] / dz[idx_z] / four_third_pi_N0_rho_w, Float(1.)/Float(3.)) : Float(0.);
            const Float rei_local = ciwp[idx] > Float(0.) ? Float(1.e6) * sig_g_fac * pow(ciwp[idx] / dz[idx_z] / four_third_pi_N0_rho_i, Float(1.)/Float(3.)) : Float(0.);

            rel[idx] = max(Float(2.5), min(rel_local, Float(21.5)));
            rei[idx] = max(Float(10.), min(rei_local, Float(180.)));

            clwp[idx] *= Float(1.e3);
            ciwp[idx] *= Float(1.e3);
        }
    }

    __global__
    void add_ghost_cells_g(
            Float* __restrict__ out, const Float* __restrict__ in,
            const int istart, const int jstart, const int kstart,
            const int iend, const int jend, const int kendh,
            const int jj, const int kk,
            const int jj_nogc, const int kk_nogc)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kendh)
        {
            const int ijk_nogc  = (i-istart) + (j-jstart)*jj_nogc + (k-kstart)*kk_nogc;
            const int ijk = i + j*jj + k*kk;

            out[ijk] = in[ijk_nogc];
        }
    }

    std::vector<std::string> get_variable_string(
            const std::string& var_name,
            std::vector<int> i_count,
            Netcdf_handle& input_nc,
            const int string_len,
            bool trim=true)
    {
        // Multiply all elements in i_count.
        int total_count = std::accumulate(i_count.begin(), i_count.end(), 1, std::multiplies<>());

        // Add the string length as the rightmost dimension.
        i_count.push_back(string_len);

        // Multiply all elements in i_count.
        // int total_count_char = std::accumulate(i_count.begin(), i_count.end(), 1, std::multiplies<>());

        // Read the entire char array;
        std::vector<char> var_char;
        var_char = input_nc.get_variable<char>(var_name, i_count);

        std::vector<std::string> var;

        for (int n=0; n<total_count; ++n)
        {
            std::string s(var_char.begin()+n*string_len, var_char.begin()+(n+1)*string_len);
            if (trim)
                boost::trim(s);
            var.push_back(s);
        }

        return var;
    }

    Gas_optics_rrtmgp_gpu load_and_init_gas_optics(
            Master& master,
            const Gas_concs_gpu& gas_concs,
            const std::string& coef_file)
    {
        // READ THE COEFFICIENTS FOR THE OPTICAL SOLVER.
        Netcdf_file coef_nc(master, coef_file, Netcdf_mode::Read);

        // Read k-distribution information.
        const int n_temps = coef_nc.get_dimension_size("temperature");
        const int n_press = coef_nc.get_dimension_size("pressure");
        const int n_absorbers = coef_nc.get_dimension_size("absorber");
        const int n_char = coef_nc.get_dimension_size("string_len");
        const int n_minorabsorbers = coef_nc.get_dimension_size("minor_absorber");
        const int n_extabsorbers = coef_nc.get_dimension_size("absorber_ext");
        const int n_mixingfracs = coef_nc.get_dimension_size("mixing_fraction");
        const int n_layers = coef_nc.get_dimension_size("atmos_layer");
        const int n_bnds = coef_nc.get_dimension_size("bnd");
        const int n_gpts = coef_nc.get_dimension_size("gpt");
        const int n_pairs = coef_nc.get_dimension_size("pair");
        const int n_minor_absorber_intervals_lower = coef_nc.get_dimension_size("minor_absorber_intervals_lower");
        const int n_minor_absorber_intervals_upper = coef_nc.get_dimension_size("minor_absorber_intervals_upper");
        const int n_contributors_lower = coef_nc.get_dimension_size("contributors_lower");
        const int n_contributors_upper = coef_nc.get_dimension_size("contributors_upper");

        // Read gas names.
        Array<std::string,1> gas_names(
                get_variable_string("gas_names", {n_absorbers}, coef_nc, n_char, true), {n_absorbers});

        Array<int,3> key_species(
                coef_nc.get_variable<int>("key_species", {n_bnds, n_layers, 2}),
                {2, n_layers, n_bnds});
        Array<Float,2> band_lims(coef_nc.get_variable<Float>("bnd_limits_wavenumber", {n_bnds, 2}), {2, n_bnds});
        Array<int,2> band2gpt(coef_nc.get_variable<int>("bnd_limits_gpt", {n_bnds, 2}), {2, n_bnds});
        Array<Float,1> press_ref(coef_nc.get_variable<Float>("press_ref", {n_press}), {n_press});
        Array<Float,1> temp_ref(coef_nc.get_variable<Float>("temp_ref", {n_temps}), {n_temps});

        Float temp_ref_p = coef_nc.get_variable<Float>("absorption_coefficient_ref_P");
        Float temp_ref_t = coef_nc.get_variable<Float>("absorption_coefficient_ref_T");
        Float press_ref_trop = coef_nc.get_variable<Float>("press_ref_trop");

        Array<Float,3> kminor_lower(
                coef_nc.get_variable<Float>("kminor_lower", {n_temps, n_mixingfracs, n_contributors_lower}),
                {n_contributors_lower, n_mixingfracs, n_temps});
        Array<Float,3> kminor_upper(
                coef_nc.get_variable<Float>("kminor_upper", {n_temps, n_mixingfracs, n_contributors_upper}),
                {n_contributors_upper, n_mixingfracs, n_temps});

        Array<std::string,1> gas_minor(get_variable_string("gas_minor", {n_minorabsorbers}, coef_nc, n_char),
                                       {n_minorabsorbers});

        Array<std::string,1> identifier_minor(
                get_variable_string("identifier_minor", {n_minorabsorbers}, coef_nc, n_char), {n_minorabsorbers});

        Array<std::string,1> minor_gases_lower(
                get_variable_string("minor_gases_lower", {n_minor_absorber_intervals_lower}, coef_nc, n_char),
                {n_minor_absorber_intervals_lower});
        Array<std::string,1> minor_gases_upper(
                get_variable_string("minor_gases_upper", {n_minor_absorber_intervals_upper}, coef_nc, n_char),
                {n_minor_absorber_intervals_upper});

        Array<int,2> minor_limits_gpt_lower(
                coef_nc.get_variable<int>("minor_limits_gpt_lower", {n_minor_absorber_intervals_lower, n_pairs}),
                {n_pairs, n_minor_absorber_intervals_lower});
        Array<int,2> minor_limits_gpt_upper(
                coef_nc.get_variable<int>("minor_limits_gpt_upper", {n_minor_absorber_intervals_upper, n_pairs}),
                {n_pairs, n_minor_absorber_intervals_upper});

        Array<Bool,1> minor_scales_with_density_lower(
                coef_nc.get_variable<Bool>("minor_scales_with_density_lower", {n_minor_absorber_intervals_lower}),
                {n_minor_absorber_intervals_lower});
        Array<Bool,1> minor_scales_with_density_upper(
                coef_nc.get_variable<Bool>("minor_scales_with_density_upper", {n_minor_absorber_intervals_upper}),
                {n_minor_absorber_intervals_upper});

        Array<Bool,1> scale_by_complement_lower(
                coef_nc.get_variable<Bool>("scale_by_complement_lower", {n_minor_absorber_intervals_lower}),
                {n_minor_absorber_intervals_lower});
        Array<Bool,1> scale_by_complement_upper(
                coef_nc.get_variable<Bool>("scale_by_complement_upper", {n_minor_absorber_intervals_upper}),
                {n_minor_absorber_intervals_upper});

        Array<std::string,1> scaling_gas_lower(
                get_variable_string("scaling_gas_lower", {n_minor_absorber_intervals_lower}, coef_nc, n_char),
                {n_minor_absorber_intervals_lower});
        Array<std::string,1> scaling_gas_upper(
                get_variable_string("scaling_gas_upper", {n_minor_absorber_intervals_upper}, coef_nc, n_char),
                {n_minor_absorber_intervals_upper});

        Array<int,1> kminor_start_lower(
                coef_nc.get_variable<int>("kminor_start_lower", {n_minor_absorber_intervals_lower}),
                {n_minor_absorber_intervals_lower});
        Array<int,1> kminor_start_upper(
                coef_nc.get_variable<int>("kminor_start_upper", {n_minor_absorber_intervals_upper}),
                {n_minor_absorber_intervals_upper});

        Array<Float,3> vmr_ref(
                coef_nc.get_variable<Float>("vmr_ref", {n_temps, n_extabsorbers, n_layers}),
                {n_layers, n_extabsorbers, n_temps});

        Array<Float,4> kmajor(
                coef_nc.get_variable<Float>("kmajor", {n_temps, n_press+1, n_mixingfracs, n_gpts}),
                {n_gpts, n_mixingfracs, n_press+1, n_temps});

        // Keep the size at zero, if it does not exist.
        Array<Float,3> rayl_lower;
        Array<Float,3> rayl_upper;

        if (coef_nc.variable_exists("rayl_lower"))
        {
            rayl_lower.set_dims({n_gpts, n_mixingfracs, n_temps});
            rayl_upper.set_dims({n_gpts, n_mixingfracs, n_temps});
            rayl_lower = coef_nc.get_variable<Float>("rayl_lower", {n_temps, n_mixingfracs, n_gpts});
            rayl_upper = coef_nc.get_variable<Float>("rayl_upper", {n_temps, n_mixingfracs, n_gpts});
        }

        // Is it really LW if so read these variables as well.
        if (coef_nc.variable_exists("totplnk"))
        {
            int n_internal_sourcetemps = coef_nc.get_dimension_size("temperature_Planck");

            Array<Float,2> totplnk(
                    coef_nc.get_variable<Float>( "totplnk", {n_bnds, n_internal_sourcetemps}),
                    {n_internal_sourcetemps, n_bnds});
            Array<Float,4> planck_frac(
                    coef_nc.get_variable<Float>("plank_fraction", {n_temps, n_press+1, n_mixingfracs, n_gpts}),
                    {n_gpts, n_mixingfracs, n_press+1, n_temps});

            // Construct the k-distribution.
            return Gas_optics_rrtmgp_gpu(
                    gas_concs,
                    gas_names,
                    key_species,
                    band2gpt,
                    band_lims,
                    press_ref,
                    press_ref_trop,
                    temp_ref,
                    temp_ref_p,
                    temp_ref_t,
                    vmr_ref,
                    kmajor,
                    kminor_lower,
                    kminor_upper,
                    gas_minor,
                    identifier_minor,
                    minor_gases_lower,
                    minor_gases_upper,
                    minor_limits_gpt_lower,
                    minor_limits_gpt_upper,
                    minor_scales_with_density_lower,
                    minor_scales_with_density_upper,
                    scaling_gas_lower,
                    scaling_gas_upper,
                    scale_by_complement_lower,
                    scale_by_complement_upper,
                    kminor_start_lower,
                    kminor_start_upper,
                    totplnk,
                    planck_frac,
                    rayl_lower,
                    rayl_upper);
        }
        else
        {
            Array<Float,1> solar_src_quiet(
                    coef_nc.get_variable<Float>("solar_source_quiet", {n_gpts}), {n_gpts});
            Array<Float,1> solar_src_facular(
                    coef_nc.get_variable<Float>("solar_source_facular", {n_gpts}), {n_gpts});
            Array<Float,1> solar_src_sunspot(
                    coef_nc.get_variable<Float>("solar_source_sunspot", {n_gpts}), {n_gpts});

            Float tsi = coef_nc.get_variable<Float>("tsi_default");
            Float mg_index = coef_nc.get_variable<Float>("mg_default");
            Float sb_index = coef_nc.get_variable<Float>("sb_default");

            return Gas_optics_rrtmgp_gpu(
                    gas_concs,
                    gas_names,
                    key_species,
                    band2gpt,
                    band_lims,
                    press_ref,
                    press_ref_trop,
                    temp_ref,
                    temp_ref_p,
                    temp_ref_t,
                    vmr_ref,
                    kmajor,
                    kminor_lower,
                    kminor_upper,
                    gas_minor,
                    identifier_minor,
                    minor_gases_lower,
                    minor_gases_upper,
                    minor_limits_gpt_lower,
                    minor_limits_gpt_upper,
                    minor_scales_with_density_lower,
                    minor_scales_with_density_upper,
                    scaling_gas_lower,
                    scaling_gas_upper,
                    scale_by_complement_lower,
                    scale_by_complement_upper,
                    kminor_start_lower,
                    kminor_start_upper,
                    solar_src_quiet,
                    solar_src_facular,
                    solar_src_sunspot,
                    tsi,
                    mg_index,
                    sb_index,
                    rayl_lower,
                    rayl_upper);
        }
        // End reading of k-distribution.
    }

    Cloud_optics_gpu load_and_init_cloud_optics(
            Master& master,
            const std::string& coef_file)
    {
        // READ THE COEFFICIENTS FOR THE OPTICAL SOLVER.
        Netcdf_file coef_nc(master, coef_file, Netcdf_mode::Read);

        // Read look-up table coefficient dimensions
        int n_band     = coef_nc.get_dimension_size("nband");
        int n_rghice   = coef_nc.get_dimension_size("nrghice");
        int n_size_liq = coef_nc.get_dimension_size("nsize_liq");
        int n_size_ice = coef_nc.get_dimension_size("nsize_ice");

        Array<Float,2> band_lims_wvn(coef_nc.get_variable<Float>("bnd_limits_wavenumber", {n_band, 2}), {2, n_band});

        // Read look-up table constants.
        Float radliq_lwr = coef_nc.get_variable<Float>("radliq_lwr");
        Float radliq_upr = coef_nc.get_variable<Float>("radliq_upr");
        Float radliq_fac = coef_nc.get_variable<Float>("radliq_fac");

        Float radice_lwr = coef_nc.get_variable<Float>("radice_lwr");
        Float radice_upr = coef_nc.get_variable<Float>("radice_upr");
        Float radice_fac = coef_nc.get_variable<Float>("radice_fac");

        Array<Float,2> lut_extliq(
                coef_nc.get_variable<Float>("lut_extliq", {n_band, n_size_liq}), {n_size_liq, n_band});
        Array<Float,2> lut_ssaliq(
                coef_nc.get_variable<Float>("lut_ssaliq", {n_band, n_size_liq}), {n_size_liq, n_band});
        Array<Float,2> lut_asyliq(
                coef_nc.get_variable<Float>("lut_asyliq", {n_band, n_size_liq}), {n_size_liq, n_band});

        Array<Float,3> lut_extice(
                coef_nc.get_variable<Float>("lut_extice", {n_rghice, n_band, n_size_ice}), {n_size_ice, n_band, n_rghice});
        Array<Float,3> lut_ssaice(
                coef_nc.get_variable<Float>("lut_ssaice", {n_rghice, n_band, n_size_ice}), {n_size_ice, n_band, n_rghice});
        Array<Float,3> lut_asyice(
                coef_nc.get_variable<Float>("lut_asyice", {n_rghice, n_band, n_size_ice}), {n_size_ice, n_band, n_rghice});

        return Cloud_optics_gpu(
                band_lims_wvn,
                radliq_lwr, radliq_upr, radliq_fac,
                radice_lwr, radice_upr, radice_fac,
                lut_extliq, lut_ssaliq, lut_asyliq,
                lut_extice, lut_ssaice, lut_asyice);
    }

    void configure_memory_pool(int nlays, int ncols, int nchunks, int ngpts, int nbnds)
    {
        #ifdef RTE_RRTMGP_GPU_MEMPOOL_OWN
        /* Heuristic way to set up memory pool queues */
        std::map<std::size_t, std::size_t> pool_queues = {
            {64, 20},
            {128, 20},
            {256, 10},
            {512, 10},
            {1024, 5},
            {2048, 5},
            {nchunks * ngpts * sizeof(Float), 16},
            {nchunks * nbnds * sizeof(Float), 16},
            {(nlays + 1) * ncols * sizeof(Float), 14},
            {(nlays + 1) * nchunks * sizeof(Float), 10},
            {(nlays + 1) * nchunks * nbnds * sizeof(Float), 4},
            {(nlays + 1) * nchunks * ngpts * sizeof(int)/2, 6},
            {(nlays + 1) * nchunks * ngpts * sizeof(Float), 18}
        };

        Memory_pool_gpu::init_instance(pool_queues);
        #endif
    }
}


#ifdef USECUDA
template<typename TF>
void Radiation_rrtmgp<TF>::prepare_device()
{
    auto& gd = grid.get_grid_data();

    // Set the memory pool.
    int ngpt_pool = 0;
    int nbnd_pool = 0;

    if (sw_longwave)
    {
        Netcdf_file coef_nc_lw(master, "coefficients_lw.nc", Netcdf_mode::Read);
        nbnd_pool = std::max(coef_nc_lw.get_dimension_size("bnd"), nbnd_pool);
        ngpt_pool = std::max(coef_nc_lw.get_dimension_size("gpt"), ngpt_pool);
    }

    if (sw_shortwave)
    {
        Netcdf_file coef_nc_sw(master, "coefficients_sw.nc", Netcdf_mode::Read);
        nbnd_pool = std::max(coef_nc_sw.get_dimension_size("bnd"), nbnd_pool);
        ngpt_pool = std::max(coef_nc_sw.get_dimension_size("gpt"), ngpt_pool);
    }

    configure_memory_pool(gd.ktot, gd.imax*gd.jmax, 512, ngpt_pool, nbnd_pool);

    // Initialize the pointers.
    this->gas_concs_gpu = std::make_unique<Gas_concs_gpu>(gas_concs);

    if (sw_longwave)
    {
        this->kdist_lw_gpu = std::make_unique<Gas_optics_rrtmgp_gpu>(
                load_and_init_gas_optics(master, *gas_concs_gpu, "coefficients_lw.nc"));

        this->cloud_lw_gpu = std::make_unique<Cloud_optics_gpu>(
                load_and_init_cloud_optics(master, "cloud_coefficients_lw.nc"));

        const int nsfcsize = gd.ijcells*sizeof(Float);
        cuda_safe_call(hipMalloc(&lw_flux_dn_sfc_g, nsfcsize));
        cuda_safe_call(hipMalloc(&lw_flux_up_sfc_g, nsfcsize));

        const int ncolgptsize = n_col*kdist_lw_gpu->get_ngpt()*sizeof(Float);
        cuda_safe_call(hipMalloc(&lw_flux_dn_inc_g, ncolgptsize));

        cuda_safe_call(hipMemcpy(lw_flux_dn_inc_g, lw_flux_dn_inc.ptr(), ncolgptsize, hipMemcpyHostToDevice));
    }

    if (sw_shortwave)
    {
        this->kdist_sw_gpu = std::make_unique<Gas_optics_rrtmgp_gpu>(
                load_and_init_gas_optics(master, *gas_concs_gpu, "coefficients_sw.nc"));

        this->cloud_sw_gpu = std::make_unique<Cloud_optics_gpu>(
                load_and_init_cloud_optics(master, "cloud_coefficients_sw.nc"));

        const int nsfcsize = gd.ijcells*sizeof(Float);
        cuda_safe_call(hipMalloc(&sw_flux_dn_sfc_g, nsfcsize));
        cuda_safe_call(hipMalloc(&sw_flux_up_sfc_g, nsfcsize));

        const int ncolgptsize = n_col*kdist_sw_gpu->get_ngpt()*sizeof(Float);
        cuda_safe_call(hipMalloc(&sw_flux_dn_dir_inc_g, ncolgptsize));
        cuda_safe_call(hipMalloc(&sw_flux_dn_dif_inc_g, ncolgptsize));

        cuda_safe_call(hipMemcpy(sw_flux_dn_dir_inc_g, sw_flux_dn_dir_inc.ptr(), ncolgptsize, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(sw_flux_dn_dif_inc_g, sw_flux_dn_dif_inc.ptr(), ncolgptsize, hipMemcpyHostToDevice));
    }
}
#endif


#ifdef USECUDA
template<typename TF>
void Radiation_rrtmgp<TF>::exec_longwave(
        Thermo<TF>& thermo, Timeloop<TF>& timeloop, Stats<TF>& stats,
        Array_gpu<Float,2>& flux_up, Array_gpu<Float,2>& flux_dn, Array_gpu<Float,2>& flux_net,
        const Array_gpu<Float,2>& t_lay, const Array_gpu<Float,2>& t_lev, const Array_gpu<Float,1>& t_sfc,
        const Array_gpu<Float,2>& h2o, const Array_gpu<Float,2>& clwp, const Array_gpu<Float,2>& ciwp,
        const bool compute_clouds, const int n_col)
{
    constexpr int n_col_block = 1024;

    auto& gd = grid.get_grid_data();

    const int n_lay = gd.ktot;
    const int n_lev = gd.ktot+1;

    const int n_blocks = n_col / n_col_block;
    const int n_col_block_residual = n_col % n_col_block;

    const int n_gpt = this->kdist_lw_gpu->get_ngpt();
    const int n_bnd = this->kdist_lw_gpu->get_nband();

    const Bool top_at_1 = 0;

    // Define the pointers for the subsetting.
    std::unique_ptr<Optical_props_arry_gpu> optical_props_subset =
            std::make_unique<Optical_props_1scl_gpu>(n_col_block, n_lay, *kdist_lw_gpu);
    std::unique_ptr<Source_func_lw_gpu> sources_subset =
            std::make_unique<Source_func_lw_gpu>(n_col_block, n_lay, *kdist_lw_gpu);
    std::unique_ptr<Optical_props_1scl_gpu> cloud_optical_props_subset =
            std::make_unique<Optical_props_1scl_gpu>(n_col_block, n_lay, *cloud_lw_gpu);

    std::unique_ptr<Optical_props_arry_gpu> optical_props_residual =
            std::make_unique<Optical_props_1scl_gpu>(n_col_block_residual, n_lay, *kdist_lw_gpu);
    std::unique_ptr<Source_func_lw_gpu> sources_residual =
            std::make_unique<Source_func_lw_gpu>(n_col_block_residual, n_lay, *kdist_lw_gpu);
    std::unique_ptr<Optical_props_1scl_gpu> cloud_optical_props_residual =
            std::make_unique<Optical_props_1scl_gpu>(n_col_block_residual, n_lay, *cloud_lw_gpu);

    // Make view to the TOD flux pointers
    auto lw_flux_dn_inc_local = Array_gpu<Float,2>(lw_flux_dn_inc_g, {1, n_gpt});

    // Make views to the base state pointer.
    auto p_lay = Array_gpu<Float,2>(thermo.get_basestate_fld_g("pref") + gd.kstart, {1, n_lay});
    auto p_lev = Array_gpu<Float,2>(thermo.get_basestate_fld_g("prefh") + gd.kstart, {1, n_lev});

    // CvH: this can be improved by creating a fill function for the GPU.
    Array<Float,2> emis_sfc_cpu(std::vector<Float>(n_bnd, this->emis_sfc), {n_bnd, 1});
    Array_gpu<Float,2> emis_sfc(emis_sfc_cpu);

    gas_concs_gpu->set_vmr("h2o", h2o);

    // CvH: This can be done better: we now allocate a complete array.
    Array_gpu<Float,2> col_dry({n_col, n_lay});
    Gas_optics_rrtmgp_gpu::get_col_dry(col_dry, gas_concs_gpu->get_vmr("h2o"), p_lev.subset({{ {1, n_col}, {1, n_lev} }}));

    // Constants for computation of liquid and ice droplet effective radius
    const Float sig_g = 1.34;
    const Float fac = std::exp(std::log(sig_g)*std::log(sig_g)); // no conversion to micron yet.

    const Float Nc0 = 100.e6;
    const Float Ni0 = 1.e5;

    const Float four_third_pi_N0_rho_w = (4./3.)*M_PI*Nc0*Constants::rho_w<Float>;
    const Float four_third_pi_N0_rho_i = (4./3.)*M_PI*Ni0*Constants::rho_i<Float>;

    const int block_col = 16;
    const int block_lay = 16;
    const int grid_col  = n_col_block/block_col + (n_col_block%block_col > 0);
    const int grid_lay  = n_lay/block_lay + (n_lay%block_lay > 0);

    dim3 gridGPU_re (grid_col, grid_lay, 1);
    dim3 blockGPU_re (block_col, block_lay, 1);

    // Lambda function for solving optical properties subset.
    auto call_kernels = [&](
            const int col_s_in, const int col_e_in,
            std::unique_ptr<Optical_props_arry_gpu>& optical_props_subset_in,
            std::unique_ptr<Optical_props_1scl_gpu>& cloud_optical_props_subset_in,
            Source_func_lw_gpu& sources_subset_in,
            const Array_gpu<Float,2>& emis_sfc_subset_in,
            const Array_gpu<Float,2>& lw_flux_dn_inc_subset_in,
            Fluxes_broadband_gpu& fluxes,
            Fluxes_broadband_gpu& bnd_fluxes)
    {
        const int n_col_in = col_e_in - col_s_in + 1;
        Gas_concs_gpu gas_concs_subset(*gas_concs_gpu, col_s_in, n_col_in);

        auto p_lev_subset = p_lev.subset({{ {col_s_in, col_e_in}, {1, n_lev} }});

        kdist_lw_gpu->gas_optics(
                p_lay.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                p_lev_subset,
                t_lay.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                t_sfc.subset({{ {col_s_in, col_e_in} }}),
                gas_concs_subset,
                optical_props_subset_in,
                sources_subset_in,
                col_dry.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                t_lev.subset({{ {col_s_in, col_e_in}, {1, n_lev} }}) );


        if (compute_clouds)
        {
            auto clwp_subset = clwp.subset({{ {col_s_in, col_e_in}, {1, n_lay} }});
            auto ciwp_subset = ciwp.subset({{ {col_s_in, col_e_in}, {1, n_lay} }});
            Array_gpu<Float,2> rel({n_col_in, n_lay});
            Array_gpu<Float,2> rei({n_col_in, n_lay});

            effective_radius_and_ciwp_to_gm2<<<gridGPU_re, blockGPU_re>>>(
                    rel.ptr(), rei.ptr(),
                    clwp_subset.ptr(), ciwp_subset.ptr(),
                    gd.dz_g,
                    n_col_in, n_lay, gd.kstart,
                    four_third_pi_N0_rho_w, four_third_pi_N0_rho_i, fac);

            cloud_lw_gpu->cloud_optics(
                    clwp_subset,
                    ciwp_subset,
                    rel,
                    rei,
                    *cloud_optical_props_subset_in);

            // Add the cloud optical props to the gas optical properties.
            add_to(
                    dynamic_cast<Optical_props_1scl_gpu&>(*optical_props_subset_in),
                    dynamic_cast<Optical_props_1scl_gpu&>(*cloud_optical_props_subset_in));
        }

        Array_gpu<Float,3> gpt_flux_up({n_col_in, n_lev, n_gpt});
        Array_gpu<Float,3> gpt_flux_dn({n_col_in, n_lev, n_gpt});

        constexpr int n_ang = 1;

        rte_lw_gpu.rte_lw(
                optical_props_subset_in,
                top_at_1,
                sources_subset_in,
                emis_sfc_subset_in,
                lw_flux_dn_inc_subset_in,
                gpt_flux_up,
                gpt_flux_dn,
                n_ang);

        fluxes.reduce(gpt_flux_up, gpt_flux_dn, optical_props_subset_in, top_at_1);

        // Copy the data to the output.
        subset_kernel_launcher_cuda::get_from_subset(
                n_col, n_lev, n_col_in, col_s_in, flux_up.ptr(), flux_dn.ptr(), flux_net.ptr(),
                fluxes.get_flux_up().ptr(), fluxes.get_flux_dn().ptr(), fluxes.get_flux_net().ptr());
    };

    for (int b=1; b<=n_blocks; ++b)
    {
        const int col_s = (b-1) * n_col_block + 1;
        const int col_e =  b    * n_col_block;

        Array_gpu<Float,2> emis_sfc_subset = emis_sfc.subset({{ {1, n_bnd}, {col_s, col_e} }});
        Array_gpu<Float,2> lw_flux_dn_inc_subset = lw_flux_dn_inc_local.subset({{ {col_s, col_e}, {1, n_gpt} }});

        std::unique_ptr<Fluxes_broadband_gpu> fluxes_subset =
                std::make_unique<Fluxes_broadband_gpu>(n_col_block, n_lev);
        std::unique_ptr<Fluxes_broadband_gpu> bnd_fluxes_subset =
                std::make_unique<Fluxes_byband_gpu>(n_col_block, n_lev, n_bnd);

        call_kernels(
                col_s, col_e,
                optical_props_subset,
                cloud_optical_props_subset,
                *sources_subset,
                emis_sfc_subset,
                lw_flux_dn_inc_subset,
                *fluxes_subset,
                *bnd_fluxes_subset);
    }

    if (n_col_block_residual > 0)
    {
        const int col_s = n_col - n_col_block_residual + 1;
        const int col_e = n_col;

        Array_gpu<Float,2> emis_sfc_residual = emis_sfc.subset({{ {1, n_bnd}, {col_s, col_e} }});
        Array_gpu<Float,2> lw_flux_dn_inc_residual = lw_flux_dn_inc_local.subset({{ {col_s, col_e}, {1, n_gpt} }});

        std::unique_ptr<Fluxes_broadband_gpu> fluxes_residual =
                std::make_unique<Fluxes_broadband_gpu>(n_col_block_residual, n_lev);
        std::unique_ptr<Fluxes_broadband_gpu> bnd_fluxes_residual =
                std::make_unique<Fluxes_byband_gpu>(n_col_block_residual, n_lev, n_bnd);

        call_kernels(
                col_s, col_e,
                optical_props_residual,
                cloud_optical_props_residual,
                *sources_residual,
                emis_sfc_residual,
                lw_flux_dn_inc_residual,
                *fluxes_residual,
                *bnd_fluxes_residual);
    }
}
#endif

#ifdef USECUDA
template<typename TF>
void Radiation_rrtmgp<TF>::exec_shortwave(
        Thermo<TF>& thermo, Timeloop<TF>& timeloop, Stats<TF>& stats,
        Array_gpu<Float,2>& flux_up, Array_gpu<Float,2>& flux_dn, Array_gpu<Float,2>& flux_dn_dir, Array_gpu<Float,2>& flux_net,
        const Array_gpu<Float,2>& t_lay, const Array_gpu<Float,2>& t_lev,
        const Array_gpu<Float,2>& h2o, const Array_gpu<Float,2>& clwp, const Array_gpu<Float,2>& ciwp,
        const bool compute_clouds, const int n_col)
{
    constexpr int n_col_block = 1024;

    auto& gd = grid.get_grid_data();

    const int n_lay = gd.ktot;
    const int n_lev = gd.ktot+1;

    const int n_blocks = n_col / n_col_block;
    const int n_col_block_residual = n_col % n_col_block;

    const int n_gpt = this->kdist_sw_gpu->get_ngpt();
    const int n_bnd = this->kdist_sw_gpu->get_nband();

    const Bool top_at_1 = 0;

    // Define the pointers for the subsetting.
    std::unique_ptr<Optical_props_arry_gpu> optical_props_subset =
            std::make_unique<Optical_props_2str_gpu>(n_col_block, n_lay, *kdist_sw_gpu);
    std::unique_ptr<Optical_props_2str_gpu> cloud_optical_props_subset =
            std::make_unique<Optical_props_2str_gpu>(n_col_block, n_lay, *cloud_sw_gpu);

    std::unique_ptr<Optical_props_arry_gpu> optical_props_residual =
            std::make_unique<Optical_props_2str_gpu>(n_col_block_residual, n_lay, *kdist_sw_gpu);
    std::unique_ptr<Optical_props_2str_gpu> cloud_optical_props_residual =
            std::make_unique<Optical_props_2str_gpu>(n_col_block_residual, n_lay, *cloud_sw_gpu);

    // Make views to the base state pointer.
    auto p_lay = Array_gpu<Float,2>(thermo.get_basestate_fld_g("pref") + gd.kstart, {1, n_lay});
    auto p_lev = Array_gpu<Float,2>(thermo.get_basestate_fld_g("prefh") + gd.kstart, {1, n_lev});

    // Make views to the TOD flux pointers
    auto sw_flux_dn_dir_inc_local = Array_gpu<Float,2>(sw_flux_dn_dir_inc_g, {1, n_gpt});
    auto sw_flux_dn_dif_inc_local = Array_gpu<Float,2>(sw_flux_dn_dif_inc_g, {1, n_gpt});

    // Create the boundary conditions
    Array<Float,1> mu0_cpu(std::vector<Float>(1, this->mu0), {1});
    Array_gpu<Float,1> mu0(mu0_cpu);
    Array<Float,2> sfc_alb_dir_cpu(std::vector<Float>(n_bnd, this->sfc_alb_dir), {n_bnd, 1});
    Array_gpu<Float,2> sfc_alb_dir(sfc_alb_dir_cpu);
    Array<Float,2> sfc_alb_dif_cpu(std::vector<Float>(n_bnd, this->sfc_alb_dif), {n_bnd, 1});
    Array_gpu<Float,2> sfc_alb_dif(sfc_alb_dif_cpu);

    gas_concs_gpu->set_vmr("h2o", h2o);

    // CvH: This can be done better: we now allocate a complete array.
    Array_gpu<Float,2> col_dry({n_col, n_lay});
    Gas_optics_rrtmgp_gpu::get_col_dry(col_dry, gas_concs_gpu->get_vmr("h2o"), p_lev.subset({{ {1, n_col}, {1, n_lev} }}));

    // Constants for computation of liquid and ice droplet effective radius
    const Float sig_g = 1.34;
    const Float fac = std::exp(std::log(sig_g)*std::log(sig_g)); // no conversion to micron yet.

    const Float Nc0 = 100.e6;
    const Float Ni0 = 1.e5;

    const Float four_third_pi_N0_rho_w = (4./3.)*M_PI*Nc0*Constants::rho_w<Float>;
    const Float four_third_pi_N0_rho_i = (4./3.)*M_PI*Ni0*Constants::rho_i<Float>;

    const int block_col = 16;
    const int block_lay = 16;
    const int grid_col  = n_col_block/block_col + (n_col_block%block_col > 0);
    const int grid_lay  = n_lay/block_lay + (n_lay%block_lay > 0);

    dim3 gridGPU_re (grid_col, grid_lay, 1);
    dim3 blockGPU_re (block_col, block_lay, 1);

    // Lambda function for solving optical properties subset.
    auto call_kernels = [&](
            const int col_s_in, const int col_e_in,
            std::unique_ptr<Optical_props_arry_gpu>& optical_props_subset_in,
            std::unique_ptr<Optical_props_2str_gpu>& cloud_optical_props_subset_in,
            const Array_gpu<Float,1>& mu0_subset_in,
            const Array_gpu<Float,2>& sw_flux_dn_dir_inc_subset_in,
            const Array_gpu<Float,2>& sfc_alb_dir_subset_in,
            const Array_gpu<Float,2>& sfc_alb_dif_subset_in,
            const Array_gpu<Float,2>& sw_flux_dn_dif_inc_subset_in,
            Fluxes_broadband_gpu& fluxes,
            Fluxes_broadband_gpu& bnd_fluxes)
    {
        const int n_col_in = col_e_in - col_s_in + 1;
        Gas_concs_gpu gas_concs_subset(*gas_concs_gpu, col_s_in, n_col_in);
        Array_gpu<Float,2> toa_src_dummy({n_col_in, n_gpt});

        auto p_lev_subset = p_lev.subset({{ {col_s_in, col_e_in}, {1, n_lev} }});
        kdist_sw_gpu->gas_optics(
                p_lay.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                p_lev_subset,
                t_lay.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                gas_concs_subset,
                optical_props_subset_in,
                toa_src_dummy,
                col_dry.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}) );


        if (compute_clouds)
        {
            auto clwp_subset = clwp.subset({{ {col_s_in, col_e_in}, {1, n_lay} }});
            auto ciwp_subset = ciwp.subset({{ {col_s_in, col_e_in}, {1, n_lay} }});
            Array_gpu<Float,2> rel({n_col_in, n_lay});
            Array_gpu<Float,2> rei({n_col_in, n_lay});

            effective_radius_and_ciwp_to_gm2<<<gridGPU_re, blockGPU_re>>>(
                    rel.ptr(), rei.ptr(),
                    clwp_subset.ptr(), ciwp_subset.ptr(),
                    gd.dz_g,
                    n_col_in, n_lay, gd.kstart,
                    four_third_pi_N0_rho_w, four_third_pi_N0_rho_i, fac);

            cloud_sw_gpu->cloud_optics(
                    clwp_subset,
                    ciwp_subset,
                    rel,
                    rei,
                    *cloud_optical_props_subset_in);

            // Add the cloud optical props to the gas optical properties.
            add_to(
                    dynamic_cast<Optical_props_2str_gpu&>(*optical_props_subset_in),
                    dynamic_cast<Optical_props_2str_gpu&>(*cloud_optical_props_subset_in));
        }

        Array_gpu<Float,3> gpt_flux_up({n_col_in, n_lev, n_gpt});
        Array_gpu<Float,3> gpt_flux_dn({n_col_in, n_lev, n_gpt});
        Array_gpu<Float,3> gpt_flux_dn_dir({n_col_in, n_lev, n_gpt});

        rte_sw_gpu.rte_sw(
                optical_props_subset_in,
                top_at_1,
                mu0_subset_in,
                sw_flux_dn_dir_inc_subset_in,
                sfc_alb_dir_subset_in,
                sfc_alb_dif_subset_in,
                sw_flux_dn_dif_inc_subset_in,
                gpt_flux_up,
                gpt_flux_dn,
                gpt_flux_dn_dir);

        fluxes.reduce(gpt_flux_up, gpt_flux_dn, gpt_flux_dn_dir, optical_props_subset_in, top_at_1);

        // Copy the data to the output.
        subset_kernel_launcher_cuda::get_from_subset(
                n_col, n_lev, n_col_in, col_s_in, flux_up.ptr(), flux_dn.ptr(), flux_dn_dir.ptr(), flux_net.ptr(),
                fluxes.get_flux_up().ptr(), fluxes.get_flux_dn().ptr(), fluxes.get_flux_dn_dir().ptr(), fluxes.get_flux_net().ptr());
    };

    for (int b=1; b<=n_blocks; ++b)
    {
        const int col_s = (b-1) * n_col_block + 1;
        const int col_e =  b    * n_col_block;

        Array_gpu<Float,1> mu0_subset = mu0.subset({{ {col_s, col_e} }});
        Array_gpu<Float,2> sw_flux_dn_dir_inc_subset = sw_flux_dn_dir_inc_local.subset({{ {col_s, col_e}, {1, n_gpt} }});
        Array_gpu<Float,2> sfc_alb_dir_subset = sfc_alb_dir.subset({{ {1, n_bnd}, {col_s, col_e} }});
        Array_gpu<Float,2> sfc_alb_dif_subset = sfc_alb_dif.subset({{ {1, n_bnd}, {col_s, col_e} }});
        Array_gpu<Float,2> sw_flux_dn_dif_inc_subset = sw_flux_dn_dif_inc_local.subset({{ {col_s, col_e}, {1, n_gpt} }});

        std::unique_ptr<Fluxes_broadband_gpu> fluxes_subset =
                std::make_unique<Fluxes_broadband_gpu>(n_col_block, n_lev);
        std::unique_ptr<Fluxes_broadband_gpu> bnd_fluxes_subset =
                std::make_unique<Fluxes_byband_gpu>(n_col_block, n_lev, n_bnd);
        call_kernels(
                col_s, col_e,
                optical_props_subset,
                cloud_optical_props_subset,
                mu0_subset,
                sw_flux_dn_dir_inc_subset,
                sfc_alb_dir_subset,
                sfc_alb_dif_subset,
                sw_flux_dn_dif_inc_subset,
                *fluxes_subset,
                *bnd_fluxes_subset);

    }

    if (n_col_block_residual > 0)
    {
        const int col_s = n_col - n_col_block_residual + 1;
        const int col_e = n_col;

        Array_gpu<Float,1> mu0_residual = mu0.subset({{ {col_s, col_e} }});
        Array_gpu<Float,2> sw_flux_dn_dir_inc_residual = sw_flux_dn_dir_inc_local.subset({{ {col_s, col_e}, {1, n_gpt} }});
        Array_gpu<Float,2> sfc_alb_dir_residual = sfc_alb_dir.subset({{ {1, n_bnd}, {col_s, col_e} }});
        Array_gpu<Float,2> sfc_alb_dif_residual = sfc_alb_dif.subset({{ {1, n_bnd}, {col_s, col_e} }});
        Array_gpu<Float,2> sw_flux_dn_dif_inc_residual = sw_flux_dn_dif_inc_local.subset({{ {col_s, col_e}, {1, n_gpt} }});

        std::unique_ptr<Fluxes_broadband_gpu> fluxes_residual =
                std::make_unique<Fluxes_broadband_gpu>(n_col_block_residual, n_lev);
        std::unique_ptr<Fluxes_broadband_gpu> bnd_fluxes_residual =
                std::make_unique<Fluxes_byband_gpu>(n_col_block_residual, n_lev, n_bnd);

        call_kernels(
                col_s, col_e,
                optical_props_residual,
                cloud_optical_props_residual,
                mu0_residual,
                sw_flux_dn_dir_inc_residual,
                sfc_alb_dir_residual,
                sfc_alb_dif_residual,
                sw_flux_dn_dif_inc_residual,
                *fluxes_residual,
                *bnd_fluxes_residual);
    }
}
#endif

#ifdef USECUDA
template <typename TF>
void Radiation_rrtmgp<TF>::exec(Thermo<TF>& thermo, double time, Timeloop<TF>& timeloop, Stats<TF>& stats)
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU_3d (gridi, gridj, gd.kmax+1);
    dim3 blockGPU_3d(blocki, blockj, 1);
    dim3 gridGPU_2d (gridi, gridj, 1);
    dim3 blockGPU_2d(blocki, blockj, 1);

    const bool do_radiation = ((timeloop.get_itime() % idt_rad == 0) && !timeloop.in_substep()) ;
    const bool do_radiation_stats = timeloop.is_stats_step();

    if (do_radiation)
    {
        // Set the tendency to zero.
        // std::fill(fields.sd.at("thlt_rad")->fld.begin(), fields.sd.at("thlt_rad")->fld.end(), Float(0.));
        hipMemset(fields.sd.at("thlt_rad")->fld_g, 0, gd.ncells*sizeof(Float));

        auto t_lay = fields.get_tmp_g();
        auto t_lev = fields.get_tmp_g();
        auto h2o   = fields.get_tmp_g(); // This is the volume mixing ratio, not the specific humidity of vapor.
        auto clwp  = fields.get_tmp_g();
        auto ciwp  = fields.get_tmp_g();

        // Set the input to the radiation on a 3D grid without ghost cells.
        thermo.get_radiation_fields_g(*t_lay, *t_lev, *h2o, *clwp, *ciwp);

        const int nmaxh = gd.imax*gd.jmax*(gd.ktot+1);
        const int ijmax = gd.imax*gd.jmax;

        // Create views on existing variables.
        Array_gpu<Float,2> t_lay_a(t_lay->fld_g, {gd.imax*gd.jmax, gd.ktot});
        Array_gpu<Float,2> t_lev_a(t_lev->fld_g, {gd.imax*gd.jmax, gd.ktot+1});
        Array_gpu<Float,1> t_sfc_a(t_lev->fld_bot_g, {gd.imax*gd.jmax});
        Array_gpu<Float,2> h2o_a(h2o->fld_g, {gd.imax*gd.jmax, gd.ktot});
        Array_gpu<Float,2> clwp_a(clwp->fld_g, {gd.imax*gd.jmax, gd.ktot});
        Array_gpu<Float,2> ciwp_a(ciwp->fld_g, {gd.imax*gd.jmax, gd.ktot});

        // Flux fields.
        Array_gpu<Float,2> flux_up ({gd.imax*gd.jmax, gd.ktot+1});
        Array_gpu<Float,2> flux_dn ({gd.imax*gd.jmax, gd.ktot+1});
        Array_gpu<Float,2> flux_net({gd.imax*gd.jmax, gd.ktot+1});

        const bool compute_clouds = true;

        try
        {
            if (sw_longwave)
            {
                const int n_col = gd.imax*gd.jmax;
                exec_longwave(
                        thermo, timeloop, stats,
                        flux_up, flux_dn, flux_net,
                        t_lay_a, t_lev_a, t_sfc_a, h2o_a, clwp_a, ciwp_a,
                        compute_clouds, n_col);

                calc_tendency<<<gridGPU_3d, blockGPU_3d>>>(
                        fields.sd.at("thlt_rad")->fld_g,
                        flux_up.ptr(), flux_dn.ptr(),
                        fields.rhoref_g, thermo.get_basestate_fld_g("exner"),
                        gd.dz_g,
                        gd.istart, gd.jstart, gd.kstart,
                        gd.iend, gd.jend, gd.kend,
                        gd.igc, gd.jgc, gd.kgc,
                        gd.icells, gd.ijcells,
                        gd.imax, gd.imax*gd.jmax);
                cuda_check_error();

                store_surface_fluxes<<<gridGPU_2d, blockGPU_2d>>>(
                        lw_flux_up_sfc_g, lw_flux_dn_sfc_g,
                        flux_up.ptr(), flux_dn.ptr(),
                        gd.istart, gd.iend,
                        gd.jstart, gd.jend,
                        gd.igc, gd.jgc,
                        gd.icells, gd.ijcells,
                        gd.imax);
                cuda_check_error();

                if (do_radiation_stats)
                {
                    // Make sure that the top boundary is taken into account in case of fluxes.
                    auto do_gcs = [&](Field3d<Float>& out, const Array_gpu<Float,2>& in)
                    {
                        add_ghost_cells_g<<<gridGPU_3d, blockGPU_3d>>>(
                                out.fld_g, in.ptr(),
                                gd.istart, gd.jstart, gd.kstart,
                                gd.iend, gd.jend, gd.kend+1,
                                gd.icells, gd.ijcells,
                                gd.imax, gd.imax*gd.jmax);
                    };

                    do_gcs(*fields.sd.at("lw_flux_up"), flux_up);
                    do_gcs(*fields.sd.at("lw_flux_dn"), flux_dn);

                    // clear sky
                    if (sw_clear_sky_stats)
                    {
                        const int n_col = gd.imax*gd.jmax;
                        exec_longwave(
                                thermo, timeloop, stats,
                                flux_up, flux_dn, flux_net,
                                t_lay_a, t_lev_a, t_sfc_a, h2o_a, clwp_a, ciwp_a,
                                !compute_clouds, n_col);

                        do_gcs(*fields.sd.at("lw_flux_up_clear"), flux_up);
                        do_gcs(*fields.sd.at("lw_flux_dn_clear"), flux_dn);
                    }
                }
            }

            if (sw_shortwave)
            {
                Array_gpu<Float,2> flux_dn_dir({gd.imax*gd.jmax, gd.ktot+1});

                // Single column solve of background profile for TOA conditions
                if (!sw_fixed_sza)
                {
                    // Update the solar zenith angle and sun-earth distance.
                    set_sun_location(timeloop);

                    if (is_day(this->mu0))
                    {
                        const int n_bnd = kdist_sw->get_nband();
                        const int n_gpt = kdist_sw->get_ngpt();

                        // Calculate new background column (on the CPU).
                        Float* ph_g = thermo.get_basestate_fld_g("prefh");
                        Float p_top;
                        hipMemcpy(&p_top, &ph_g[gd.kend], sizeof(TF), hipMemcpyDeviceToHost);

                        set_background_column_shortwave(p_top);

                        // Copy TOD fluxes to GPU
                        const int ncolgptsize = n_col * n_gpt * sizeof(Float);
                        cuda_safe_call(hipMemcpy(sw_flux_dn_dir_inc_g, sw_flux_dn_dir_inc.ptr(), ncolgptsize, hipMemcpyHostToDevice));
                        cuda_safe_call(hipMemcpy(sw_flux_dn_dif_inc_g, sw_flux_dn_dif_inc.ptr(), ncolgptsize, hipMemcpyHostToDevice));
                    }
                }

                if (is_day(this->mu0))
                {
                    const int n_col = gd.imax*gd.jmax;
                    exec_shortwave(
                            thermo, timeloop, stats,
                            flux_up, flux_dn, flux_dn_dir, flux_net,
                            t_lay_a, t_lev_a, h2o_a, clwp_a, ciwp_a,
                            compute_clouds, n_col);

                    calc_tendency<<<gridGPU_3d, blockGPU_3d>>>(
                            fields.sd.at("thlt_rad")->fld_g,
                            flux_up.ptr(), flux_dn.ptr(),
                            fields.rhoref_g, thermo.get_basestate_fld_g("exner"),
                            gd.dz_g,
                            gd.istart, gd.jstart, gd.kstart,
                            gd.iend, gd.jend, gd.kend,
                            gd.igc, gd.jgc, gd.kgc,
                            gd.icells, gd.ijcells,
                            gd.imax, gd.imax*gd.jmax);
                    cuda_check_error();

                    store_surface_fluxes<<<gridGPU_2d, blockGPU_2d>>>(
                            sw_flux_up_sfc_g, sw_flux_dn_sfc_g,
                            flux_up.ptr(), flux_dn.ptr(),
                            gd.istart, gd.iend,
                            gd.jstart, gd.jend,
                            gd.igc, gd.jgc,
                            gd.icells, gd.ijcells,
                            gd.imax);
                    cuda_check_error();
                }
                else
                {
                    // Set the surface fluxes to zero, for (e.g.) the land-surface model.
                    hipMemset(sw_flux_dn_sfc_g, 0, gd.ijcells*sizeof(Float));
                    hipMemset(sw_flux_up_sfc_g, 0, gd.ijcells*sizeof(Float));
                }

                if (do_radiation_stats)
                {
                    // Make sure that the top boundary is taken into account in case of fluxes.
                    auto do_gcs = [&](Field3d<Float>& out, const Array_gpu<Float,2>& in)
                    {
                        add_ghost_cells_g<<<gridGPU_3d, blockGPU_3d>>>(
                                out.fld_g, in.ptr(),
                                gd.istart, gd.jstart, gd.kstart,
                                gd.iend, gd.jend, gd.kend+1,
                                gd.icells, gd.ijcells,
                                gd.imax, gd.imax*gd.jmax);
                    };

                    if (!is_day(this->mu0))
                    {
                        flux_up.fill(Float(0.));
                        flux_dn.fill(Float(0.));
                        flux_dn_dir.fill(Float(0.));
                    }

                    do_gcs(*fields.sd.at("sw_flux_up"), flux_up);
                    do_gcs(*fields.sd.at("sw_flux_dn"), flux_dn);
                    do_gcs(*fields.sd.at("sw_flux_dn_dir"), flux_dn_dir);

                    // clear sky
                    if (sw_clear_sky_stats)
                    {
                        if (is_day(this->mu0))
                        {
                            const int n_col = gd.imax*gd.jmax;
                            exec_shortwave(
                                    thermo, timeloop, stats,
                                    flux_up, flux_dn, flux_dn_dir, flux_net,
                                    t_lay_a, t_lev_a, h2o_a, clwp_a, ciwp_a,
                                    !compute_clouds, n_col);
                        }

                        do_gcs(*fields.sd.at("sw_flux_up_clear"), flux_up);
                        do_gcs(*fields.sd.at("sw_flux_dn_clear"), flux_dn);
                        do_gcs(*fields.sd.at("sw_flux_dn_dir_clear"), flux_dn_dir);
                    }
                 }
             }
         } // End try block.
         catch (std::exception& e)
         {
             #ifdef USEMPI
            std::cout << "SINGLE PROCESS EXCEPTION: " << e.what() << std::endl;
            MPI_Abort(MPI_COMM_WORLD, 1);
            #else
            throw;
            #endif
        }

        fields.release_tmp_g(t_lay);
        fields.release_tmp_g(t_lev);
        fields.release_tmp_g(h2o);
        fields.release_tmp_g(clwp);
        fields.release_tmp_g(ciwp);
    }

    // Always add the tendency.
    add_tendency<<<gridGPU_3d, blockGPU_3d>>>(
            fields.st.at("thl")->fld_g,
            fields.sd.at("thlt_rad")->fld_g,
            gd.istart, gd.jstart, gd.kstart,
            gd.iend, gd.jend, gd.kend,
            gd.icells, gd.ijcells);
    cuda_check_error();

    hipDeviceSynchronize();
    stats.calc_tend(*fields.st.at("thl"), tend_name);
}


template <typename TF>
std::vector<TF>& Radiation_rrtmgp<TF>::get_surface_radiation(const std::string& name)
{
    throw std::runtime_error("Radiation_rrtmgp is not implemented yet on the GPU");
}


template <typename TF>
TF* Radiation_rrtmgp<TF>::get_surface_radiation_g(const std::string& name)
{
    if (name == "sw_down")
        return sw_flux_dn_sfc_g;
    else if (name == "sw_up")
        return sw_flux_up_sfc_g;
    else if (name == "lw_down")
        return lw_flux_dn_sfc_g;
    else if (name == "lw_up")
        return lw_flux_up_sfc_g;
    else
    {
        std::string error = "Variable \"" + name + "\" is not a valid surface radiation field";
        throw std::runtime_error(error);
    }
}


template <typename TF>
void Radiation_rrtmgp<TF>::clear_device()
{
    cuda_safe_call(hipFree(lw_flux_dn_sfc_g));
    cuda_safe_call(hipFree(lw_flux_up_sfc_g));
    cuda_safe_call(hipFree(sw_flux_dn_sfc_g));
    cuda_safe_call(hipFree(sw_flux_up_sfc_g));
}


template<typename TF>
void Radiation_rrtmgp<TF>::exec_individual_column_stats(
        Column<TF>& column, Thermo<TF>& thermo, Timeloop<TF>& timeloop, Stats<TF>& stats)
{
    auto& gd = grid.get_grid_data();

    const int n_stat_col = column.get_n_columns();

    // We can safely do nothing if there are no columns on this proc.
    if (n_stat_col == 0)
        return;

    auto tmp = fields.get_tmp();

    // Get the column indices on CPU and GPU.
    std::vector<int> col_i;
    std::vector<int> col_j;
    column.get_column_locations(col_i, col_j);

    int* col_i_g = column.get_column_location_g("i");
    int* col_j_g = column.get_column_location_g("j");

    // Retrieve the thermo fields for the output columns.
    auto tmp_g = fields.get_tmp_g();
    thermo.get_radiation_columns_g(*tmp_g, col_i_g, col_j_g, n_stat_col);

    // Create Array_gpu views on the thermo columns.
    int offset = 0;
    Array_gpu<Float,2> t_lay_a(&tmp_g->fld_g[offset], {n_stat_col, gd.ktot  }); offset += n_stat_col * gd.ktot;
    Array_gpu<Float,2> t_lev_a(&tmp_g->fld_g[offset], {n_stat_col, gd.ktot+1}); offset += n_stat_col * (gd.ktot+1);
    Array_gpu<Float,1> t_sfc_a(&tmp_g->fld_g[offset], {n_stat_col           }); offset += n_stat_col;
    Array_gpu<Float,2> h2o_a  (&tmp_g->fld_g[offset], {n_stat_col, gd.ktot  }); offset += n_stat_col * gd.ktot;
    Array_gpu<Float,2> clwp_a (&tmp_g->fld_g[offset], {n_stat_col, gd.ktot  }); offset += n_stat_col * gd.ktot;
    Array_gpu<Float,2> ciwp_a (&tmp_g->fld_g[offset], {n_stat_col, gd.ktot  });

    // Flux fields.
    Array_gpu<Float,2> flux_up ({n_stat_col, gd.ktot+1});
    Array_gpu<Float,2> flux_dn ({n_stat_col, gd.ktot+1});
    Array_gpu<Float,2> flux_net({n_stat_col, gd.ktot+1});

    bool compute_clouds = true;

    // Lambda function to set the column data and save column statistics.
    auto save_column = [&](
            const Array_gpu<Float,2>& array, const std::string& name)
    {
        const int size = array.dim(2);

        for (int n=0; n<n_stat_col; ++n)
        {
            // Copy data from GPU.
            //Array_gpu<Float,2> array_col(array.subset({{ {n+1, n+1}, {1, size} }}));
            //cuda_safe_call(hipMemcpy(
            //    &tmp->fld_mean.data()[gd.kstart], array_col.ptr(), size*sizeof(Float), hipMemcpyDeviceToHost));

            cuda_safe_call(hipMemcpy2D(
                        &tmp->fld_mean.data()[gd.kstart], sizeof(Float),
                        &array.ptr()[n], n_stat_col*sizeof(Float),
                        sizeof(Float), size, hipMemcpyDeviceToHost));

            const TF no_offset = 0;
            column.set_individual_column(name, tmp->fld_mean.data(), no_offset, col_i[n], col_j[n]);
        }
    };

    if (sw_longwave)
    {
        exec_longwave(
                thermo, timeloop, stats,
                flux_up, flux_dn, flux_net,
                t_lay_a, t_lev_a, t_sfc_a, h2o_a, clwp_a, ciwp_a,
                compute_clouds, n_stat_col);

        save_column(flux_up, "lw_flux_up");
        save_column(flux_dn, "lw_flux_dn");

        if (sw_clear_sky_stats)
        {
            exec_longwave(
                    thermo, timeloop, stats,
                    flux_up, flux_dn, flux_net,
                    t_lay_a, t_lev_a, t_sfc_a, h2o_a, clwp_a, ciwp_a,
                    !compute_clouds, n_stat_col);

            save_column(flux_up, "lw_flux_up_clear");
            save_column(flux_dn, "lw_flux_dn_clear");
        }
    }

    if (sw_shortwave)
    {
        Array_gpu<Float,2> flux_dn_dir({n_stat_col, gd.ktot+1});

        // Single column solve of background profile for TOA conditions
        if (!sw_fixed_sza)
        {
            // Update the solar zenith angle and sun-earth distance.
            set_sun_location(timeloop);

            if (is_day(this->mu0))
            {
                const int n_bnd = kdist_sw->get_nband();
                const int n_gpt = kdist_sw->get_ngpt();

                // Calculate new background column (on the CPU).
                Float* ph_g = thermo.get_basestate_fld_g("prefh");
                Float p_top;
                hipMemcpy(&p_top, &ph_g[gd.kend], sizeof(TF), hipMemcpyDeviceToHost);

                set_background_column_shortwave(p_top);

                // Copy TOD fluxes to GPU
                const int ncolgptsize = n_col * n_gpt * sizeof(Float);
                cuda_safe_call(hipMemcpy(sw_flux_dn_dir_inc_g, sw_flux_dn_dir_inc.ptr(), ncolgptsize, hipMemcpyHostToDevice));
                cuda_safe_call(hipMemcpy(sw_flux_dn_dif_inc_g, sw_flux_dn_dif_inc.ptr(), ncolgptsize, hipMemcpyHostToDevice));
            }
        }

        if (is_day(this->mu0))
        {
            exec_shortwave(
                    thermo, timeloop, stats,
                    flux_up, flux_dn, flux_dn_dir, flux_net,
                    t_lay_a, t_lev_a, h2o_a, clwp_a, ciwp_a,
                    compute_clouds, n_stat_col);
        }
        else
        {
            flux_up.fill(Float(0.));
            flux_dn.fill(Float(0.));
            flux_dn_dir.fill(Float(0.));
        }

        save_column(flux_up, "sw_flux_up");
        save_column(flux_dn, "sw_flux_dn");
        save_column(flux_dn_dir, "sw_flux_dn_dir");

        // clear sky
        if (sw_clear_sky_stats)
        {
            if (is_day(this->mu0))
            {
                exec_shortwave(
                        thermo, timeloop, stats,
                        flux_up, flux_dn, flux_dn_dir, flux_net,
                        t_lay_a, t_lev_a, h2o_a, clwp_a, ciwp_a,
                        !compute_clouds, n_stat_col);
            }

            save_column(flux_up, "sw_flux_up_clear");
            save_column(flux_dn, "sw_flux_dn_clear");
            save_column(flux_dn_dir, "sw_flux_dn_dir_clear");
        }
    }

    fields.release_tmp_g(tmp_g);
    fields.release_tmp(tmp);
}
#endif

#ifdef FLOAT_SINGLE
template class Radiation_rrtmgp<float>;
#else
template class Radiation_rrtmgp<double>;
#endif
