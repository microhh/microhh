#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2020 Chiel van Heerwaarden
 * Copyright (c) 2011-2020 Thijs Heus
 * Copyright (c) 2014-2020 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <cstdio>
#include <iostream>
#include "master.h"
#include "grid.h"
#include "fields.h"
#include "defines.h"
#include "constants.h"
#include "thermo.h"
#include "model.h"
#include "master.h"
#include "tools.h"
#include "timedep.h"
#include "monin_obukhov.h"
#include "boundary_surface.h"
#include "boundary_surface_kernels_gpu.h"
#include "fast_math.h"
#include "column.h"

namespace
{
    namespace most = Monin_obukhov;
    namespace fm = Fast_math;
    namespace bsk = Boundary_surface_kernels_g;

    const int nzL = 10000; // Size of the lookup table for MO iterations.

    template<typename TF> __device__
    TF find_obuk_g(
            const float* const __restrict__ zL,
            const float* const __restrict__ f,
            int &n,
            const TF Ri,
            const TF zsl)
    {
        // Determine search direction.
        if ((f[n]-Ri) > 0.f)
            while ( (f[n-1]-Ri) > 0.f && n > 0) { --n; }
        else
            while ( (f[n]-Ri) < 0.f && n < (nzL-1) ) { ++n; }

        const TF zL0 = (n == 0 || n == nzL-1) ? zL[n] : zL[n-1] + (Ri-f[n-1]) / (f[n]-f[n-1]) * (zL[n]-zL[n-1]);

        return zsl/zL0;
    }


    template<typename TF> __device__
    TF calc_obuk_noslip_flux_g(
            const float* const __restrict__ zL,
            const float* const __restrict__ f,
            int& n,
            const TF du,
            const TF bfluxbot,
            const TF zsl)
    {
        // Calculate the appropriate Richardson number.
        const TF Ri = -Constants::kappa<TF> * bfluxbot * zsl / fm::pow3(du);
        return find_obuk_g(zL, f, n, Ri, zsl);
    }

    template<typename TF> __device__
    TF calc_obuk_noslip_dirichlet_g(
            const float* const __restrict__ zL,
            const float* const __restrict__ f,
            int& n,
            const TF du,
            const TF db,
            const TF zsl)
    {
        // Calculate the appropriate Richardson number.
        const TF Ri = Constants::kappa<TF> * db * zsl / fm::pow2(du);
        return find_obuk_g(zL, f, n, Ri, zsl);
    }

    template<typename TF> __global__
    void stability_g(
            TF* const __restrict__ ustar,
            TF* const __restrict__ obuk,
            int* const __restrict__ nobuk_g,
            const TF* const __restrict__ b,
            const TF* const __restrict__ bbot,
            const TF* const __restrict__ bfluxbot,
            const TF* const __restrict__ dutot,
            const TF* const __restrict__ z0m,
            const float* const __restrict__ zL_sl_g,
            const float* const __restrict__ f_sl_g,
            const TF db_ref,
            const TF zsl,
            const int icells, const int jcells,
            const int kstart, const int jj, int kk,
            const Boundary_type mbcbot,
            const Boundary_type thermobc)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        const int j = blockIdx.y*blockDim.y + threadIdx.y;

        if (i < icells && j < jcells)
        {
            const int ij  = i + j*jj;
            const int ijk = i + j*jj + kstart*kk;

            // case 1: fixed buoyancy flux and fixed ustar
            if (mbcbot == Boundary_type::Ustar_type && thermobc == Boundary_type::Flux_type)
            {
                obuk[ij] = -fm::pow3(ustar[ij]) / (Constants::kappa<TF>*bfluxbot[ij]);
            }
            // case 2: fixed buoyancy flux and free ustar
            else if (mbcbot == Boundary_type::Dirichlet_type && thermobc == Boundary_type::Flux_type)
            {
                obuk [ij] = calc_obuk_noslip_flux_g(zL_sl_g, f_sl_g, nobuk_g[ij], dutot[ij], bfluxbot[ij], zsl);
                ustar[ij] = dutot[ij] * most::fm(zsl, z0m[ij], obuk[ij]);
            }
            // case 3: fixed buoyancy surface value and free ustar
            else if (mbcbot == Boundary_type::Dirichlet_type && thermobc == Boundary_type::Dirichlet_type)
            {
                TF db = b[ijk] - bbot[ij] + db_ref;
                obuk [ij] = calc_obuk_noslip_dirichlet_g(zL_sl_g, f_sl_g, nobuk_g[ij], dutot[ij], db, zsl);
                ustar[ij] = dutot[ij] * most::fm(zsl, z0m[ij], obuk[ij]);
            }
        }
    }

    template<typename TF> __global__
    void stability_neutral_g(
            TF* const __restrict__ ustar,
            TF* const __restrict__ obuk,
            const TF* const __restrict__ dutot,
            const TF* const __restrict__ z0m,
            const TF zsl,
            const int icells, const int jcells, const int jj,
            const Boundary_type mbcbot,
            const Boundary_type thermobc)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        const int j = blockIdx.y*blockDim.y + threadIdx.y;

        if (i < icells && j < jcells)
        {
            const int ij  = i + j*jj;

            // case 1: fixed buoyancy flux and fixed ustar
            if (mbcbot == Boundary_type::Ustar_type && thermobc == Boundary_type::Flux_type)
            {
                obuk[ij] = -Constants::dbig;
            }
            // case 2: fixed buoyancy flux and free ustar
            else if (mbcbot == Boundary_type::Dirichlet_type && thermobc == Boundary_type::Flux_type)
            {
                obuk [ij] = -Constants::dbig;
                ustar[ij] = dutot[ij] * most::fm(zsl, z0m[ij], obuk[ij]);
            }
            // case 3: fixed buoyancy surface value and free ustar
            else if (mbcbot == Boundary_type::Dirichlet_type && thermobc == Boundary_type::Dirichlet_type)
            {
                obuk [ij] = -Constants::dbig;
                ustar[ij] = dutot[ij] * most::fm(zsl, z0m[ij], obuk[ij]);
            }
        }
    }

    template<typename TF> __global__
    void surfm_flux_g(
            TF* const __restrict__ ufluxbot,
            TF* const __restrict__ vfluxbot,
            const TF* const __restrict__ u,
            const TF* const __restrict__ v,
            const TF* const __restrict__ ubot,
            const TF* const __restrict__ vbot,
            const TF* const __restrict__ ustar,
            const TF* const __restrict__ obuk,
            const TF* const __restrict__ z0m,
            const TF zsl,
            const int istart, const int iend,
            const int jstart, const int jend,
            const int kstart,
            const int jj, const int kk,
            const Boundary_type bcbot)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;

        if (i < iend && j < jend)
        {
            const int ii  = 1;
            const int ij  = i + j*jj;
            const int ijk = i + j*jj + kstart*kk;

            if (bcbot == Boundary_type::Dirichlet_type)
            {
                // interpolate the whole stability function rather than ustar or obuk
                ufluxbot[ij] = -(u[ijk]-ubot[ij])*TF(0.5)*(ustar[ij-ii]*most::fm(zsl, z0m[ij], obuk[ij-ii]) +
                                                           ustar[ij]   *most::fm(zsl, z0m[ij], obuk[ij]));
                vfluxbot[ij] = -(v[ijk]-vbot[ij])*TF(0.5)*(ustar[ij-jj]*most::fm(zsl, z0m[ij], obuk[ij-jj]) +
                                                           ustar[ij]   *most::fm(zsl, z0m[ij], obuk[ij]));
            }
            else if (bcbot == Boundary_type::Ustar_type)
            {
                const TF minval = 1.e-2;

                // minimize the wind at 0.01, thus the wind speed squared at 0.0001
                const TF vonu2 = fmax(minval, TF(0.25)*( fm::pow2(v[ijk-ii]-vbot[ij-ii]) + fm::pow2(v[ijk-ii+jj]-vbot[ij-ii+jj])
                                                       + fm::pow2(v[ijk   ]-vbot[ij   ]) + fm::pow2(v[ijk   +jj]-vbot[ij   +jj])) );
                const TF uonv2 = fmax(minval, TF(0.25)*( fm::pow2(u[ijk-jj]-ubot[ij-jj]) + fm::pow2(u[ijk+ii-jj]-ubot[ij+ii-jj])
                                                       + fm::pow2(u[ijk   ]-ubot[ij   ]) + fm::pow2(u[ijk+ii   ]-ubot[ij+ii   ])) );

                const TF u2 = fmax(minval, fm::pow2(u[ijk]-ubot[ij]));
                const TF v2 = fmax(minval, fm::pow2(v[ijk]-vbot[ij]));

                const TF ustaronu4 = TF(0.5)*(fm::pow4(ustar[ij-ii]) + fm::pow4(ustar[ij]));
                const TF ustaronv4 = TF(0.5)*(fm::pow4(ustar[ij-jj]) + fm::pow4(ustar[ij]));

                ufluxbot[ij] = -copysign(TF(1.), u[ijk]-ubot[ij]) * pow(ustaronu4 / (TF(1.) + vonu2 / u2), TF(0.5));
                vfluxbot[ij] = -copysign(TF(1.), v[ijk]-vbot[ij]) * pow(ustaronv4 / (TF(1.) + uonv2 / v2), TF(0.5));
            }
        }
    }

    template<typename TF> __global__
    void surfm_grad_g(
            TF* const __restrict__ ugradbot,
            TF* const __restrict__ vgradbot,
            const TF* const __restrict__ u,
            const TF* const __restrict__ v,
            const TF* const __restrict__ ubot,
            const TF* const __restrict__ vbot,
            const TF zsl,
            const int icells, const int jcells,
            const int kstart, const int jj, const int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        const int j = blockIdx.y*blockDim.y + threadIdx.y;

        if (i < icells && j < jcells)
        {
            const int ij  = i + j*jj;
            const int ijk = i + j*jj + kstart*kk;

            ugradbot[ij] = (u[ijk]-ubot[ij])/zsl;
            vgradbot[ij] = (v[ijk]-vbot[ij])/zsl;
        }
    }

    template<typename TF> __global__
    void surfs_g(
            TF* const __restrict__ varfluxbot,
            TF* const __restrict__ vargradbot,
            TF* const __restrict__ varbot,
            const TF* const __restrict__ var,
            const TF* const __restrict__ ustar,
            const TF* const __restrict__ obuk,
            const TF* const __restrict__ z0h,
            const TF zsl,
            const int icells, const int jcells,
            const int kstart, const int jj, const int kk,
            const Boundary_type bcbot)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        const int j = blockIdx.y*blockDim.y + threadIdx.y;

        if (i < icells && j < jcells)
        {
            const int ij  = i + j*jj;
            const int ijk = i + j*jj + kstart*kk;

            if (bcbot == Boundary_type::Dirichlet_type)
            {
                varfluxbot[ij] = -(var[ijk]-varbot[ij])*ustar[ij]*most::fh(zsl, z0h[ij], obuk[ij]);
                vargradbot[ij] = (var[ijk]-varbot[ij])/zsl;
            }
            else if (bcbot == Boundary_type::Flux_type)
            {
                varbot[ij]     = varfluxbot[ij] / (ustar[ij]*most::fh(zsl, z0h[ij], obuk[ij])) + var[ijk];
                vargradbot[ij] = (var[ijk]-varbot[ij])/zsl;
            }
        }
    }
}

template<typename TF>
void Boundary_surface<TF>::prepare_device()
{
    auto& gd = grid.get_grid_data();

    const int dmemsize2d = gd.ijcells*sizeof(TF);
    const int imemsize2d = gd.ijcells*sizeof(int);
    const int dimemsize  = gd.icells*sizeof(TF);
    const int iimemsize  = gd.icells*sizeof(int);

    cuda_safe_call(hipMalloc(&obuk_g,  dmemsize2d));
    cuda_safe_call(hipMalloc(&ustar_g, dmemsize2d));
    cuda_safe_call(hipMalloc(&z0m_g,   dmemsize2d));
    cuda_safe_call(hipMalloc(&z0h_g,   dmemsize2d));

    cuda_safe_call(hipMalloc(&dudz_mo_g, dmemsize2d));
    cuda_safe_call(hipMalloc(&dvdz_mo_g, dmemsize2d));
    cuda_safe_call(hipMalloc(&dbdz_mo_g, dmemsize2d));

    cuda_safe_call(hipMalloc(&nobuk_g, imemsize2d));

    cuda_safe_call(hipMalloc(&zL_sl_g, nzL*sizeof(float)));
    cuda_safe_call(hipMalloc(&f_sl_g,  nzL*sizeof(float)));

    cuda_safe_call(hipMemcpy2D(obuk_g,  dimemsize, obuk.data(),  dimemsize, dimemsize, gd.jcells, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy2D(ustar_g, dimemsize, ustar.data(), dimemsize, dimemsize, gd.jcells, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy2D(z0m_g,   dimemsize, z0m.data(),   dimemsize, dimemsize, gd.jcells, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy2D(z0h_g,   dimemsize, z0h.data(),   dimemsize, dimemsize, gd.jcells, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy2D(nobuk_g, iimemsize, nobuk.data(), iimemsize, iimemsize, gd.jcells, hipMemcpyHostToDevice));

    cuda_safe_call(hipMemcpy(zL_sl_g, zL_sl.data(), nzL*sizeof(float), hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(f_sl_g,  f_sl.data(),  nzL*sizeof(float), hipMemcpyHostToDevice));
}

template<typename TF>
void Boundary_surface<TF>::forward_device()
{
    auto& gd = grid.get_grid_data();

    const int dimemsize   = gd.icells  * sizeof(TF);
    const int iimemsize   = gd.icells  * sizeof(int);

    cuda_safe_call(hipMemcpy2D(obuk_g,  dimemsize, obuk.data(),  dimemsize, dimemsize, gd.jcells, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy2D(ustar_g, dimemsize, ustar.data(), dimemsize, dimemsize, gd.jcells, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy2D(z0m_g,   dimemsize, z0m.data(),   dimemsize, dimemsize, gd.jcells, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy2D(z0h_g,   dimemsize, z0h.data(),   dimemsize, dimemsize, gd.jcells, hipMemcpyHostToDevice));

    cuda_safe_call(hipMemcpy2D(dudz_mo_g, dimemsize, z0h.data(), dimemsize, dimemsize, gd.jcells, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy2D(dvdz_mo_g, dimemsize, z0h.data(), dimemsize, dimemsize, gd.jcells, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy2D(dbdz_mo_g, dimemsize, z0h.data(), dimemsize, dimemsize, gd.jcells, hipMemcpyHostToDevice));

    cuda_safe_call(hipMemcpy2D(nobuk_g, iimemsize, nobuk.data(), iimemsize, iimemsize, gd.jcells, hipMemcpyHostToDevice));
}

template<typename TF>
void Boundary_surface<TF>::backward_device()
{
    auto& gd = grid.get_grid_data();

    const int dimemsize = gd.icells * sizeof(TF);
    const int iimemsize = gd.icells * sizeof(int);

    cuda_safe_call(hipMemcpy2D(obuk.data(),  dimemsize, obuk_g,  dimemsize, dimemsize, gd.jcells, hipMemcpyDeviceToHost));
    cuda_safe_call(hipMemcpy2D(ustar.data(), dimemsize, ustar_g, dimemsize, dimemsize, gd.jcells, hipMemcpyDeviceToHost));

    cuda_safe_call(hipMemcpy2D(dudz_mo.data(), dimemsize, dudz_mo_g, dimemsize, dimemsize, gd.jcells, hipMemcpyDeviceToHost));
    cuda_safe_call(hipMemcpy2D(dvdz_mo.data(), dimemsize, dvdz_mo_g, dimemsize, dimemsize, gd.jcells, hipMemcpyDeviceToHost));
    cuda_safe_call(hipMemcpy2D(dbdz_mo.data(), dimemsize, dbdz_mo_g, dimemsize, dimemsize, gd.jcells, hipMemcpyDeviceToHost));

    cuda_safe_call(hipMemcpy2D(nobuk.data(), iimemsize, nobuk_g, iimemsize, iimemsize, gd.jcells, hipMemcpyDeviceToHost));
}

template<typename TF>
void Boundary_surface<TF>::clear_device()
{
    cuda_safe_call(hipFree(obuk_g ));
    cuda_safe_call(hipFree(ustar_g));
    cuda_safe_call(hipFree(z0m_g));
    cuda_safe_call(hipFree(z0h_g));

    cuda_safe_call(hipFree(dudz_mo_g));
    cuda_safe_call(hipFree(dvdz_mo_g));
    cuda_safe_call(hipFree(dbdz_mo_g));

    cuda_safe_call(hipFree(nobuk_g));
    cuda_safe_call(hipFree(zL_sl_g));
    cuda_safe_call(hipFree(f_sl_g ));
}

#ifdef USECUDA
template<typename TF>
void Boundary_surface<TF>::exec(
        Thermo<TF>& thermo, Radiation<TF>& radiation,
        Microphys<TF>& microphys, Timeloop<TF>& timeloop)
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;

    // For 2D field excluding ghost cells
    int gridi = gd.imax/blocki + (gd.imax%blocki > 0);
    int gridj = gd.jmax/blockj + (gd.jmax%blockj > 0);
    dim3 gridGPU (gridi,  gridj,  1);
    dim3 blockGPU(blocki, blockj, 1);

    // For 2D field including ghost cells
    gridi = gd.icells/blocki + (gd.icells%blocki > 0);
    gridj = gd.jcells/blockj + (gd.jcells%blockj > 0);
    dim3 gridGPU2 (gridi,  gridj,  1);
    dim3 blockGPU2(blocki, blockj, 1);

    // Calculate dutot in tmp2
    auto dutot = fields.get_tmp_g();

    bsk::calc_dutot_g<TF><<<gridGPU, blockGPU>>>(
        dutot->fld_g,
        fields.mp.at("u")->fld_g,
        fields.mp.at("v")->fld_g,
        fields.mp.at("u")->fld_bot_g,
        fields.mp.at("v")->fld_bot_g,
        gd.istart, gd.iend,
        gd.jstart, gd.jend,
        gd.kstart,
        gd.icells, gd.ijcells);
    cuda_check_error();

    // 2D cyclic boundaries on dutot
    boundary_cyclic.exec_2d_g(dutot->fld_g);

    // start with retrieving the stability information
    if (thermo.get_switch() == "0")
    {
        // Calculate ustar and Obukhov length, including ghost cells
        stability_neutral_g<TF><<<gridGPU2, blockGPU2>>>(
            ustar_g, obuk_g,
            dutot->fld_g, z0m_g, gd.z[gd.kstart],
            gd.icells, gd.jcells, gd.icells,
            mbcbot, thermobc);
        cuda_check_error();
    }
    else
    {
        auto buoy = fields.get_tmp_g();
        thermo.get_buoyancy_surf_g(*buoy);
        const TF db_ref = thermo.get_db_ref();

        // Calculate ustar and Obukhov length, including ghost cells
        stability_g<TF><<<gridGPU2, blockGPU2>>>(
            ustar_g, obuk_g, nobuk_g,
            buoy->fld_g, buoy->fld_bot_g, buoy->flux_bot_g,
            dutot->fld_g, z0m_g,
            zL_sl_g, f_sl_g,
            db_ref, gd.z[gd.kstart],
            gd.icells, gd.jcells,
            gd.kstart, gd.icells,
            gd.ijcells,
            mbcbot, thermobc);
        cuda_check_error();

        fields.release_tmp_g(buoy);
    }

    fields.release_tmp_g(dutot);

    // Calculate the surface value, gradient and flux depending on the chosen boundary condition.
    // Momentum:
    surfm_flux_g<TF><<<gridGPU, blockGPU>>>(
        fields.mp.at("u")->flux_bot_g,
        fields.mp.at("v")->flux_bot_g,
        fields.mp.at("u")->fld_g,
        fields.mp.at("v")->fld_g,
        fields.mp.at("u")->fld_bot_g,
        fields.mp.at("v")->fld_bot_g,
        ustar_g, obuk_g, z0m_g, gd.z[gd.kstart],
        gd.istart, gd.iend,
        gd.jstart, gd.jend,
        gd.kstart,
        gd.icells, gd.ijcells,
        mbcbot);
    cuda_check_error();

    // 2D cyclic boundaries on the surface fluxes
    boundary_cyclic.exec_2d_g(fields.mp.at("u")->flux_bot_g);
    boundary_cyclic.exec_2d_g(fields.mp.at("v")->flux_bot_g);

    // Calculate surface gradients, including ghost cells
    surfm_grad_g<TF><<<gridGPU2, blockGPU2>>>(
        fields.mp.at("u")->grad_bot_g,
        fields.mp.at("v")->grad_bot_g,
        fields.mp.at("u")->fld_g,
        fields.mp.at("v")->fld_g,
        fields.mp.at("u")->fld_bot_g,
        fields.mp.at("v")->fld_bot_g,
        gd.z[gd.kstart], gd.icells, gd.jcells,
        gd.kstart, gd.icells, gd.ijcells);
    cuda_check_error();

    // Scalars:
    for (auto it : fields.sp)
        surfs_g<TF><<<gridGPU2, blockGPU2>>>(
            it.second->flux_bot_g,
            it.second->grad_bot_g,
            it.second->fld_bot_g,
            it.second->fld_g,
            ustar_g, obuk_g, z0h_g, gd.z[gd.kstart],
            gd.icells,  gd.jcells, gd.kstart,
            gd.icells, gd.ijcells,
            sbc.at(it.first).bcbot);
    cuda_check_error();

    // Calc MO gradients, for subgrid scheme
    bsk::calc_duvdz_mo_g<TF><<<gridGPU2, blockGPU2>>>(
            dudz_mo_g, dvdz_mo_g,
            fields.mp.at("u")->fld_g,
            fields.mp.at("v")->fld_g,
            fields.mp.at("u")->fld_bot_g,
            fields.mp.at("v")->fld_bot_g,
            fields.mp.at("u")->flux_bot_g,
            fields.mp.at("v")->flux_bot_g,
            ustar_g, obuk_g, z0m_g,
            gd.z[gd.kstart],
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            gd.kstart,
            gd.icells, gd.ijcells);
    cuda_check_error();

    auto buoy = fields.get_tmp_g();
    thermo.get_buoyancy_fluxbot_g(*buoy);

    bsk::calc_dbdz_mo_g<TF><<<gridGPU2, blockGPU2>>>(
            dbdz_mo_g, buoy->flux_bot_g,
            ustar_g, obuk_g,
            gd.z[gd.kstart],
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            gd.icells);
    cuda_check_error();

    fields.release_tmp_g(buoy);
}

template<typename TF>
void Boundary_surface<TF>::exec_column(Column<TF>& column)
{
    const TF no_offset = 0.;
    column.calc_time_series("obuk", obuk_g, no_offset);
    column.calc_time_series("ustar", ustar_g, no_offset);
}
#endif

template class Boundary_surface<double>;
template class Boundary_surface<float>;
