#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2020 Chiel van Heerwaarden
 * Copyright (c) 2011-2020 Thijs Heus
 * Copyright (c) 2014-2020 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <algorithm>
#include "master.h"
#include "grid.h"
#include "fields.h"
#include "field3d_operators.h"
#include "timeloop.h"
#include "timedep.h"
#include "stats.h"
#include "finite_difference.h"
#include "constants.h"
#include "tools.h"
#include "boundary.h"
#include "thermo.h"
#include "force.h"

using namespace Finite_difference::O2;

namespace
{
    template<typename TF> __global__
    void add_pressure_force_g(TF* const __restrict__ ut,
                       const TF fbody,
                       const int jj, const int kk,
                       const int istart, const int jstart, const int kstart,
                       const int iend,   const int jend,   const int kend)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            ut[ijk] += fbody;
        }
    }

    template<typename TF> __global__
    void coriolis_2nd_g(TF* const __restrict__ ut, TF* const __restrict__ vt,
                        TF* const __restrict__ u,  TF* const __restrict__ v,
                        TF* const __restrict__ ug, TF* const __restrict__ vg,
                        const TF fc, const TF ugrid, const TF vgrid,
                        const int jj, const int kk,
                        const int istart, const int jstart, const int kstart,
                        const int iend,   const int jend,   const int kend)
    {
        const int i  = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j  = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k  = blockIdx.z + kstart;
        const int ii = 1;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            ut[ijk] += fc * (TF(0.25)*(v[ijk-ii] + v[ijk] + v[ijk-ii+jj] + v[ijk+jj]) + vgrid - vg[k]);
            vt[ijk] -= fc * (TF(0.25)*(u[ijk-jj] + u[ijk] + u[ijk+ii-jj] + u[ijk+ii]) + ugrid - ug[k]);
        }
    }

    template<typename TF> __global__
    void coriolis_4th_g(TF* const __restrict__ ut, TF* const __restrict__ vt,
                        TF* const __restrict__ u,  TF* const __restrict__ v,
                        TF* const __restrict__ ug, TF* const __restrict__ vg,
                        const TF fc, const TF ugrid, const TF vgrid,
                        const int jj, const int kk,
                        const int istart, const int jstart, const int kstart,
                        const int iend,   const int jend,   const int kend)
    {
        using namespace Finite_difference::O4;

        const int i   = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j   = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k   = blockIdx.z + kstart;
        const int ii  = 1;
        const int ii2 = 2;
        const int jj2 = 2*jj;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            ut[ijk] += fc * ( ( ci0<TF>*(ci0<TF>*v[ijk-ii2-jj ] + ci1<TF>*v[ijk-ii-jj ] + ci2<TF>*v[ijk-jj    ] + ci3<TF>*v[ijk+ii-jj  ])
                              + ci1<TF>*(ci0<TF>*v[ijk-ii2    ] + ci1<TF>*v[ijk-ii    ] + ci2<TF>*v[ijk       ] + ci3<TF>*v[ijk+ii     ])
                              + ci2<TF>*(ci0<TF>*v[ijk-ii2+jj ] + ci1<TF>*v[ijk-ii+jj ] + ci2<TF>*v[ijk+jj    ] + ci3<TF>*v[ijk+ii+jj  ])
                              + ci3<TF>*(ci0<TF>*v[ijk-ii2+jj2] + ci1<TF>*v[ijk-ii+jj2] + ci2<TF>*v[ijk+jj2   ] + ci3<TF>*v[ijk+ii+jj2 ]) )
                       + vgrid - vg[k] );

            vt[ijk] -= fc * ( ( ci0<TF>*(ci0<TF>*u[ijk-ii-jj2 ] + ci1<TF>*u[ijk-jj2   ] + ci2<TF>*u[ijk+ii-jj2] + ci3<TF>*u[ijk+ii2-jj2])
                              + ci1<TF>*(ci0<TF>*u[ijk-ii-jj  ] + ci1<TF>*u[ijk-jj    ] + ci2<TF>*u[ijk+ii-jj ] + ci3<TF>*u[ijk+ii2-jj ])
                              + ci2<TF>*(ci0<TF>*u[ijk-ii     ] + ci1<TF>*u[ijk       ] + ci2<TF>*u[ijk+ii    ] + ci3<TF>*u[ijk+ii2    ])
                              + ci3<TF>*(ci0<TF>*u[ijk-ii+jj  ] + ci1<TF>*u[ijk+jj    ] + ci2<TF>*u[ijk+ii+jj ] + ci3<TF>*u[ijk+ii2+jj ]) )
                       + ugrid - ug[k]);
        }
    }

    template<typename TF> __global__
    void advec_wls_2nd_g(TF* const __restrict__ st, TF* const __restrict__ s,
                         const TF* const __restrict__ wls, const TF* const __restrict__ dzhi,
                         const int istart, const int jstart, const int kstart,
                         const int iend,   const int jend,   const int kend,
                         const int jj,     const int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;

            if (wls[k] > 0.)
                st[ijk] -=  wls[k] * (s[k]-s[k-1])*dzhi[k];
            else
                st[ijk] -=  wls[k] * (s[k+1]-s[k])*dzhi[k+1];
        }
    }

    template<typename TF> __global__
    void large_scale_source_g(TF* const __restrict__ st, TF* const __restrict__ sls,
                              const int istart, const int jstart, const int kstart,
                              const int iend,   const int jend,   const int kend,
                              const int jj,     const int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            st[ijk] += sls[k];
        }
    }

    template<typename TF> __global__
    void nudging_tendency_g(TF* const __restrict__ st, TF* const __restrict__ smn,
                            TF* const __restrict__ snudge, TF* const __restrict__ nudge_fac,
                            const int istart, const int jstart, const int kstart,
                            const int iend,   const int jend,   const int kend,
                            const int jj,     const int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;

            st[ijk] += - nudge_fac[k] * (smn[k]-snudge[k]);

        }
    }

    template<typename TF>
    int calc_zi(const TF* const restrict fldmean, const int kstart, const int kend, const int plusminus)
    {
        TF maxgrad = 0.;
        TF grad = 0.;
        int kinv = kstart;
        for (int k=kstart+1; k<kend; ++k)
        {
            grad = plusminus * (fldmean[k] - fldmean[k-1]);
            if (grad > maxgrad)
            {
                maxgrad = grad;
                kinv = k;
            }
        }
        return kinv;
    }

    template<typename TF>
    void rescale_nudgeprof(TF* const restrict fldmean, const int kinv, const int kstart, const int kend)
    {
        for (int k=kstart+1; k<kinv; ++k)
            fldmean[k] = fldmean[kstart];

        for (int k=kinv+1; k<kend-2; ++k)
            fldmean[k] = fldmean[kend-1];
    }

    template<typename TF> __global__
    void calc_time_dependent_prof_g(TF* const __restrict__ prof, const TF* const __restrict__ data,
                                    const double fac0, const double fac1,
                                    const int index0,  const int index1,
                                    const int kmax,    const int kgc)
    {
        const int k = blockIdx.x*blockDim.x + threadIdx.x;
        const int kk = kmax;

        if (k < kmax)
            prof[k+kgc] = fac0*data[index0*kk+k] + fac1*data[index1*kk+k];
    }
} // end namespace

template<typename TF>
void Force<TF>::prepare_device()
{
    auto& gd = grid.get_grid_data();

    const int nmemsize  = gd.kcells*sizeof(TF);

    if (swlspres == Large_scale_pressure_type::Geo_wind)
    {
        cuda_safe_call(hipMalloc(&ug_g, nmemsize));
        cuda_safe_call(hipMalloc(&vg_g, nmemsize));

        cuda_safe_call(hipMemcpy(ug_g, ug.data(), nmemsize, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(vg_g, vg.data(), nmemsize, hipMemcpyHostToDevice));
    }

    if (swls == Large_scale_tendency_type::Enabled)
    {
        for (auto& it : lslist)
        {
            lsprofs_g.emplace(it, nullptr);
            cuda_safe_call(hipMalloc(&lsprofs_g.at(it), nmemsize));
            cuda_safe_call(hipMemcpy(lsprofs_g.at(it), lsprofs.at(it).data(), nmemsize, hipMemcpyHostToDevice));
        }
    }

    if (swnudge == Nudging_type::Enabled)
    {
        for (auto& it : nudgelist)
        {
            nudgeprofs_g.emplace(it, nullptr);
            cuda_safe_call(hipMalloc(&nudgeprofs_g.at(it), nmemsize));
            cuda_safe_call(hipMemcpy(nudgeprofs_g.at(it), nudgeprofs.at(it).data(), nmemsize, hipMemcpyHostToDevice));
        }
        cuda_safe_call(hipMalloc(&nudge_factor_g, nmemsize));
        cuda_safe_call(hipMemcpy(nudge_factor_g, nudge_factor.data(), nmemsize, hipMemcpyHostToDevice));
    }

    if (swwls == Large_scale_subsidence_type::Mean_field)
    {
        cuda_safe_call(hipMalloc(&wls_g, nmemsize));
        cuda_safe_call(hipMemcpy(wls_g, wls.data(), nmemsize, hipMemcpyHostToDevice));
    }
}

template<typename TF>
void Force<TF>::clear_device()
{
    if (swlspres == Large_scale_pressure_type::Geo_wind)
    {
        cuda_safe_call(hipFree(ug_g));
        cuda_safe_call(hipFree(vg_g));
        for (auto& it : tdep_geo)
            it.second->clear_device();
    }

    if (swls == Large_scale_tendency_type::Enabled)
    {
        for (auto& it : lsprofs_g)
            cuda_safe_call(hipFree(it.second));
        for (auto& it : tdep_ls)
            it.second->clear_device();
    }

    if (swnudge == Nudging_type::Enabled)
    {
        for (auto& it : nudgeprofs_g)
            cuda_safe_call(hipFree(it.second));
        cuda_safe_call(hipFree(nudge_factor_g));
        for (auto& it : tdep_nudge)
            it.second->clear_device();

    }

    if (swwls == Large_scale_subsidence_type::Mean_field)
    {
        cuda_safe_call(hipFree(wls_g));
        tdep_wls->clear_device();
    }
}

#ifdef USECUDA
template<typename TF>
void Force<TF>::exec(double dt, Thermo<TF>& thermo, Stats<TF>& stats)
{
    auto& gd = grid.get_grid_data();
    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, gd.kcells);
    dim3 blockGPU(blocki, blockj, 1);

    if (swlspres == Large_scale_pressure_type::Fixed_flux)
    {
        auto tmp = fields.get_tmp_g();

        TF uavg  = field3d_operators.calc_mean_g(fields.mp.at("u")->fld_g);
        TF utavg = field3d_operators.calc_mean_g(fields.mt.at("u")->fld_g);

        fields.release_tmp_g(tmp);

        const TF fbody = (uflux - uavg - grid.utrans) / dt - utavg;

        add_pressure_force_g<TF><<<gridGPU, blockGPU>>>(
            fields.mt.at("u")->fld_g,
            fbody,
            gd.icells, gd.ijcells,
            gd.istart, gd.jstart, gd.kstart,
            gd.iend,   gd.jend,   gd.kend);
        cuda_check_error();
        hipDeviceSynchronize();

        stats.calc_tend(*fields.mt.at("u"), tend_name_pres);
    }
    else if (swlspres == Large_scale_pressure_type::Pressure_gradient)
    {
        const TF fbody = TF(-1.)*dpdx;
        add_pressure_force_g<TF><<<gridGPU, blockGPU>>>(
            fields.mt.at("u")->fld_g, fbody,
            gd.icells, gd.ijcells,
            gd.istart, gd.jstart, gd.kstart,
            gd.iend,   gd.jend,   gd.kend);
        cuda_check_error();
        hipDeviceSynchronize();

        stats.calc_tend(*fields.mt.at("u"), tend_name_pres);
    }
    else if (swlspres == Large_scale_pressure_type::Geo_wind)
    {
        if (grid.get_spatial_order() == Grid_order::Second)
        {
            coriolis_2nd_g<TF><<<gridGPU, blockGPU>>>(
                fields.mt.at("u")->fld_g, fields.mt.at("v")->fld_g,
                fields.mp.at("u")->fld_g, fields.mp.at("v")->fld_g,
                ug_g, vg_g, fc, grid.utrans, grid.vtrans,
                gd.icells, gd.ijcells,
                gd.istart, gd.jstart, gd.kstart,
                gd.iend,   gd.jend,   gd.kend);
            cuda_check_error();
        }
        else if (grid.get_spatial_order() == Grid_order::Fourth)
        {
            coriolis_4th_g<TF><<<gridGPU, blockGPU>>>(
                fields.mt.at("u")->fld_g, fields.mt.at("v")->fld_g,
                fields.mp.at("u")->fld_g, fields.mp.at("v")->fld_g,
                ug_g, vg_g, fc, grid.utrans, grid.vtrans,
                gd.icells, gd.ijcells,
                gd.istart, gd.jstart, gd.kstart,
                gd.iend,   gd.jend,   gd.kend);
            cuda_check_error();
        }
        hipDeviceSynchronize();

        stats.calc_tend(*fields.mt.at("u"), tend_name_cor);
        stats.calc_tend(*fields.mt.at("v"), tend_name_cor);
    }

    if (swls == Large_scale_tendency_type::Enabled)
    {
        for (auto& it : lslist)
        {
            large_scale_source_g<TF><<<gridGPU, blockGPU>>>(
                fields.at.at(it)->fld_g, lsprofs_g.at(it),
                gd.istart, gd.jstart, gd.kstart,
                gd.iend,   gd.jend,   gd.kend,
                gd.icells, gd.ijcells);
            cuda_check_error();
            hipDeviceSynchronize();
            stats.calc_tend(*fields.at.at(it), tend_name_ls);
        }
    }

    if (swnudge == Nudging_type::Enabled)
    {
        for (auto& it : nudgelist)
        {
            auto it1 = std::find(scalednudgelist.begin(), scalednudgelist.end(), it);
            if (it1 != scalednudgelist.end())
            {
                hipMemcpy(fields.ap.at(it)->fld_mean.data(), fields.ap.at(it)->fld_mean_g, gd.kcells*sizeof(TF), hipMemcpyDeviceToHost);
                const int kinv = thermo.get_bl_depth();
                rescale_nudgeprof(nudgeprofs.at(it).data(), kinv, gd.kstart, gd.kend);
                hipMemcpy(nudgeprofs_g.at(it), nudgeprofs.at(it).data(), gd.kcells*sizeof(TF), hipMemcpyHostToDevice);
            }

            nudging_tendency_g<TF><<<gridGPU, blockGPU>>>(
                fields.at.at(it)->fld_g, fields.ap.at(it)->fld_mean_g,
                nudgeprofs_g.at(it), nudge_factor_g,
                gd.istart, gd.jstart, gd.kstart,
                gd.iend,   gd.jend,   gd.kend,
                gd.icells, gd.ijcells);
            cuda_check_error();
            hipDeviceSynchronize();
            stats.calc_tend(*fields.at.at(it), tend_name_nudge);
        }
    }

    if (swwls == Large_scale_subsidence_type::Mean_field)
    {
        for (auto& it : fields.st)
        {
            advec_wls_2nd_g<TF><<<gridGPU, blockGPU>>>(
                fields.st.at(it.first)->fld_g, fields.sp.at(it.first)->fld_mean_g, wls_g, gd.dzhi_g,
                gd.istart, gd.jstart, gd.kstart,
                gd.iend,   gd.jend,   gd.kend,
                gd.icells, gd.ijcells);
            cuda_check_error();

            hipDeviceSynchronize();
            stats.calc_tend(*it.second, tend_name_subs);
        }
    }
}
#endif

#ifdef USECUDA
template <typename TF>
void Force<TF>::update_time_dependent(Timeloop<TF>& timeloop)
{
    if (swls == Large_scale_tendency_type::Enabled)
    {
        for (auto& it : tdep_ls)
            it.second->update_time_dependent_prof_g(lsprofs_g.at(it.first), timeloop);
    }

    if (swnudge == Nudging_type::Enabled)
    {
        for (auto& it : tdep_nudge)
            it.second->update_time_dependent_prof_g(nudgeprofs_g.at(it.first), timeloop);
    }

    if (swlspres == Large_scale_pressure_type::Geo_wind)
    {
        tdep_geo.at("u_geo")->update_time_dependent_prof_g(ug_g, timeloop);
        tdep_geo.at("v_geo")->update_time_dependent_prof_g(vg_g, timeloop);
    }

    if (swwls == Large_scale_subsidence_type::Mean_field)
        tdep_wls->update_time_dependent_prof_g(wls_g, timeloop);
}
#endif

template class Force<double>;
template class Force<float>;
