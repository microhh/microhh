#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2020 Chiel van Heerwaarden
 * Copyright (c) 2011-2020 Thijs Heus
 * Copyright (c) 2014-2020 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <iostream>

#include "boundary.h"
#include "boundary_surface_lsm.h"
#include "land_surface_kernels_gpu.h"
#include "boundary_surface_kernels_gpu.h"
#include "soil_kernels_gpu.h"
#include "tools.h"
#include "grid.h"
#include "soil_grid.h"
#include "fields.h"
#include "soil_field3d.h"
#include "radiation.h"
#include "thermo.h"
#include "microphys.h"
#include "column.h"

namespace
{
    namespace lsmk = Land_surface_kernels_g;
    namespace bsk = Boundary_surface_kernels_g;
    namespace sk = Soil_kernels_g;
}

namespace
{
    template<typename TF>
    void dump_field(
        TF* const restrict fld,
        TF* const restrict tmp,
        std::string name,
        const int size)
    {
        std::cout << "Saving: " << name << std::endl;
        cuda_safe_call(hipMemcpy(tmp, fld, size*sizeof(TF), hipMemcpyDeviceToHost));
        FILE *pFile;
        pFile = fopen(name.c_str(), "wb");
        if (pFile == NULL)
            std::cout << "Error opening file" << std::endl;
        fwrite(tmp, sizeof(TF), size, pFile);
        fclose(pFile);
    }
}

#ifdef USECUDA
template<typename TF>
void Boundary_surface_lsm<TF>::exec(
        Thermo<TF>& thermo, Radiation<TF>& radiation,
        Microphys<TF>& microphys, Timeloop<TF>& timeloop)
{
    auto& gd = grid.get_grid_data();
    auto& sgd = soil_grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;

    // For 2D field excluding ghost cells
    int gridi = gd.imax/blocki + (gd.imax%blocki > 0);
    int gridj = gd.jmax/blockj + (gd.jmax%blockj > 0);
    dim3 grid_gpu_2d (gridi,  gridj,  1);
    dim3 block_gpu_2d(blocki, blockj, 1);

    // For 2D field including ghost cells
    gridi = gd.icells/blocki + (gd.icells%blocki > 0);
    gridj = gd.jcells/blockj + (gd.jcells%blockj > 0);
    dim3 grid_gpu_2d_gc (gridi,  gridj,  1);
    dim3 block_gpu_2d_gc(blocki, blockj, 1);

    // Calculate filtered wind speed difference surface-atmosphere.
    auto tmp1 = fields.get_tmp_g();
    // Aarrghh, TODO: replace with `get_tmp_xy_g()......`.
    TF* du_tot = tmp1->fld_bot_g;

    bsk::calc_dutot_g<TF><<<grid_gpu_2d, block_gpu_2d>>>(
        du_tot,
        fields.mp.at("u")->fld_g,
        fields.mp.at("v")->fld_g,
        fields.mp.at("u")->fld_bot_g,
        fields.mp.at("v")->fld_bot_g,
        gd.istart, gd.iend,
        gd.jstart, gd.jend,
        gd.kstart,
        gd.icells, gd.ijcells);
    boundary_cyclic.exec_2d_g(du_tot);
    cuda_check_error();

    //
    // Retrieve necessary data from other classes.
    //
    // Get references to surface radiation fluxes
    TF* sw_dn = radiation.get_surface_radiation_g("sw_down");
    TF* sw_up = radiation.get_surface_radiation_g("sw_up");
    TF* lw_dn = radiation.get_surface_radiation_g("lw_down");
    TF* lw_up = radiation.get_surface_radiation_g("lw_up");

    // Get (near-) surface thermo.
    // Aarrghh, TODO: replace with `get_tmp_xy_g()......`.
    TF* T_bot = tmp1->flux_bot_g;
    TF* T_a = tmp1->grad_bot_g;
    TF* vpd = tmp1->fld_top_g;
    TF* qsat_bot = tmp1->flux_top_g;
    TF* dqsatdT_bot = tmp1->grad_top_g;

    thermo.get_land_surface_fields_g(
        T_bot, T_a, vpd, qsat_bot, dqsatdT_bot);

    // Get (near-) surface buoyancy.
    auto buoy = fields.get_tmp_g();
    thermo.get_buoyancy_surf_g(*buoy);
    const TF db_ref = thermo.get_db_ref();

    // Get basestate vectors.
    TF* rhorefh = thermo.get_basestate_fld_g("rhoh");
    TF* thvrefh = thermo.get_basestate_fld_g("thvh");
    TF* exnrefh = thermo.get_basestate_fld_g("exnerh");
    TF* prefh   = thermo.get_basestate_fld_g("prefh");

    // Get surface precipitation (positive downwards, kg m-2 s-1 = mm s-1)
    auto tmp2 = fields.get_tmp_g();
    TF* rain_rate = tmp2->fld_bot_g;
    microphys.get_surface_rain_rate_g(rain_rate);

    // XY tmp fields for intermediate calculations
    // Aarrghh, TODO: replace with `get_tmp_xy_g()......`.
    TF* f1  = tmp2->flux_bot_g;
    TF* f2  = tmp2->grad_bot_g;
    TF* f2b = tmp2->fld_top_g;
    TF* f3  = tmp2->flux_top_g;
    TF* theta_mean_n = tmp2->grad_top_g;

    const double subdt = timeloop.get_sub_time_step();

    const int iter = timeloop.get_iteration();
    const int subs = timeloop.get_substep();
    const int mpiid = master.get_mpiid();

    //
    // LSM calculations
    //
    lsmk::calc_tile_fractions_g<<<grid_gpu_2d, block_gpu_2d>>>(
            tiles.at("veg").fraction_g,
            tiles.at("soil").fraction_g,
            tiles.at("wet").fraction_g,
            fields.ap2d.at("wl")->fld_g,
            c_veg_g, lai_g,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            gd.icells);
    cuda_check_error();

    // Calculate root fraction weighted mean soil water content
    sk::calc_root_weighted_mean_theta_g<<<grid_gpu_2d, block_gpu_2d>>>(
            theta_mean_n,
            fields.sps.at("theta")->fld_g,
            soil_index_g,
            root_fraction_g,
            theta_wp_g,
            theta_fc_g,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            sgd.kstart, sgd.kend,
            gd.icells, gd.ijcells);
    cuda_check_error();

    // Calculate vegetation/soil resistance functions `f`.
    lsmk::calc_resistance_functions_g<<<grid_gpu_2d, block_gpu_2d>>>(
            f1, f2, f2b, f3,
            sw_dn,
            fields.sps.at("theta")->fld_g,
            theta_mean_n, vpd,
            gD_coeff_g,
            c_veg_g,
            theta_wp_g,
            theta_fc_g,
            theta_res_g,
            soil_index_g,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            sgd.kend,
            gd.icells, gd.ijcells);
    cuda_check_error();

    // Calculate canopy resistance for veg and soil tiles.
    lsmk::calc_canopy_resistance_g<<<grid_gpu_2d, block_gpu_2d>>>(
            tiles.at("veg").rs_g,
            rs_veg_min_g, lai_g,
            f1, f2, f3,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            gd.icells);
    cuda_check_error();

    lsmk::calc_soil_resistance_g<<<grid_gpu_2d, block_gpu_2d>>>(
            tiles.at("soil").rs_g,
            rs_soil_min_g, f2b,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            gd.icells);
    cuda_check_error();

    // Loop over tiles, and calculate tile properties and fluxes
    for (auto& tile : tiles)
    {
        bool use_cs_veg = (tile.first == "veg");

        //
        // 1) Calculate obuk/ustar/ra using thl_bot and qt_bot
        // from previous time step (= old method, similar to DALES).
        // 2) Calculate new thl_bot such that SEB closes.
        //
        thermo.get_buoyancy_surf_g(
                buoy->fld_bot_g,
                tile.second.thl_bot_g,
                tile.second.qt_bot_g);

        // Calculate Obuk, ustar, and ra.
        if (sw_constant_z0)
            lsmk::calc_stability_g<TF, true><<<grid_gpu_2d_gc, block_gpu_2d_gc>>>(
                    tile.second.ustar_g,
                    tile.second.obuk_g,
                    tile.second.bfluxbot_g,
                    tile.second.ra_g,
                    tile.second.nobuk_g,
                    du_tot,
                    buoy->fld_g,
                    buoy->fld_bot_g,
                    z0m_g, z0h_g,
                    zL_sl_g,
                    f_sl_g,
                    db_ref,
                    gd.z[gd.kstart],
                    gd.istart, gd.iend,
                    gd.jstart, gd.jend,
                    gd.kstart,
                    gd.icells, gd.jcells,
                    gd.ijcells);
        else
            lsmk::calc_stability_g<TF, false><<<grid_gpu_2d_gc, block_gpu_2d_gc>>>(
                    tile.second.ustar_g,
                    tile.second.obuk_g,
                    tile.second.bfluxbot_g,
                    tile.second.ra_g,
                    tile.second.nobuk_g,
                    du_tot,
                    buoy->fld_g,
                    buoy->fld_bot_g,
                    z0m_g, z0h_g,
                    zL_sl_g,
                    f_sl_g,
                    db_ref,
                    gd.z[gd.kstart],
                    gd.istart, gd.iend,
                    gd.jstart, gd.jend,
                    gd.kstart,
                    gd.icells, gd.jcells,
                    gd.ijcells);
        cuda_check_error();

        //auto tmp_cpu = fields.get_tmp();
        //dump_field(tile.second.ustar_g, tmp_cpu->fld_bot.data(), "dump_gpu", gd.ijcells);
        //fields.release_tmp(tmp_cpu);
        //hipDeviceSynchronize();
        //throw 1;

        // Calculate surface fluxes
        lsmk::calc_fluxes_g<TF><<<grid_gpu_2d, block_gpu_2d>>>(
                tile.second.H_g,
                tile.second.LE_g,
                tile.second.G_g,
                tile.second.S_g,
                tile.second.thl_bot_g,
                tile.second.qt_bot_g,
                T_a,
                fields.sp.at("qt")->fld_g,
                fields.sps.at("t")->fld_g,
                qsat_bot, dqsatdT_bot,
                tile.second.ra_g,
                tile.second.rs_g,
                lambda_stable_g,
                lambda_unstable_g,
                cs_veg_g,
                sw_dn,
                sw_up,
                lw_dn,
                lw_up,
                buoy->fld_g,
                buoy->fld_bot_g,
                rhorefh,
                exnrefh,
                db_ref, emis_sfc,
                TF(subdt),
                gd.istart, gd.iend,
                gd.jstart, gd.jend,
                gd.kstart, sgd.kend,
                gd.icells, gd.ijcells,
                use_cs_veg);
        cuda_check_error();
    }

    // Override grid point with water
    if (sw_water)
    {
        // Set BCs for water grid points
        lsmk::set_water_tiles<TF><<<grid_gpu_2d, block_gpu_2d>>>(
                tiles.at("veg").fraction_g,
                tiles.at("soil").fraction_g,
                tiles.at("wet").fraction_g,
                tiles.at("veg").H_g,
                tiles.at("soil").H_g,
                tiles.at("wet").H_g,
                tiles.at("veg").LE_g,
                tiles.at("soil").LE_g,
                tiles.at("wet").LE_g,
                tiles.at("veg").G_g,
                tiles.at("soil").G_g,
                tiles.at("wet").G_g,
                tiles.at("veg").rs_g,
                tiles.at("soil").rs_g,
                tiles.at("wet").rs_g,
                tiles.at("wet").thl_bot_g,
                tiles.at("wet").qt_bot_g,
                water_mask_g,
                t_bot_water_g,
                fields.sp.at("thl")->fld_g,
                fields.sp.at("qt")->fld_g,
                fields.sp.at("thl")->fld_bot_g,
                fields.sp.at("qt")->fld_bot_g,
                tiles.at("wet").ra_g,
                rhorefh,
                prefh,
                exnrefh,
                gd.istart, gd.iend,
                gd.jstart, gd.jend,
                gd.kstart,
                gd.icells, gd.ijcells);
        cuda_check_error();
    }

    // Calculate tile averaged surface fluxes and values.
    const TF rhoref_bot = thermo.get_basestate_vector("rhoh")[gd.kstart];
    const TF rhocpi = TF(1) / (rhoref_bot * Constants::cp<TF>);
    const TF rholvi = TF(1) / (rhoref_bot * Constants::Lv<TF>);
    const TF no_scaling = TF(1);

    // Surface fluxes.
    get_tiled_mean_g(fields.sp.at("thl")->flux_bot_g, "H", rhocpi);
    get_tiled_mean_g(fields.sp.at("qt")->flux_bot_g, "LE", rholvi);
    get_tiled_mean_g(ustar_g, "ustar", no_scaling);
    get_tiled_mean_g(buoy->flux_bot_g, "bfluxbot", no_scaling);

    // Surface values.
    get_tiled_mean_g(fields.sp.at("thl")->fld_bot_g, "thl_bot", TF(1));
    get_tiled_mean_g(fields.sp.at("qt")->fld_bot_g, "qt_bot", TF(1));

    // Set ghost cells `thl_bot`, `qt_bot`, needed for surface scheme
    boundary_cyclic.exec_2d_g(fields.sp.at("thl")->fld_bot_g);
    boundary_cyclic.exec_2d_g(fields.sp.at("qt")->fld_bot_g);

    // Calculate bulk Obukhov length.
    lsmk::calc_bulk_obuk_g<TF><<<grid_gpu_2d, block_gpu_2d>>>(
            obuk_g,
            buoy->flux_bot_g,
            ustar_g,
            gd.z[gd.kstart],
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            gd.icells);
    cuda_check_error();

    boundary_cyclic.exec_2d_g(ustar_g);
    boundary_cyclic.exec_2d_g(obuk_g);

    // Redistribute ustar over `uw` and `vw`.
    lsmk::set_bcs_momentum_g<TF><<<grid_gpu_2d, block_gpu_2d>>>(
            fields.mp.at("u")->flux_bot_g,
            fields.mp.at("v")->flux_bot_g,
            fields.mp.at("u")->grad_bot_g,
            fields.mp.at("v")->grad_bot_g,
            ustar_g,
            fields.mp.at("u")->fld_g,
            fields.mp.at("u")->fld_bot_g,
            fields.mp.at("v")->fld_g,
            fields.mp.at("v")->fld_bot_g,
            z0m_g, gd.z[gd.kstart],
            gd.istart, gd.iend,
            gd.jstart, gd.jend, gd.kstart,
            gd.icells, gd.jcells, gd.ijcells);
    cuda_check_error();

    boundary_cyclic.exec_2d_g(fields.mp.at("u")->flux_bot_g);
    boundary_cyclic.exec_2d_g(fields.mp.at("v")->flux_bot_g);
    boundary_cyclic.exec_2d_g(fields.mp.at("u")->grad_bot_g);
    boundary_cyclic.exec_2d_g(fields.mp.at("v")->grad_bot_g);

    // Set BCs (gradients) thl + qt
    lsmk::set_bcs_thl_qt_g<TF><<<grid_gpu_2d_gc, block_gpu_2d_gc>>>(
            fields.sp.at("thl")->grad_bot_g,
            fields.sp.at("qt")->grad_bot_g,
            fields.sp.at("thl")->fld_g,
            fields.sp.at("qt")->fld_g,
            fields.sp.at("thl")->fld_bot_g,
            fields.sp.at("qt")->fld_bot_g,
            gd.z[gd.kstart], gd.kstart,
            gd.icells, gd.jcells, gd.ijcells);
    cuda_check_error();

    // Set BCs other scalars
    for (auto& it : fields.sp)
        if (it.first != "thl" and it.first != "qt")
        {
            if (sbc.at(it.first).bcbot == Boundary_type::Dirichlet_type)
                lsmk::set_bcs_scalars_dirichlet_g<TF><<<grid_gpu_2d_gc, block_gpu_2d_gc>>>(
                    it.second->fld_bot_g,
                    it.second->grad_bot_g,
                    it.second->flux_bot_g,
                    ustar_g, obuk_g,
                    it.second->fld_g, z0h_g,
                    gd.z[gd.kstart],
                    gd.istart, gd.iend,
                    gd.jstart, gd.jend, gd.kstart,
                    gd.icells, gd.jcells, gd.ijcells);

            else if (sbc.at(it.first).bcbot == Boundary_type::Flux_type)
                lsmk::set_bcs_scalars_flux_g<TF><<<grid_gpu_2d_gc, block_gpu_2d_gc>>>(
                    it.second->fld_bot_g,
                    it.second->grad_bot_g,
                    it.second->flux_bot_g,
                    ustar_g, obuk_g,
                    it.second->fld_g, z0h_g,
                    gd.z[gd.kstart],
                    gd.istart, gd.iend,
                    gd.jstart, gd.jend, gd.kstart,
                    gd.icells, gd.jcells, gd.ijcells);
            cuda_check_error();
        }

    // Calc MO gradients, for subgrid scheme
    bsk::calc_duvdz_mo_g<TF><<<grid_gpu_2d_gc, block_gpu_2d_gc>>>(
            dudz_mo_g, dvdz_mo_g,
            fields.mp.at("u")->fld_g,
            fields.mp.at("v")->fld_g,
            fields.mp.at("u")->fld_bot_g,
            fields.mp.at("v")->fld_bot_g,
            fields.mp.at("u")->flux_bot_g,
            fields.mp.at("v")->flux_bot_g,
            ustar_g, obuk_g, z0m_g,
            gd.z[gd.kstart],
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            gd.kstart,
            gd.icells, gd.ijcells);
    cuda_check_error();

    bsk::calc_dbdz_mo_g<TF><<<grid_gpu_2d_gc, block_gpu_2d_gc>>>(
            dbdz_mo_g, buoy->flux_bot_g,
            ustar_g, obuk_g,
            gd.z[gd.kstart],
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            gd.icells);
    cuda_check_error();

    // Calculate changes in the liquid water reservoir
    lsmk::calc_liquid_water_reservoir_g<<<grid_gpu_2d, block_gpu_2d>>>(
            fields.at2d.at("wl")->fld_g,
            interception_g,
            throughfall_g,
            fields.ap2d.at("wl")->fld_g,
            tiles.at("veg").LE_g,
            tiles.at("soil").LE_g,
            tiles.at("wet").LE_g,
            tiles.at("veg").fraction_g,
            tiles.at("soil").fraction_g,
            tiles.at("wet").fraction_g,
            rain_rate,
            c_veg_g,
            lai_g, subdt,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            gd.icells);
    cuda_check_error();

    fields.release_tmp_g(buoy);
    fields.release_tmp_g(tmp2);

    //
    // Calculate soil tendencies
    //
    // Only soil moisture has a source and conductivity term
    const bool sw_source_term_t = false;
    const bool sw_conductivity_term_t = false;
    const bool sw_source_term_theta = true;
    const bool sw_conductivity_term_theta = true;

    // Soil GPU grid without ghost cells.
    gridi = gd.imax/blocki + (gd.imax%blocki > 0);
    gridj = gd.jmax/blockj + (gd.jmax%blockj > 0);
    dim3 grid_gpu_3d (gridi,  gridj,  sgd.kmax);
    dim3 block_gpu_3d(blocki, blockj, 1);

    //
    // Soil temperature
    //
    // Calculate the thermal diffusivity at full levels
    sk::calc_thermal_properties_g<<<grid_gpu_3d, block_gpu_3d>>>(
            diffusivity_g,
            conductivity_g,
            soil_index_g,
            fields.sps.at("theta")->fld_g,
            theta_sat_g,
            gamma_T_dry_g,
            rho_C_g,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            sgd.kstart, sgd.kend,
            gd.icells, gd.ijcells);
    cuda_check_error();

    // Linear interpolation diffusivity to half levels
    sk::interp_2_vertical_g<TF, Soil_interpolation_type::Harmonic_mean><<<grid_gpu_3d, block_gpu_3d>>>(
            diffusivity_h_g,
            diffusivity_g,
            sgd.dz_g,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            sgd.kstart, sgd.kend,
            gd.icells, gd.ijcells);
    cuda_check_error();

    // Set flux boundary conditions at top and bottom of soil column
    // Top = soil heat flux (G) averaged over all tiles, bottom = zero flux.
    get_tiled_mean_g(tmp1->fld_bot_g, "G", TF(1));

    sk::set_bcs_temperature_g<TF><<<grid_gpu_2d, block_gpu_2d>>>(
            fields.sps.at("t")->flux_top_g,
            fields.sps.at("t")->flux_bot_g,
            tmp1->fld_bot_g,
            rho_C_g,
            soil_index_g,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            sgd.kend,
            gd.icells, gd.ijcells);
    cuda_check_error();

    // Calculate diffusive tendency
    sk::diff_explicit_g<TF, sw_source_term_t, sw_conductivity_term_t><<<grid_gpu_3d, block_gpu_3d>>>(
            fields.sts.at("t")->fld_g,
            fields.sps.at("t")->fld_g,
            diffusivity_h_g,
            conductivity_h_g,
            source_g,
            fields.sps.at("t")->flux_top_g,
            fields.sps.at("t")->flux_bot_g,
            sgd.dzi_g, sgd.dzhi_g,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            sgd.kstart, sgd.kend,
            gd.icells, gd.ijcells);
    cuda_check_error();

    //
    // Soil moisture
    //
    // Calculate the hydraulic diffusivity and conductivity at full levels
    sk::calc_hydraulic_properties_g<<<grid_gpu_3d, block_gpu_3d>>>(
            diffusivity_g,
            conductivity_g,
            soil_index_g,
            fields.sps.at("theta")->fld_g,
            theta_sat_g,
            theta_res_g,
            vg_a_g,
            vg_l_g,
            vg_m_g,
            gamma_theta_sat_g,
            gamma_theta_min_g,
            gamma_theta_max_g,
            kappa_theta_min_g,
            kappa_theta_max_g,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            sgd.kstart, sgd.kend,
            gd.icells, gd.ijcells);
    cuda_check_error();

    // Interpolation diffusivity and conductivity to half levels,
    // using the IFS method, which uses the max value from the
    // two surrounding grid points.
    sk::interp_2_vertical_g<TF, Soil_interpolation_type::Max><<<grid_gpu_3d, block_gpu_3d>>>(
            diffusivity_h_g,
            diffusivity_g,
            sgd.dz_g,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            sgd.kstart, sgd.kend,
            gd.icells, gd.ijcells);
    cuda_check_error();

    sk::interp_2_vertical_g<TF, Soil_interpolation_type::Max><<<grid_gpu_3d, block_gpu_3d>>>(
            conductivity_h_g,
            conductivity_g,
            sgd.dz_g,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            sgd.kstart, sgd.kend,
            gd.icells, gd.ijcells);
    cuda_check_error();

    // Calculate infiltration/runoff
    sk::calc_infiltration_g<<<grid_gpu_2d, block_gpu_2d>>>(
            infiltration_g,
            runoff_g,
            throughfall_g,
            fields.sps.at("theta")->fld_g,
            theta_sat_g,
            kappa_theta_max_g,
            gamma_theta_max_g,
            sgd.dz_g,
            soil_index_g,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            sgd.kend,
            gd.icells, gd.ijcells);
    cuda_check_error();

    // Set the boundary conditions.
    // Top = evaporation from bare soil tile.
    // Bottom = optionally free drainage (or else closed)
    sk::set_bcs_moisture_g<<<grid_gpu_2d, block_gpu_2d>>>(
            fields.sps.at("theta")->flux_top_g,
            fields.sps.at("theta")->flux_bot_g,
            conductivity_h_g,
            tiles.at("soil").LE_g,
            tiles.at("soil").fraction_g,
            infiltration_g,
            sw_free_drainage,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            sgd.kstart, sgd.kend,
            gd.icells, gd.ijcells);
    cuda_check_error();

    // Calculate root water extraction
    lsmk::scale_tile_with_fraction_g<TF><<<grid_gpu_2d, block_gpu_2d>>>(
            tmp1->fld_bot_g,
            tiles.at("veg").LE_g,
            tiles.at("veg").fraction_g,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            gd.icells);
    cuda_check_error();

    sk::calc_root_water_extraction_g<TF><<<grid_gpu_2d, block_gpu_2d>>>(
            source_g,
            tmp1->fld_top_g,
            fields.sps.at("theta")->fld_g,
            root_fraction_g,
            tmp1->fld_bot_g,
            sgd.dzi_g,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            sgd.kstart, sgd.kend,
            gd.icells, gd.ijcells);
    cuda_check_error();

    // Calculate diffusive tendency
    sk::diff_explicit_g<TF, sw_source_term_theta, sw_conductivity_term_theta><<<grid_gpu_3d, block_gpu_3d>>>(
            fields.sts.at("theta")->fld_g,
            fields.sps.at("theta")->fld_g,
            diffusivity_h_g,
            conductivity_h_g,
            source_g,
            fields.sps.at("theta")->flux_top_g,
            fields.sps.at("theta")->flux_bot_g,
            sgd.dzi_g, sgd.dzhi_g,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            sgd.kstart, sgd.kend,
            gd.icells, gd.ijcells);

    fields.release_tmp_g(tmp1);
}

template<typename TF>
void Boundary_surface_lsm<TF>::exec_column(Column<TF>& column)
{
    const TF no_offset = 0.;

    auto tmp = fields.get_tmp_g();

    column.calc_time_series("obuk", obuk_g, no_offset);
    column.calc_time_series("ustar", ustar_g, no_offset);
    column.calc_time_series("wl", fields.ap2d.at("wl")->fld_g, no_offset);

    get_tiled_mean_g(tmp->fld_bot_g, "H", TF(1));
    column.calc_time_series("H", tmp->fld_bot_g, no_offset);

    get_tiled_mean_g(tmp->fld_bot_g, "LE", TF(1));
    column.calc_time_series("LE", tmp->fld_bot_g, no_offset);

    get_tiled_mean_g(tmp->fld_bot_g, "G", TF(1));
    column.calc_time_series("G", tmp->fld_bot_g, no_offset);

    get_tiled_mean_g(tmp->fld_bot_g, "S", TF(1));
    column.calc_time_series("S", tmp->fld_bot_g, no_offset);

    if (sw_tile_stats_col)
        for (auto& tile : tiles)
        {
            column.calc_time_series("c_"+tile.first, tile.second.fraction_g, no_offset);

            column.calc_time_series("ustar_"+tile.first, tile.second.ustar_g, no_offset);
            column.calc_time_series("obuk_"+tile.first, tile.second.obuk_g, no_offset);

            column.calc_time_series("rs_"+tile.first, tile.second.rs_g, no_offset);
            column.calc_time_series("ra_"+tile.first, tile.second.ra_g, no_offset);

            column.calc_time_series("thl_bot_"+tile.first, tile.second.thl_bot_g, no_offset);
            column.calc_time_series("qt_bot_"+tile.first, tile.second.qt_bot_g, no_offset);

            column.calc_time_series("H_"+tile.first, tile.second.H_g, no_offset);
            column.calc_time_series("LE_"+tile.first, tile.second.LE_g, no_offset);
            column.calc_time_series("G_"+tile.first, tile.second.G_g, no_offset);
            column.calc_time_series("S_"+tile.first, tile.second.S_g, no_offset);
        }

    fields.release_tmp_g(tmp);
}

template<typename TF>
void Boundary_surface_lsm<TF>::get_tiled_mean_g(
    TF* const __restrict__ fld_out, std::string name, const TF scale_factor)
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;

    // For 2D field excluding ghost cells
    int gridi = gd.imax/blocki + (gd.imax%blocki > 0);
    int gridj = gd.jmax/blockj + (gd.jmax%blockj > 0);
    dim3 gridGPU (gridi,  gridj,  1);
    dim3 blockGPU(blocki, blockj, 1);

    TF* fld_veg;
    TF* fld_soil;
    TF* fld_wet;

    // Yikes..
    if (name == "H")
    {
        fld_veg  = tiles.at("veg").H_g;
        fld_soil = tiles.at("soil").H_g;
        fld_wet  = tiles.at("wet").H_g;
    }
    else if (name == "LE")
    {
        fld_veg  = tiles.at("veg").LE_g;
        fld_soil = tiles.at("soil").LE_g;
        fld_wet  = tiles.at("wet").LE_g;
    }
    else if (name == "G")
    {
        fld_veg  = tiles.at("veg").G_g;
        fld_soil = tiles.at("soil").G_g;
        fld_wet  = tiles.at("wet").G_g;
    }
    else if (name == "S")
    {
        fld_veg  = tiles.at("veg").S_g;
        fld_soil = tiles.at("soil").S_g;
        fld_wet  = tiles.at("wet").S_g;
    }
    else if (name == "bfluxbot")
    {
        fld_veg  = tiles.at("veg").bfluxbot_g;
        fld_soil = tiles.at("soil").bfluxbot_g;
        fld_wet  = tiles.at("wet").bfluxbot_g;
    }
    else if (name == "ustar")
    {
        fld_veg  = tiles.at("veg").ustar_g;
        fld_soil = tiles.at("soil").ustar_g;
        fld_wet  = tiles.at("wet").ustar_g;
    }
    else if (name == "thl_bot")
    {
        fld_veg  = tiles.at("veg").thl_bot_g;
        fld_soil = tiles.at("soil").thl_bot_g;
        fld_wet  = tiles.at("wet").thl_bot_g;
    }
    else if (name == "qt_bot")
    {
        fld_veg  = tiles.at("veg").qt_bot_g;
        fld_soil = tiles.at("soil").qt_bot_g;
        fld_wet  = tiles.at("wet").qt_bot_g;
    }
    else
        throw std::runtime_error("Cannot calculate tiled mean for variable \"" + name + "\"\\n");

    lsmk::calc_tiled_mean_g<<<gridGPU, blockGPU>>>(
            fld_out,
            tiles.at("veg").fraction_g,
            tiles.at("soil").fraction_g,
            tiles.at("wet").fraction_g,
            fld_veg,
            fld_soil,
            fld_wet,
            scale_factor,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            gd.icells);
}


template<typename TF>
void Boundary_surface_lsm<TF>::print_ij(
    const TF* const __restrict__ fld_g)
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;

    int gridi = gd.imax/blocki + (gd.imax%blocki > 0);
    int gridj = gd.jmax/blockj + (gd.jmax%blockj > 0);
    dim3 gridGPU (gridi,  gridj,  1);
    dim3 blockGPU(blocki, blockj, 1);

    lsmk::print_ij<<<gridGPU, blockGPU>>>(
        fld_g,
        gd.istart, gd.iend,
        gd.jstart, gd.jend,
        gd.icells);
    cuda_check_error();
    hipDeviceSynchronize();
}

template<typename TF>
void Boundary_surface_lsm<TF>::prepare_device()
{
    auto& gd = grid.get_grid_data();
    auto& sgd = soil_grid.get_grid_data();

    // Prepare base boundary, for inflow profiles.
    Boundary<TF>::prepare_device();

    const int tf_memsize_ij  = gd.ijcells*sizeof(TF);
    const int int_memsize_ij = gd.ijcells*sizeof(int);
    const int float_memsize_mo_lut = nzL_lut*sizeof(float);

    // Surface layer / Monin-Obukhov:
    cuda_safe_call(hipMalloc(&obuk_g,  tf_memsize_ij));
    cuda_safe_call(hipMalloc(&ustar_g, tf_memsize_ij));

    cuda_safe_call(hipMalloc(&z0m_g,   tf_memsize_ij));
    cuda_safe_call(hipMalloc(&z0h_g,   tf_memsize_ij));

    cuda_safe_call(hipMalloc(&dudz_mo_g, tf_memsize_ij));
    cuda_safe_call(hipMalloc(&dvdz_mo_g, tf_memsize_ij));
    cuda_safe_call(hipMalloc(&dbdz_mo_g, tf_memsize_ij));

    if (sw_constant_z0)
    {
        cuda_safe_call(hipMalloc(&nobuk_g, int_memsize_ij));
        cuda_safe_call(hipMalloc(&zL_sl_g, float_memsize_mo_lut));
        cuda_safe_call(hipMalloc(&f_sl_g,  float_memsize_mo_lut));
    }

    // Land-surface:
    // 1. Init tiles:
    for (auto& tile : tiles)
        lsmk::init_tile(tile.second, gd.ijcells);

    // 2. Init 2D surface properties:
    cuda_safe_call(hipMalloc(&gD_coeff_g, tf_memsize_ij));
    cuda_safe_call(hipMalloc(&c_veg_g, tf_memsize_ij));
    cuda_safe_call(hipMalloc(&lai_g, tf_memsize_ij));
    cuda_safe_call(hipMalloc(&rs_veg_min_g, tf_memsize_ij));
    cuda_safe_call(hipMalloc(&rs_soil_min_g, tf_memsize_ij));
    cuda_safe_call(hipMalloc(&lambda_stable_g, tf_memsize_ij));
    cuda_safe_call(hipMalloc(&lambda_unstable_g, tf_memsize_ij));
    cuda_safe_call(hipMalloc(&cs_veg_g, tf_memsize_ij));

    if (sw_water)
    {
        cuda_safe_call(hipMalloc(&water_mask_g, int_memsize_ij));
        cuda_safe_call(hipMalloc(&t_bot_water_g, tf_memsize_ij));
    }

    cuda_safe_call(hipMalloc(&interception_g, tf_memsize_ij));
    cuda_safe_call(hipMalloc(&throughfall_g, tf_memsize_ij));
    cuda_safe_call(hipMalloc(&infiltration_g, tf_memsize_ij));
    cuda_safe_call(hipMalloc(&runoff_g, tf_memsize_ij));

    // 3. Init 3D soil properties:
    const int tf_memsize_ijk  = sgd.ncells*sizeof(TF);
    const int tf_memsizeh_ijk = sgd.ncellsh*sizeof(TF);
    const int int_memsize_ijk = sgd.ncells*sizeof(int);

    cuda_safe_call(hipMalloc(&soil_index_g, int_memsize_ijk));
    cuda_safe_call(hipMalloc(&diffusivity_g, tf_memsize_ijk));
    cuda_safe_call(hipMalloc(&diffusivity_h_g, tf_memsizeh_ijk));
    cuda_safe_call(hipMalloc(&conductivity_g, tf_memsize_ijk));
    cuda_safe_call(hipMalloc(&conductivity_h_g, tf_memsizeh_ijk));
    cuda_safe_call(hipMalloc(&source_g, tf_memsize_ijk));
    cuda_safe_call(hipMalloc(&root_fraction_g, tf_memsize_ijk));

    // 4. Init lookup table with van Genuchten parameters:
    const int memsize_vg_lut = theta_res.size() * sizeof(TF);

    cuda_safe_call(hipMalloc(&theta_res_g, memsize_vg_lut));
    cuda_safe_call(hipMalloc(&theta_wp_g, memsize_vg_lut));
    cuda_safe_call(hipMalloc(&theta_fc_g, memsize_vg_lut));
    cuda_safe_call(hipMalloc(&theta_sat_g, memsize_vg_lut));

    cuda_safe_call(hipMalloc(&gamma_theta_sat_g, memsize_vg_lut));

    cuda_safe_call(hipMalloc(&vg_a_g, memsize_vg_lut));
    cuda_safe_call(hipMalloc(&vg_l_g, memsize_vg_lut));
    cuda_safe_call(hipMalloc(&vg_n_g, memsize_vg_lut));
    cuda_safe_call(hipMalloc(&vg_m_g, memsize_vg_lut));

    cuda_safe_call(hipMalloc(&kappa_theta_max_g, memsize_vg_lut));
    cuda_safe_call(hipMalloc(&kappa_theta_min_g, memsize_vg_lut));
    cuda_safe_call(hipMalloc(&gamma_theta_max_g, memsize_vg_lut));
    cuda_safe_call(hipMalloc(&gamma_theta_min_g, memsize_vg_lut));
    cuda_safe_call(hipMalloc(&gamma_T_dry_g, memsize_vg_lut));
    cuda_safe_call(hipMalloc(&rho_C_g, memsize_vg_lut));

    // Copy data from host to device
    forward_device();
}

template<typename TF>
void Boundary_surface_lsm<TF>::forward_device()
{
    auto& gd = grid.get_grid_data();
    auto& sgd = soil_grid.get_grid_data();

    const int tf_memsize_ij  = gd.ijcells*sizeof(TF);
    const int int_memsize_ij = gd.ijcells*sizeof(int);
    const int float_memsize_lut = nzL_lut*sizeof(float);

    // Surface layer / Monin-Obukhov:
    cuda_safe_call(hipMemcpy(obuk_g,  obuk.data(),  tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(ustar_g, ustar.data(), tf_memsize_ij, hipMemcpyHostToDevice));

    cuda_safe_call(hipMemcpy(z0m_g, z0m.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(z0h_g, z0h.data(), tf_memsize_ij, hipMemcpyHostToDevice));

    cuda_safe_call(hipMemcpy(dudz_mo_g, dudz_mo.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(dvdz_mo_g, dvdz_mo.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(dbdz_mo_g, dbdz_mo.data(), tf_memsize_ij, hipMemcpyHostToDevice));

    if (sw_constant_z0)
    {
        cuda_safe_call(hipMemcpy(nobuk_g, nobuk.data(), int_memsize_ij, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(zL_sl_g, zL_sl.data(), float_memsize_lut, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(f_sl_g,  f_sl.data(),  float_memsize_lut, hipMemcpyHostToDevice));
    }

    // Land-surface:
    // 1. Copy tiles:
    for (auto& tile : tiles)
        lsmk::forward_device_tile(tile.second, gd.ijcells);

    // 2. Copy 2D surface properties:
    cuda_safe_call(hipMemcpy(gD_coeff_g, gD_coeff.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(c_veg_g, c_veg.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(lai_g, lai.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(rs_veg_min_g, rs_veg_min.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(rs_soil_min_g, rs_soil_min.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(lambda_stable_g, lambda_stable.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(lambda_unstable_g, lambda_unstable.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(cs_veg_g, cs_veg.data(), tf_memsize_ij, hipMemcpyHostToDevice));

    if (sw_water)
    {
        cuda_safe_call(hipMemcpy(water_mask_g, water_mask.data(), int_memsize_ij, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(t_bot_water_g, t_bot_water.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    }

    cuda_safe_call(hipMemcpy(interception_g, interception.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(throughfall_g, throughfall.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(infiltration_g, infiltration.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(runoff_g, runoff.data(), tf_memsize_ij, hipMemcpyHostToDevice));

    // 3. Copy 3D soil properties:
    const int tf_memsize_ijk  = sgd.ncells*sizeof(TF);
    const int int_memsize_ijk = sgd.ncells*sizeof(int);

    cuda_safe_call(hipMemcpy(soil_index_g, soil_index.data(), int_memsize_ijk, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(diffusivity_g, diffusivity.data(), tf_memsize_ijk, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(diffusivity_h_g, diffusivity_h.data(), tf_memsize_ijk, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(conductivity_g, conductivity.data(), tf_memsize_ijk, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(conductivity_h_g, conductivity_h.data(), tf_memsize_ijk, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(source_g, source.data(), tf_memsize_ijk, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(root_fraction_g, root_fraction.data(), tf_memsize_ijk, hipMemcpyHostToDevice));

    // 4. Copy lookup table with van Genuchten parameters:
    const int memsize_vg_lut = theta_res.size() * sizeof(TF);

    cuda_safe_call(hipMemcpy(theta_res_g, theta_res.data(), memsize_vg_lut, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(theta_wp_g, theta_wp.data(), memsize_vg_lut, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(theta_fc_g, theta_fc.data(), memsize_vg_lut, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(theta_sat_g, theta_sat.data(), memsize_vg_lut, hipMemcpyHostToDevice));

    cuda_safe_call(hipMemcpy(gamma_theta_sat_g, gamma_theta_sat.data(), memsize_vg_lut, hipMemcpyHostToDevice));

    cuda_safe_call(hipMemcpy(vg_a_g, vg_a.data(), memsize_vg_lut, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(vg_l_g, vg_l.data(), memsize_vg_lut, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(vg_n_g, vg_n.data(), memsize_vg_lut, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(vg_m_g, vg_m.data(), memsize_vg_lut, hipMemcpyHostToDevice));

    cuda_safe_call(hipMemcpy(kappa_theta_max_g, kappa_theta_max.data(), memsize_vg_lut, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(kappa_theta_min_g, kappa_theta_min.data(), memsize_vg_lut, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(gamma_theta_max_g, gamma_theta_max.data(), memsize_vg_lut, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(gamma_theta_min_g, gamma_theta_min.data(), memsize_vg_lut, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(gamma_T_dry_g, gamma_T_dry.data(), memsize_vg_lut, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(rho_C_g, rho_C.data(), memsize_vg_lut, hipMemcpyHostToDevice));
}

template<typename TF>
void Boundary_surface_lsm<TF>::backward_device()
{
    auto& gd = grid.get_grid_data();

    const int tf_memsize_ij  = gd.ijcells*sizeof(TF);
    const int int_memsize_ij = gd.ijcells*sizeof(int);

    // NOTE: only copy back the required/useful data...
    cuda_safe_call(hipMemcpy(obuk.data(),  obuk_g,  tf_memsize_ij, hipMemcpyDeviceToHost));
    cuda_safe_call(hipMemcpy(ustar.data(), ustar_g, tf_memsize_ij, hipMemcpyDeviceToHost));

    cuda_safe_call(hipMemcpy(dudz_mo.data(), dudz_mo_g, tf_memsize_ij, hipMemcpyDeviceToHost));
    cuda_safe_call(hipMemcpy(dvdz_mo.data(), dvdz_mo_g, tf_memsize_ij, hipMemcpyDeviceToHost));
    cuda_safe_call(hipMemcpy(dbdz_mo.data(), dbdz_mo_g, tf_memsize_ij, hipMemcpyDeviceToHost));

    // TODO: which fields are needed from the land-surface?
    // Nearly all tile fields are used in the statistics:
    for (auto& tile : tiles)
        lsmk::backward_device_tile(tile.second, gd.ijcells);
}

template<typename TF>
void Boundary_surface_lsm<TF>::clear_device()
{
    //
    // De-llocate fields on GPU
    //
    // Monin-Obukhov stuff:
    cuda_safe_call(hipFree(obuk_g));
    cuda_safe_call(hipFree(ustar_g));

    cuda_safe_call(hipFree(z0m_g));
    cuda_safe_call(hipFree(z0h_g));

    cuda_safe_call(hipFree(dudz_mo_g));
    cuda_safe_call(hipFree(dvdz_mo_g));
    cuda_safe_call(hipFree(dbdz_mo_g));

    if (sw_constant_z0)
    {
        cuda_safe_call(hipFree(nobuk_g));
        cuda_safe_call(hipFree(zL_sl_g));
        cuda_safe_call(hipFree(f_sl_g));
    }
    // Land-surface stuff:
}
#endif

template class Boundary_surface_lsm<double>;
template class Boundary_surface_lsm<float>;
