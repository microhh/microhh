#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2020 Chiel van Heerwaarden
 * Copyright (c) 2011-2020 Thijs Heus
 * Copyright (c) 2014-2020 Bart van Stratum
 *
 * The cuda_safe_call() and cuda_check_error() are from
 * http://choorucode.com/2011/03/02/how-to-do-error-checking-in-cuda/
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <stdio.h>
#include "float.h"
#include "tools.h"

namespace Tools_g
{
    static std::string format_exception_message(hipError_t err, const char *file, const int line)
    {
        char output[1024];
        snprintf(output, sizeof output, "CUDA error: %s (%s) at %s:%d",
                 hipGetErrorName(err),
                 hipGetErrorString(err),
                 file,
                 line);

        return output;
    }

    cuda_exception::cuda_exception(hipError_t err, const char *file, const int line):
        err_(err),
        line_(line),
        file_(file),
        message_(format_exception_message(err, file, line))
    {
        //
    }

    const char *cuda_exception::what() const throw()
    {
        return message_.c_str();
    }

    int cuda_exception::line() const {
        return line_;
    }

    const char *cuda_exception::file() const {
        return file_.c_str();
    }

    hipError_t cuda_exception::error() const {
        return err_;
    }

    template <typename TF, Reduce_type function> __device__
    TF reduction(TF v1, TF v2)
    {
        TF rval;
        if (function == Sum_type)
            rval = v1+v2;
        else if (function == Max_type)
            rval = fmax(v1,v2);
        return rval;
    }

    // Reduce one block of data
    template <typename TF, Reduce_type function, int blockSize> __device__
    void reduce_block_kernel(volatile TF* as, const int tid)
    {
        /* Loop is completely unrolled for performance */
        if (blockSize >= 512) { if (tid < 256) { as[tid] = reduction<TF, function>(as[tid],as[tid + 256]); } __syncthreads(); }
        if (blockSize >= 256) { if (tid < 128) { as[tid] = reduction<TF, function>(as[tid],as[tid + 128]); } __syncthreads(); }
        if (blockSize >= 128) { if (tid <  64) { as[tid] = reduction<TF, function>(as[tid],as[tid +  64]); } __syncthreads(); }

        /* Once we get to the last 32 values (1 thread warp), the __syncthreads() is no longer necessary */
        if (tid < 32)
        {
            if (blockSize >=  64) { if (tid < 32) { as[tid] = reduction<TF, function>(as[tid],as[tid + 32]); }}
            if (blockSize >=  32) { if (tid < 16) { as[tid] = reduction<TF, function>(as[tid],as[tid + 16]); }}
            if (blockSize >=  16) { if (tid <  8) { as[tid] = reduction<TF, function>(as[tid],as[tid +  8]); }}
            if (blockSize >=   8) { if (tid <  4) { as[tid] = reduction<TF, function>(as[tid],as[tid +  4]); }}
            if (blockSize >=   4) { if (tid <  2) { as[tid] = reduction<TF, function>(as[tid],as[tid +  2]); }}
            if (blockSize >=   2) { if (tid <  1) { as[tid] = reduction<TF, function>(as[tid],as[tid +  1]); }}
        }
    }

    // Reduce field from 3D to 2D, excluding ghost cells and padding
    template <typename TF, Reduce_type function, int blockSize> __global__
    void reduce_interior_kernel(const TF* a, TF* a2d,
                        int istart, int jstart, int kstart,
                        int iend,   int jend,
                        int icells, int ijcells)
    {
        // See https://stackoverflow.com/a/27570775/3581217
        extern __shared__ unsigned char as_tmp[];
        TF *as = reinterpret_cast<TF*>(as_tmp);

        const int tid  = threadIdx.x;
        const int i    = istart + threadIdx.x;
        const int j    = jstart + blockIdx.y;
        const int k    = kstart + blockIdx.z;
        const int jk   = blockIdx.y+blockIdx.z*(jend-jstart);   // Index in 2D "a2d"
        const int ijk  = i + j*icells + k*ijcells;              // Index in 3D "a"
        const int ijkm = iend + j*icells + k*ijcells;    // Max index in X-direction

        TF tmpval;
        if (function == Max_type)
            tmpval = -FLT_MAX;  // This should ideally be a TF_MAX
        else if (function == Sum_type)
            tmpval = 0;

        int ii = ijk;
        while (ii < ijkm)
        {
            tmpval = reduction<TF, function>(tmpval,a[ii]);
            if (ii + blockDim.x < ijkm)
                tmpval = reduction<TF, function>(tmpval,a[ii+blockDim.x]);
            ii += 2*blockDim.x;
        }
        as[tid] = tmpval;

        __syncthreads();

        reduce_block_kernel<TF, function, blockSize>(as, tid);

        if (tid == 0)
            a2d[jk] = as[0];
    }

    // Reduce array, not accounting from ghost cells or padding
    template <typename TF, Reduce_type function, int blockSize> __global__
    void reduce_all_kernel(const TF* a, TF* aout, int ncells, int nvaluesperblock, TF scalefac)
    {
        // See https://stackoverflow.com/a/27570775/3581217
//        extern __shared__ __align__(sizeof(TF)) unsigned char as_tmp[];
        extern __shared__ unsigned char as_tmp[];
        TF *as = reinterpret_cast<TF*>(as_tmp);

        const int tid = threadIdx.x;
        const int iim = nvaluesperblock * (blockIdx.x+1);
        int ii        = nvaluesperblock *  blockIdx.x + threadIdx.x;

        TF tmpval;
        if (function == Max_type)
            tmpval = -FLT_MAX;  // This should ideally be a TF_MAX
        else if (function == Sum_type)
            tmpval = 0;

        while (ii < iim)
        {
            tmpval = reduction<TF, function>(tmpval,a[ii]);
            if (ii + blockDim.x < iim && ii + blockDim.x < ncells)
                tmpval = reduction<TF, function>(tmpval,a[ii+blockDim.x]);
            ii += 2*blockDim.x;
        }
        as[tid] = tmpval * scalefac;

        // Make sure all threads are synchronised before reducing the shared array
        __syncthreads();

        // Reduce block in shared memory
        reduce_block_kernel<TF, function, blockSize>(as, tid);

        // First value in shared array now holds the reduced value. Write back to global memory
        if (tid == 0)
            aout[blockIdx.x] = as[0];
    }

    template<typename TF> __global__
    void set_to_val(TF* __restrict__ a, int nsize, TF val)
    {
        const int n = blockIdx.x*blockDim.x + threadIdx.x;

        if (n < nsize)
            a[n] = val;
    }

    template<typename TF> __global__
    void mult_by_val(TF* __restrict__ a, int nsize, TF val)
    {
        const int n = blockIdx.x*blockDim.x + threadIdx.x;

        if (n < nsize)
            a[n] *= val;
    }

    int next_pow_of_2(unsigned int x)
    {
        return (int)pow(2,ceil(log(x)/log(2)));
    }


    template<typename TF>
    void reduce_interior(const TF* a, TF* a2d,
                         int itot, int istart, int iend,
                         int jtot, int jstart, int jend,
                         int ktot, int kstart,
                         int icells, int ijcells, Reduce_type mode)
    {
        const int nthreads = max(16,min(reduce_max_threads, next_pow_of_2(itot/2)));

        dim3 gridGPU (1, jtot, ktot);
        dim3 blockGPU(nthreads, 1, 1);

        if (mode == Max_type)
        {
            switch (nthreads)
            {
                case 512:
                    reduce_interior_kernel<TF, Max_type, 512><<<gridGPU, blockGPU, nthreads*sizeof(TF)>>>(a, a2d, istart, jstart, kstart, iend, jend, icells, ijcells); break;
                case 256:
                    reduce_interior_kernel<TF, Max_type, 256><<<gridGPU, blockGPU, nthreads*sizeof(TF)>>>(a, a2d, istart, jstart, kstart, iend, jend, icells, ijcells); break;
                case 128:
                    reduce_interior_kernel<TF, Max_type, 128><<<gridGPU, blockGPU, nthreads*sizeof(TF)>>>(a, a2d, istart, jstart, kstart, iend, jend, icells, ijcells); break;
                case 64:
                    reduce_interior_kernel<TF, Max_type,  64><<<gridGPU, blockGPU, nthreads*sizeof(TF)>>>(a, a2d, istart, jstart, kstart, iend, jend, icells, ijcells); break;
                case 32:
                    reduce_interior_kernel<TF, Max_type,  32><<<gridGPU, blockGPU, nthreads*sizeof(TF)>>>(a, a2d, istart, jstart, kstart, iend, jend, icells, ijcells); break;
                case 16:
                    reduce_interior_kernel<TF, Max_type,  16><<<gridGPU, blockGPU, nthreads*sizeof(TF)>>>(a, a2d, istart, jstart, kstart, iend, jend, icells, ijcells); break;
            }
        }
        else if (mode == Sum_type)
        {
            switch (nthreads)
            {
                case 512:
                    reduce_interior_kernel<TF, Sum_type, 512><<<gridGPU, blockGPU, nthreads*sizeof(TF)>>>(a, a2d, istart, jstart, kstart, iend, jend, icells, ijcells); break;
                case 256:
                    reduce_interior_kernel<TF, Sum_type, 256><<<gridGPU, blockGPU, nthreads*sizeof(TF)>>>(a, a2d, istart, jstart, kstart, iend, jend, icells, ijcells); break;
                case 128:
                    reduce_interior_kernel<TF, Sum_type, 128><<<gridGPU, blockGPU, nthreads*sizeof(TF)>>>(a, a2d, istart, jstart, kstart, iend, jend, icells, ijcells); break;
                case 64:
                    reduce_interior_kernel<TF, Sum_type,  64><<<gridGPU, blockGPU, nthreads*sizeof(TF)>>>(a, a2d, istart, jstart, kstart, iend, jend, icells, ijcells); break;
                case 32:
                    reduce_interior_kernel<TF, Sum_type,  32><<<gridGPU, blockGPU, nthreads*sizeof(TF)>>>(a, a2d, istart, jstart, kstart, iend, jend, icells, ijcells); break;
                case 16:
                    reduce_interior_kernel<TF, Sum_type,  16><<<gridGPU, blockGPU, nthreads*sizeof(TF)>>>(a, a2d, istart, jstart, kstart, iend, jend, icells, ijcells); break;
            }
        }
        cuda_check_error();
    }

    template<typename TF>
    void reduce_all(const TF* a, TF* aout, int ncells, int nblocks, int nvaluesperblock, Reduce_type mode, TF scalefac)
    {
       const int nthreads = max(16,min(reduce_max_threads, next_pow_of_2(nvaluesperblock/2)));

        dim3 gridGPU (nblocks,  1, 1);
        dim3 blockGPU(nthreads, 1, 1);

        if (mode == Max_type)
        {
            switch (nthreads)
            {
                case 512:
                    reduce_all_kernel<TF, Max_type, 512><<<gridGPU, blockGPU, nthreads*sizeof(TF)>>>(a, aout, ncells, nvaluesperblock, scalefac); break;
                case 256:
                    reduce_all_kernel<TF, Max_type, 256><<<gridGPU, blockGPU, nthreads*sizeof(TF)>>>(a, aout, ncells, nvaluesperblock, scalefac); break;
                case 128:
                    reduce_all_kernel<TF, Max_type, 128><<<gridGPU, blockGPU, nthreads*sizeof(TF)>>>(a, aout, ncells, nvaluesperblock, scalefac); break;
                case 64:
                    reduce_all_kernel<TF, Max_type,  64><<<gridGPU, blockGPU, nthreads*sizeof(TF)>>>(a, aout, ncells, nvaluesperblock, scalefac); break;
                case 32:
                    reduce_all_kernel<TF, Max_type,  32><<<gridGPU, blockGPU, nthreads*sizeof(TF)>>>(a, aout, ncells, nvaluesperblock, scalefac); break;
                case 16:
                    reduce_all_kernel<TF, Max_type,  16><<<gridGPU, blockGPU, nthreads*sizeof(TF)>>>(a, aout, ncells, nvaluesperblock, scalefac); break;
            }
        }
        else if (mode == Sum_type)
        {
            switch (nthreads)
            {
                case 512:
                    reduce_all_kernel<TF, Sum_type, 512><<<gridGPU, blockGPU, nthreads*sizeof(TF)>>>(a, aout, ncells, nvaluesperblock, scalefac); break;
                case 256:
                    reduce_all_kernel<TF, Sum_type, 256><<<gridGPU, blockGPU, nthreads*sizeof(TF)>>>(a, aout, ncells, nvaluesperblock, scalefac); break;
                case 128:
                    reduce_all_kernel<TF, Sum_type, 128><<<gridGPU, blockGPU, nthreads*sizeof(TF)>>>(a, aout, ncells, nvaluesperblock, scalefac); break;
                case 64:
                    reduce_all_kernel<TF, Sum_type,  64><<<gridGPU, blockGPU, nthreads*sizeof(TF)>>>(a, aout, ncells, nvaluesperblock, scalefac); break;
                case 32:
                    reduce_all_kernel<TF, Sum_type,  32><<<gridGPU, blockGPU, nthreads*sizeof(TF)>>>(a, aout, ncells, nvaluesperblock, scalefac); break;
                case 16:
                    reduce_all_kernel<TF, Sum_type,  16><<<gridGPU, blockGPU, nthreads*sizeof(TF)>>>(a, aout, ncells, nvaluesperblock, scalefac); break;
            }
        }
        cuda_check_error();
    }

}

template void Tools_g::reduce_interior<double>(const double*, double*, int, int, int, int, int, int, int, int, int, int, Tools_g::Reduce_type);
template void Tools_g::reduce_interior<float>(const float*, float*, int, int, int, int, int, int, int, int, int, int, Tools_g::Reduce_type);
template void Tools_g::reduce_all<double>(const double*, double*, int, int, int, Tools_g::Reduce_type, double);
template void Tools_g::reduce_all<float>(const float*, float*, int, int, int, Tools_g::Reduce_type, float);
template  __global__ void Tools_g::set_to_val(double* __restrict__, int, double);
template  __global__ void Tools_g::set_to_val(float* __restrict__, int, float);
template  __global__ void Tools_g::mult_by_val(double* __restrict__, int, double);
template  __global__ void Tools_g::mult_by_val(float* __restrict__, int, float);
