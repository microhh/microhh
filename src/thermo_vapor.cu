#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2020 Chiel van Heerwaarden
 * Copyright (c) 2011-2020 Thijs Heus
 * Copyright (c) 2014-2020 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <cstdio>
#include "grid.h"
#include "fields.h"
#include "thermo_vapor.h"
#include "defines.h"
#include "constants.h"
#include "finite_difference.h"
#include "master.h"
#include "tools.h"
#include "column.h"
#include "stats.h"

#include "thermo_moist_functions.h"

namespace
{
    using namespace Constants;
    using namespace Finite_difference::O2;
    using namespace Thermo_moist_functions;

    template<typename TF> __global__
    void calc_buoyancy_tend_2nd_g(TF* __restrict__ wt, TF* __restrict__ th, TF* __restrict__ qt,
                                  TF* __restrict__ thvrefh, TF* __restrict__ exnh, TF* __restrict__ ph,
                                  int istart, int jstart, int kstart,
                                  int iend,   int jend,   int kend,
                                  int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;

            // Half level temperature and vaporure content
            const TF thh = TF(0.5) * (th[ijk-kk] + th[ijk]);         // Half level liq. water pot. temp.
            const TF qth = TF(0.5) * (qt[ijk-kk] + qt[ijk]);         // Half level specific hum.
            wt[ijk] += buoyancy_no_ql(thh, qth, thvrefh[k]);
        }
    }

    template<typename TF> __global__
    void calc_buoyancy_g(TF* __restrict__ b,  TF* __restrict__ th,
                         TF* __restrict__ qt, TF* __restrict__ thvref,
                         TF* __restrict__ p,  TF* __restrict__ exn,
                         int istart, int jstart, int kstart,
                         int iend,   int jend,   int kcells,
                         int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z;

        if (i < iend && j < jend && k < kcells)
        {
            const int ijk   = i + j*jj + k*kk;
            b[ijk] = buoyancy_no_ql(th[ijk], qt[ijk], thvref[k]);
        }
    }

    template<typename TF> __global__
    void calc_buoyancy_bot_g(TF* __restrict__ b,      TF* __restrict__ bbot,
                             TF* __restrict__ th,     TF* __restrict__ thbot,
                             TF* __restrict__ qt,     TF* __restrict__ qtbot,
                             TF* __restrict__ thvref, TF* __restrict__ thvrefh,
                             int kstart, int icells, int jcells,
                             int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        const int j = blockIdx.y*blockDim.y + threadIdx.y;

        if (i < icells && j < jcells)
        {
            const int ij  = i + j*jj;
            const int ijk = i + j*jj + kstart*kk;

            bbot[ij ] = buoyancy_no_ql(thbot[ij], qtbot[ij], thvrefh[kstart]);
            b   [ijk] = buoyancy_no_ql(th[ijk],   qt[ijk],   thvref[kstart]);
        }
    }

    template<typename TF> __global__
    void calc_buoyancy_flux_bot_g(TF* __restrict__ bfluxbot,
                                  TF* __restrict__ th, TF* __restrict__ thfluxbot,
                                  TF* __restrict__ qt, TF* __restrict__ qtfluxbot,
                                  TF* __restrict__ thvrefh,
                                  int kstart, int icells, int jcells,
                                  int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        const int j = blockIdx.y*blockDim.y + threadIdx.y;

        if (i < icells && j < jcells)
        {
            const int ij  = i + j*jj;
            const int ijk = i + j*jj + kstart*kk;
            bfluxbot[ij] = buoyancy_flux_no_ql(th[ijk], thfluxbot[ij], qt[ijk], qtfluxbot[ij], thvrefh[kstart]);
        }
    }

    template<typename TF> __global__
    void calc_N2_g(TF* __restrict__ N2, TF* __restrict__ th,
                   TF* __restrict__ thvref, TF* __restrict__ dzi,
                   int istart, int jstart, int kstart,
                   int iend,   int jend,   int kend,
                   int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            N2[ijk] = grav<TF>/thvref[k]*TF(0.5)*(th[ijk+kk] - th[ijk-kk])*dzi[k];
        }
    }

} // end name    space

template<typename TF>
void Thermo_vapor<TF>::prepare_device()
{
    auto& gd = grid.get_grid_data();
    const int nmemsize = gd.kcells*sizeof(TF);

    // Allocate fields for Boussinesq and anelastic solver
    cuda_safe_call(hipMalloc(&bs.thvref_g,  nmemsize));
    cuda_safe_call(hipMalloc(&bs.thvrefh_g, nmemsize));
    cuda_safe_call(hipMalloc(&bs.pref_g,    nmemsize));
    cuda_safe_call(hipMalloc(&bs.prefh_g,   nmemsize));
    cuda_safe_call(hipMalloc(&bs.exnref_g,  nmemsize));
    cuda_safe_call(hipMalloc(&bs.exnrefh_g, nmemsize));

    // Copy fields to device
    cuda_safe_call(hipMemcpy(bs.thvref_g,  bs.thvref.data(),  nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(bs.thvrefh_g, bs.thvrefh.data(), nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(bs.pref_g,    bs.pref.data(),    nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(bs.prefh_g,   bs.prefh.data(),   nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(bs.exnref_g,  bs.exnref.data(),  nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(bs.exnrefh_g, bs.exnrefh.data(), nmemsize, hipMemcpyHostToDevice));
}

template<typename TF>
void Thermo_vapor<TF>::clear_device()
{
    cuda_safe_call(hipFree(bs.thvref_g ));
    cuda_safe_call(hipFree(bs.thvrefh_g));
    cuda_safe_call(hipFree(bs.pref_g   ));
    cuda_safe_call(hipFree(bs.prefh_g  ));
    cuda_safe_call(hipFree(bs.exnref_g ));
    cuda_safe_call(hipFree(bs.exnrefh_g));
    tdep_pbot->clear_device();
}

template<typename TF>
void Thermo_vapor<TF>::forward_device()
{
    // Copy fields to device
    auto& gd = grid.get_grid_data();
    const int nmemsize = gd.kcells*sizeof(TF);
    cuda_safe_call(hipMemcpy(bs.pref_g,    bs.pref.data(),    nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(bs.prefh_g,   bs.prefh.data(),   nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(bs.exnref_g,  bs.exnref.data(),  nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(bs.exnrefh_g, bs.exnrefh.data(), nmemsize, hipMemcpyHostToDevice));
}

template<typename TF>
void Thermo_vapor<TF>::backward_device()
{
    auto& gd = grid.get_grid_data();
    const int nmemsize = gd.kcells*sizeof(TF);
    hipMemcpy(bs.pref_g,    bs.pref.data(),    nmemsize, hipMemcpyHostToDevice);
    hipMemcpy(bs.prefh_g,   bs.prefh.data(),   nmemsize, hipMemcpyHostToDevice);
    hipMemcpy(bs.exnref_g,  bs.exnref.data(),  nmemsize, hipMemcpyHostToDevice);
    hipMemcpy(bs.exnrefh_g, bs.exnrefh.data(), nmemsize, hipMemcpyHostToDevice);

    bs_stats = bs;

}

#ifdef USECUDA
template<typename TF>
void Thermo_vapor<TF>::exec(const double dt, Stats<TF>& stats)
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, gd.kmax);
    dim3 blockGPU(blocki, blockj, 1);


    // Re-calculate hydrostatic pressure and exner
    if (bs.swupdatebasestate)
    {
        //calc_hydrostatic_pressure<TF><<<1, 1>>>(bs.pref_g, bs.prefh_g, bs.exnref_g, bs.exnrefh_g,
        //                                        fields.sp.at("thl")->fld_mean_g, fields.sp.at("qt")->fld_mean_g,
        //                                        gd.z_g, gd.dz_g, gd.dzh_g, bs.pbot, gd.kstart, gd.kend);
        //cuda_check_error();

        // BvS: Calculating hydrostatic pressure on GPU is extremely slow. As temporary solution, copy back mean profiles to host,
        //      calculate pressure there and copy back the required profiles.
        hipMemcpy(fields.sp.at("thl")->fld_mean.data(), fields.sp.at("thl")->fld_mean_g, gd.kcells*sizeof(TF), hipMemcpyDeviceToHost);
        hipMemcpy(fields.sp.at("qt")->fld_mean.data(),  fields.sp.at("qt")->fld_mean_g,  gd.kcells*sizeof(TF), hipMemcpyDeviceToHost);

        auto tmp = fields.get_tmp();

        calc_base_state_no_ql(bs.pref.data(), bs.prefh.data(),
                        &tmp->fld[0*gd.kcells], &tmp->fld[1*gd.kcells], &tmp->fld[2*gd.kcells], &tmp->fld[3*gd.kcells],
                        bs.exnref.data(), bs.exnrefh.data(), fields.sp.at("thl")->fld_mean.data(), fields.sp.at("qt")->fld_mean.data(),
                        bs.pbot, gd.kstart, gd.kend, gd.z.data(), gd.dz.data(), gd.dzh.data());

        fields.release_tmp(tmp);

        // Only half level pressure and bs.exner needed for BuoyancyTend()
        hipMemcpy(bs.prefh_g,   bs.prefh.data(),   gd.kcells*sizeof(TF), hipMemcpyHostToDevice);
        hipMemcpy(bs.exnrefh_g, bs.exnrefh.data(), gd.kcells*sizeof(TF), hipMemcpyHostToDevice);
    }

    calc_buoyancy_tend_2nd_g<TF><<<gridGPU, blockGPU>>>(
        fields.mt.at("w")->fld_g, fields.sp.at("thl")->fld_g,
        fields.sp.at("qt")->fld_g, bs.thvrefh_g, bs.exnrefh_g, bs.prefh_g,
        gd.istart,  gd.jstart, gd.kstart+1,
        gd.iend,    gd.jend,   gd.kend,
        gd.icells, gd.ijcells);
    cuda_check_error();

    hipDeviceSynchronize();
    stats.calc_tend(*fields.mt.at("w"), tend_name);

}
#endif

#ifdef USECUDA
template<typename TF>
void Thermo_vapor<TF>::get_thermo_field_g(
        Field3d<TF>& fld, const std::string& name, const bool cyclic )
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, gd.kcells);
    dim3 blockGPU(blocki, blockj, 1);

    dim3 gridGPU2 (gridi, gridj, gd.kmax);
    dim3 blockGPU2(blocki, blockj, 1);

    // Re-calculate hydrostatic pressure and exner
    if (bs.swupdatebasestate)
    {
        //calc_hydrostatic_pressure<TF><<<1, 1>>>(bs.pref_g, bs.prefh_g, bs.exnref_g, bs.exnrefh_g,
        //                                        fields.sp.at("thl")->fld_mean_g, fields.sp.at("qt")->fld_mean_g,
        //                                        gd.z_g, gd.dz_g, gd.dzh_g, bs.pbot, gd.kstart, gd.kend);
        //cuda_check_error();

        // BvS: Calculating hydrostatic pressure on GPU is extremely slow. As temporary solution, copy back mean profiles to host,
        //      calculate pressure there and copy back the required profiles.
        hipMemcpy(fields.sp.at("thl")->fld_mean.data(), fields.sp.at("thl")->fld_mean_g, gd.kcells*sizeof(TF), hipMemcpyDeviceToHost);
        hipMemcpy(fields.sp.at("qt")->fld_mean.data(),  fields.sp.at("qt")->fld_mean_g,  gd.kcells*sizeof(TF), hipMemcpyDeviceToHost);

        auto tmp = fields.get_tmp();

        calc_base_state_no_ql(bs.pref.data(), bs.prefh.data(),
                        &tmp->fld[0*gd.kcells], &tmp->fld[1*gd.kcells], &tmp->fld[2*gd.kcells], &tmp->fld[3*gd.kcells],
                        bs.exnref.data(), bs.exnrefh.data(), fields.sp.at("thl")->fld_mean.data(), fields.sp.at("qt")->fld_mean.data(),
                        bs.pbot, gd.kstart, gd.kend, gd.z.data(), gd.dz.data(), gd.dzh.data());

        fields.release_tmp(tmp);

        // Only half level pressure and bs.exner needed for BuoyancyTend()
        hipMemcpy(bs.prefh_g,   bs.prefh.data(),   gd.kcells*sizeof(TF), hipMemcpyHostToDevice);
        hipMemcpy(bs.exnrefh_g, bs.exnrefh.data(), gd.kcells*sizeof(TF), hipMemcpyHostToDevice);
    }


    if (name == "b")
    {
        calc_buoyancy_g<TF><<<gridGPU, blockGPU>>>(
            fld.fld_g, fields.sp.at("thl")->fld_g, fields.sp.at("qt")->fld_g,
            bs.thvref_g, bs.pref_g, bs.exnref_g,
            gd.istart,  gd.jstart, gd.kstart,
            gd.iend, gd.jend, gd.kcells,
            gd.icells, gd.ijcells);
        cuda_check_error();
    }
    else if (name == "N2")
    {
        calc_N2_g<TF><<<gridGPU2, blockGPU2>>>(
            fld.fld_g, fields.sp.at("thl")->fld_g, bs.thvref_g, gd.dzi_g,
            gd.istart,  gd.jstart, gd.kstart,
            gd.iend,    gd.jend,   gd.kend,
            gd.icells, gd.ijcells);
        cuda_check_error();
    }
    else
    {
            std::string msg = "get_thermo_field \"" + name + "\" not supported";
            throw std::runtime_error(msg);
    }

    if (cyclic)
        boundary_cyclic.exec_g(fld.fld_g);
}
#endif

#ifdef USECUDA
template<typename TF>
TF* Thermo_vapor<TF>::get_basestate_fld_g(std::string name)
{
    // BvS TO-DO: change std::string to enum
    if (name == "pref")
        return bs.pref_g;
    else if (name == "prefh")
        return bs.prefh_g;
    else if (name == "exner")
        return bs.exnref_g;
    else if (name == "exnerh")
        return bs.exnrefh_g;
    else
    {
        std::string error_message = "Can not get basestate field \"" + name + "\" from thermo_moist";
        throw std::runtime_error(error_message);
    }
}
#endif

#ifdef USECUDA
template<typename TF>
void Thermo_vapor<TF>::get_buoyancy_fluxbot_g(Field3d<TF>& bfield)
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.icells/blocki + (gd.icells%blocki > 0);
    const int gridj  = gd.jcells/blockj + (gd.jcells%blockj > 0);

    dim3 gridGPU (gridi, gridj, 1);
    dim3 blockGPU(blocki, blockj, 1);

    calc_buoyancy_flux_bot_g<TF><<<gridGPU, blockGPU>>>(
        bfield.flux_bot_g,
        fields.sp.at("thl")->fld_g, fields.sp.at("thl")->flux_bot_g,
        fields.sp.at("qt")->fld_g, fields.sp.at("qt")->flux_bot_g,
        bs.thvrefh_g, gd.kstart, gd.icells, gd.jcells,
        gd.icells, gd.ijcells);
    cuda_check_error();
}
#endif

#ifdef USECUDA
template<typename TF>
void Thermo_vapor<TF>::get_buoyancy_surf_g(Field3d<TF>& bfield)
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.icells/blocki + (gd.icells%blocki > 0);
    const int gridj  = gd.jcells/blockj + (gd.jcells%blockj > 0);

    dim3 gridGPU (gridi, gridj, 1);
    dim3 blockGPU(blocki, blockj, 1);

    calc_buoyancy_bot_g<TF><<<gridGPU, blockGPU>>>(
        bfield.fld_g, bfield.fld_bot_g,
        fields.sp.at("thl")->fld_g, fields.sp.at("thl")->fld_bot_g,
        fields.sp.at("qt")->fld_g, fields.sp.at("qt")->fld_bot_g,
        bs.thvref_g, bs.thvrefh_g, gd.kstart, gd.icells, gd.jcells,
        gd.icells, gd.ijcells);
    cuda_check_error();

    calc_buoyancy_flux_bot_g<TF><<<gridGPU, blockGPU>>>(
        bfield.flux_bot_g,
        fields.sp.at("thl")->fld_g, fields.sp.at("thl")->flux_bot_g,
        fields.sp.at("qt")->fld_g, fields.sp.at("qt")->flux_bot_g,
        bs.thvrefh_g, gd.kstart, gd.icells, gd.jcells,
        gd.icells, gd.ijcells);
    cuda_check_error();
}
#endif

#ifdef USECUDA
template<typename TF>
void Thermo_vapor<TF>::exec_column(Column<TF>& column)
{
    const TF no_offset = 0.;
    auto output = fields.get_tmp_g();

    get_thermo_field_g(*output, "b", false);
    column.calc_column("b", output->fld_g, no_offset);

    fields.release_tmp_g(output);
}
#endif
template class Thermo_vapor<double>;
template class Thermo_vapor<float>;
